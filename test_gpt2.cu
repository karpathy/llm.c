#include "hip/hip_runtime.h"
#define TESTING
#include "train_gpt2.cu"

// poor man's tensor checker
int check_tensor(float *a, float *b, int n, const char* label, float threshold=1e-0) {
    // a is the calculated tensor, b is the reference tensor
    int print_upto = 10;
    int ok = 1;
    float max_diff = 0.0f;
    float max_rel_error = 0.0f;
    float max_to_threshold = 0.f;
    float max_a = 0.0f;
    float max_b = 0.0f;
    float epsilon = 0.079;      // BF16 epsilon value
    printf("---\n");
    printf("checking tensor: %s\n", label);
    for (int i = 0; i < n; i++) {
        float t_eff = threshold + fabs(b[i]) * epsilon;
        float diff = fabsf(a[i] - b[i]);
        max_to_threshold = max(max_to_threshold, diff / t_eff);
        if (diff > max_diff) {
            max_diff = diff;
            float denom = fabsf(b[i]);
            max_rel_error = (denom == 0.0f) ? 0.0f : diff / denom;
            max_a = a[i];
            max_b = b[i];
        }
        if (diff > t_eff) {
            ok = 0;
        }
        // print the first few elements so we can visually assess the "proof" of the comparison
        if (i < print_upto) {
            printf(diff <= t_eff ? "OK " :  "NOT OK ");
            printf("%f %f\n", a[i], b[i]);
        }
    }
    // print the final result
    if (ok) {
        printf("TENSOR OK, max diff: %.3e, with rel error: %.3e (calculated=%10f, ref=%10f), %.2f%% of maximum error\n",
                max_diff, max_rel_error, max_a, max_b, max_to_threshold*100);
    } else {
        printf("TENSOR NOT OK, max diff: %.3e, with rel error: %.3e (calculated=%10f, ref=%10f), %.2f%% of maximum error\n",
                max_diff, max_rel_error, max_a, max_b, max_to_threshold*100);
    }
    return ok;
}

// the same tensors as in the train file, but in float, which are used as reference
typedef struct {
    float*  wte; // (Vp, C)
    float*  wpe; // (maxT, C)
    float*  ln1w; // (L, C)
    float*  ln1b; // (L, C)
    float*  qkvw; // (L, 3*C, C)
    float*  qkvb; // (L, 3*C)
    float*  attprojw; // (L, C, C)
    float*  attprojb; // (L, C)
    float*  ln2w; // (L, C)
    float*  ln2b; // (L, C)
    float*  fcw; // (L, 4*C, C)
    float*  fcb; // (L, 4*C)
    float*  fcprojw; // (L, C, 4*C)
    float*  fcprojb; // (L, C)
    float*  lnfw; // (C)
    float*  lnfb; // (C)
} FloatParameterTensors;
static_assert(sizeof(FloatParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

// malloc_and_point, but in float and on CPU, because we use this data to check correctness on CPU
float* float_cpu_malloc_and_point_parameters(FloatParameterTensors* params, size_t* param_sizes) {
    // calculate the total number of parameters
    size_t num_parameters = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += param_sizes[i];
    }
    // everything is float so number of bytes to allocate is a simple multiplication
    float* params_memory = (float*)mallocCheck(num_parameters * sizeof(float));
    float** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    float* params_memory_iterator = params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = params_memory_iterator;
        params_memory_iterator += param_sizes[i];
    }
    return params_memory;
}

int main(int argc, char *argv[]) {
    char nccl_init_method[256] = "mpi";  // "tcp" or "fs" or "mpi"
    int num_processes = -1;  // doesn't matter when using MPI
    int process_rank = -1;  // doesn't matter when using MPI
    int gpus_per_node = -1;  // doesn't matter when using MPI
    char server_ip[256] = "";  // doesn't matter when using MPI
    char fs_path[256] = "";  // doesn't matter when using MPI
    multi_gpu_config = multi_gpu_config_init(num_processes, process_rank, gpus_per_node, server_ip, fs_path, nccl_init_method);
    common_start(false, true);

    // set the right paths
    #if defined(ENABLE_BF16)
    const char* load_filename = "gpt2_124M_bf16.bin";
    #else
    const char* load_filename = "gpt2_124M.bin";
    #endif

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_init_common(&model);

    gpt2_build_from_checkpoint(&model, load_filename);
    size_t V = model.config.vocab_size;
    size_t Vp = model.config.padded_vocab_size;
    size_t maxT = model.config.max_seq_len;
    size_t L = model.config.num_layers;
    size_t C = model.config.channels;

    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { exit(EXIT_FAILURE);  } // must have arg after flag
        if (!(strlen(argv[i]) == 2 || strlen(argv[i]) == 3)) { exit(EXIT_FAILURE); } // must be -x[y] (one dash, one or two letters)
        if (argv[i][0] != '-') { exit(EXIT_FAILURE); } // must start with dash
        if (argv[i][1] == 'w') { model.use_master_weights = atoi(argv[i+1]); }
        else if (argv[i][1] == 'r') { model.recompute = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g' && argv[i][2] == 'e') { model.gelu_fusion = atoi(argv[i+1]); }
    }

    // load additional information that we will use for debugging and error checking
    FILE *state_file = fopenCheck("gpt2_124M_debug_state.bin", "rb");
    int state_header[256];
    freadCheck(state_header, sizeof(int), 256, state_file);
    if (state_header[0] != 20240327) { fprintf(stderr, "Bad magic state file\n"); exit(EXIT_FAILURE); }
    if (state_header[1] != 2) {
        fprintf(stderr, "Bad version in state file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }
    int B = state_header[2]; // batch size, e.g. 4
    int T = state_header[3]; // time / sequence length (e.g. 64, up to maxT)
    assert(0 <= T && T <= maxT);
    printf("[State]\n");
    printf("batch_size: %d\n", B);
    printf("seq_len: %d\n", T);

    set_zero_configs(&multi_gpu_config, 0, model.num_parameters);

    // read reference information from the file saved from Python/PyTorch side
    // 1) input x and y
    int* x = (int*)mallocCheck(B * T * sizeof(int));
    int* y = (int*)mallocCheck(B * T * sizeof(int));
    freadCheck(x, sizeof(int), B*T, state_file);
    freadCheck(y, sizeof(int), B*T, state_file);
    // 2) results of forward pass (logits and loss)
    float* expected_logits = (float*) mallocCheck(B * T * V * sizeof(float));
    float* expected_loss = (float*) mallocCheck(1 * sizeof(float));
    freadCheck(expected_logits, sizeof(float), B*T*V, state_file);
    freadCheck(expected_loss, sizeof(float), 1, state_file);
    // 3) results of backward pass (parameter gradients)
    FloatParameterTensors expected_grads; // will be read from file. right now: all in fp32
    float* expected_grads_memory = float_cpu_malloc_and_point_parameters(&expected_grads, model.param_elements);
    freadCheck(expected_grads_memory, sizeof(float), model.num_parameters, state_file);
    fcloseCheck(state_file);

    // this memory will be used to do one single copy of all (mixed precision) GPU grads to CPU grads
    void* grads_memory_cpu = mallocCheck(model.num_parameters_bytes);
    float* grads_memory_cpu_float = (float*)mallocCheck(model.num_parameters * sizeof(float));

    // overall OK signal for the test
    int allok = 1;

    gpt2_allocate_state(&model, B, T);

    // First, do target-free forward pass to validate logits
    gpt2_forward(&model, x, B, T);
    // at this point, target should be equal to expected_logits, let's compare
    // copy logits to CPU so we can compare them
    floatX* logits_cpu_raw = (floatX*)mallocCheck(B * T * Vp * sizeof(floatX));
    float* logits_cpu = (float*)mallocCheck(B * T * Vp * sizeof(float));
    cudaCheck(hipMemcpy(logits_cpu_raw, model.acts.output, B * T * Vp * sizeof(floatX), hipMemcpyDeviceToHost));
    for (int i = 0; i < B * T * Vp; i++) {
        logits_cpu[i] = (float)logits_cpu_raw[i];
    }

    float logit_accuracy_threshold = 1e-3f;
    float loss_diff_threshold = 1e-5f;
    // FP16 and lower require very high tolerances unfortunately. TODO look into more
    #if defined(ENABLE_BF16) || defined(ENABLE_F16)
    logit_accuracy_threshold = 25.0f; // 15.0f was too low even without cuDNN?! :(
    loss_diff_threshold = 0.05f;
    #endif

    // compare the output logits from the forward pass
    // also careful that we don't access and compare the padded columns of logits
    int logits_ok = 1;
    float max_diff = 0.0f;
    for (int bt = 0; bt < B*T; bt++) {
        for (int v = 0; v < V; v++) {
            int i = bt * Vp + v; // linearized index
            if (i < 10) {
                printf("%f, %f\n", expected_logits[i], logits_cpu[i]);
            }
            float diff = fabsf(expected_logits[bt*V + v] - logits_cpu[i]);
            max_diff = fmaxf(max_diff, diff);
            if (diff >= logit_accuracy_threshold) {
                printf("MISMATCH AT INDEX %d,%d: ", bt, v);
                printf("%f %f\n", expected_logits[bt*V + v], logits_cpu[i]);
                logits_ok = 0;
                bt = B*T; // to break out of both loops
                break;
            }
        }
    }
    allok = allok && logits_ok;
    if(!logits_ok) { printf("NOT "); }
    printf("OK (LOGITS)\n");
    printf("logit max diff: %f\n", max_diff);

    // let's do 10 training iterations, following the pytorch code
    float losses[10];
    for (int step = 0; step < 10; step++) {
        struct timespec start, end;
        clock_gettime(CLOCK_MONOTONIC, &start);
        gpt2_forward(&model, x, B, T);
        gpt2_backward_and_reduce(&model, x, y, 1, 0);
        clock_gettime(CLOCK_MONOTONIC, &end);
        double time_elapsed_s = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

        if (step == 0) {
            // error checking at step 0 for reference activations

            // move the (mixed precision) grads from GPU to CPU
            cudaCheck(hipMemcpy(grads_memory_cpu, model.grads_memory, model.num_parameters_bytes, hipMemcpyDeviceToHost));

            // convert all gradients to float on the CPU
            char* src_iterator = (char*)grads_memory_cpu; // can be lower precision, so we use char*
            float* dst_iterator = (float*)grads_memory_cpu_float; // float*
            float* exp_iterator = expected_grads_memory; // float* of expected gradients from Python
            float* tensors1[NUM_PARAMETER_TENSORS];
            float* tensors2[NUM_PARAMETER_TENSORS];
            for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
                if (model.param_sizeof[i] == sizeof(float)) {
                    // float tensor => copy over directly
                    memcpy(dst_iterator, src_iterator, model.param_elements[i] * sizeof(float));
                } else {
                    // low-precision tensor => convert to float
                    assert(model.param_sizeof[i] == sizeof(floatX)); // floatX is the single non-float supported atm
                    for (size_t j = 0; j < model.param_elements[i]; j++) {
                        dst_iterator[j] = ((floatX*)src_iterator)[j]; // convert to float
                    }
                }
                // for convenience record the position of comparison for reality vs. expectation
                tensors1[i] = dst_iterator; // reality
                tensors2[i] = exp_iterator; // expectation
                // advance the iterators
                src_iterator += model.param_elements[i] * model.param_sizeof[i];
                dst_iterator += model.param_elements[i];
                exp_iterator += model.param_elements[i];
            }

            // compare the gradients on the parameters all at once, in fp32
            // I set the tolerances manually by inspecting the gradient differences for
            // a few elements of each tensor. bf16 looks ok but not amazing here.
            // It's possible we have bugs lurking, or maybe it is bf16. Not 100% sure.
            // Also, if code changes and some of these get tripped, it could be ok if it's not by too much,
            // because our use of stochastic rounding is adding some non-determinism "pepper noise".
            // In that case it's ok to extend the tolerance by a bit, after a manual review.
            // Also, different GPUs may use different matrix multiplication algorithms, so the
            // actual errors can be hardware specific.

            float grad_thresholds[NUM_PARAMETER_TENSORS] = {5e-1f, 4e-3f, 1e-1f, 3.5e-2f, 2e-2f, 3e-2f, 5e-2f, 5e-2f, 5e-2f, 1.5e-2f, 5e-4f, 8e-3f, 1.5e-3f, 2.5e-3f, 1e-1f, 2e-2f};
            #if defined(ENABLE_FP32)
            for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
                grad_thresholds[i] = 1e-6f;  // we can be much more precise in FP32
            }
            #endif

            allok = allok & check_tensor(tensors1[0], tensors2[0], V * C, "wte", grad_thresholds[0]);
            allok = allok & check_tensor(tensors1[1], tensors2[1], maxT * C, "wpe", grad_thresholds[1]);
            allok = allok & check_tensor(tensors1[2], tensors2[2], L * 3*C * C, "qkvw", grad_thresholds[2]);
            allok = allok & check_tensor(tensors1[3], tensors2[3], L * 3*C, "qkvb", grad_thresholds[3]);
            allok = allok & check_tensor(tensors1[4], tensors2[4], L * C * C, "attprojw", grad_thresholds[4]);
            allok = allok & check_tensor(tensors1[5], tensors2[5], L * C, "attprojb", grad_thresholds[5]);
            allok = allok & check_tensor(tensors1[6], tensors2[6], L * 4*C * C, "fcw", grad_thresholds[6]);
            allok = allok & check_tensor(tensors1[7], tensors2[7], L * 4*C, "fcb", grad_thresholds[7]);
            allok = allok & check_tensor(tensors1[8], tensors2[8], L * C * 4*C, "fcprojw", grad_thresholds[8]);
            allok = allok & check_tensor(tensors1[9], tensors2[9], L * C, "fcprojb", grad_thresholds[9]);
            allok = allok & check_tensor(tensors1[10], tensors2[10], L * C, "ln1w", grad_thresholds[10]);
            allok = allok & check_tensor(tensors1[11], tensors2[11], L * C, "ln1b", grad_thresholds[11]);
            allok = allok & check_tensor(tensors1[12], tensors2[12], L * C, "ln2w", grad_thresholds[12]);
            allok = allok & check_tensor(tensors1[13], tensors2[13], L * C, "ln2b", grad_thresholds[13]);
            allok = allok & check_tensor(tensors1[14], tensors2[14], C, "lnfw", grad_thresholds[14]);
            allok = allok & check_tensor(tensors1[15], tensors2[15], C, "lnfb", grad_thresholds[15]);
        }

        float grad_norm = gpt2_calculate_grad_norm(&model, &multi_gpu_config);
        float grad_scale = (grad_norm > 1.0f) ? 1.0f / grad_norm : 1.0f;
        gpt2_update(&model, 1e-4f, 0.9f, 0.95f, 1e-8f, 0.0f, grad_scale, step+1, &multi_gpu_config);

        // print the timing information at the end
        printf("step %d: loss %f (took %f ms)\n", step+1, model.mean_loss, time_elapsed_s * 1000);
        // the expected losses from PyTorch were copied over after the print formatting rounded
        // them to 6 decimal places, so we do the same here
        float rounded_loss = roundf(model.mean_loss * 1000000) / 1000000;
        losses[step] = rounded_loss;
    }

    // expected losses are as follows, from Python
    float expected_losses[10] = {
        5.270009f,
        4.060681f,
        3.320085f,
        2.717550f,
        2.181066f,
        1.653923f,
        1.168050f,
        0.736873f,
        0.401021f,
        0.187493f
    };

    // compare
    for (int i = 0; i < 10; i++) {
        if (fabsf(losses[i] - expected_losses[i]) >= loss_diff_threshold) {
            printf("LOSS MISMATCH AT STEP %d: %f %f\n", i+1, losses[i], expected_losses[i]);
            allok = 0;
        } else {
            printf("loss ok at step %d: %f %f\n", i+1, losses[i], expected_losses[i]);
        }
    }

    // Finally, let's check determinism
    gpt2_write_to_checkpoint(&model, "test_gpt2cu_model.ckpt");

    DataLoader loader;
    dataloader_init(&loader, "dev/data/tinyshakespeare/tiny_shakespeare_val.bin", B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes, 1);
    save_state("test_gpt2cu_state.ckpt", 10, &model, &loader);
    int tokens[10];
    for (int step = 0; step < 10; step++) {
        dataloader_next_batch(&loader);
        gpt2_forward(&model, loader.inputs, B, T);
        gpt2_backward_and_reduce(&model, loader.inputs, loader.targets, 1, 0);
        gpt2_update(&model, 1e-4f, 0.9f, 0.95f, 1e-8f, 0.0f, 1.0f, step+11, &multi_gpu_config);
        losses[step] = model.mean_loss;
        tokens[step] = loader.inputs[0];
    }

    // reload
    gpt2_free(&model);
    gpt2_build_from_checkpoint(&model, "test_gpt2cu_model.ckpt");
    int ld_step;
    gpt2_allocate_state(&model, B, T);
    load_state(&ld_step, &model, &loader, "test_gpt2cu_state.ckpt");
    for (int step = 0; step < 10; step++) {
        dataloader_next_batch(&loader);
        gpt2_forward(&model, loader.inputs, B, T);
        gpt2_backward_and_reduce(&model, loader.inputs, loader.targets, 1, 0);
        gpt2_update(&model, 1e-4f, 0.9f, 0.95f, 1e-8f, 0.0f, 1.0f, step+11, &multi_gpu_config);

        if(loader.inputs[0] != tokens[step]) {
            printf("Nondeterminism! Token mismatch at step %d: %d vs %d\n", step, tokens[step], loader.inputs[0]);
            allok = false;
            break;
        }

        if(losses[step] != model.mean_loss) {
            printf("Nondeterminism! Loss mismatch at step %d: %.15f vs %.15f\n", step, losses[step], model.mean_loss);
            allok = false;
            break;
        } else {
            printf("loss ok at step %d: %f %f\n", step, losses[step], model.mean_loss);
        }
    }

    // final approval
    printf("overall okay: %d\n", allok);

    // delete intermediate test files
    remove("test_gpt2cu_model.ckpt");
    remove("test_gpt2cu_state.ckpt");

    // free everything
    dataloader_free(&loader);
    gpt2_free(&model);
    common_free(model);
    free(x);
    free(y);
    free(logits_cpu_raw);
    free(logits_cpu);
    free(expected_logits);
    free(expected_loss);
    free(expected_grads_memory);
    free(grads_memory_cpu);
    free(grads_memory_cpu_float);
    return allok ? EXIT_SUCCESS : EXIT_FAILURE;
}
