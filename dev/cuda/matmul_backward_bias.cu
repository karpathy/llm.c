#include "hip/hip_runtime.h"
/*
Kernels for matmul backward pass bias only.

Compile example:
nvcc -O3 -lcublas -lcublasLt -std=c++17 matmul_backward_bias.cu -lineinfo -o matmul_backward_bias

./matmul_backward_bias 1
........................
./matmul_backward_bias 10

101/102/104/108/116 correspond to kernel 10 with blockDim.x of 1/2/4/8/16

ncu:
sudo ncu --set full --import-source yes -o bias -f ./matmul_backward_bias 1
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <type_traits>

#define ENABLE_BF16
#include "common.h"


// ----------------------------------------------------------------------------
// utility functions
__host__ __device__ bool isPowerOfTwo(int n) {
    return (n > 0) && ((n & (n - 1)) == 0);
}

__host__ __device__ int largestPowerOfTwoLessOrEqual(int n) {
    // Return the largest power of 2 less than or equal to n
    if (n < 1) {
        return 0;
    }

    while ((n & (n - 1)) > 0) {
        n = n & (n - 1);
    }

    return n;
}

// ----------------------------------------------------------------------------
// CPU code reference

void matmul_backward_bias_cpu(float* dinp, float* dweight, float* dbias,
                     float* dout, float* inp, float* weight,
                     int B, int T, int C, int OC) {
    for (int o = 0; o < OC; o++) {
        double sum = 0.0;
        for (int b = 0; b < B; b++) {
            for (int t = 0; t < T; t++) {
                float* dout_bt = dout + b * T * OC + t * OC;
                sum += dout_bt[o];
            }
        }
        dbias[o] = sum;
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

float* dbias_buffer;

__global__ void matmul_backward_bias_kernel1(floatX* dbias, const floatX* dout, int B, int T, int OC) {
    extern __shared__ float shared[];
    int o = blockIdx.x; // range [0, OC)
    int tid = threadIdx.x; // range [0, block_size)
    int block_size = blockDim.x;
    const floatX* x = dout + o;
    // thread coarsening
    float sum = 0.0;
    for (int i = tid; i < B * T; i += block_size) {
        sum += (float)x[i * OC];
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        dbias[o] = (floatX)((float)dbias[o] + shared[0]);
    }
}

// cooperative groups solution, one warp per output channel
__global__ void matmul_backward_bias_kernel2(floatX* dbias, const floatX* dout, int B, int T, int OC) {
    // dout is (B, T, OC), dbias is (OC)
    // e.g. if block_size = 128, then we have 4 warps per block, each in charge of one output channel
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    // meta_group_size is the number of warps in a block (e.g. 4), meta_group_rank is the warp index (0,1,2,3)
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= OC) { return; }
    int BT = B * T; // number of elements to reduce in total, per channel
    // first, thread coarsening to sum reduce the problem size from B*T to 32
    float sum = 0.0f;
    for(int i = warp.thread_rank(); i < BT; i += warp.size()) {
        sum += (float)dout[i * OC + idx];
    }
    // now do a warp-level reduce to get the sum across the 32 threads in this warp
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    // write the result to output (global memory)
    if(warp.thread_rank() == 0) {
        dbias[idx] = (float)dbias[idx] + sum;
    }
}

__global__ void matmul_backward_bias_kernel3(floatX* dbias, const floatX* dout, int B, int T, int OC) {
    // dout is (B, T, OC), dbias is (OC)
    // in this version of the kernel the entire block of block_size is dedicated to one output channel
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float shared_sum[32]; // block_size max is 1024 = 32 * 32 warps
    int BT = B * T; // number of elements to reduce in total, per channel
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    int idx = blockIdx.x; // simply one block per row
    // round 1: thread coarsening to reduce the problem size from B*T to block_size
    float thread_sum = 0.0f;
    for(int i = threadIdx.x; i < BT; i += blockDim.x) {
        thread_sum += (float)dout[i * OC + idx];
    }
    // now do a warp-level reduce to get the sum across the 32 threads in each warp
    // reduce the problem size from block_size to block_size/32 i.e. `num_warps`
    float warp_sum = cg::reduce(warp, thread_sum, cg::plus<float>{});
    // store the warp sum in shared memory (we could have lane_id == 0 guard but not needed)
    shared_sum[warp_id] = warp_sum;
    __syncthreads();
    // load results from shared memory to threads, pad with zeros for threads that are out of bounds
    warp_sum = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
    // now reduce the warp-level reductions
    float block_sum = cg::reduce(warp, warp_sum, cg::plus<float>{}); // sum(x)
    // write the result to output (global memory)
    if(threadIdx.x == 0) {
        dbias[idx] = (float)dbias[idx] + block_sum;
    }
}

// this kernel performs a column-wise reduction over dout, in PyTorch equivalent to:
// dbias = dout.sum((0,1))
// the idea is to employ one block to reduce along several columns,
// where each block has a width of 32 columns to ensure coalesced access.
// at the end we accumulate the reductions performed by the warps in each block via shared memory
__global__ void matmul_backward_bias_kernel4(floatX* dbias, const floatX* dout, int B, int T, int OC) {
    // this kernel is launched with 1D grid_dim of OC/32
    // for example let's say block_size is 128
    extern __shared__ float smem[]; // of size block_size (128)
    const int warp_id = threadIdx.x / warpSize; // warp index in the block, 0,1,2,3
    const int lane_id = threadIdx.x % warpSize; // thread index in the warp, 0,1,2,...,31
    const int tl = blockIdx.x * warpSize; // pointer to the start column for this block
    const int vstep = blockDim.x / warpSize; // number of warps in a block, e.g. 4

    // pointer to the start of the column for one lane of threads
    // so e.g. 4 (`vstep`) threads (of the same lane_id) will reduce this one column
    const floatX* dout_col = dout + tl + lane_id;

    // column reductions by looping through the rows
    // each of the 4 threads offsets by its warp_id and then skips by vstep
    // together these 4 threads cover all B*T rows of this (lane_id) column
    // importantly, consecutive threads (in threadId) are processing adjacent columns,
    // leading to a coalesced memory access pattern
    float dout_sum = 0.0f;
    for (int row = warp_id; row < B * T; row += vstep) {
        dout_sum += (float)dout_col[row * OC];
    }
    smem[lane_id + warp_id * warpSize] = dout_sum;
    __syncthreads();

    // warp_id 0 reduces the shared memory column-wise, linearly
    dout_sum = 0.0f;
    if (warp_id == 0) {
        for (int j = 0; j < vstep; j++) {
            dout_sum += smem[lane_id + j * warpSize];
        }
        dbias[tl + lane_id] = (float)dbias[tl + lane_id] + dout_sum;
    }
}

#ifndef ENABLE_BF16
__global__ void matmul_backward_bias_kernel5(floatX* dbias, const floatX* dout, int B, int T, int OC) {
    int oc = blockIdx.x * blockDim.x + threadIdx.x;
    if(oc >= OC) return;
    float sum = 0.0;
    // grid-wide loop for maximum parallelism
    for (int i = blockIdx.y; i < B * T; i += gridDim.y) {
        sum += (float)dout[i * OC + oc];
    }
    // and atomically add everything together. atomics within one block are conflict-free!
    atomicAdd(dbias + oc, sum);
}
#endif


__global__ void cast_and_add_kernel(floatX* dst, const float* src, size_t n) {
    // used only for matmul_backward_bias kernel, a little bit embarassing TODO delete later
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) { dst[idx] = (floatX)((float)dst[idx] + src[idx]); } // have to += because dbias is a paramater
}

__global__ void matmul_backward_bias_kernel7(float* dbias, const floatX* dout, int B, int T, int OC, const int block_size) {
    // note: this kernel reads in floatX, but it writes to float!
    // this is because we're using atomics, which are super slow in < fp32 precision on < H100 GPUs
    // so the trick is do fp32 atomics to a buffer, and then copy_and_cast the result to floatX
    // (this also results in higher accuracy than doing accumulation directly in floatX)

    // see comments in matmul_backward() for an explanation of block/grid dimensions etc.
    const int block_size_x = 32;
    const int block_size_y = block_size / block_size_x; // 16
    const int OC_per_warp = block_size_x * x128::size;  // 256 at BF16

    int local_oc = threadIdx.x * x128::size;
    int global_oc = blockIdx.x * OC_per_warp + local_oc;
    float accumulators[x128::size];
    extern __shared__ float shared[];

    for (int k = 0; k < x128::size; k++) {
        accumulators[k] = 0.0f;
    }
    int thread_id = threadIdx.y * block_size_x + threadIdx.x;
    for (int idx = thread_id; idx < OC_per_warp; idx += block_size) {
        shared[idx] = 0.0f;
    }
    __syncthreads();
    if(global_oc < OC) {
        for (int idx = blockIdx.y*block_size_y + threadIdx.y; idx < B * T; idx += gridDim.y*block_size_y) {
            x128 packed_dout = load128(dout + global_oc + idx*OC);
            for (int k = 0; k < x128::size; k++) {
                accumulators[k] += (float)packed_dout[k];
            }
        }
        // we need to avoid shared memory bank conflicts for the atomicAdd to maximise performance,
        // so we accumulate in a conflict-free order, then reorder to match the global memory order
        for (int k = 0; k < x128::size; k++) {
            atomicAdd(shared + threadIdx.x + (k * block_size_x), accumulators[k]);
        }
    }
    if (threadIdx.y >= x128::size) { return; } // only need this many warps to reorder the data
    __syncthreads();
    // read the accumulated values in the conflict-free order
    int i = threadIdx.x + (threadIdx.y * block_size_x);
    float tmp = shared[i];
    __syncthreads();
    // write them back to shared memory in the global memory order
    // 8-way bank conflict for BF16 x128, but only 8x per threadblock (rather than 8x per warp)
    shared[local_oc + threadIdx.y] = tmp;
    __syncthreads();
    // now we do a perfectly coalesced atomic add to global memory (1x 128-byte cacheline per warp)
    if (i + blockIdx.x*OC_per_warp < OC) {
        atomicAdd(dbias + i + blockIdx.x*OC_per_warp, shared[i]);
    }
}

// We want to decrease the amount of channels handled by each block, so that we need fewer across-block reductions.
// We do this by realizing the following: For scalar memory access, we need to read one element per thread in a warp
// to read an entire cacheline, but for vectorized memory access, with 128 bit of data per thread, we only need eight
// threads to fetch a cacheline, which means that we can already operate on a "depth" of four within a single warp.
// => blockDim.x == 4, blockDim.y == 32/4 = 8
//
template<typename OutFloat, bool Atomic>
__global__ void matmul_backward_bias_kernel8(OutFloat* dbias, const floatX* dout, int B, int T, int OC,
                                             std::bool_constant<Atomic>) {
    constexpr const int bdx = 4;
    constexpr const int bdy = 32 / bdx;
    assert(blockDim.x == bdx);
    assert(blockDim.y == bdy);

    int warp_d = (int)threadIdx.x;
    int warp_c = (int)threadIdx.y;
    int block_d = (int)threadIdx.z;

    const int OC_per_warp = bdy * x128::size;  // 64 at BF16

    int local_oc = warp_c * x128::size;
    int global_oc = blockIdx.x * OC_per_warp + local_oc;

    int local_bt = warp_d + bdx * block_d;
    int bt_per_block = bdx * blockDim.z;

    float accumulators[x128::size];
    for (int k = 0; k < x128::size; k++) {
        accumulators[k] = 0.0f;
    }

    if(global_oc < OC) {
        // sum up over all bt within registers
        for (int idx = blockIdx.y * bt_per_block + local_bt; idx < B * T; idx += gridDim.y * bt_per_block) {
            x128 packed_dout = load128(dout + global_oc + idx*OC);
            for (int k = 0; k < x128::size; k++) {
                accumulators[k] += (float)packed_dout[k];
            }
        }
    }

    __shared__ float sub_results[x128::size][32][bdy];

    // reduce within-warp results
    for (int k = 0; k < x128::size; k++) {
        float v = accumulators[k];
        v += __shfl_down_sync(0xffffffff, v, 1, 4);
        v += __shfl_down_sync(0xffffffff, v, 2, 4);
        if(warp_d == 0) {
            sub_results[k][block_d][warp_c] = v;
        }
    }
    __syncthreads();

    // block-wide reductions
    for (int k = block_d; k < x128::size; k += blockDim.z) {
        float a = 0.f;
        for (int r = warp_d; r < blockDim.z; r += bdx) {
            float v = sub_results[k][r][warp_c];
            v += __shfl_down_sync(0xffffffff, v, 1, 4);
            v += __shfl_down_sync(0xffffffff, v, 2, 4);
            a += v;
        }
        if(warp_d == 0 && global_oc < OC) {
            // coalesced, but not cacheline-sized
            if constexpr (!Atomic) {
                dbias[global_oc + k] = (OutFloat)(a + (float)dbias[global_oc + k]);
            } else {
                atomicAdd(dbias + global_oc + k, a);
            }
        }
    }
}

// Like kernel 8, but instead of accumulating to the auxiliary buffer, it writes
// multiple values that need to be summed up in a separate kernel call.
// If UseAuxBuffer is false, gridDim.y has to be one, and results are added directly
// to dbias.
template<typename OutFloat, bool UseAuxBuffer>
__global__ void matmul_backward_bias_kernel9(OutFloat* dbias, const floatX* dout, int B, int T, int OC,
                                             std::bool_constant<UseAuxBuffer>) {
    constexpr const int bdx = 4;
    constexpr const int bdy = 32 / bdx;
    assert(blockDim.x == bdx);
    assert(blockDim.y == bdy);

    int warp_d = (int)threadIdx.x;
    int warp_c = (int)threadIdx.y;
    int block_d = (int)threadIdx.z;

    const int OC_per_warp = bdy * x128::size;  // 64 at BF16

    int local_oc = warp_c * x128::size;
    int global_oc = blockIdx.x * OC_per_warp + local_oc;

    int local_bt = warp_d + bdx * block_d;
    int bt_per_block = bdx * blockDim.z;

    float accumulators[x128::size];
    for (int k = 0; k < x128::size; k++) {
        accumulators[k] = 0.0f;
    }

    if(global_oc < OC) {
        // sum up over all bt within registers
        for (int idx = blockIdx.y * bt_per_block + local_bt; idx < B * T; idx += gridDim.y * bt_per_block) {
            x128 packed_dout = load128(dout + global_oc + idx*OC);
            for (int k = 0; k < x128::size; k++) {
                accumulators[k] += (float)packed_dout[k];
            }
        }
    }

    __shared__ float sub_results[x128::size][32][bdy];

    // reduce within-warp results
    for (int k = 0; k < x128::size; k++) {
        float v = accumulators[k];
        v += __shfl_down_sync(0xffffffff, v, 1, 4);
        v += __shfl_down_sync(0xffffffff, v, 2, 4);
        if(warp_d == 0) {
            sub_results[k][block_d][warp_c] = v;
        }
    }
    __syncthreads();

    // block-wide reductions
    for (int k = block_d; k < x128::size; k += blockDim.z) {
        float a = 0.f;
        for (int r = warp_d; r < blockDim.z; r += bdx) {
            float v = sub_results[k][r][warp_c];
            v += __shfl_down_sync(0xffffffff, v, 1, 4);
            v += __shfl_down_sync(0xffffffff, v, 2, 4);
            a += v;
        }
        if(warp_d == 0 && global_oc < OC) {
            // coalesced, but not cacheline-sized
            if constexpr (!UseAuxBuffer) {
                dbias[global_oc + k] = (OutFloat)(a + (float)dbias[global_oc + k]);
            } else {
                dbias[global_oc + k + blockIdx.y * OC] = a;
            }
        }
    }
}


__global__ void reduce_add_sum_kernel(floatX* dst, const float* src, size_t n, size_t m) {
    const size_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * f128::size;
    assert(n % x128::size == 0);
    if (idx < n) {
        f128 acc;
        for(int k = 0; k < f128::size; ++k) {
            acc[k] = 0.f;
        }

        for(int l = 0; l < m; ++l) {
            f128 s = load128(src + idx + n * l);
            for(int k = 0; k < f128::size; ++k) {
                acc[k] += s[k];
            }
        }
        for(int k = 0; k < f128::size; ++k) {
            dst[idx + k] = (floatX) ((float)dst[idx + k] + acc[k]);
        }
    }
}

// (extra-long explanation for /dev/cuda/ only)
//
// (in the description below, we assume "column major" layout like cuBLAS/FORTRAN)
// this is really a general purpose column reduction kernel, it's not just for bias!
// it could easily be generalised if we needed this for something else in the future...
//
// this kernel performs a column-wise reduction over dout, in PyTorch equivalent to:
// dbias = dout.sum((0,1))
// we want our memory loads to be coalesced, but the data we need is not contiguous.
// the solution is to employ one block to reduce along several columns in parallel,
// each block is responsible for (blockIdx.x * x128::size) columns.
//
// NVIDIA has 32B L1 cache sectors, 64B DRAM accesses, and 128B L1/L2 cachelines.
// with x128, we load 16B per thread, so we need blockIdx.x to be between 2 and 8.
// i.e. we process 16 to 64 columns per block with BF16.
//
// we also process blockIdx.y rows in parallel to maximise GPU utilization,
// we need to do this rather than just loop across BT in every thread because
// OC can be as low as 768 for some ops on GPT2 124M, which would starve the GPU!
// e.g. 768/16 = 48 (for BF16 and blockIdx.x=2) so only 48 of 132 SMs active on H100
//
// at the end we combine the reductions of these rows using shared memory.
// with a block size of 1024, that means e.g. blockIdx.x=8 and blockIdx.y=128
// ==> 64 columns (for BF16) and 128 rows processed in parallel in a single block
//
// in total, there are OC columns and B*T rows to process, so we need to launch
// "OC / (blockIdx.x * x128::size)" blocks per grid.
// we try to adjust blockIdx.x according to some heuristics in the launcher function.
// block dimensions are given at compile time via templating to enable loop unrolling
//
// (end of extra-long explanation for /dev/cuda/ only)
//
// (in the description below, we assume "column major" layout like cuBLAS/FORTRAN but unlike C/C++)
// general reduction kernel for any minor axis, could be used for other things than bias backward!
// bias backward: OC columns and B*T rows ==> per-column sum reduction with OC outputs in dbias
// each block handles (blockIdx.x * x128::size) columns and (blockIdx.y) rows
// data layout is column major => contiguous for column X and X+1 (row_stride elements across rows)
// ==> 128B coalesced loads with BF16 require blockIdx.x >= 8 (64 columns per block)
// with few columns (OC), we want smaller blockIdx.x to get more blocks and better GPU utilisation
template <int block_dim_x=2, int block_dim_y=512, bool accumulate=true, typename OutFloat=floatX>
__global__ void column_reduction_kernel(OutFloat* output, const floatX* input,
                                        int num_rows, int num_columns, int row_stride) {
    assert(block_dim_x == blockDim.x && block_dim_y == blockDim.y); // check template parameters
    assert(num_columns == gridDim.x * block_dim_x * x128::size); // must match, no partial blocks
    constexpr int block_size = block_dim_x * block_dim_y;
    __shared__ float smem[block_size * x128::size];

    float column_sum[x128::size] = {0.0f}; // per-thread (partial column) FP32 accumulator
    int column_idx = (blockIdx.x * block_dim_x + threadIdx.x) * x128::size;
    int smem_idx = threadIdx.x + threadIdx.y * block_dim_x; // smem idx for this thread with k=0

    #pragma unroll 4
    for (int row = threadIdx.y; row < num_rows; row += block_dim_y) {
        x128 packed_dout = load128(input + column_idx + row * row_stride);
        for (int k = 0; k < x128::size; k++) {
            column_sum[k] += (float)packed_dout[k];
        }
    }
    // todo - currently don't use f128 for smem, so we stride by block_size to avoid bank conflicts
    for (int k = 0; k < x128::size; k++) {
        smem[smem_idx + k * block_size] = column_sum[k]; // write column partial sums to shared mem
    }

    // blockDim.y threads are all processing the same column, so we need to add up their sums
    // i.e. we calculate (blockDim.x * x128::size) final sums in parallel (one per column)
    // so with blockDim.x = 8, we avoid the parts of the reduction with only 1/2/4 active threads
    for (int stride = block_size/2; stride >= block_dim_x; stride /= 2) {
        __syncthreads();
        if (threadIdx.y * block_dim_x < stride) {
            for (int k = 0; k < x128::size; k++) {
                int smem_idx_k = smem_idx + k * block_size;
                smem[smem_idx_k] = smem[smem_idx_k] + smem[smem_idx_k + stride];
            }
        }
    } // no __syncthreads() needed because smem read below was written by the same thread

    if (threadIdx.y == 0) {
        // accumulate if necessary (e.g. gradient accumulation for multiple micro-batches per batch)
        // one output per column (e.g. 1 bias parameter gradient per OC)
        x128 output128 = accumulate ? load128(output + column_idx) : x128::zeros();
        for (int k = 0; k < x128::size; k++) {
            output128[k] = (OutFloat)((float)output128[k] + smem[threadIdx.x + k * block_size]);
        }
        store128(output + column_idx, output128);
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

// version1: simple cuBLAS calls
void matmul_backward_bias1(floatX* dbias, const floatX* dout,
                      int B, int T, int OC, int block_size) {
    block_size = largestPowerOfTwoLessOrEqual(block_size);
    assert(isPowerOfTwo(block_size)); // block_size needs to be power of 2 due to the reduction
    dim3 block_dim(block_size);
    dim3 grid_dim(OC);
    size_t shared_mem_size = block_size * sizeof(float);
    matmul_backward_bias_kernel1<<<grid_dim, block_dim, shared_mem_size>>>(dbias, dout, B, T, OC);
    cudaCheck(hipGetLastError());
}

void matmul_backward_bias2(floatX* dbias, const floatX* dout,
                      int B, int T, int OC, int block_size) {
    // block_size 512 seems best
    const int grid_size = ceil_div(OC * 32, block_size);
    matmul_backward_bias_kernel2<<<grid_size, block_size>>>(dbias, dout, B, T, OC);
    cudaCheck(hipGetLastError());
}

void matmul_backward_bias3(floatX* dbias, const floatX* dout,
                      int B, int T, int OC, int block_size) {
    // block_size 256 seems best
    matmul_backward_bias_kernel3<<<OC, block_size>>>(dbias, dout, B, T, OC);
    cudaCheck(hipGetLastError());
}

void matmul_backward_bias4(floatX* dbias, const floatX* dout,
                      int B, int T, int OC, int block_size) {
    assert(OC % 32 == 0); // OC must be divisible by 32 for this kernel
    const int grid_size = OC / 32;
    matmul_backward_bias_kernel4<<<grid_size, block_size, block_size * sizeof(float)>>>(dbias, dout, B, T, OC);
    cudaCheck(hipGetLastError());
}

#ifndef ENABLE_BF16
void matmul_backward_bias5(floatX* dbias, const floatX* dout,
                      int B, int T, int OC, int block_size) {
    const int grid_size_x = ceil_div(OC, block_size);
    const int grid_size_y = max(1, cuda_threads_per_SM * cuda_num_SMs / block_size);
    matmul_backward_bias_kernel5<<<dim3(grid_size_x, grid_size_y), dim3(block_size)>>>(dbias, dout, B, T, OC);
    cudaCheck(hipGetLastError());
}
#endif

void matmul_backward_bias7(floatX* dbias, const floatX* dout,
                      int B, int T, int OC, int block_size) {
    if(block_size < 256) {
        block_size = 256;
    }
    // Each warp is responsible for 32 * "x128::size" = 256 OCs at BF16 (OC must be a multiple of 256!)
    // Block size is 512 threads (16 warps) and we reduce those 16 values into 1 at the end
    // blockDim.x is 32 --> single warp being responsible for those 256 OCs
    // blockDim.y is 16 --> 16 parallel independent warps processing the same OCs for different BTs
    // gridDim.x is OC / 256 --> each block processes 256 OCs
    // grimDim.y is max(1, (cuda_num_SMs * threads_per_SM) / (512 * gridDim.x)); --> fill up the entire GPU!
    const int warp_size = 32;
    const int OC_per_warp = warp_size * x128::size; // 256 at BF16
    const int block_size_x = 32;
    const int block_size_y = block_size / block_size_x; // 16
    const int grid_size_x = ceil_div(OC, OC_per_warp); // e.g. 3 horizontal blocks for 768 OCs at BF16
    const int grid_size_y = max(1, cuda_threads_per_SM * cuda_num_SMs / (block_size * grid_size_x)); // full GPU!

    assert(block_size_y >= x128::size); // part of the kernel assumes this is large enough to avoid loops

    cudaCheck(hipMemset(dbias_buffer, 0, OC * sizeof(float)));
    matmul_backward_bias_kernel7<<<dim3(grid_size_x, grid_size_y),
        dim3(block_size_x, block_size_y), OC_per_warp * sizeof(float)>>>(dbias_buffer, dout, B, T, OC, block_size);
    cudaCheck(hipGetLastError());
    cast_and_add_kernel<<<ceil_div(OC, 256), 256, 0>>>(dbias, dbias_buffer, OC);
    cudaCheck(hipGetLastError());
}

void matmul_backward_bias8(floatX* dbias, const floatX* dout,
                      int B, int T, int OC, int block_size) {
    dim3 block_dim = {4, 8, (unsigned)block_size/32};
    const int OC_per_warp = block_dim.y * x128::size; // 64 at BF16
    const int grid_size_x = ceil_div(OC, OC_per_warp); // e.g. 12 horizontal blocks for 768 OCs at BF16
    const int grid_size_y = max(1, cuda_threads_per_SM * cuda_num_SMs / (block_size * grid_size_x)); // full GPU!

    // If we have enough OC that we don't need cross-block reductions, we can skip the bias_buffer accumulation
    // and write results directly to the output.
    if(grid_size_y == 1) {
        matmul_backward_bias_kernel8<<<dim3(grid_size_x, grid_size_y), block_dim>>>(dbias, dout, B, T, OC, std::bool_constant<false>{});
        cudaCheck(hipGetLastError());
    } else {
        cudaCheck(hipMemset(dbias_buffer, 0, OC * sizeof(float)));
        matmul_backward_bias_kernel8<<<dim3(grid_size_x, grid_size_y), block_dim>>>(dbias_buffer, dout, B, T, OC, std::bool_constant<true>{});
        cudaCheck(hipGetLastError());
        cast_and_add_kernel<<<ceil_div(OC, 256), 256, 0>>>(dbias, dbias_buffer, OC);
        cudaCheck(hipGetLastError());
    }
}


void matmul_backward_bias9(floatX* dbias, const floatX* dout,
                           int B, int T, int OC, int block_size) {
    dim3 block_dim = {4, 8, (unsigned)block_size/32};
    const int OC_per_warp = block_dim.y * x128::size; // 64 at BF16
    const int grid_size_x = ceil_div(OC, OC_per_warp); // e.g. 12 horizontal blocks for 768 OCs at BF16
    const int grid_size_y = max(1, cuda_threads_per_SM * cuda_num_SMs / (block_size * grid_size_x)); // full GPU!

    // If we have enough OC that we don't need cross-block reductions, we can skip the bias_buffer accumulation
    // and write results directly to the output.
    if(grid_size_y == 1) {
        matmul_backward_bias_kernel9<<<dim3(grid_size_x, grid_size_y), block_dim>>>(dbias, dout, B, T, OC, std::bool_constant<false>{});
        cudaCheck(hipGetLastError());
    } else {
        // kernel 9 overwrites temp buffer, so no need to memset
        matmul_backward_bias_kernel9<<<dim3(grid_size_x, grid_size_y), block_dim>>>(dbias_buffer, dout, B, T, OC, std::bool_constant<true>{});
        cudaCheck(hipGetLastError());
        reduce_add_sum_kernel<<<ceil_div(OC, 256 * f128::size), 256, 0>>>(dbias, dbias_buffer, OC, grid_size_y);
        cudaCheck(hipGetLastError());
    }
}

template <int bs_x=2> // aka block_size.x
void matmul_backward_bias10(floatX* dbias, const floatX* dout,
                            int B, int T, int OC, int desired_block_size) {
    const dim3 grid_size = dim3(OC / (bs_x * x128::size)); // block_size.x columns are processed per block
    assert(OC % (bs_x * x128::size) == 0); // should always be true for sensible values of block_size.x

    switch (desired_block_size) {
        case 32: column_reduction_kernel<bs_x, 32/bs_x><<<grid_size, dim3(bs_x, 32/bs_x)>>>(dbias, dout, B*T, OC, OC); break;
        case 64: column_reduction_kernel<bs_x, 64/bs_x><<<grid_size, dim3(bs_x, 64/bs_x)>>>(dbias, dout, B*T, OC, OC); break;
        case 128: column_reduction_kernel<bs_x, 128/bs_x><<<grid_size, dim3(bs_x, 128/bs_x)>>>(dbias, dout, B*T, OC, OC); break;
        case 256: column_reduction_kernel<bs_x, 256/bs_x><<<grid_size, dim3(bs_x, 256/bs_x)>>>(dbias, dout, B*T, OC, OC); break;
        case 512: column_reduction_kernel<bs_x, 512/bs_x><<<grid_size, dim3(bs_x, 512/bs_x)>>>(dbias, dout, B*T, OC, OC); break;
        case 768: column_reduction_kernel<bs_x, 768/bs_x><<<grid_size, dim3(bs_x, 768/bs_x)>>>(dbias, dout, B*T, OC, OC); break;
        case 1024: column_reduction_kernel<bs_x, 1024/bs_x><<<grid_size, dim3(bs_x, 1024/bs_x)>>>(dbias, dout, B*T, OC, OC); break;
        default: break;
    }
    cudaCheck(hipGetLastError());
}

void matmul_backward_bias(int kernel_num, floatX* dbias, floatX* dout,
                     int B, int T, int OC, int block_size) {
    switch (kernel_num) {
        case 1:
            matmul_backward_bias1(dbias, dout, B, T, OC, block_size);
            break;
        case 2:
            matmul_backward_bias2(dbias, dout, B, T, OC, block_size);
            break;
        case 3:
            matmul_backward_bias3(dbias, dout,  B, T, OC, block_size);
            break;
        case 4:
            matmul_backward_bias4(dbias, dout, B, T, OC, block_size);
            break;
        case 5:
#ifndef ENABLE_BF16
            matmul_backward_bias5(dbias, dout, B, T, OC, block_size);
#else
            fprintf(stderr, "Kernel 5 is only supported for fp32");
            exit(1);
#endif
            break;
        case 7:
            matmul_backward_bias7(dbias, dout, B, T, OC, block_size);
            break;
        case 8:
            matmul_backward_bias8(dbias, dout, B, T, OC, block_size);
            break;
        case 9:
            matmul_backward_bias9(dbias, dout, B, T, OC, block_size);
            break;
        case 10:
            {
                // same heuristic as in llm.c for blockDim.x:
                // 1 block per SM and blockIdx.x=2 ==> need (2*2*x128::size) columns per SM ==> 16 at BF16
                // 768/16 ==> 48 SMs (out of 132 on H100) active for small bias kernels on 124M GPT2 models
                // 3072/16 ==> 192 which is good but 96 with blockIdx.x=4 is faster due to better coalescing
                // ===>
                // Set block_size_x = 8. If we get less than 0.5 or 0.25 blocks per SM, reduce to 4 or 2.
                int block_size_x = 8;
                int total_blocks = OC / (block_size_x * x128::size);
                if (total_blocks <= cuda_num_SMs / 4) { block_size_x = 2, total_blocks *= 4; }
                else if (total_blocks <= cuda_num_SMs / 2) { block_size_x = 4, total_blocks *= 2; }
                assert(OC == total_blocks * block_size_x * x128::size);

                switch (block_size_x) {
                    case 2: matmul_backward_bias10<2>(dbias, dout, B, T, OC, block_size); break;
                    case 4: matmul_backward_bias10<4>(dbias, dout, B, T, OC, block_size); break;
                    case 8: matmul_backward_bias10<8>(dbias, dout, B, T, OC, block_size); break;
                    default: break;
                }
            }
            break;
            // 101 to 116: kernel 10 but with specific forced blockDim.x values
        case 101:
            matmul_backward_bias10<1>(dbias, dout, B, T, OC, block_size);
            break;
        case 102:
            matmul_backward_bias10<2>(dbias, dout, B, T, OC, block_size);
            break;
        case 104:
            matmul_backward_bias10<4>(dbias, dout, B, T, OC, block_size);
            break;
        case 108:
            matmul_backward_bias10<8>(dbias, dout, B, T, OC, block_size);
            break;
        case 116:
            matmul_backward_bias10<16>(dbias, dout, B, T, OC, block_size);
            break;

        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    setup_main();

    int B = 4;
    int T = 1024;
    int C = 768;
    int OC = 768 * 4; // expansion of 4, e.g. in the MLP

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // create host memory of random numbers
    float* dbias = make_zeros_float(OC);
    float* dout = make_random_float(B * T * OC);

    // move to GPU
    floatX* d_dbias;
    floatX* d_dout;
    cudaCheck(hipMalloc(&d_dbias, OC * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_dout, B * T * OC * sizeof(floatX)));
    cudaCheck(hipMalloc(&dbias_buffer, OC * sizeof(float) * 32));
    cudaCheck(memcpy_convert(d_dbias, dbias, OC));
    cudaCheck(memcpy_convert(d_dout, dout, B * T * OC));

    // ncu debugging / profiling, do a single call
    // int block_size_debug;
    // if (kernel_num == 1) { block_size_debug = 512;
    // } else if (kernel_num == 2) { block_size_debug = 512;
    // } else { block_size_debug = 256; }
    // printf("kernel %d, block_size %d\n", kernel_num, block_size_debug);
    // matmul_backward_bias(kernel_num, NULL, NULL, d_dbias, d_dout, NULL, NULL, NULL, B, T, C, OC, block_size_debug);
    // exit(EXIT_SUCCESS);

    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    // calculate the CPU reference
    matmul_backward_bias_cpu(NULL, NULL, dbias, dout, NULL, NULL, B, T, C, OC);

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        // memset the bias to zero
        cudaCheck(hipMemset(d_dbias, 0, OC * sizeof(floatX)));
        // calculate the GPU version
        matmul_backward_bias(kernel_num, d_dbias, d_dout, B, T, OC, block_size);
        // compare
        printf("Checking correctness...\n");
        float tol = std::is_same_v<floatX, float> ? 5e-3f : 1.0f;
        validate_result(d_dbias, dbias, "dbias", OC, tol);
        printf("All results match for block_size=%d.\n\n", block_size);
    }

    // now benchmark the kernel
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 2000;
        float elapsed_time = benchmark_kernel(repeat_times, matmul_backward_bias, kernel_num,
                                            d_dbias, d_dout, B, T, OC, block_size);
        printf("block_size %d time %.4f ms\n", block_size, elapsed_time);
    }

    // cleanups
    free(dbias);
    free(dout);
    cudaCheck(hipFree(dbias_buffer));
    cudaCheck(hipFree(d_dbias));
    cudaCheck(hipFree(d_dout));

    return 0;
}