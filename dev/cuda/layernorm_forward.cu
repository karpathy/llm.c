#include "hip/hip_runtime.h"
/*
Kernels for layernorm forward pass.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt layernorm_forward.cu -o layernorm_forward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
./layernorm_forward 1

version 2 parallelizes over all of B,T,C
./layernorm_forward 2

version 3 uses cooperative groups to parallelize over all of B,T,C
./layernorm_forward 3

version 4 uses a more clever way to estimate variance, var(x) = mean(x**2) - mean(x)**2
          (allowing us to do a single pass over x on load)
./layernorm_forward 4

verstion 5 allocates blocks per row instead of warps per row, same alg as 4 otherwise
./layernorm_forward 5
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"
// ----------------------------------------------------------------------------
// CPU code reference

// GPT-2 layernorm forward pass
void layernorm_forward_cpu(float* out, float* mean, float* rstd,
                       const float* inp, const float* weight, const float* bias,
                       int B, int T, int C) {
    float eps = 1e-5f;
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b,t,:]
            const float* x = inp + b * T * C + t * C;
            // calculate the mean
            float m = 0.0f;
            for (int i = 0; i < C; i++) {
                m += x[i];
            }
            m = m/C;
            // calculate the variance (without any bias correction)
            float v = 0.0f;
            for (int i = 0; i < C; i++) {
                float xshift = x[i] - m;
                v += xshift * xshift;
            }
            v = v/C;
            // calculate the rstd
            float s = 1.0f / sqrtf(v + eps);
            // seek to the output position in out[b,t,:]
            float* out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = (s * (x[i] - m)); // normalized output
                float o = n * weight[i] + bias[i]; // scale and shift it
                out_bt[i] = o; // write
            }
            // cache the mean and rstd for the backward pass later
            mean[b * T + t] = m;
            rstd[b * T + t] = s;
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// naive drag and drop implementation into kernel, parallelize over B,T, loop over C
__global__ void layernorm_forward_kernel1(float* out, float* mean, float* rstd,
                                 const float* inp, const float* weight, const float* bias,
                                 int N, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float eps = 1e-5f;

    if (idx < N) {
        // seek to the input position inp[idx,:]
        const float* x = inp + idx * C;
        // calculate the mean
        float m = 0.0f;
        for (int i = 0; i < C; i++) {
            m += x[i];
        }
        m = m / C;
        // calculate the variance (without any bias correction)
        float v = 0.0f;
        for (int i = 0; i < C; i++) {
            float xshift = x[i] - m;
            v += xshift * xshift;
        }
        v = v / C;
        // calculate the rstd
        float s = 1.0f / sqrtf(v + eps);
        // seek to the output position in out[idx,:]
        float* out_idx = out + idx * C;
        for (int i = 0; i < C; i++) {
            float n = (s * (x[i] - m)); // normalized output
            float o = n * weight[i] + bias[i]; // scale and shift it
            out_idx[i] = o; // write
        }
        // cache the mean and rstd for the backward pass later
        mean[idx] = m;
        rstd[idx] = s;
    }
}

__global__ void mean_kernel(float* mean, const float* inp, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    const float* x = inp + idx * C;
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        sum += x[i];
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        mean[idx] = shared[0] / C;
    }
}

__global__ void rstd_kernel(float* rstd, const float* inp, const float* mean, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    const float* x = inp + idx * C;
    float m = mean[idx];
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        rstd[idx] = 1.0f / sqrtf(shared[0] / C + 1e-5f);
    }
}

__global__ void normalization_kernel(float* out, const float* inp, float* mean, float* rstd,
                                     const float* weight, const float* bias, int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int bt = idx / C;
    int c = idx % C;

    float m = mean[bt];
    float s = rstd[bt];
    float xi = inp[idx];
    float n = s * (xi - m);
    float o = n * weight[c] + bias[c];

    out[idx] = o;
}

__global__ void layernorm_forward_kernel3(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    // meta_group_size is the number of warps in a block, and meta_group_rank is the warp index
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        sum += x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float m = sum / C;
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }

    // rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float s = rsqrtf(sum / C + 1e-5f);
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }

    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * (__ldcs(x+c) - m);
        __stcs(o+c, n * weight[c] + bias[c]);
    }
}

// same as kernel 3 but uses var(x) == mean(x**2) - mean(x)**2
__global__ void layernorm_forward_kernel4(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;

    // thread coarsening through the row, reduce the sum in series
    float sum = 0.0; // stores sum(x)
    float sum2 = 0.0; // stores sum(x**2)
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float xi = x[i];
        sum += xi;
        sum2 += xi * xi;
    }
    // warp-level reduction at the end
    sum = cg::reduce(warp, sum, cg::plus<float>{}); // sum(x)
    sum2 = cg::reduce(warp, sum2, cg::plus<float>{}); // sum(x**2)
    sum /= C; // mean(x)
    sum2 /= C; // mean(x**2)

    // mean, var, rstd
    float m = sum;
    float var = sum2 - sum * sum;
    float s = rsqrtf(var + 1e-5f);

    // store the mean, no need to cache it
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }
    // store the rstd, no need to cache it
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }
    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        float n = s * (__ldcs(x+c) - m);
        __stcs(o+c, n * weight[c] + bias[c]);
    }
}

// like 4, but in kernel 5 we have each block doing one row, not just a single warp
__global__ void layernorm_forward_kernel5(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float shared_sum[32]; // block_size max is 1024 = 32 * 32 warps
    __shared__ float shared_sum2[32]; // warps will be writing into shared memeory after warp-reduce
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    int idx = blockIdx.x; // simply one block per row
    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;
    // thread coarsening through the row, reduce the sum in series
    float thread_sum = 0.0; // stores sum(x)
    float thread_sum2 = 0.0; // stores sum(x**2)
    // for (int i = C + threadIdx.x - blockDim.x; i >= 0; i -= blockDim.x) {
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float xi = x[i];
        thread_sum += xi;
        thread_sum2 += xi * xi;
    }
    // warp-level reduction
    float warp_sum = cg::reduce(warp, thread_sum, cg::plus<float>{}); // sum(x)
    float warp_sum2 = cg::reduce(warp, thread_sum2, cg::plus<float>{}); // sum(x**2)
    // store the warp-level reduction in shared memory (we could have lane_id == 0 guard but not needed)
    shared_sum[warp_id] = warp_sum;
    shared_sum2[warp_id] = warp_sum2;
    __syncthreads();
    // load results from shared memory to threads, pad with zeros for threads that are out of bounds
    warp_sum = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
    warp_sum2 = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;
    // now reduce the warp-level reductions
    float block_sum = cg::reduce(warp, warp_sum, cg::plus<float>{}); // sum(x)
    float block_sum2 = cg::reduce(warp, warp_sum2, cg::plus<float>{}); // sum(x**2)
    // mean, var, rstd
    block_sum /= C; // mean(x)
    block_sum2 /= C; // mean(x**2)
    float m = block_sum;
    float var = block_sum2 - m * m;
    float s = rsqrtf(var + 1e-5f);
    // store the mean, no need to cache it
    if(threadIdx.x == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }
    // store the rstd, no need to cache it
    if(threadIdx.x == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }
    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float n = s * (__ldcs(x+i) - m);
        __stcs(o+i, n * weight[i] + bias[i]);
    }
}

// Inspired by `fused_residual_forward_kernel5` in fused_residual_forward.cu
__global__ void layernorm_forward_kernel6(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    assert(blockDim.x == WARP_SIZE);

    // load weights and biases into shared memory
    // do this before we allow any threads to exit!
    extern __shared__ char params[];
    // load128/store128 sometimes generated multiple instructions when the types here were floatX*, so
    // let's keep everything as x128
    x128* s_weight = reinterpret_cast<x128*>(params);
    x128* s_bias = reinterpret_cast<x128*>(params) + (C / x128::size);
    x128* s_in = reinterpret_cast<x128*>(params) + ((2 + threadIdx.y) * C / x128::size);

    int sidx = (threadIdx.x + WARP_SIZE * threadIdx.y) * x128::size;
    for(int i = sidx; i < C; i += blockDim.y * WARP_SIZE * x128::size) {
        s_weight[i/x128::size] = load128(weight + i);
        s_bias[i/x128::size] = load128(bias + i);
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    if(idx >= N) { return; } // guard

    // adjust pointers to current token
    inp += idx * C;
    out += idx * C;

    const float eps = 1e-5f;
    float sum = 0.0f;
    float sum2 = 0.0f;
    for(int c = threadIdx.x * x128::size; c < C; c += WARP_SIZE * x128::size) {
        const x128 in_data = load128cs(inp + c);
        for(int k = 0; k < x128::size; ++k) {
            float xi = (float)in_data[k];
            sum += xi;
            sum2 += xi * xi;
        }
        s_in[c / x128::size] = in_data;
    }

    sum = warpReduceSum(sum);
    sum2 = warpReduceSum(sum2);
    sum /= C;
    sum2 /= C;
    float m = sum;
    float v = sum2 - sum * sum;
    float s = rsqrtf(v + eps);

    for(int c = threadIdx.x * x128::size; c < C; c += WARP_SIZE * x128::size) {
        const x128 in_data = s_in[c / x128::size];
        const x128 w = s_weight[c / x128::size];
        const x128 b = s_bias[c / x128::size];
        x128 out_data;
        for(int k = 0; k < x128::size; ++k) {
            float n = s * ((float)in_data[k] - m); // normalized output
            float o = n * (float)w[k] + (float)b[k]; // scale and shift it
            out_data[k] = o;
        }

        store128cs(out + c, out_data);
    }
    // cache the mean and rstd for the backward pass later
    if(threadIdx.x == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }
    // store the rstd, no need to cache it
    if(threadIdx.x == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void layernorm_forward1(float* out, float* mean, float* rstd,
                           const float* inp, const float* weight, const float* bias,
                           int B, int T, int C,
                           const int block_size) {
    const int N = B * T;
    const int grid_size = ceil_div(N, block_size);
    layernorm_forward_kernel1<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward2(float* out, float* mean, float* rstd,
                       const float* inp, const float* weight, const float* bias,
                       int B, int T, int C,
                       const int block_size) {
    int N = B * T;
    // in mean and rstd, threads cooperate within blocks via reductions
    mean_kernel<<<N, block_size, block_size * sizeof(float)>>>(mean, inp, N, C, block_size);
    cudaCheck(hipGetLastError());
    rstd_kernel<<<N, block_size, block_size * sizeof(float)>>>(rstd, inp, mean, N, C, block_size);
    cudaCheck(hipGetLastError());
    // in the normalization, everything just gets flattened out
    const int block_size2 = 256;
    const int grid_size = ceil_div(B * T * C, block_size2);
    normalization_kernel<<<grid_size, block_size2>>>(out, inp, mean, rstd, weight, bias, B, T, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward3(float* out, float* mean, float* rstd,
                       const float* inp, const float* weight, const float* bias,
                       int B, int T, int C,
                       const int block_size) {
    assert(block_size % 32 == 0);
    const int N = B * T;
    const int grid_size = ceil_div(N * 32, block_size);
    layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward4(float* out, float* mean, float* rstd,
                       const float* inp, const float* weight, const float* bias,
                       int B, int T, int C,
                       const int block_size) {
    assert(block_size % 32 == 0);
    const int N = B * T;
    const int grid_size = ceil_div(N * 32, block_size);
    layernorm_forward_kernel4<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward5(float* out, float* mean, float* rstd,
                       const float* inp, const float* weight, const float* bias,
                       int B, int T, int C,
                       const int block_size) {
    assert(block_size % 32 == 0);
    assert(block_size <= 1024);
    const int N = B * T;
    const int grid_size = N;
    layernorm_forward_kernel5<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward6(float* out, float* mean, float* rstd,
                       const float* inp, const float* weight, const float* bias,
                       int B, int T, int C,
                       int block_size) {
    assert(block_size % 32 == 0);
    const int N = B * T;
    int block_y = block_size / WARP_SIZE;
    const int grid_size = ceil_div(N, block_y);
    size_t smem = (2 + block_y) * C * sizeof(float);

    // in order to use more than 48 KiB of smem, need to call hipFuncSetAttribute
    // this may fail, in which case we fall back to the smem free implementation.
    cudaCheck(hipGetLastError());
    auto status = hipFuncSetAttribute(reinterpret_cast<const void*>(layernorm_forward_kernel6), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
    hipGetLastError();
    if (status == hipSuccess) {
        layernorm_forward_kernel6<<<grid_size, dim3(32, block_y), smem>>>(out, mean, rstd, inp, weight, bias, N, C);
    } else {
        const int grid_size = N;
        // fall back to the version without shared memory
        layernorm_forward_kernel5<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    }
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void layernorm_forward(int kernel_num,
                    float* out, float* mean, float* rstd,
                    const float* inp, const float* weight, const float* bias,
                    int B, int T, int C,
                    const int block_size) {
    switch (kernel_num) {
        case 1:
            layernorm_forward1(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 2:
            layernorm_forward2(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 3:
            layernorm_forward3(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 4:
            layernorm_forward4(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 5:
            layernorm_forward5(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        case 6:
            layernorm_forward6(out, mean, rstd, inp, weight, bias, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* mean = (float*)malloc(B * T * sizeof(float));
    float* rstd = (float*)malloc(B * T * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(C);
    float* bias = make_random_float(C);

    // move to GPU
    float* d_out;
    float* d_mean;
    float* d_rstd;
    float* d_inp;
    float* d_weight;
    float* d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_mean, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_rstd, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 2;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    layernorm_forward_cpu(out, mean, rstd, inp, weight, bias, B, T, C);

    // check the correctness of the kernel at all block sizes
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);

        layernorm_forward(kernel_num, d_out, d_mean, d_rstd, d_inp, d_weight, d_bias, B, T, C, block_size);

        validate_result(d_out, out, "out", B * T * C, 1e-5f);
        validate_result(d_mean, mean, "mean", B * T, 1e-5f);
        validate_result(d_rstd, rstd, "rstd", B * T, 1e-5f);
    }

    printf("All results match. Starting benchmarks.\n\n");

    // time the kernel at different block sizes
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 2000;
        float elapsed_time = benchmark_kernel(repeat_times, layernorm_forward,
                                              kernel_num, d_out, d_mean, d_rstd, d_inp, d_weight, d_bias,
                                              B, T, C, block_size);

        // napkin math: estimate the memory bandwidth achieved
        // e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = (2 * B * T * C) * 4; // *4 for float
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(out);
    free(mean);
    free(rstd);
    free(inp);
    free(weight);
    free(bias);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_mean));
    cudaCheck(hipFree(d_rstd));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));

    return 0;
}
