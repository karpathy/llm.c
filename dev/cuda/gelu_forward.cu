#include "hip/hip_runtime.h"
/*
Kernels for gelu forward pass.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt gelu_forward.cu -o gelu_forward

If encountering "error: identifier "M_PI" is undefined", add the following lines to the top of the file:

#define _USE_MATH_DEFINES
#include <math.h>  OR  #include <cmath>

version 1 is naive CPU port
./gelu_forward 1

version 2 is bfloat16 with the Packed128 data structure
./gelu_forward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)

void gelu_forward_cpu(float* out, const float* inp, int N) {
    for (int i = 0; i < N; i++) {
        float x = inp[i];
        float cube = 0.044715f * x * x * x;
        out[i] = 0.5f * x * (1.0f + tanhf(GELU_SCALING_FACTOR * (x + cube)));
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// elementwise ops are nice and ez
__global__ void gelu_forward_kernel1(floatX* out, const floatX* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
    }
}

// elementwise ops are nice and ez
__global__ void gelu_forward_kernel2(floatX* out, const floatX* inp, int N) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    if (i < N) {
        x128 packed_out;
        x128 packed_inp = load128cs(inp + i); // load and do not keep in cache
        for(int k = 0; k < packed_inp.size; ++k) {
            float xi = (float)packed_inp[k];
            float cube = 0.044715f * xi * xi * xi;
            packed_out[k] = (floatX)(0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube))));
        }
        // store instead of storecs (without cache streaming) in case it is useful for the
        // data to be in the cache for the next operation after this GeLU
        store128(out + i, packed_out);
    }
}

// use hardware intrinsics for marginally faster performance
//  use restrict keyword on data pointers
__global__ void gelu_kernel_2(
  float* __restrict__ out, 
  const float* __restrict__ inp, 
  int N
) {
    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    int chunk_size = blockDim.x * gridDim.x;

    float s = __fsqrt_rn(__fdiv_rn(2.0f, M_PI));
    
    #pragma unroll
    for (int i = ti; i < N; i += chunk_size) {
        const float xi = inp[i];
        float cdf = 1.0f + tanhf(s * (xi + 0.044715f * (xi * xi * xi)));
        cdf *= 0.5f;
        out[i] = xi * cdf;
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void gelu_forward1(floatX* out, const floatX* inp, int N, const int block_size) {
    const int grid_size = ceil_div(N, block_size);
    gelu_forward_kernel1<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward2(floatX* out, const floatX* inp, int N, const int block_size) {
    const int grid_size = ceil_div(N, block_size * x128::size);
    gelu_forward_kernel2<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward2(float* out, float* inp, int N, const int block_size) {
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_kernel_2<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void gelu_forward(int kernel_num,
                  floatX* out,
                  const floatX* inp,
                  int B, int T, int C,
                  int block_size) {
    switch (kernel_num) {
        case 1:
            gelu_forward1(out, inp, B * T * C, block_size);
            break;
        case 2:
            gelu_forward2(out, inp, B * T * C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, const char **argv) {
    setup_main();

    int B = 8;
    int T = 1024;
    int C = 768;

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* inp = make_random_float(B * T * C);

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    gelu_forward_cpu(out, inp, B * T * C);

    // move to GPU
    floatX* d_out;
    floatX* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(floatX)));
    cudaCheck(memcpy_convert(d_inp, inp, B * T * C));

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        gelu_forward(kernel_num, d_out, d_inp, B, T, C, block_size);
#if !defined(ENABLE_BF16) && !defined(ENABLE_FP16)
        float tol = 1e-5;
#else
        float tol = 1e-2f;
#endif
        validate_result(d_out, out, "out", B * T * C, tol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, gelu_forward,
                                              kernel_num, d_out, d_inp,
                                              B, T, C, block_size);

        // napkin math: estimate the memory bandwidth achieved
        // for each (B,T,C) output element, we do 1 read and 1 write, 4 bytes each
        // and e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = B * T * C * 2 * (int)sizeof(floatX);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(out);
    free(inp);

    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    return 0;
}