#include "hip/hip_runtime.h"
/*
Kernels for gelu forward pass.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt gelu_forward.cu -o gelu_forward

If encountering "error: identifier "M_PI" is undefined", add the following lines to the top of the file:

#define _USE_MATH_DEFINES
#include <math.h>  OR  #include <cmath>

version 1 is naive CPU port
./gelu_forward 1

version 2 is bfloat16 with the Packed128 data structure
./gelu_forward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)

void gelu_forward_cpu(float* out, const float* inp, int N) {
    for (int i = 0; i < N; i++) {
        float x = inp[i];
        float cube = 0.044715f * x * x * x;
        out[i] = 0.5f * x * (1.0f + tanhf(GELU_SCALING_FACTOR * (x + cube)));
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// elementwise ops are nice and ez
__global__ void gelu_forward_kernel1(floatX* out, const floatX* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
    }
}

// elementwise ops are nice and ez
__global__ void gelu_forward_kernel2(floatX* out, const floatX* inp, int N) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    if (i < N) {
        x128 packed_out;
        x128 packed_inp = load128cs(inp + i); // load and do not keep in cache
        for(int k = 0; k < packed_inp.size; ++k) {
            float xi = (float)packed_inp[k];
            float cube = 0.044715f * xi * xi * xi;
            packed_out[k] = (floatX)(0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube))));
        }
        // store instead of storecs (without cache streaming) in case it is useful for the
        // data to be in the cache for the next operation after this GeLU
        store128(out + i, packed_out);
    }
}

// Optimised with option to use optimised HW TANH instruction by default
__global__ void gelu_forward_kernel3(floatX* out, const floatX* inp, int N) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    if (idx >= N) { return; }

    x128 packed_out;
    x128 packed_inp = load128cs(inp + idx); // load and do not keep in cache
    for(int k = 0; k < packed_inp.size; ++k) {
        float xi = (float)packed_inp[k];
        float cube = 0.044715f * xi * xi * xi;

        float tanh_in_out = GELU_SCALING_FACTOR * (xi + cube);
        #if !defined(PRECISE_GELU_TANH) && __CUDA_ARCH__ >= 750
        asm ("tanh.approx.f32 %0,%1;" : "=f"(tanh_in_out) : "f"(tanh_in_out));
        #else
        tanh_in_out = tanhf(tanh_in_out);
        #endif

        // the following uses FMUL+FMA instead of FMUL+FADD+FMUL for "0.5f * x * (1.0f + tanh_out)"
        float half_xi = 0.5f * xi;
        packed_out[k] = (floatX)(half_xi * tanh_in_out + half_xi);
    }
    // store instead of storecs (without cache streaming) in case it is useful for the
    // data to be in the cache for the next operation after this GeLU
    store128(out + idx, packed_out);
}

// ----------------------------------------------------------------------------
// kernel launcher

void gelu_forward1(floatX* out, const floatX* inp, int N, const int block_size) {
    const int grid_size = ceil_div(N, block_size);
    gelu_forward_kernel1<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward2(floatX* out, const floatX* inp, int N, const int block_size) {
    const int grid_size = ceil_div(N, block_size * x128::size);
    gelu_forward_kernel2<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward3(floatX* out, const floatX* inp, int N, const int block_size) {
    const int grid_size = ceil_div(N, block_size * x128::size);
    gelu_forward_kernel3<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void gelu_forward(int kernel_num,
                  floatX* out,
                  const floatX* inp,
                  int B, int T, int C,
                  int block_size) {
    switch (kernel_num) {
        case 1:
            gelu_forward1(out, inp, B * T * C, block_size);
            break;
        case 2:
            gelu_forward2(out, inp, B * T * C, block_size);
            break;
        case 3:
            gelu_forward3(out, inp, B * T * C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, const char **argv) {
    setup_main();

    int B = 128;
    int T = 1024;
    int C = 768;

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* inp = make_random_float(B * T * C);

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    gelu_forward_cpu(out, inp, B * T * C);

    // move to GPU
    floatX* d_out;
    floatX* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(floatX)));
    cudaCheck(memcpy_convert(d_inp, inp, B * T * C));

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        gelu_forward(kernel_num, d_out, d_inp, B, T, C, block_size);
#if !defined(ENABLE_BF16) && !defined(ENABLE_FP16)
        float tol = 1e-5f;
#else
        float tol = 1e-3f;
#endif
        validate_result(d_out, out, "out", B * T * C, tol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, gelu_forward,
                                              kernel_num, d_out, d_inp,
                                              B, T, C, block_size);

        // napkin math: estimate the memory bandwidth achieved
        // for each (B,T,C) output element, we do 1 read and 1 write, 4 bytes each
        // and e.g. A100 40GB PCIe is advertised at 1,555GB/s
        long memory_ops = B * T * C * 2 * (int)sizeof(floatX);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(out);
    free(inp);

    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    return 0;
}