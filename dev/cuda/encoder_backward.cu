#include "hip/hip_runtime.h"
/*
Kernels for the positional encoder forward pass in GPT-2.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt encoder_backward.cu -o encoder_backward

version 1 is naive port from CPU code to kernel
parallelizes over B,T,C, uses atomics to add to dwte, dwpe
./encoder_backward 1

version 2 is another naive port
parallelizes over C, loops over B,T; much slower than version 1
./encoder_backward 2

version 3 uses shared memory to reduce global memory accesses and improve performance
parallelizes over B,T,C, uses atomics to add to dwte, dwpe, utilizes shared memory for intermediate results
./encoder_backward 3

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

// GPT-2 positional encoder forward pass
void encoder_backward_cpu(float* dwte, float* dwpe,
                            float* dout, int* inp,
                            int B, int T, int C) {
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            float* dout_bt = dout + b * T * C + t * C;
            int ix = inp[b * T + t];
            float* dwte_ix = dwte + ix * C;
            float* dwpe_t = dwpe + t * C;
            for (int i = 0; i < C; i++) {
                float d = dout_bt[i];
                dwte_ix[i] += d;
                dwpe_t[i] += d;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// naive implementation with atomics
__global__ void encoder_backward_kernel1(float* dwte, float* dwpe,
                                        const float* dout, const int* inp,
                                        int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        const float* dout_btc = dout + b * T * C + t * C + c;
        float* dwte_ix = dwte + ix * C + c;
        float* dwpe_tc = dwpe + t * C + c;

        atomicAdd(dwte_ix, *dout_btc);
        atomicAdd(dwpe_tc, *dout_btc);
    }
}

// naive implementation that parallelizes over C and loops over B,T
// but it gets rid of atomics
__global__ void encoder_backward_kernel2(float* dwte, float* dwpe,
                                        const float* dout, const int* inp,
                                        int B, int T, int C) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= C) { return; } // guard
    int BT = B * T;
    for (int i = 0; i < BT; i++) {
        int t = i % T;
        int ix = inp[i];
        float dout_btc = dout[i * C + c];
        dwte[ix * C + c] += dout_btc;
        dwpe[t * C + c] += dout_btc;
    }
}

// Optimized implementation using shared memory
__global__ void encoder_backward_kernel3(float* dwte, float* dwpe,
                                         const float* dout, const int* inp,
                                         int B, int T, int C) {
    extern __shared__ float shared_mem[];
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (global_tid < N) {
        int bt = global_tid / C;
        int b = bt / T;
        int t = bt % T;
        int c = global_tid % C;

        float d = dout[global_tid];
        int ix = inp[b * T + t];

        atomicAdd(&dwte[ix * C + c], d);
        atomicAdd(&dwpe[t * C + c], d);
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void encoder_backward1(float* dwte, float* dwpe,
                    const float* dout, const int* inp,
                    int B, int T, int C,
                    const int block_size) {
    const int N = B * T * C;
    const int grid_size = ceil_div(N, block_size);
    encoder_backward_kernel1<<<grid_size, block_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

void encoder_backward2(float* dwte, float* dwpe,
                    const float* dout, const int* inp,
                    int B, int T, int C,
                    const int block_size) {
    const int grid_size = ceil_div(C, block_size);
    encoder_backward_kernel2<<<grid_size, block_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

void encoder_backward3(float* dwte, float* dwpe,
                       const float* dout, const int* inp,
                       int B, int T, int C,
                       const int block_size) {
    const int N = B * T * C;
    const int grid_size = ceil_div(N, block_size);
    size_t shared_mem_size = block_size * sizeof(float);
    encoder_backward_kernel3<<<grid_size, block_size, shared_mem_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void encoder_backward(int kernel_num,
                     float* dwte, float* dwpe,
                    const float* dout, const int* inp,
                    int B, int T, int C,
                    const int block_size) {
    switch (kernel_num) {
        case 1:
            encoder_backward1(dwte, dwpe, dout, inp, B, T, C, block_size);
            break;
        case 2:
            encoder_backward2(dwte, dwpe, dout, inp, B, T, C, block_size);
            break;
         case 3:
            encoder_backward3(dwte, dwpe, dout, inp, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;
    int V = 50257;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* dout = make_random_float(B * T * C);
    int* inp = make_random_int(B * T, V);
    float* dwte = make_zeros_float(V * C);
    float* dwpe = make_zeros_float(T * C);

    // move to GPU
    float* d_dout;
    int* d_inp;
    float* d_dwte;
    float* d_dwpe;
    cudaCheck(hipMalloc(&d_dout, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * sizeof(int)));
    cudaCheck(hipMalloc(&d_dwte, V * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dwpe, T * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_dout, dout, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * sizeof(int), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    encoder_backward_cpu(dwte, dwpe, dout, inp, B, T, C);

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        cudaCheck(hipMemset(d_dwte, 0, V * C * sizeof(float)));
        cudaCheck(hipMemset(d_dwpe, 0, T * C * sizeof(float)));
        printf("Checking block size %d.\n", block_size);
        encoder_backward(kernel_num, d_dwte, d_dwpe, d_dout, d_inp, B, T, C, block_size);
        validate_result(d_dwte, dwte, "dwte", V * C, 1e-5f);
        validate_result(d_dwpe, dwpe, "dwpe", T * C, 1e-5f);
    }
    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 1000;
        float elapsed_time = benchmark_kernel(repeat_times, encoder_backward,
                                              kernel_num, d_dwte, d_dwpe, d_dout, d_inp, B, T, C, block_size);
        printf("block_size %4d | time %.4f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(dout);
    free(inp);
    free(dwte);
    free(dwpe);
    hipFree(d_dout);
    hipFree(d_inp);
    hipFree(d_dwte);
    hipFree(d_dwpe);

    return 0;
}
