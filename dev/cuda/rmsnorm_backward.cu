#include "hip/hip_runtime.h"
/*
Kernels for RMSNorm backward pass.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt rmsnorm_backward.cu -o rmsnorm_backward

./rmsnorm_backward 1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

void rmsnorm_forward_cpu(float* out, float* rstd,
                       const float* inp, const float* weight,
                       int B, int T, int C) {
    float eps = 1e-5f;
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b,t,:]
            const float* x = inp + b * T * C + t * C;
            // calculate the variance (without any bias correction)
            float v = 0.0f;
            for (int i = 0; i < C; i++) {
                float xi = x[i];
                v += xi * xi;
            }
            v = v/C;
            // calculate the rstd (reciprocal standard deviation)
            float s = 1.0f / sqrtf(v + eps);
            // seek to the output position in out[b,t,:]
            float* out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = (s * x[i]); // normalize
                float o = n * weight[i]; // scale and shift
                out_bt[i] = o; // write
            }
            // cache the rstd for the backward pass later
            rstd[b * T + t] = s;
        }
    }
}

void rmsnorm_backward_cpu(float* dinp, float* dweight,
                        const float* dout, const float* inp, const float* weight, const float* rstd,
                        int B, int T, int C) {
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            const float* dout_bt = dout + b * T * C + t * C;
            const float* inp_bt = inp + b * T * C + t * C;
            float* dinp_bt = dinp + b * T * C + t * C;
            const float rstd_bt = rstd[b * T + t];

            // first: the reduce operation
            float dnorm_norm_mean = 0.0f;
            for (int i = 0; i < C; i++) {
                float norm_bti = inp_bt[i] * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                dnorm_norm_mean += dnorm_i * norm_bti;
            }
            dnorm_norm_mean = dnorm_norm_mean / C;

            // now iterate again and accumulate all the gradients
            for (int i = 0; i < C; i++) {
                float norm_bti = inp_bt[i] * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                // gradient contribution to weight
                dweight[i] += norm_bti * dout_bt[i];
                // gradient contribution to input
                float dval = 0.0f;
                dval += dnorm_i; // term 1
                dval -= norm_bti * dnorm_norm_mean; // term 2
                dval *= rstd_bt; // final scale
                dinp_bt[i] += dval;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernel

__global__ void __launch_bounds__(512, 2) // todo - any warnings on Turing with only 1024 threads?
    rmsnorm_backward_kernel10(floatX* dinp, floatX* dweight, float* scratch,
                                const floatX* dout, const floatX* inp, const floatX* weight,
                                const float* rstd, int B, int T, int C) {
    // TODO: this kernel uses too much shared memory due to historical reasons of it coming from layernorm_backward.cu
    // this memory use can be reduced by half later
    int BLOCK_SIZE = blockDim.x;
    int warpsInBlock = BLOCK_SIZE / WARP_SIZE; //number of warps in block
    extern __shared__ float shared[];

    int warpId = threadIdx.x / WARP_SIZE; // warp index within a block
    int baseIdx = blockIdx.x * warpsInBlock + warpId;
    int warpThreadIdx = threadIdx.x % WARP_SIZE; // Thread index within the warp
    int warpsInGrid = gridDim.x * warpsInBlock;
    int C_per_iteration = WARP_SIZE * x128::size;
    int iterations_C = ceil_div(C, C_per_iteration); // + 2;

    // the first half of shared memory is bias, second is weight
    size_t rounded_C = ceil_div(C, (32 * x128::size)) * (32 * x128::size);
    float* dweight_shared = shared + rounded_C;
    // warp zero doesn't actually write to the _tmp_shared memory locations, so we don't need to reserve memory
    // the obvious solution is to change the addressing below to use (threadId.x-32) as offset, but that causes
    // register spills, so instead we mess with the base pointer here, which doesn't increase register usage.
    float* dweight_tmp_shared = shared + 2 * rounded_C + f128::size * BLOCK_SIZE - 2 * WARP_SIZE * f128::size;

    // init shared memory to zero
    for(int i = threadIdx.x * f128::size; i < rounded_C; i += BLOCK_SIZE * f128::size) {
        store128(dweight_shared + i, f128::zeros());
    }
    __syncthreads();

    for (int bt = baseIdx; bt < B * T; bt += warpsInGrid) {
        const floatX* dout_bt = dout + bt * C;
        const floatX* inp_bt = inp +bt * C;
        floatX* dinp_bt = dinp + bt * C;

        // first: two reduce operations
        float dnorm_mean = 0.0f;
        float dnorm_norm_mean = 0.0f;
        for (int i = warpThreadIdx * x128::size; i < C; i += WARP_SIZE * x128::size) {
            x128 dout128_i   = load128(dout_bt + i);
            x128 inp128_i    = load128(inp_bt  + i);
            x128 weight128_i = load128(weight  + i);
            for (int k = 0; k < x128::size; k++) {
                float dnorm_i = (float)weight128_i[k] * (float)dout128_i[k];
                dnorm_mean += dnorm_i;
                dnorm_norm_mean += dnorm_i * (float)inp128_i[k];
            }
        }

        const float rstd_bt = rstd[bt];
        dnorm_norm_mean = warpReduceSum(dnorm_norm_mean) / C * rstd_bt;

        for (int c = 0; c < iterations_C; c++) {
            int global_index = (warpThreadIdx * x128::size) + (c * C_per_iteration);

            x128 dout128   = x128::zeros();
            x128 inp128    = x128::zeros();
            x128 dinp128   = x128::zeros();
            x128 weight128 = x128::zeros();

            if(global_index < C) {
                dout128 = load128cs(dout_bt + global_index);
                inp128 = load128cs(inp_bt + global_index);
                dinp128 = load128(dinp_bt + global_index);
                weight128 = load128(weight + global_index);
            }

            for(int o = 0; o < x128::size / f128::size; ++o) {
                f128 dweight_f;
                for(int i = 0; i < f128::size; ++i) {
                    int x = o * f128::size + i;
                    float dout_i = (float)dout128[x];
                    float norm_bti = ((float)inp128[x]) * rstd_bt;
                    dweight_f[i] = norm_bti * dout_i;

                    float dval = 0.0f;
                    dval += (float) weight128[x] * (float)dout128[x]; // term 1
                    dval -= norm_bti * dnorm_norm_mean; // term 2
                    dval *= rstd_bt; // final scale
                    dinp128[x] = (floatX) ((float) dinp128[x] + dval);
                }

                if (warpId != 0) {
                    // this seems to generate a 64-bit store, instead of 128-bit.
                    // however, forcing 128-bit (e.g., using inline ptx), results in register
                    // spilling and much worse performance, so we'll keep it like this for now
                    // but ideally, we could reduce the register pressure a little.
                    store128(dweight_tmp_shared + threadIdx.x * f128::size, dweight_f);
                }
                __syncthreads();
                if (warpId == 0) {
                    for (int j = 1; j < warpsInBlock; j++) {
                        f128 dweight_tmp = load128(dweight_tmp_shared + f128::size * (threadIdx.x + j * WARP_SIZE));
                        for(int i = 0; i < f128::size; ++i) {
                            dweight_f[i] += dweight_tmp[i];
                        }
                    }
                }
                __syncthreads();
                if (warpId == 0) {
                    f128 dw_old = load128(dweight_shared + global_index + f128::size * o);
                    for(int i = 0; i < f128::size; ++i) {
                        dweight_f[i] += dw_old[i];
                    }
                    store128(dweight_shared + global_index + f128::size * o, dweight_f);
                }
            }
            if(global_index < C) {
                // cache in L2 as this is read by the next kernel, but bypass L1 to minimise thrashing
                store128cg(dinp_bt + global_index, dinp128);
            }
        }
    }
    __syncthreads();
    // Each block writes its partial sum to global memory
    // The last block to finish becomes responsible for summing up all the partial sums
    // This is done by atomically incrementing a flag (cleared to 0 before launching the kernel)
    unsigned int* scratchFlag = (unsigned int*)(scratch);
    // Increment scratch pointer by a full cacheline so that everything remains cacheline aligned
    scratch += 32;
    float* scratch_dweight = scratch + C;
    for(int i = threadIdx.x * f128::size; i < C; i += BLOCK_SIZE * f128::size) {
        // Write to global memory in the same "shared memory banking friendly" order
        store128(scratch_dweight + i + 2*C*blockIdx.x, load128(dweight_shared + i));
    }
    __syncthreads();
    // that portion of shared memory is no longer used, so we can repurpose it for the scratch flag.
    unsigned int *tmp_flag = (unsigned int*)(shared + 2*rounded_C);
    if (threadIdx.x == 0) {
        *tmp_flag = atomicInc(scratchFlag, gridDim.x);
    }
    __syncthreads();
    if (*tmp_flag == gridDim.x-1) {
        // Reduction of the partial sums by the final block
        // todo - there isn't enough parallelism even inside that single SM...
        // ==> so could maybe split into another kernel with YET ANOTHER level of reduction?!
        for(int i = threadIdx.x * f128::size; i < C; i += BLOCK_SIZE * f128::size) {
            f128 dweight_accum = f128::zeros();

            for (int read_block_idx = 0; read_block_idx < gridDim.x; read_block_idx++) {
                int offset = i + 2*C*read_block_idx;
                f128 dweight128 = load128(scratch_dweight + offset);
                for(int k = 0; k < f128::size; k++) {
                    dweight_accum[k] += dweight128[k];
                }
            }
            store128(dweight_shared + i, dweight_accum);
        }
        __syncthreads();

        // convert from float/FP32 to floatX/BF16 for the final write
        // this is separate because it cannot use as many warps as the above (f128 vs x128)
        // todo - if we split this code into another kernel, we could maybe do it at the same time?
        for (int c = warpId; c < iterations_C; c += warpsInBlock) {
            int global_index = (warpThreadIdx * x128::size) + (c * C_per_iteration);
            if (global_index >= C) {
                break;
            }
            x128 dweight128 = load128(dweight + global_index);
            for(int o = 0; o < x128::size / f128::size; ++o) {
                f128 s_dw = load128(dweight_shared + global_index + o * f128::size);
                for(int i = 0; i < f128::size; ++i) {
                    int x = o * f128::size + i;
                    dweight128[x] = (floatX)(s_dw[i] + (float)dweight128[x]);
                }
            }
            store128(dweight + global_index, dweight128);
        }
    }
}

// ----------------------------------------------------------------------------
// Kernel launcher

void rmsnorm_backward(floatX* dinp, floatX* dweight, float* scratch,
                        const floatX* dout, const floatX* inp, const floatX* weight, const float* rstd,
                        int B, int T, int C, hipStream_t stream) {
    const int block_size = 512;
    const int blocks_per_sm = 2; // supported on every architecture and less cache thrashing than 3
    // const int grid_size = blocks_per_sm * deviceProp.multiProcessorCount;
    const int grid_size = blocks_per_sm * cuda_num_SMs;
    size_t rounded_C = ceil_div(C, (32 * x128::size)) * (32 * x128::size);
    size_t shared_mem_size = (2 * rounded_C + 2 * (block_size - 32) * f128::size) * sizeof(float);

    cudaCheck(hipMemsetAsync(scratch, 0, 1 * sizeof(float), stream)); // only need to reset the flag to 0
    rmsnorm_backward_kernel10<<<grid_size, block_size, shared_mem_size, stream>>>(dinp, dweight, scratch, dout, inp, weight, rstd, B, T, C);
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    setup_main();

    int B = 8;
    int T = 1024;
    int C = 1024;

    // first do the forward pass in CPU
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* rstd = (float*)malloc(B * T * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(C);
    rmsnorm_forward_cpu(out, rstd, inp, weight, B, T, C);

    // now do the backward pass, again on CPU
    float *dout = make_random_float(B * T * C);
    float *dinp = make_zeros_float(B * T * C);
    float *dweight = make_zeros_float(C);
    rmsnorm_backward_cpu(dinp, dweight, dout, inp, weight, rstd, B, T, C);

    // the above calculations act as the reference
    // now let's do the same on the GPU

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // move all the variables we need for backward pass onto the GPU
    floatX* d_dinp;
    floatX* d_dweight;
    floatX* d_dout;
    floatX* d_inp;
    floatX* d_weight;
    float* d_rstd;
    float* d_scratch;
    cudaCheck(hipMalloc(&d_dinp, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_dweight, C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_dout, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(floatX)));
    cudaCheck(hipMalloc(&d_rstd, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_scratch, (1024/32) * cuda_num_SMs * (2 * C + 1) * sizeof(float)));

    // copy over the "inputs" to the backward call
    cudaCheck(memcpy_convert(d_dout, dout, B * T * C));
    cudaCheck(memcpy_convert(d_inp, inp, B * T * C));
    cudaCheck(memcpy_convert(d_weight, weight, C));
    cudaCheck(memcpy_convert(d_rstd, rstd, B * T));

    // launch the kernel
    int block_sizes[] = {32, 64, 128, 256, 512, /*768,*/ 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        // init the "outputs" of the backward call to zeros
        cudaCheck(hipMemset(d_dinp, 0, B * T * C * sizeof(floatX)));
        cudaCheck(hipMemset(d_dweight, 0, C * sizeof(floatX)));

        rmsnorm_backward(d_dinp, d_dweight, d_scratch, d_dout, d_inp, d_weight, d_rstd, B, T, C, 0);

        // check the correctness of the kernel
        float error_threshold_dinp = sizeof(floatX) == 4 ? 1e-3f : 1e-1f; // allow larger errors for BF16/FP16
        float error_threshold_dparams = sizeof(floatX) == 4 ? 1e-3f : 5e-1f; // much, much larger...
        printf("Checking correctness...\n");
        printf("dinp:\n");
        validate_result(d_dinp, dinp, "dinp", B * T * C, error_threshold_dinp);
        printf("dweight:\n");
        validate_result(d_dweight, dweight, "dweight", C, error_threshold_dparams);

        printf("All results match for block_size=%d.\n\n", block_size);
    }
}
