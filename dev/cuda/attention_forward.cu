#include "hip/hip_runtime.h"
/*
Kernels for attention forward pass.

If you do not have CUDNN, you can remove ENABLE_CUDNN to run the other kernels

See the README for cuDNN install instructions

Compile example with cuDNN:
nvcc -I/PATH/TO/cudnn-frontend/include -DENABLE_CUDNN -O3 --use_fast_math --lcublas -lcublasLt -lcudnn attention_forward.cu -o attention_forward

Compile example without cuDNN:
nvcc -O3 --use_fast_math -lcublas -lcublasLt attention_forward.cu -o attention_forward

version 1 is naive port from CPU code to kernel, parallelize over batch, time, heads only
./attention_forward 1

version 2 is a naive implementation of flash attention, taken, adapted from
https://github.com/tspeterkim/flash-attention-minimal
and with help from
https://github.com/leloykun/flash-hyperbolic-attention-minimal
sadly, this flash attention version seems about 3X slower than the naive version
./attention_forward 2

version 3 is a cuBLAS + softmax version, similar to the PyTorch implementation
cuBLAS is used both to calculate the QK^T and the final weighted sum
the softmax is calculated using a custom, efficient kernel as well
this turns out to be ~20X faster than (1) nice
./attention_forward 3

version 4 is a further optimized kernel that fuses the scale operation,
uses a directly autoregressive softmax, and uses the online softmax algorithm.
./attention_forward 4

version 5 is a FP16 version of kernel 4
./attention_forward 5

version 6 is kernel 5 skipping (un)permute (unrealistic but useful comparison point)

version 10 is using cuDNN Flash Attention using FP16 or BF16, see:
https://github.com/NVIDIA/cudnn-frontend/blob/main/docs/operations/Attention.md
./attention_forward 10

version 11 is kernel 10 skipping FP16/FP32 conversions (full FP16/BF16 network)
./attention_forward 11
*/
//#define ENABLE_CUDNN // can be enabled via nvcc "-DENABLE_CUDNN"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#define ENABLE_BF16
#include "common.h"

// ----------------------------------------------------------------------------
// CUDA & cuDNN setup
static bool first_run_validation = true; // always run e.g. permute on 1st run

#ifdef ENABLE_CUDNN
#include <cudnn_frontend.h>
namespace fe = cudnn_frontend;
#if CUBLAS_LOWP == HIP_R_16BF
#define CUDNN_16BIT fe::DataType_t::BFLOAT16
#else
#define CUDNN_16BIT fe::DataType_t::HALF
#endif

static hipdnnHandle_t cudnn_handle;
static size_t cudnn_workspace_size = 0; // dynamically allocated as needed (up to 256MiB!)
static void* cudnn_workspace = NULL;

#define checkCudaErr(err) assert((int)err == 0);
#define checkCudnnErr(err) assert((int)err == 0);
#endif // ENABLE_CUDNN
// ----------------------------------------------------------------------------
// CPU code reference

void attention_forward_cpu(float* out, float* preatt, float* att,
                       const float* inp,
                       int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q,K,V
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = C*3;
    int hs = C / NH; // head size
    float scale = 1.0 / sqrtf(hs);

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            for (int h = 0; h < NH; h++) {
                const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
                float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
                float* att_bth = att + b*NH*T*T + h*T*T + t*T;

                // pass 1: calculate query dot key and maxval
                float maxval = -FLT_MAX;
                for (int t2 = 0; t2 <= t; t2++) {
                    const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

                    // (query_t) dot (key_t2)
                    float val = 0.0f;
                    for (int i = 0; i < hs; i++) {
                        val += query_t[i] * key_t2[i];
                    }
                    val *= scale;
                    if (val > maxval) {
                        maxval = val;
                    }

                    preatt_bth[t2] = val;
                }
                // pad with -INFINITY outside of autoregressive region for debugging comparisons
                for (int t2 = t+1; t2 < T; t2++) {
                    preatt_bth[t2] = -INFINITY;
                }

                // pass 2: calculate the exp and keep track of sum
                float expsum = 0.0f;
                for (int t2 = 0; t2 <= t; t2++) {
                    float expv = expf(preatt_bth[t2] - maxval);
                    expsum += expv;
                    att_bth[t2] = expv;
                }
                float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

                // pass 3: normalize to get the softmax
                for (int t2 = 0; t2 < T; t2++) {
                    if (t2 <= t) {
                        att_bth[t2] *= expsum_inv;
                    } else {
                        // causal attention mask. not strictly necessary to set to zero here
                        // only doing this explicitly for debugging and checking to PyTorch
                        att_bth[t2] = 0.0f;
                    }
                }

                // pass 4: accumulate weighted values into the output of attention
                float* out_bth = out + b * T * C + t * C + h * hs;
                for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
                for (int t2 = 0; t2 <= t; t2++) {
                    const float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
                    float att_btht2 = att_bth[t2];
                    for (int i = 0; i < hs; i++) {
                        out_bth[i] += att_btht2 * value_t2[i];
                    }
                }
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void attention_query_key_kernel1(float* preatt, const float* inp,
                                           int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * NH * T * T;

    if (idx < total_threads) {
        int t2 = idx % T;
        int t = (idx / T) % T;
        if (t2 > t) {
            // autoregressive mask
            preatt[idx] = -INFINITY;
            return;
        }
        int h = (idx / (T * T)) % NH;
        int b = idx / (NH * T * T);

        int C3 = C*3;
        int hs = C / NH; // head size
        const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
        const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

        // (query_t) dot (key_t2)
        float val = 0.0f;
        for (int i = 0; i < hs; i++) {
            val += query_t[i] * key_t2[i];
        }
        val *= 1.0 / sqrtf(hs);

        preatt[idx] = val;
    }
}

__global__ void attention_softmax_kernel1(float* att, const float* preatt,
                                         int B, int T, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        const float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
        float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        // find maxval
        float maxval = -FLT_MAX;
        for (int t2 = 0; t2 <= t; t2++) {
            if (preatt_bth[t2] > maxval) {
                maxval = preatt_bth[t2];
            }
        }

        // calculate the exp and keep track of sum
        float expsum = 0.0f;
        for (int t2 = 0; t2 <= t; t2++) {
            float expv = expf(preatt_bth[t2] - maxval);
            expsum += expv;
            att_bth[t2] = expv;
        }
        float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

        // normalize to get the softmax
        for (int t2 = 0; t2 < T; t2++) {
            if (t2 <= t) {
                att_bth[t2] *= expsum_inv;
            } else {
                // causal attention mask. not strictly necessary to set to zero here
                // only doing this explicitly for debugging and checking to PyTorch
                att_bth[t2] = 0.0f;
            }
        }
    }
}

// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

__global__ void softmax_forward_kernel4(float* out, const float* inp, int N, int C) {
    // out is (N, C) just like inp. Each row of inp will get softmaxed.
    // same as kernel3, but can handle any block size (multiple of 32)
    // each row of C elements is handled by block_size threads
    // furthermore, each block_size threads get executed in warps of 32 threads

    // special reduction operations warpReduceMax/warpReduceSum are used for intra-warp reductions
    // shared memory is used for inter-warp reduction
    extern __shared__ float shared[];
    int idx = blockIdx.x;
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32; // warp index within a block
    int laneId = threadIdx.x % 32; // thread index within a warp

    // the number of warps per block. recall that blockDim.x is block_size
    int warpsPerBlock = blockDim.x / 32;

    // shared[] must be allocated to have 2 * warpsPerBlock elements
    // first half for max values, the second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // one row of inp, i.e. inp[idx, :] of shape (C,)
    const float* x = inp + idx * C;

    // first, thread coarsening by directly accessing global memory in series
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += blockDim.x) {
        maxval = fmaxf(maxval, x[i]);
    }
    // now within-warp reductions for maxval
    maxval = warpReduceMax(maxval);

    // the 0th thread of each warp writes the maxval of that warp to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // now the 0th thread reduces the maxvals in shared memory, i.e. across warps
    if (tid == 0) {
        float val = maxvals[tid];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        // store the final max in the first position
        maxvals[0] = val;
    }
    __syncthreads();
    // broadcast the max to all threads
    float offset = maxvals[0];

    // compute expf and write the result to global memory
    for (int i = tid; i < C; i += blockDim.x) {
        // subtract max for numerical stability
        out[idx * C + i] = expf(x[i] - offset);
    }

    // okay now we calculated exp(x - max(x))
    // step 2: sum all the values and divide by the sum

    // thread coarsening for sum
    x = out + idx * C;
    float sumval = 0.0f;
    for (int i = tid; i < C; i += blockDim.x) {
        sumval += x[i];
    }
    // within-warp reduction for sumval
    sumval = warpReduceSum(sumval);

    // write sumval to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // inter-thread reduction of sum
    if (tid == 0) {
        float val = sumvals[tid];
        for (int i = 1; i < warpsPerBlock; ++i) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();
    // broadcast the sum to all threads
    float sum = sumvals[0];

    // divide the whole row by the sum
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = x[i] / sum;
    }
}


__device__ float& vec_at(float4& vec, int index) {
    return reinterpret_cast<float*>(&vec)[index];
}

__device__ float vec_at(const float4& vec, int index) {
    return reinterpret_cast<const float*>(&vec)[index];
}

__global__ void softmax_forward_kernel5(float* out, float inv_temperature, const float* inp, int N, int T) {
    // inp, out shape: (N, T, T), where N = B * NH
    // fuses the multiplication by scale inside attention
    // directly autoregressive, so we only compute the lower triangular part
    // uses the online softmax algorithm
    assert(T % 4  == 0);
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N * T) {
        return;
    }
    int own_pos = idx % T;
    int pos_by_4 = own_pos / 4;

    // one row of inp, i.e. inp[idx, :] of shape (T,)
    const float* x = inp + idx * T;

    // not INF, so we don't get NaNs accidentally when subtracting two values.
    float maxval = -FLT_MAX;
    float sumval = 0.0f;

    const float4* x_vec = reinterpret_cast<const float4*>(x);
    for (int i = warp.thread_rank(); i < pos_by_4; i += warp.size()) {
        float4 v = x_vec[i];
        float old_maxval = maxval;
        for(int k = 0; k < 4; ++k) {
            maxval = fmaxf(maxval, vec_at(v, k));
        }
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        for(int k = 0; k < 4; ++k) {
            sumval += expf(inv_temperature * (vec_at(v, k) - maxval));
        }
    }

    if(4*pos_by_4 + warp.thread_rank() <= own_pos) {
        float old_maxval = maxval;
        maxval = fmaxf(maxval, x[4*pos_by_4 + warp.thread_rank()]);
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        sumval += expf(inv_temperature * (x[4*pos_by_4 + warp.thread_rank()] - maxval));
    }

    float global_maxval = cg::reduce(warp, maxval, cg::greater<float>{});
    sumval *= expf(inv_temperature * (maxval - global_maxval));

    float sum = cg::reduce(warp, sumval, cg::plus<float>{});
    float norm = 1.f / sum;

    // divide the whole row by the sum
    for (int i = warp.thread_rank(); i <= own_pos; i += warp.size()) {
        // recalculation is faster than doing the round-trip through memory.
        float ev = expf(inv_temperature * (__ldcs(x + i) - global_maxval));
        __stcs(out + idx * T + i, ev * norm);
    }
}


__global__ void attention_value_kernel1(float* out, const float* att, const float* inp,
                                       int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        int C3 = C*3;
        int hs = C / NH; // head size

        float* out_bth = out + b * T * C + t * C + h * hs;
        const float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
        for (int t2 = 0; t2 <= t; t2++) {
           const  float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
            float att_btht2 = att_bth[t2];
            for (int i = 0; i < hs; i++) {
                out_bth[i] += att_btht2 * value_t2[i];
            }
        }
    }
}

__global__
void attention_forward_kernel2(
    const float* Q,
    const float* K,
    const float* V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float* l,
    float* m,
    float* O
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    // Offset into Q,K,V,O,l,m - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for l and m

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d;  // size of Qi, Kj, Vj
    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size * 2];
    float* S = &sram[tile_size * 3];

    for (int j = 0; j < Tc; j++) {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++) {
            Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
        }
        __syncthreads();  // such that the inner loop can use the correct Kj, Vj

        for (int i = 0; i < Tr; i++)  {
            // if past the end of the sequence, break
            if (i * Br + tx >= N) {
                break;
            }

            // Load Qi to SRAM, l and m to registers
            for (int x = 0; x < d; x++) {
                Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
            }
            float row_m_prev = m[lm_offset + (Br * i) + tx];
            float row_l_prev = l[lm_offset + (Br * i) + tx];

            // S = QK^T, row_m = rowmax(S)
            // S[tx][y] = Sum_{x = 0}^{d-1} {Qi[tx][x] * Kj[y][x]}
            // row_m = Max_{y = 0}^{Bc-1} S[tx][y]
            // with causal masking
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // implement softmax with causal masking
            // P = exp(S - row_m), row_l = rowsum(P)
            // P[tx][y] = exp(S[tx][y] - row_m)
            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);
                row_l += S[(Bc * tx) + y];
            }

            // Compute new m and l
            float row_m_new = max(row_m_prev, row_m);
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

            // Write O, l, m to HBM
            for (int x = 0; x < d; x++) {
                float pv = 0;  // Pij * Vj
                for (int y = 0; y < Bc; y++) {
                    if (j * Bc + y >= N) {
                        break;
                    }
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = (1 / row_l_new) \
                    * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tx * d) + x]) \
                    + (__expf(row_m - row_m_new) * pv));
            }
            m[lm_offset + (Br * i) + tx] = row_m_new;
            l[lm_offset + (Br * i) + tx] = row_l_new;
        }
        __syncthreads();  // otherwise, thread can use the wrong Kj, Vj in inner loop
    }
}

__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]

    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = \
            (b * N * 3 * NH * d)
            +   (n * 3 * NH * d)
            +       (0 * NH * d)
            +          (nh_ * d)
            +                d_;

        q[idx] = inp[inp_idx];
        k[idx] = inp[inp_idx + NH * d];
        v[idx] = inp[inp_idx + 2 * (NH * d)];
    }
}

__global__ void unpermute_kernel(const float* inp, float *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}

__global__ void scale_kernel(float* inp, float scale, int B, int NH, int T) {
    // scales the pre-softmax attention scores by scale
    // and sets the autoregressive locations to -INFINITY
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * T * T) {
        int rest = idx % (NH * T * T);
        rest = rest % (T * T);
        int t2 = rest / T;
        int t = rest % T;
        if (t > t2) {
            inp[idx] = -INFINITY;
        } else {
            inp[idx] *= scale;
        }
    }
}

// direct translation of the CPU kernel. Each warp handles ont (b, h, t) combination.
// The important changes compared to the CPU version:
//  - each inner loop is handled by a warp
//  - don't write non-autoregressive parts
//  - reordered the last loops so that we can do all writing in the outer loop.
__global__ void attention_forward_fused1(float* out, float* preatt, float* att,
                                         const float* inp,
                                         int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q,K,V
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = C*3;
    int hs = C / NH; // head size
    float scale = 1.0 / sqrtf(hs);

    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int t = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    int h = blockIdx.y;
    int b = blockIdx.z;

    if(t >= T) return;

    const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
    float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
    float* att_bth = att + b*NH*T*T + h*T*T + t*T;

    // pass 1: calculate query dot key and maxval
    float maxval = -INFINITY;
    for (int t2 = 0; t2 <= t; t2++) {
        const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

        // (query_t) dot (key_t2)
        float val = 0.0f;
        for (int i = warp.thread_rank(); i < hs; i += warp.size()) {
            val += query_t[i] * key_t2[i];
        }
        val = cg::reduce(warp, val, cg::plus<float>{});
        val *= scale;
        maxval = max(maxval, val);
        if(warp.thread_rank() == 0) {
            preatt_bth[t2] = val;
        }
    }

    // pass 2: calculate the exp and keep track of sum
    float expsum = 0.0f;
    for (int t2 = warp.thread_rank(); t2 <= t; t2 += warp.size()) {
        float expv = expf(preatt_bth[t2] - maxval);
        expsum += expv;
    }

    expsum = cg::reduce(warp, expsum, cg::plus<float>{});

    float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

    // pass 3: normalize to get the softmax is combined with the next loop to reduce memory round-trips
    for (int t2 = warp.thread_rank(); t2 <= t; t2 += warp.size()) {
        att_bth[t2] = expf(preatt_bth[t2] - maxval) * expsum_inv;
    }

    // pass 4: accumulate weighted values into the output of attention
    float* out_bth = out + b * T * C + t * C + h * hs;
    for (int i = warp.thread_rank(); i < hs; i += warp.size()) {
        float o = 0.f;
        for (int t2 = 0; t2 <= t; t2++) {
            const float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C * 2; // +C*2 because it's value
            float att_btht2 = att_bth[t2];
            o += att_btht2 * value_t2[i];
        }
        out_bth[i] = o;
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void attention_forward1(float* out, float* preatt, float* att,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // attention calculation
    int total_threads = B * NH * T * T;
    int num_blocks = ceil_div(total_threads, block_size);
    attention_query_key_kernel1<<<num_blocks, block_size>>>(preatt, inp, B, T, C, NH);
    // softmax and value accumulation
    total_threads = B * T * NH;
    num_blocks = ceil_div(total_threads, block_size);
    attention_softmax_kernel1<<<num_blocks, block_size>>>(att, preatt, B, T, NH);
    attention_value_kernel1<<<num_blocks, block_size>>>(out, att, inp, B, T, C, NH);
}


void attention_forward2(float* out,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // TODO there should be no mallocs inside any of these functions!
    // not fixing this because we don't intend to use attention_forward2,
    // it seems to be way too slow as is

    // these are hardcoded to 32 for now
    const int Bc = 32;
    const int Br = 32;
    // renaming these to be consistent with the kernel
    // const int B = B;
    const int nh = NH;
    const int N = T;
    const int d = C / NH;
    // more
    const int Tc = ceil((float) N / Bc);
    const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);
    // create some temporary memory
    float* l;
    float* m;
    cudaCheck(hipMalloc(&l, B * nh * N * sizeof(float)));
    cudaCheck(hipMalloc(&m, B * nh * N * sizeof(float)));
    cudaCheck(hipMemset(l, 0, B * nh * N * sizeof(float)));
    cudaCheck(hipMemset(m, -10000.0f, B * nh * N * sizeof(float)));

    // calculate SRAM size needed per block, ensure we have enough shared memory
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi
    const int sram_size =
        (2 * col_tile_size * sizeof(float))  // SRAM size for Kj, Vj
        + (row_tile_size * sizeof(float))  // SRAM size for Qi
        + (Bc * Br * sizeof(float));  // SRAM size for S
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    if (sram_size > max_sram_size) {
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
        printf("SRAM size exceeds maximum shared memory per block\n");
        printf("Try decreasing col_tile_size or row_tile_size further\n");
        exit(1);
    }

    // grid and block dims
    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(Br);  // Br threads per block

    // okay so now, this kernel wants Q,K,V to all be of shape (B, nh, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, nh, d)
    // so we have to permute the tensor using a kernel with block_size
    float *q, *k, *v;
    cudaCheck(hipMalloc(&q, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&k, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&v, B * T * C * sizeof(float)));
    int total_threads = B * N * nh * d;
    int num_blocks = ceil_div(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, N, nh, d);

    // now actually call the flash attention kernel
    attention_forward_kernel2<<<grid_dim, block_dim, sram_size>>>(
        q, k, v,
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        l, m, out
    );

    // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    unpermute_kernel<<<num_blocks, block_size>>>(out, q, B, N, nh, d);
    cudaCheck(hipMemcpy(out, q, B * T * C * sizeof(float), hipMemcpyDeviceToDevice));

    // free memory
    cudaCheck(hipFree(l));
    cudaCheck(hipFree(m));
    cudaCheck(hipFree(q));
    cudaCheck(hipFree(k));
    cudaCheck(hipFree(v));
}

void attention_forward3(float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = ceil_div(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);

    // batched matrix multiply with cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            T, T, HS,
                            &alpha,
                            k, HS, T * HS,
                            q, HS, T * HS,
                            &beta,
                            preatt, T, T * T,
                            B * NH));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0f / sqrtf(HS);
    total_threads = B * NH * T * T;
    num_blocks = ceil_div(total_threads, block_size);
    scale_kernel<<<num_blocks, block_size>>>(preatt, scale, B, NH, T);

    // softmax. preatt is (B, NH, T, T) but we view it as (B * NH * T, T) and use the softmax kernel
    int softmax_block_size = 256;
    int grid_size = B * NH * T;
    size_t shared_mem_size = 2 * softmax_block_size / 32 * sizeof(float);
    softmax_forward_kernel4<<<grid_size, softmax_block_size, shared_mem_size>>>(att, preatt, B * NH * T, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            HS, T, T,
                            &alpha,
                            v, HS, T * HS,
                            att, T, T * T,
                            &beta,
                            vaccum, HS, T * HS,
                            B * NH));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = ceil_div(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
}

void attention_forward4(float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                        const float* inp,
                        int B, int T, int C, int NH,
                        const int block_size) {
    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = ceil_div(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);

    // batched matrix multiply with cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;

    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                                     HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     T, T, HS,
                                     &alpha,
                                     k, HS, T * HS,
                                     q, HS, T * HS,
                                     &beta,
                                     preatt, T, T * T,
                                     B * NH));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    int softmax_block_size = 256;
    int grid_size = ceil_div(B * NH * T * 32, softmax_block_size);
    softmax_forward_kernel5<<<grid_size, softmax_block_size>>>(att, scale, preatt, B * NH, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                                     HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     HS, T, T,
                                     &alpha,
                                     v, HS, T * HS,
                                     att, T, T * T,
                                     &beta,
                                     vaccum, HS, T * HS,
                                     B * NH));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = ceil_div(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
}


__global__ void softmax_forward_kernel5_lowp(floatX* out, float inv_temperature,
                                             const floatX* inp, int N, int T) {
    // inp, out shape: (N, T, T), where N = B * NH
    // fuses the multiplication by scale inside attention
    // directly autoregressive, so we only compute the lower triangular part
    // uses the online softmax algorithm
    assert(T % 4  == 0);
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N * T) {
        return;
    }
    int own_pos = idx % T;
    int pos_by_4 = own_pos / 4;

    // one row of inp, i.e. inp[idx, :] of shape (T,)
    const floatX* x = inp + idx * T;

    // not INF, so we don't get NaNs accidentally when subtracting two values.
    float maxval = -FLT_MAX;
    float sumval = 0.0f;

    // Same thing but without float4, one at a time
    for (int i = warp.thread_rank(); i < pos_by_4; i += warp.size()) {
        float old_maxval = maxval;
        for(int k = 0; k < 4; ++k) {
            maxval = fmaxf(maxval, (float)x[4*i + k]);
        }
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        for(int k = 0; k < 4; ++k) {
            sumval += expf(inv_temperature * ((float)x[4*i + k] - maxval));
        }
    }

    if(4*pos_by_4 + warp.thread_rank() <= own_pos) {
        float old_maxval = maxval;
        maxval = fmaxf(maxval, (float)x[4*pos_by_4 + warp.thread_rank()]);
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        sumval += expf(inv_temperature * ((float)x[4*pos_by_4 + warp.thread_rank()] - maxval));
    }

    float global_maxval = cg::reduce(warp, maxval, cg::greater<float>{});
    sumval *= expf(inv_temperature * (maxval - global_maxval));

    float sum = cg::reduce(warp, sumval, cg::plus<float>{});
    float norm = 1.f / sum;

    // divide the whole row by the sum
    for (int i = warp.thread_rank(); i <= own_pos; i += warp.size()) {
        // recalculation is faster than doing the round-trip through memory.
        float ev = expf(inv_temperature * ((float)__ldcs(x + i) - global_maxval));
        __stcs(out + idx * T + i, (floatX)(ev * norm));
    }
}

__global__ void permute_kernel_lowp(floatX* q, floatX* k, floatX* v,
                                    const float* inp,
                                    int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = \
            (b * N * 3 * NH * d)
            +   (n * 3 * NH * d)
            +       (0 * NH * d)
            +          (nh_ * d)
            +                d_;

        q[idx] = (floatX)inp[inp_idx];
        k[idx] = (floatX)inp[inp_idx + NH * d];
        v[idx] = (floatX)inp[inp_idx + 2 * (NH * d)];
    }
}

__global__ void unpermute_kernel_lowp(const floatX* inp, float *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = (float)inp[idx];
    }
}

void attention_forward5(float* out, floatX* vaccum, floatX* qkvr, floatX* preatt, floatX* att,
                        const float* inp,
                        int B, int T, int C, int NH,
                        const int block_size, bool skip_permute=false) {
    // FP16 version of kernel 4 (with permute/unpermute doing FP32<->FP16)
    // That permute can be skipped on perf runs to analyse its performance impact
    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    int HS = C / NH; // head size
    floatX *q = qkvr + 0 * B * T * C;
    floatX *k = qkvr + 1 * B * T * C;
    floatX* v = qkvr + 2 * B * T * C;

    int total_threads = B * NH * T * HS;
    int num_blocks = ceil_div(total_threads, block_size);
    if (!skip_permute || first_run_validation) {
        permute_kernel_lowp<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);
    }

    // IMPORTANT: alpha/beta are FP32 for HIPBLAS_COMPUTE_32F even if FP16 inputs/outputs
    // But need FP16 scale for HIPBLAS_COMPUTE_16F (no errors otherwise, just garbage results *sigh*)
    const float alpha = 1.0f;
    const float beta = 0.0f;
    const floatX alpha_lowp = (floatX)alpha;
    const floatX beta_lowp = (floatX)beta;
    void* alpha_ptr = CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F ? (void*)&alpha_lowp : (void*)&alpha;
    void* beta_ptr = CUBLAS_LOWP_COMPUTE == HIPBLAS_COMPUTE_16F ? (void*)&beta_lowp : (void*)&beta;

    // batched matrix multiply with cuBLAS
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     T, T, HS,
                                     alpha_ptr,
                                     k, CUBLAS_LOWP, HS, T * HS,
                                     q, CUBLAS_LOWP, HS, T * HS,
                                     beta_ptr,
                                     preatt, CUBLAS_LOWP, T, T * T,
                                     B * NH,
                                     CUBLAS_LOWP_COMPUTE,
                                     HIPBLAS_GEMM_DEFAULT));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0f / sqrtf(HS);
    int softmax_block_size = 256;
    int grid_size = ceil_div(B * NH * T * 32, softmax_block_size);
    softmax_forward_kernel5_lowp<<<grid_size, softmax_block_size>>>(att, scale, preatt, B * NH, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     HS, T, T,
                                     alpha_ptr,
                                     v, CUBLAS_LOWP, HS, T * HS,
                                     att, CUBLAS_LOWP, T, T * T,
                                     beta_ptr,
                                     vaccum, CUBLAS_LOWP, HS, T * HS,
                                     B * NH,
                                     CUBLAS_LOWP_COMPUTE,
                                     HIPBLAS_GEMM_DEFAULT));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = ceil_div(B * T * C, block_size);
    if(!skip_permute || first_run_validation) {
        unpermute_kernel_lowp<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
    }
}

#ifdef ENABLE_CUDNN
using graph_tensors_fwd = std::tuple<std::shared_ptr<fe::graph::Graph>,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // Q,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // K,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // V,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // Attn_scale,
                                     std::shared_ptr<fe::graph::Tensor_attributes>,  // O
                                     std::shared_ptr<fe::graph::Tensor_attributes>>; // Stats

// Need a cache because graph->build_operation_graph() is slow but everything else seems fast
using cache_type_fwd = std::unordered_map<std::size_t, graph_tensors_fwd>;

// Loosely based on cuDNN frontend samples functions and massively simplified
template <typename... Args>
auto lookup_cache_or_build_graph_fwd(Args... args) {
    static cache_type_fwd user_maintained_cache_fwd;
    auto [B, H, T, HS, is_inference_only] = std::make_tuple(args...);

    auto graph = std::make_shared<fe::graph::Graph>();
    graph->set_io_data_type(CUDNN_16BIT)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);

    // QKV is (B, T, 3, NH, HS) which cuDNN can handle directly without an external permute
    auto Q = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("Q")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T,  HS, 3 * H * HS, 1}));
    auto K = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("K")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto V = graph->tensor(fe::graph::Tensor_attributes()
                               .set_name("V")
                               .set_dim({B, H, T, HS})
                               .set_stride({3 * H * HS * T, HS, 3 * H * HS, 1}));
    auto attn_scale = graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("attn_scale")
                                .set_dim({1, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_is_pass_by_value(true)
                                .set_data_type(fe::DataType_t::FLOAT));

    auto sdpa_options = fe::graph::SDPA_attributes().set_name("flash_attention");
    sdpa_options.set_is_inference(is_inference_only);
    sdpa_options.set_attn_scale(attn_scale);
    sdpa_options.set_causal_mask(true);

    // Create the graph operation and get the output tensors back
    auto [O, stats] = graph->sdpa(Q, K, V, sdpa_options);

    // Output is (B, T, NH, HS) BF16/FP16 and stats for backward pass is (B, NH, T) FP32
    O->set_output(true).set_dim({B, H, T, HS}).set_stride({H * HS * T, HS, H * HS, 1});

    assert(stats == nullptr || is_inference_only == false);
    if (is_inference_only == false) {
        stats->set_output(true).set_data_type(fe::DataType_t::FLOAT)
                               .set_dim({B, H, T, 1})
                               .set_stride({H * T, T, 1, 1});
    }

    assert(graph->validate().is_good());
    auto key = graph->key();
    auto it = user_maintained_cache_fwd.find(key);
    if (it != user_maintained_cache_fwd.end()) {
        return it->second;
    }

    // Build the operation graph and execution part (this is the VERY SLOW PART)
    assert(graph->build_operation_graph(cudnn_handle).is_good());
    auto plans = graph->create_execution_plans({fe::HeurMode_t::A});
    assert(graph->check_support(cudnn_handle).is_good());
    assert(graph->build_plans(cudnn_handle).is_good());

    auto tuple = std::make_tuple(graph, Q, K, V, attn_scale, O, stats);
    user_maintained_cache_fwd.insert({key, tuple});
    return tuple;
}

// Used on first run only so we can validate against the CPU results
__global__ void fp32_to_lowp_kernel(floatX* out, const float* inp) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    out[idx] = (floatX)inp[idx];
}

__global__ void lowp_to_fp32_kernel(const floatX* inp, float *out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    out[idx] = (float)inp[idx];
}

void attention_forward_cudnn(floatX* out,  // output: (B, T, NH, HS)
                             float* stats, // output for backward pass: (B, NH, T)
                             floatX* inp,  // input: (B, T, 3, NH, HS) QKV
                             float* in_fp32,  // fp32 input
                             float* out_fp32, // fp32 output for validation
                             int B, int T, int C, int NH) {
    static bool first_run_validation = true;
    int HS = C / NH; // number of features per head
    bool is_inference_only = (stats == nullptr);

    // Convert from FP32 to FP16/BF16 on 1st run to get correct results
    const int block_size = 64; // smallest full occupancy block size on modern GPUs
    if (first_run_validation) {
        int total_threads = B * T * C * 3;
        assert(total_threads % block_size == 0);
        int num_blocks = total_threads / block_size;
        fp32_to_lowp_kernel<<<num_blocks, block_size>>>(inp, in_fp32);
    }

    // Get graph and tensors from cache (or generate it on first use)
    auto [graph, Q, K, V, attn_scale, O, softmax_stats] =
        lookup_cache_or_build_graph_fwd(B, NH, T, HS, is_inference_only);

    // Prepare all the tensor pointers for executing the graph
    void* devPtrQ = inp;
    void* devPtrK = (inp + C);
    void* devPtrV = (inp + 2 * C);
    float attn_scale_cpu = 1.0 / sqrtf(HS);
    void* devPtrO = out;

    // Build variant pack
    std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void*> variant_pack = {
        {Q, devPtrQ}, {K, devPtrK}, {V, devPtrV}, {attn_scale, &attn_scale_cpu}, {O, devPtrO}};

    // Add the stats tensor unless we are only doing inference (only needed for backward pass)
    if (is_inference_only == false) {
        variant_pack[softmax_stats] = stats;
    }

    // Reallocate the workspace if the required size is greater than the current workspace
    // By default, cuDNN uses up to 256MiB of workspace, so we don't want to just allocate the maximum
    if (graph->get_workspace_size() > cudnn_workspace_size) {
        if (cudnn_workspace_size > 0) {
            cudaCheck(hipFree(cudnn_workspace));
        }
        cudnn_workspace_size = graph->get_workspace_size();
        cudaCheck(hipMalloc(&cudnn_workspace, cudnn_workspace_size));
    }

    // Execute graph
    assert(graph->execute(cudnn_handle, variant_pack, cudnn_workspace).is_good());
    cudaCheck(hipGetLastError());

    // Optionally convert back from FP16/BF16 to FP32
    if (first_run_validation) {
        int total_threads = B * T * C;
        assert(total_threads % block_size == 0);
        int num_blocks = total_threads / block_size;
        lowp_to_fp32_kernel<<<num_blocks, block_size>>>(out, out_fp32);
    }
    cudaCheck(hipGetLastError());
    first_run_validation = false;
}

#endif // ENABLE_CUDNN

// kernel version dispatch
void attention_forward(int kernel_num,
                       float* out, float* stats, float* vaccum,
                       float* qkvr, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    switch (kernel_num) {
        case 1:
            attention_forward1(out, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 2:
            attention_forward2(out, inp, B, T, C, NH, block_size);
            break;
        case 3:
            attention_forward3(out, vaccum, qkvr, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 4:
            attention_forward4(out, vaccum, qkvr, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 5:
            attention_forward5(out, (floatX*)vaccum, (floatX*)qkvr,
                               (floatX*)preatt, (floatX*)att,
                               inp, B, T, C, NH, block_size, false);
            break;
        case 6: // skip permutes for perf passes (to analyse perf as if in/out were truly 16-bit)
            attention_forward5(out, (floatX*)vaccum, (floatX*)qkvr,
                               (floatX*)preatt, (floatX*)att,
                               inp, B, T, C, NH, block_size, true);
            break;
        #ifdef ENABLE_CUDNN
        case 10:
            // note: validation only cares about out, which is out_fp32 of the function
            // inp is hackily converted to FP16 into qkvr only on the first run
            // similarly, vaccum is converted to FP32 into out only on the first run
            attention_forward_cudnn((floatX*)vaccum, stats, (floatX*)qkvr, inp, out, B, T, C, NH);
            break;
        #endif
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}
// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    setup_main();

    int B = 8;
    int T = 1024;
    int C = 768;
    int NH = 12;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);

    // setup cuBLAS (and cuDNN if needed)
    hipblasCreate(&cublas_handle);
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    printf("enable_tf32: %d\n", enable_tf32);
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));

    #ifdef ENABLE_CUDNN
    checkCudnnErr(hipdnnCreate(&cudnn_handle));
    #endif

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* preatt = (float*)malloc(B * NH * T * T * sizeof(float));
    float* att = (float*)malloc(B * NH * T * T * sizeof(float));
    //float* inp = make_random_float(B * T * 3 * C, 10.0f);
    float* inp = make_random_float(B * T * 3 * C);

    // move to GPU
    float* d_out;
    float* d_stats; // for cuDNN
    float* d_vaccum;
    float* d_qkvr;
    float* d_preatt;
    float* d_att;
    float* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_stats, B * NH * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_vaccum, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_qkvr, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_preatt, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_att, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * 3 * C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);
    int block_sizes[] = {32, 64, 128, 256, 512};

    // Lower accuracy requirements for FP16 (1e-4f also too much for TF32 on kernels 3 & 4)
    float accuracy_threshold = (kernel_num <= 4) ? 1e-3f : 1e-2f;

    // first check the correctness of the kernel
    attention_forward_cpu(out, preatt, att, inp, B, T, C, NH);
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        attention_forward(kernel_num, d_out, d_stats, d_vaccum, d_qkvr, d_preatt, d_att, d_inp, B, T, C, NH, block_size);
        // all kernels should produce the correct output out
        // todo - make accuracy threshold dynamic and depend on FP16 vs FP32?
        validate_result(d_out, out, "out", B * T * C, accuracy_threshold);
        // but as for preatt and att, things get a bit more complicated:
        if (kernel_num != 2 && kernel_num < 5) {
            // kernel 2 (knowingly) fails att/preatt because it uses a different algorithm
            // that estimates the softmax online and never materializes preatt/att
            validate_result(d_att, att, "att", B * NH * T * T, accuracy_threshold);
        }
        if (kernel_num != 2 && kernel_num < 4) {
            // kernel 4 (knowingly) fails preatt because it fuses the scale normalization
            // into the softmax, so preatt is off by 1.0f / sqrt(HS)
            // but att and out (checked below) should match.
            validate_result(d_preatt, preatt, "preatt", B * NH * T * T, accuracy_threshold);
        }
    }
    printf("All results match. Starting benchmarks.\n\n");
    first_run_validation = false;

    // benchmark speed of the kernel
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 100;

        float elapsed_time = benchmark_kernel(repeat_times, attention_forward,
                                              kernel_num, d_out, d_stats, d_vaccum, d_qkvr, d_preatt, d_att,
                                              d_inp, B, T, C, NH, block_size);

        printf("block_size %4d | time %f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(out);
    free(preatt);
    free(att);
    free(inp);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_vaccum));
    cudaCheck(hipFree(d_qkvr));
    cudaCheck(hipFree(d_preatt));
    cudaCheck(hipFree(d_att));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_stats));
    hipblasDestroy(cublas_handle);

    #ifdef ENABLE_CUDNN
    hipdnnDestroy(cudnn_handle);
    if (cudnn_workspace_size > 0) {
        cudaCheck(hipFree(cudnn_workspace));
    }
    #endif

    return 0;
}