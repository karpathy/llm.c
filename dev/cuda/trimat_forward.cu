#include "hip/hip_runtime.h"
/*
Triangular matrix multiplication as in autoregressive attention. A short story.
by @ngc92

Compile:
nvcc -O3 --use_fast_math -lcublas -lcublasLt trimat_forward.cu -o trimat_forward -lcublas

Run:

cuBLAS baseline kernel
./trimat_forward 0

naive
./trimat_forward 1

registers
./trimat_forward 2

tri3
./trimat_forward 3

tri4
./trimat_forward 4
*/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

static float* d_qkvr;   // scratch for the cublas kernel

/*                    ** Chapter I - Introduction **
 *
 *  You are Trimul. You've always wanted to do fast matrix multiplication, but they said
 *  "Don't bother, big dumb Cublas is much faster than you!"
 *  "I don't need to be faster than Cublas", you replied, "I can be smarter. Cublas calculates
 *  the entire matrix, but I need only half. If I'm more than half as fast as Cublas, I'm
 *  going to win."
 *
 *  So to prove everyone wrong, you enter the TriMatlon, the most prestigious competition
 *  for anyone paying Attention.
 *
 *  Before you start preparing, lets have a look at the players involved
 *
 *  First, there is the Referee (`trimul_cpu`), slow and ponderous, but producing results
 *  beyond reproof.
 *  Then, there is Cublas. Cublas' mind is so inflexible, it doesn't actually comprehend
 *  what we are trying to do here, so Cublas has brought an assistant (`permute_kernel`)
 *  that translates the competition into a task that it can solve. But once it recognizes
 *  the problem, its muscle memory kicks in, and matrix products are produced faster than
 *  the eye can see. Stuck in its routine, Cublas doesn't realize the task is already
 *  finished with the lower triangle, though.
 *
 *  If you can do without an assistant, and can solve the right task, then that's your opportunity
 *  to shine!
 */


// taken from then attention forward pass
void trimul_cpu(float* out, const float* inp,
                int B, int T, int C, int NH) {
    // inp shape: (B, T, 3, NH, HS)
    // out shape: (B, NH, T, T)
    int C3 = C*3;
    int HS = C / NH; // head size
    float scale = 1.0 / sqrtf(HS);

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            for (int nh = 0; nh < NH; nh++) {
                // Q[b][nh][t][:] = inp[b][t][0][nh][:] (where : is the slice operator for hs)
                const float* query_t = inp + b * T * C3 + t * C3 + nh * HS;
                // out[b][nh][t][:]
                float* out_bth = out + b * NH * T * T + nh * T * T + t * T;

                // pass 1: calculate query dot key and maxval
                for (int t2 = 0; t2 <= t; t2++) {
                    // K[b][nh][t2][:] = inp[b][t2][1][nh][:]
                    const float* key_t2 = inp + b * T * C3 + t2 * C3 + nh * HS + C; // +C because it's key

                    // Q[b][nh][t][:] dot K[b][nh][t2][:]
                    float val = 0.0f;
                    for (int i = 0; i < HS; i++) {
                        val += query_t[i] * key_t2[i];
                    }
                    val *= scale;

                     // out[b][nh][t][t2] = val
                    out_bth[t2] = val;
                }
                for(int t2 = t + 1; t2 < T; ++t2) {
                    // causal mask, using NAN to supress warnings -> it could be -inf
                    // but it doesn't matter because in validate_result we ignore infinities/NANs
                    out_bth[t2] = NAN;
                }
            }
        }
    }
}

__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int T, int NH, int HS) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, T, HS)
    // but instead, we have a single tensor QKV (inp) of shape (B, T, 3, NH, HS)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh][t][hs] = inp[b][t][0][nh][hs]

    if (idx < B * NH * T * HS) {
        int b = idx / (NH * T * HS);
        int rest = idx % (NH * T * HS);
        int nh = rest / (T * HS);
        rest = rest % (T * HS);
        int t = rest / HS;
        int hs = rest % HS;

        int inp_idx = \
            (b * T * 3 * NH * HS)
            +   (t * 3 * NH * HS)
            +       (0 * NH * HS)
            +          (nh * HS)
            +                hs;

        q[idx] = inp[inp_idx];
        k[idx] = inp[inp_idx + NH * HS];
        v[idx] = inp[inp_idx + 2 * (NH * HS)];
    }
}


void trimul_cublas(float* preatt,
                   const float* inp,
                   int B, int T, int C, int NH) {
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float* q, * k, * v;
    q = d_qkvr + 0 * B * T * C;
    k = d_qkvr + 1 * B * T * C;
    v = d_qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = ceil_div(total_threads, 256);
    permute_kernel<<<num_blocks, 256>>>(q, k, v, inp, B, T, NH, HS);
    cudaCheck(hipGetLastError());

    // batched matrix multiply with cuBLAS
    const float alpha = 1.0f / sqrtf(HS);
    const float beta = 0.0f;
    // This schedules in parallel B*NH matmuls of shape q@k^t = (T, HS) @ (HS, T) = (T, T).
    // IMPORTANT NOTE: Cublas uses a column-major (and we use row-major in our codebase) representation,
    // so this call might look confusing to you if you look at the `hipblasSgemmStridedBatched` signature.
    //
    // In order to avoid having to do an additional transpose operation after this func call,
    // we need to pass in K as the first argument and Q as the second argument, which might make you think we're computing K^T @ Q.
    // That combined with the shapes we got after the permute kernel - (B, NH, T, HS) (I'll omit B, NH for brevity going forward)
    // and you might think we end up with (HS, T) @ (T, HS) = (HS, HS).
    // This is not the case. :)
    //
    // Cublas sees our row-major matrix (T, HS) as (HS, T), hence we set the lead dimensions to HS (see function signature).
    // We transpose K and end up computing K^T @ Q = (T, HS) @ (HS, T) = (T, T).
    // If you were to interpret the above formula K^T @ Q you might think we end up with:
    // -----------------------------------
    // k1.dot(q1) k1.dot(q2) ... k1.dot(qT)
    // k2.dot(q1) k2.dot(q2) ... k2.dot(qT)
    // ...
    // kT.dot(q1) kT.dot(q2) ... kT.dot(qT)
    // -----------------------------------
    // But as I mentioned, Cublas is column-major!
    // So given that the dot product is symmetric we can write k1.dot(q1) as q1.dot(k1) and transposing the above
    // representation we can see what we actually end up with in the row-major format:
    // -----------------------------------
    // q1.dot(k1) q1.dot(k2) ... q1.dot(kT)
    // q2.dot(k1) q2.dot(k2) ... q2.dot(kT)
    // ...
    // qT.dot(k1) qT.dot(k2) ... qT.dot(kT)
    // -----------------------------------
    // which is exactly what we wanted! :)
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                                          HIPBLAS_OP_T, HIPBLAS_OP_N,
                                          T, T, HS,
                                          &alpha,
                                          k, HS, T * HS,
                                          q, HS, T * HS,
                                          &beta,
                                          preatt, T, T * T,
                                          B * NH));
}

/*                    ** Chapter II - Getting a Team **
 *
 *  OK, you've registered for the competition, now what to do. TriMatlon is a team competition, so first, you need
 *  to figure out what kind of team you need, and how to organize it. The individual instances and heads of the
 *  problem are completely independent, so you just can send separate teams to work there completely independently.
 *
 *  To figure out how to organize each team, you take out your spyglass (`Nsight Compute`) and look how the Cublas teams
 *  are handling their work.
 *  Turns out, you need 256 athletes in each group, and those handle 128 x 128 of the tasks. They work together in
 *  a tight square formation, 16 wide and 16 deep.
 *
 *  So, you went out and got your 100 000 friends, and split them into groups (`trimul_launcher`). Each group gets
 *  informed about where they should work (`trimul_global`) and goes off to do their thing (`matmul_tri_naive`).
 *  Let's observe how we're doing.
 */

// using creates an alias for a function pointer
using matmul_fn_ptr = void(*)(float* p, int PS, const float* k, int KS, const float* q, int QS, int T, int HS, float alpha);

template<matmul_fn_ptr matmul_tri>
__global__ void __launch_bounds__(256, 2) trimul_global(float* out, const float* inp, int T, int C, int NH) {
    // skip above the diagonal
    if(blockIdx.y > blockIdx.x)
        return;

    // set up indices
    int C3 = C*3;
    int HS = C / NH; // head size
    float scale = 1.0 / sqrtf(HS);

    // we put the "batch x head" dimension into the z block index.
    int b = blockIdx.z / NH;
    int nh = blockIdx.z % NH;

    // Get the base address for the current batch and head
    // shapes -> inp (B, T, 3, NH, HS), Q (B, NH, T, HS), K (B, NH, T, HS)
    const float* q = inp + b * T * C3 + nh * HS;  // Q[b][nh][:][:] = inp[b][:][0][nh][:]
    const float* k = inp + b * T * C3 + nh * HS + C;  // K[b][nh][:][:] = inp[b][:][1][nh][:]
    float* r = out + (b*NH + nh)*T*T;  // out[b][nh][:][:]

    // start the multiplication
    matmul_tri(r, T, k, C3, q, C3, T, HS, scale);
}

template<matmul_fn_ptr matmul_tri>
void trimul_launcher(float* out, const float* inp, int B, int T, int C, int NH) {
    // we assume nice shapes here. Let's not make the code a mess by supporting weird shapes that you
    // wouldn't want to use anyway.
    assert(T % 128 == 0);
    // No need to ceil_div, if it's not a multiple of 128, we would get wrong results anyway.
    trimul_global<matmul_tri><<<dim3(T / 128, T / 128, NH * B), dim3(16, 16)>>>(out, inp, T, C, NH);
    cudaCheck(hipGetLastError());
}

/*                     ** Chapter III - ... **
 *
 *  You go over to the playing field. On one end of the field, there is a huge pile of funnily shaped cookie cutters.
 *  Some in the shape of animals, some in the shape of a landscape. Each group of workers has assigned some runners,
 *  fetching the cookie cutters for them. The workers seem very relaxing, chatting with each other, lounging about.
 *  You focus in on one of them.
 *
 *  He seems to be giving an instruction to a runner, and then turns back to reading a novel. The runner, meanwhile,
 *  crosses the field and back, handing him an elephant shape. Then she's off again to pick up a savannah background.
 *  Having received the two shapes, pressed them into the dough, and makes an elephant-in-the-savannah cookie. He hands
 *  the cutters back to the runner. "Can you please fetch me an elephant and a jungle next?"
 *  While she's on her way, he takes a sip off his cocktail.
 *  This time, she's making only one trip, keeping the elephant in her pocket (_Cache_). Still, it seems to take forever.
 *  You keep observing:
 *  - Elephant and zoo
 *  - Elephant and island
 *  ...
 *  - Lion and savannah
 *  - Lion and jungle
 *  - Lion and zoo
 *  ...
 *
 *  The worker has his poor runner fetch the same things over and over again, looking like she's about to faint from exhaustion.
 *  Even though she realizes this and always keeps one of them in her pocket, there is so much running,
 *  and little actual work happening.
 *
 *  Clearly, this isn't going to be effective, so you call a team meeting.
 */

// baseline implementation: 20 ms
__device__ void matmul_tri_naive(float* p, int PS, const float* k, int KS, const float* q, int QS, int T, int HS, float alpha) {
    // coordinate system:
    // | - - - - - > j
    // |
    // |
    // v
    // i
    // get coordinates of our block - each thread is responsible for a single 8x8 block.
    int i_base = 128 * blockIdx.x + 8 * threadIdx.x;
    int j_base = 128 * blockIdx.y + 8 * threadIdx.y;

    // One more check to skip the upper diagonal in blocks that are on the diagonal.
    // Note: we deliberately waste some compute on the jagged diagonal i.e. elements that belong
    // to the upper triangle that should be masked out. This will be ignored due to the causal mask
    // in the reference CPU implementation when used in the `validate_result` function.
    // Alternatively this check should be done in the nested for loop below -> if (i > j) return.
    if(j_base > i_base)
        return;

    // Simple nested loop that calculates 8x8 results in one thread.
    for(int io = 0; io < 8; ++io) {
        int i = i_base + io;
        for(int jo = 0; jo < 8; ++jo) {
            int j = j_base + jo;
            float val = 0;
            for (int s = 0; s < HS; ++s) {
                val += q[i * QS + s] * k[j * KS + s];
            }
            p[i * PS + j] = val * alpha;
        }
    }
}

/*                     ** Chapter IV - ... **
 *
 *  Each worker is producing 64 combined cookies from 8 animals and 8 landscapes. They send their runners 64 times
 *  to fetch the corresponding shapes. This is terribly inefficient; The runners need a minute or so for each trip,
 *  but making a cookie can be done in just a second.
 *
 *  "Let's try something different tomorrow: Just get all 16 cookie cutters that you need, and do all 64 combinations
 *  of them! See all this free space on your workbench (_registers_), you can keep them all there for easy access."
 *
 *  The next morning, you come back to the field for another practice session. Initially, there is bustling activity
 *  with the runners, picking up 16 shapes for each worker. But then, the workers have to put down their newspapers
 *  and start making cookies. Now there are 64 combinations, so it takes them a full minute.
 *
 *  Not all groups of workers are equally fast. When the first group finishes with all animal-landscape combinations,
 *  they already start asking the runners for the next set of cookie cutters, combining plants and houses. Even though
 *  the workers are much busier than before, they are still spending most of their time just waiting.
 *
 *  Still, instead of being busy for 20 hours, your team is now done with the task in just 3h 30 minutes; already, this
 *  is five times faster.
 *
 *  You think to yourself: "Why should we stop at 8 x 8 combinations? Lets to 16 x 16, that's only twice the work for
 *  the runners, but four times as much for the actual workers."
 *  You head over to the baking area, and make that suggestion to one of your team leaders.
 *  "In theory, that sounds great", she agrees, "but see, we only have limited space on our workbenches (_registers_).
 *  There is still some room left, but we simply cannot bake 256 cookies at the same time, sorry."
 *
 *  A different strategy is needed, then.
 */

// reorganize loops to enable data reuse: 3.5 ms
__device__ void matmul_tri_registers(float* p, int PS, const float* k, int KS, const float* q, int QS, int T, int HS, float alpha) {
    int i_base = 128 * blockIdx.x + 8 * threadIdx.x;
    int j_base = 128 * blockIdx.y + 8 * threadIdx.y;

    if (j_base > i_base)
        return;

    // shift our pointers to the sub-block this thread is responsible for
    q += i_base * QS;
    k += j_base * KS;
    p += i_base * PS + j_base;

    float vals[8][8] = {};
    for (int hs = 0; hs < HS; ++hs) {
        float lhs[8];
        float rhs[8];
        for (int u = 0; u < 8; ++u) {
            lhs[u] = q[u * QS + hs];
            rhs[u] = k[u * KS + hs];
        }

        for (int i = 0; i < 8; ++i) {
            for (int j = 0; j < 8; ++j) {
                vals[i][j] += lhs[i] * rhs[j];
            }
        }
    }

    for (int i = 0; i < 8; ++i) {
        for (int j = 0; j < 8; ++j) {
            p[i * PS + j] = vals[i][j] * alpha;
        }
    }
}

/*                     ** Chapter IV - By the Bucketload **
 *
 *  Despite the hectic activity, you pick out one of the runners. "Why are you always brining just one shape? Wouldn't
 *  it be much more efficient if you took more than one?"
 *  "Of course", the runner answers, "but they've asked me for an elephant, a lion, a zebra, and a goldfish. These
 *  are all over the place, I can't just pick them up at one spot (_strided acccess_).
 *  "But the lion is right next to the palm tree. You could bring those two together?", you confirm.
 *  "Yes", he says, "if they just asked for the different categories at the same time, that would make things
 *  so much easier. See, I have this bucket, I could carry lots of things in one go if I could just scoop them up
 *  from the same place (_coalesced access_).
 *
 *  OK, then lets fetch the first animal, first plant, first vehicle, and first landmark shape in one go (_vectorized load_).
 *  [Here, the metaphor breaks down a bit: Since we're accumulating all the results, getting more data at the same time
 *  depth-wise doesn't require more space on the workbench. We're stacking the cookies!]
 *
 *  You also streamline the shape combination further. Instead of picking up all animals and landscapes at one, it is
 *  more efficient, using less workbench space, to just pick up all animals. Then, you get one landscape, combine it
 *  will all animals, get the next landscape, combine, and so on.
 *
 *  In this way, instead of 2 x 8 x 4 cookie cutters that take up space, you only need (8+1) x 4 at the same time.
 *
 *  With these optimizations, you are down to 100 minutes for this task. Still slower than Cublas, but not by much.
 *
 *  In the arena, each team also has access to a small storage hut, much closer to their workbenches than the piles of
 *  cookie cutters on the other side. Cublas is using them heavily, so maybe you should, too.
 */

// convenient helper functions to make the code below more readable
__device__ float4 ld_vec(const float* address) {
    return *reinterpret_cast<const float4*>(address);
}

__device__ void st_vec(float* address, float4 val) {
    *reinterpret_cast<float4*>(address) = val;
}

// vector instructions for coalesced memory access: 1.7 ms
__device__ void matmul_tri3(float* p, int PS, const float* k, int KS, const float* q, int QS, int T, int HS, float alpha) {
    // Same logic as previous kernel we just load in float4 to improve coalescing
    int i_base = 128 * blockIdx.x + 8 * threadIdx.x;
    int j_base = 128 * blockIdx.y + 8 * threadIdx.y;

    if (j_base > i_base)
        return;

    // shift our pointers to the sub-block this thread is responsible for
    q += i_base * QS;
    k += j_base * KS;
    p += i_base * PS + j_base;

    float vals[8][8] = {};
    for (int hs = 0; hs < HS; hs += 4) {
        // load in float4 to improve coalescing
        float4 rhs[8];
        for (int u = 0; u < 8; ++u) {
            rhs[u] = ld_vec(k + u * KS + hs);
        }

        for (int i = 0; i < 8; ++i) {
            // no need to keep lhs around for the i loop, it's only reused in the j loop anyway.
            float4 lhs = ld_vec(q + i * QS + hs);
            for (int j = 0; j < 8; ++j) {
                vals[i][j] += lhs.x * rhs[j].x;
                vals[i][j] += lhs.y * rhs[j].y;
                vals[i][j] += lhs.z * rhs[j].z;
                vals[i][j] += lhs.w * rhs[j].w;
            }
        }
    }

    for (int i = 0; i < 8; ++i) {
        for (int j = 0; j < 8; j += 4) {
            float4 result;
            result.x = vals[i][j + 0] * alpha;
            result.y = vals[i][j + 1] * alpha;
            result.z = vals[i][j + 2] * alpha;
            result.w = vals[i][j + 3] * alpha;
            st_vec(p + i * PS + j, result);
        }
    }
}

/*                     ** Chapter V - Sharing is Caring **
 *
 *  You take a look around the shed, and see that there are 32 shelves there. They are much larger than the workbenches,
 *  giving you enough space for all the cookie cutters needed by the entire team.
 *
 *  Within the team, workers have banded together in groups of 32. They are always doing the same thing, reducing the
 *  amount of effort required for coordination. However, that also means that if you send them all to pick up different
 *  cookie cutters from the same shelf, they will have to wait and queue up (_shared memory bank conflict_).
 *
 *  In order to achieve maximum efficiency, we send the runners fetching cutters with the maximum bucket size: 32 different
 *  categories at the same time.
 *
 *  [I'm having trouble getting the specifics into the story in a sensible way. For now, please read the code for more
 *  details.]
 *
 */
__device__ void matmul_tri4(float* p, int PS, const float* k, int KS, const float* q, int QS, int T, int HS, float alpha) {
    int i_base = 128 * blockIdx.x + 8 * threadIdx.x;
    int j_base = 128 * blockIdx.y + 8 * threadIdx.y;

    // we need all threads for loading data, so none of them can chicken out early, even
    // if they are not responsible for any useful result.
    if (blockIdx.y > blockIdx.x)
        return;

    q += 128 * blockIdx.x * QS;
    k += 128 * blockIdx.y * KS;

    __shared__ float lhs_s[128][32];
    __shared__ float rhs_s[128][32];

    float vals[8][8] = {};
    for (int so = 0; so < HS; so += 32) {
        // Read a large slice of the input, worked on together by all threads.
        // They are organized differently for this part. We want to ensure
        // fully coalesced loads, so we let a single warp handle consecutive
        // addresses, which means we need to combine two threadIdx.y values
        // in one read operation.
        // note: threads may read data here that they don't need themselves.
        //       this really is a block-level operation.
        // note2: 16x16 threads (i.e. the block) will, through this for loop, fetch 32 dims from 128 keys and 128 queries
        // i.e. from Q/K, of shape (T, HS) take q[:128, so*32:(so+1)*32] and k[:128, so*32:(so+1)*32]
        __syncthreads();
        for(int y = threadIdx.y / 2; y < 128; y += 8) {
            int xo = (threadIdx.y % 2) * 16;
            lhs_s[y][threadIdx.x + xo] = q[y * QS + so + threadIdx.x + xo];
            rhs_s[y][threadIdx.x + xo] = k[y * KS + so + threadIdx.x + xo];
        }
        __syncthreads();

        // Now we compute a partial dot product (only 32 dims) for all combinations of keys and queries (128x128).
        // Each thread does 8x8 of these partial dot products.
        // E.g. thread (0,0) covers queries 0-7 and keys 0-7. More generally first row of threads
        // (0,:) covers queries 0-7 with keys 0-127 and so on.
        // In the next iterations of the outer (`so`) loop we'll be accumulating values to `vals` until we
        // get the full dot product. We then later deposit it into the output matrix for all 8x8 blocks
        // that are below the diagonal.
        for (int si = 0; si < 32; ++si) {
            float rhs[8];
            for (int u = 0; u < 8; ++u) {
                rhs[u] = rhs_s[u + 8 * threadIdx.y][(si + threadIdx.x) % 32];
            }

            for (int ii = 0; ii < 8; ++ii) {
                float lhs = lhs_s[ii + 8 * threadIdx.x][(si + threadIdx.x) % 32];
                for (int ji = 0; ji < 8; ++ji) {
                    vals[ii][ji] += lhs * rhs[ji];
                }
            }
        }
    }

    // don't write above the diagonal
    if (j_base > i_base)
        return;

    for (int ii = 0; ii < 8; ++ii) {
        for (int ji = 0; ji < 8; ji += 4) {
            int i = i_base + ii;
            int j = j_base + ji;
            float4 result;
            result.x = vals[ii][ji + 0] * alpha;
            result.y = vals[ii][ji + 1] * alpha;
            result.z = vals[ii][ji + 2] * alpha;
            result.w = vals[ii][ji + 3] * alpha;
            st_vec(p + i * PS + j, result);
        }
    }
}

/*                     ** Chapter VI - Competition Day **
 *
 * Finally, you feel ready to take on Cublas. You hand out tickets to the event for you friends to see.
 *
 *    ---------------------------------------------------------------------------------
 *    |           CuBLAS vs TriMul - Fight of the Century                             |
 *    |                                                                               |
 *    |   Ticket code:                                                                |
 *    |   > nvcc -O3 --use_fast_math trimat_forward.cu -o trimat_forward -lcublas     |
 *    |   > ./trimat 4                                                                |
 *    |                                                                               |
 *    ---------------------------------------------------------------------------------
 */

void trimul_gpu(int kernel_num,
                float* out,  const float* inp,
                int B, int T, int C, int NH) {
    switch (kernel_num) {
        case 0:
            trimul_cublas(out, inp, B, T, C, NH);
            break;
        case 1:
            trimul_launcher<matmul_tri_naive>(out, inp, B, T, C, NH);
            break;
        case 2:
            trimul_launcher<matmul_tri_registers>(out, inp, B, T, C, NH);
            break;
        case 3:
            trimul_launcher<matmul_tri3>(out, inp, B, T, C, NH);
            break;
        case 4:
            trimul_launcher<matmul_tri4>(out, inp, B, T, C, NH);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}



int main(int argc, char **argv) {
    setup_main();

    int B = 8;
    int T = 1024;
    int C = 768;
    int NH = 12;

    // create host memory of random numbers
    float* out = (float*)malloc(B * NH * T * T * sizeof(float));
    float* inp = make_random_float(B * T * 3 * C);

    // move to GPU
    float* d_out;
    float* d_inp;
    cudaCheck(hipMalloc(&d_out, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * 3 * C * sizeof(float), hipMemcpyHostToDevice));

    // buffer for cublas
    cudaCheck(hipMalloc(&d_qkvr, B * T * 3 * C * sizeof(float)));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    trimul_cpu(out, inp, B, T, C, NH);
    trimul_gpu(kernel_num, d_out, d_inp, B, T, C, NH);
    validate_result(d_out, out, "out", B * NH * T * T, 1e-4f);

    printf("All results match. Starting benchmarks.\n\n");

    // benchmark speed of the kernel
    int repeat_times = 100;

    float elapsed_time = benchmark_kernel(repeat_times, trimul_gpu,
                                          kernel_num, d_out, d_inp,
                                          B, T, C, NH);


    float cublas_time = benchmark_kernel(repeat_times, trimul_gpu,
                                         0, d_out, d_inp,
                                         B, T, C, NH);

    printf("time %.2f ms vs %.2f ms for CuBLAS\n", elapsed_time, cublas_time);

    // free memory
    free(out);
    free(inp);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_qkvr));
    hipblasDestroy(cublas_handle);

    return 0;
}
