#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net trained in raw CUDA
Non-trivial notes to be aware of:

We are being clever in the backward pass to conserve memory.
In particular, all parameters use a += in the backward pass, so we
can later do gradient accumulation. But all activations have = instead of +=
because these are faster (just read, no write). This is okay for all activations
except for those in the residual stream, where the gradients have to add. We make
sure that those parts work out ok and that we do a += as necessary. E.g.,
the layernorms are connected to the residuals so we += in layernorm backward.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <float.h>
#include <string.h>
#include <unistd.h>

// GPU / CUDA related
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
// our own utilities
// defines: fopenCheck, freadCheck, fcloseCheck, fseekCheck, mallocCheck
#include "llmc/utils.h"
// defines: tokenizer_init, tokenizer_decode, tokenizer_free
#include "llmc/tokenizer.h"
// defines: dataloader_init, dataloader_reset, dataloader_next_batch, dataloader_free
#include "llmc/dataloader.h"

// ----------------------------------------------------------------------------
// CUDA utils

// convenience macro for calculating grid/block dimensions for kernels
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// CUDA error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// cuBLAS error checking
void cublasCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("[cuBLAS ERROR]: %d %s %d\n", status, file, line);
        exit(EXIT_FAILURE);
    }
}
#define cublasCheck(status) { cublasCheck((status), __FILE__, __LINE__); }

static hipblasComputeType_t cublas_compute_type;
hipblasHandle_t cublas_handle;

namespace cg = cooperative_groups;

// ----------------------------------------------------------------------------
// all the kernels

__device__ inline float4 add_float4(const float4& a, const float4& b) {
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

// use of float4 leads to using 128-bit LDG / STG instructions in SASS,
// very helpful in memory-bound kernels like encoder_forward
__global__ void encoder_forward_kernel3(float4* out,
                               const int* inp, const float4* wte, const float4* wpe,
                               int B, int T, int C) {
    int C4 = C / 4;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C4;
    if (idx < N) {
        int bt = idx / C4;
        int b = bt / T;
        int t = bt % T;
        int c4 = idx % C4;
        int ix = inp[b * T + t];
        out[b * T * C4 + t * C4 + c4] = add_float4(wte[ix * C4 + c4], wpe[t * C4 + c4]);
    }
}

// really bad naive kernel with atomicAdd
__global__ void encoder_backward_kernel(float* dwte, float* dwpe,
                                        const float* dout, const int* inp,
                                        int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        const float* dout_btc = dout + b * T * C + t * C + c;
        float* dwte_ix = dwte + ix * C + c;
        float* dwpe_tc = dwpe + t * C + c;

        atomicAdd(dwte_ix, *dout_btc);
        atomicAdd(dwpe_tc, *dout_btc);
    }
}

__global__ void layernorm_forward_kernel3(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        sum += x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float m = sum / C;
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }

    // rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float s = rsqrtf(sum / C + 1e-5f);
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }

    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * (__ldcs(x+c) - m);
        __stcs(o+c, n * weight[c] + bias[c]);
    }
}

__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
        q[idx] = __ldcs(&inp[inp_idx]);
        k[idx] = __ldcs(&inp[inp_idx + NH * d]);
        v[idx] = __ldcs(&inp[inp_idx + 2 * (NH * d)]);
    }
}

__global__ void permute_kernel_backward(float* dinp,
                                        const float* dq, const float* dk, const float* dv,
                                        int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
        dinp[inp_idx] = dq[idx];
        dinp[inp_idx + NH * d] = dk[idx];
        dinp[inp_idx + 2 * (NH * d)] = dv[idx];
    }
}

__global__ void unpermute_kernel(float* inp, float *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = __ldcs(&inp[idx]);
    }
}

__global__ void unpermute_kernel_backward(float* dinp, const float *dout, int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;
        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        dinp[idx] = dout[other_idx];
    }
}

__device__ float& vec_at(float4& vec, int index) {
    return reinterpret_cast<float*>(&vec)[index];
}

__device__ float vec_at(const float4& vec, int index) {
    return reinterpret_cast<const float*>(&vec)[index];
}

__global__ void softmax_forward_kernel5(float* out, float inv_temperature, const float* inp, int N, int T) {
    // inp, out shape: (N, T, T), where N = B * NH
    // fuses the multiplication by scale inside attention
    // directly autoregressive, so we only compute the lower triangular part
    // uses the online softmax algorithm
    assert(T % 4  == 0);
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    // micro-optimization: we iterate backwards so that
    // after the softmax backward operation completes, the cache retains the
    // part of the matrix close to the upper left corner, which benefits the
    // matmul operation that immediately follows.
    // int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank(); // forward order
    int idx = (gridDim.x - blockIdx.x - 1) * warp.meta_group_size() + warp.meta_group_rank(); // backward order
    if(idx >= N * T) {
        return;
    }
    int own_pos = idx % T;
    int pos_by_4 = own_pos / 4;

    // one row of inp, i.e. inp[idx, :] of shape (T,)
    const float* x = inp + idx * T;

    // not INF, so we don't get NaNs accidentally when subtracting two values.
    float maxval = -FLT_MAX;
    float sumval = 0.0f;

    const float4* x_vec = reinterpret_cast<const float4*>(x);
    for (int i = warp.thread_rank(); i < pos_by_4; i += warp.size()) {
        float4 v = x_vec[i];
        float old_maxval = maxval;
        for(int k = 0; k < 4; ++k) {
            maxval = fmaxf(maxval, vec_at(v, k));
        }
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        for(int k = 0; k < 4; ++k) {
            sumval += expf(inv_temperature * (vec_at(v, k) - maxval));
        }
    }

    if(4*pos_by_4 + warp.thread_rank() <= own_pos) {
        float old_maxval = maxval;
        maxval = fmaxf(maxval, x[4*pos_by_4 + warp.thread_rank()]);
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        sumval += expf(inv_temperature * (x[4*pos_by_4 + warp.thread_rank()] - maxval));
    }

    float global_maxval = cg::reduce(warp, maxval, cg::greater<float>{});
    sumval *= expf(inv_temperature * (maxval - global_maxval));

    float sum = cg::reduce(warp, sumval, cg::plus<float>{});
    float norm = 1.f / sum;

    // divide the whole row by the sum
    for (int i = warp.thread_rank(); i <= own_pos; i += warp.size()) {
        // recalculation is faster than doing the round-trip through memory.
        float ev = expf(inv_temperature * (__ldcs(x + i) - global_maxval));
        __stcs(out + idx * T + i, ev * norm);
    }
}

__global__ void residual_forward_kernel(float* out, float* inp1, float* inp2, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = __ldcs(&inp1[idx]) + __ldcs(&inp2[idx]);
    }
}

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)
__global__ void gelu_forward_kernel(float* out, const float* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
    }
}

__global__ void gelu_backward_kernel(float* dinp, const float* inp, const float* dout, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float x = inp[i];
        float cube = 0.044715f * x * x * x;
        float tanh_arg = GELU_SCALING_FACTOR * (x + cube);
        float tanh_out = tanhf(tanh_arg);
        float coshf_out = coshf(tanh_arg);
        float sech_out = 1.0f / (coshf_out * coshf_out);
        float local_grad = 0.5f * (1.0f + tanh_out) + x * 0.5f * sech_out * GELU_SCALING_FACTOR * (1.0f + 3.0f * 0.044715f * x * x);
        dinp[i] = local_grad * dout[i];
    }
}

// this kernel performs a column-wise reduction over dout, in PyTorch equivalent to:
// dbias = dout.sum((0,1))
// the idea is to employ one block to reduce along several columns,
// where each block has a width of 32 columns to ensure coalesced access.
// at the end we accumulate the reductions performed by the warps in each block via shared memory
__global__ void matmul_backward_bias_kernel4(float* dbias, const float* dout, int B, int T, int OC) {
    // this kernel is launched with 1D grid_dim of OC/32
    // for example let's say block_size is 128
    extern __shared__ float smem[]; // of size block_size (128)
    const int warp_id = threadIdx.x / warpSize; // warp index in the block, 0,1,2,3
    const int lane_id = threadIdx.x % warpSize; // thread index in the warp, 0,1,2,...,31
    const int tl = blockIdx.x * warpSize; // pointer to the start column for this block
    const int vstep = blockDim.x / warpSize; // number of warps in a block, e.g. 4

    // pointer to the start of the column for one lane of threads
    // so e.g. 4 threads (of the same lane_id) will reduce this one column
    const float* dout_col = dout + tl + lane_id;

    // column reductions by looping through the rows
    // each of the 4 threads offsets by its warp_id and then skips by vstep
    // together these 4 threads cover all B*T rows of this (lane_id) column
    // importantly, consecutive threads (in threadId) are processing adjacent columns,
    // leading to a coalesced memory access pattern
    float dout_sum = 0.0f;
    for (int row = warp_id; row < B * T; row += vstep) {
        dout_sum += dout_col[row * OC];
    }
    smem[lane_id + warp_id * warpSize] = dout_sum;
    __syncthreads();

    // warp_id 0 reduces the shared memory column-wise, linearly
    dout_sum = 0.0f;
    if (warp_id == 0) {
        for (int j = 0; j < vstep; j++) {
            dout_sum += smem[lane_id + j * warpSize];
        }
        dbias[tl + lane_id] += dout_sum;
    }
}

// uses shared memory instead for the reduces
__global__ void layernorm_backward_kernel2(float* dinp, float* dweight, float* dbias,
                                           const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                                           int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2 * C

    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    int N = B * T;
    if(idx >= N) { return; } // thread guards

    int b = idx / T;
    int t = idx % T;

    const float* dout_bt = dout + b * T * C + t * C;
    const float* inp_bt = inp + b * T * C + t * C;
    float* dinp_bt = dinp + b * T * C + t * C;
    const float mean_bt = mean[b * T + t];
    const float rstd_bt = rstd[b * T + t];

    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;

    // init shared memory to zero
    #pragma unroll
	for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    __syncthreads();

    // first: two reduce operations
    float dnorm_mean = 0.0f;
    float dnorm_norm_mean = 0.0f;
    for (int i = warp.thread_rank(); i < C; i  += warp.size()) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        dnorm_mean += dnorm_i;
        dnorm_norm_mean += dnorm_i * norm_bti;
    }
    dnorm_mean = cg::reduce(warp, dnorm_mean, cg::plus<float>{});
    dnorm_norm_mean = cg::reduce(warp, dnorm_norm_mean, cg::plus<float>{});
    dnorm_mean = dnorm_mean / C;
    dnorm_norm_mean = dnorm_norm_mean / C;

    // now iterate again and accumulate all the gradients
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        // gradient contribution to bias
        atomicAdd(&dbias_shared[i], dout_bt[i]);
        // gradient contribution to weight
        atomicAdd(&dweight_shared[i], norm_bti * dout_bt[i]);
        // gradient contribution to input
        float dval = 0.0f;
        dval += dnorm_i; // term 1
        dval -= dnorm_mean; // term 2
        dval -= norm_bti * dnorm_norm_mean; // term 3
        dval *= rstd_bt; // final scale
        dinp_bt[i] += dval;
    }
    __syncthreads();

    // write to global memory
	for(int i = threadIdx.x; i < C; i+= blockDim.x){
        atomicAdd(&dbias[i], dbias_shared[i]);
        atomicAdd(&dweight[i], dweight_shared[i]);
	}
}

__global__ void softmax_autoregressive_backward_kernel(float* dpreatt, const float* datt, const float* att,
                                                       int B, int T, int C, float scale) {
    constexpr const int BlockSize = 256;
    constexpr int T_per_block = 4;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float block_acc[32];

    int idx = blockIdx.y;
    // go through blocks in reverse order, so the slowest block starts first
    int t0 = T - 1 - T_per_block*blockIdx.x;

    att += idx * T * T;
    datt += idx * T * T;
    dpreatt += idx * T * T;

    if (warp.meta_group_rank() == 0) {
        block_acc[warp.thread_rank()] = 0;
    }

    for(int to = 0; to < T_per_block; ++to) {
        int t = t0 - to;
        if(t < 0) return;
        const float* att_bth = att + t * T;
        const float* datt_bth = datt + t * T;
        float* dpreatt_bth = dpreatt + t * T;

        float local_sum = 0;
        for (int t2 = block.thread_rank(); t2 <= t; t2 += BlockSize) {
            local_sum += att_bth[t2] * datt_bth[t2];
        }

        block_acc[warp.meta_group_rank()] = cg::reduce(warp, local_sum, cg::plus<float>{});
        block.sync();
        local_sum = cg::reduce(warp, block_acc[warp.thread_rank()], cg::plus<float>{});

        for (int t3 = block.thread_rank(); t3 <= t; t3 += BlockSize) {
            // don't touch the cache. Some parts will still be here from the previous loop, and
            // we want to exploit those.
            float acc = __ldcs(att_bth + t3) * (__ldcs(datt_bth + t3) - local_sum);
            __stcs(dpreatt_bth + t3, scale * acc);
        }
    }
}

// Implements linear interpolation using only two floating-point operations (as opposed to three in a naive implementation).
// Reference: https://developer.nvidia.com/blog/lerp-faster-cuda
__device__ inline float lerp(float start, float end, float weight) {
    return fma(weight, end, fma(-weight, start, start));
}

__global__ void adamw_kernel2(float* params_memory, float* grads_memory, float* m_memory, float* v_memory, long num_parameters,
                              float learning_rate, float beta1, float beta2, float beta1_correction, float beta2_correction, float eps, float weight_decay) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i >= num_parameters) return;  // guard
   float grad = grads_memory[i];
   float m = m_memory[i];
   float v = v_memory[i];
   // update the first moment (momentum)
   m = lerp(grad, m, beta1);
   m_memory[i] = m;
   // update the second moment (RMSprop)
   v = lerp(grad * grad, v, beta2);
   v_memory[i] = v;
   m /= beta1_correction;  // m_hat
   v /= beta2_correction;  // v_hat
   params_memory[i] -= learning_rate * (m / (sqrtf(v) + eps) + weight_decay * params_memory[i]);
}

struct SoftmaxParams {
    float Scale;
    float Offset;
};


__device__ SoftmaxParams prepare_softmax_blockwide_nofloat4(cg::thread_block_tile<32>& warp,
                                                   int idx, const float* inp, int V, int P) {
    // same but not float4
    // one row of inp, i.e. inp[idx, :] of shape (V,)

    const float* x = inp + idx * P;
    float thread_maxval = -INFINITY;
    float thread_sumval = 0.0f;
    // do the loop in reverse to maximise probability of L2 cache hits
    // so even small L2s get some hits on the 2nd read of the same thread
    for (int i = V + threadIdx.x - blockDim.x; i >= 0; i -= blockDim.x) {
        float v = x[i];
        float old_maxval = thread_maxval;
        thread_maxval = fmaxf(thread_maxval, v);
        thread_sumval *= expf((old_maxval - thread_maxval));
        thread_sumval += expf(v - thread_maxval);
    }

    // two reductions of up to 1024 threads:
    // 1) inside warp (shuffle), 2) cross-warp (shared memory), 3) inside warp (shuffle)
    // this results in much cleaner assembly than a multi-warp cg::reduce
    __shared__ float shared_maxval[32];
    __shared__ float shared_sumval[32];
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    // reduce maxval within each warp
    float warp_maxval = cg::reduce(warp, thread_maxval, cg::greater<float>{});
    // thread 0 in each warp writes to shared memory
    if (lane_id == 0) { shared_maxval[warp_id] = warp_maxval; }
    __syncthreads();
    // each thread now loads the maxval across previous warps
    // if the thread is "out of range" of data, use -FLT_MAX as the maxval
    warp_maxval = (lane_id < num_warps) ? shared_maxval[lane_id] : -FLT_MAX;
    // now reduce the maxval among the warp threads
    float block_maxval = cg::reduce(warp, warp_maxval, cg::greater<float>{});
    // each thread uses maxval to scale sumval to avoid numerical instability / overflow
    thread_sumval *= expf(thread_maxval - block_maxval);
    // (warp-level) reduce sumval, thread 0 in each warp saves result in shared memory
    float warp_sumval = cg::reduce(warp, thread_sumval, cg::plus<float>{});
    if (lane_id == 0) { shared_sumval[warp_id] = warp_sumval; }
    __syncthreads();
    // same strategy, now reduce sumval across warps
    warp_sumval = (lane_id < num_warps) ? shared_sumval[lane_id] : 0.0f;
    float block_sumval = cg::reduce(warp, warp_sumval, cg::plus<float>{});
    // return the softmax parameters
    return SoftmaxParams{1.f / block_sumval, block_maxval};
}

// same as 2 but not using float4 (see dev/cuda/classifier_fused.cu)
// will _update_ logits to logit gradients
__global__ void fused_classifier_kernel3(float* logits, float* losses, float* probs,
                                         const float* dlosses, const int* targets,
                                         int B, int T, int V, int P) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x;
    int ix = targets[idx];

    // softmax (reading B * T * V, same logits read again below, hopefully still in cache)
    SoftmaxParams sp = prepare_softmax_blockwide_nofloat4(warp, idx, logits, V, P);

    // calculate the probability needed for the loss and update (single-threaded)
    if(threadIdx.x == 0) {
        float prob = expf(logits[idx * P + ix] - sp.Offset) * sp.Scale;
        losses[idx] = -logf(prob);
    }

    // very sensible default for dlosses is 1/(B*T), which is the uniform loss
    float dloss = dlosses != NULL ? dlosses[idx] : 1.0f / (B*T);
    // calculate the gradients directly, saves bandwidth from probs during training
    // but also supports writing probs for inference-only and debugging
    const float* logits_vec = logits + idx * P;
    for (int i = threadIdx.x; i < V; i += blockDim.x) {
        // this is the 2nd read of logits after the one in prepare_softmax2
        // this data will never be needed again, so we reduce cache persistence
        float v = __ldcs(&logits_vec[i]);
        float prob = expf(v - sp.Offset) * sp.Scale;
        if (probs != NULL) {
            probs[idx * P + i] = prob;
        }
        float indicator = (i == ix) ? 1.0f : 0.0f;
        logits[idx * P + i] = (prob - indicator) * dloss;
    }
}

__device__ float4 ld_vec(const float* address) {
    return *reinterpret_cast<const float4*>(address);
}

__device__ void st_vec(float* address, float4 val) {
    *reinterpret_cast<float4*>(address) = val;
}

__global__ void __launch_bounds__(16*16, 2) matmul_forward_kernel4(float* out,
                                                                   const float* inp, const float* weight, const float* bias,
                                                                   int C, int OC) {
    // out is (B,T,OC). OC is short for "output channels", e.g. OC = 4 * C
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    // each thread handles 8x8 elements; each block 128 by 128 elements.
    int oc = 8*(blockIdx.y * blockDim.y + threadIdx.y);

    // buffers to cache chunks of the input matrices
    __shared__ float lhs_s[128][32];
    __shared__ float rhs_s[128][32];

    // adjust our pointers for the current block
    inp += 128 * blockIdx.x * C;
    weight += 128 * blockIdx.y * C;
    out += 128 * blockIdx.x * OC + 128 * blockIdx.y;

    float vals[8][8] = {};
    if(bias != NULL) {
        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j += 4) {
                float4 b = ld_vec(bias + oc + j);
                vals[i][j+0] = b.x;
                vals[i][j+1] = b.y;
                vals[i][j+2] = b.z;
                vals[i][j+3] = b.w;
            }
        }
    }

    int si_start = 4*(16 * threadIdx.y + threadIdx.x);
    for (int so = 0; so < C; so += 32) {
        __syncthreads();
        int xmod8 = threadIdx.x % 8;
        int xby8 = threadIdx.x / 8;
        int xo = 4 * xmod8;
        for(int y = 2 * threadIdx.y + xby8; y < 128; y += 32) {
            st_vec(&lhs_s[y][xo], ld_vec(inp + y * C + so + xo));
            st_vec(&rhs_s[y][xo], ld_vec(weight + y * C + so + xo));
        }
        __syncthreads();

        for (int si = si_start; si < si_start + 32; si += 4) {
            float4 rhs[8];
            for (int u = 0; u < 8; ++u) {
                rhs[u] = ld_vec(&rhs_s[u + 8 * threadIdx.y][si % 32]);
            }

            for (int ii = 0; ii < 8; ++ii) {
                float4 lhs = ld_vec(&lhs_s[ii + 8 * threadIdx.x][si % 32]);
                for (int ji = 0; ji < 8; ++ji) {
                    vals[ii][ji] += lhs.x * rhs[ji].x;
                    vals[ii][ji] += lhs.y * rhs[ji].y;
                    vals[ii][ji] += lhs.z * rhs[ji].z;
                    vals[ii][ji] += lhs.w * rhs[ji].w;
                }
            }
        }
    }

    for (int i = 0; i < 8; ++i) {
        for (int j = 0; j < 8; j += 4) {
            float4 result;
            result.x = vals[i][j + 0];
            result.y = vals[i][j + 1];
            result.z = vals[i][j + 2];
            result.w = vals[i][j + 3];
            st_vec(out + (8*threadIdx.x+i) * OC + 8*threadIdx.y + j, result);
        }
    }
}


#define TILE_SIZE 128
#define HEAD_SIZE 64
#define gQ(i,j) gQ[(i) * 3 * NH * HS + (j)]
#define gK(i,j) gK[(i) * 3 * NH * HS + (j)]
#define gV(i,j) gV[(i) * 3 * NH * HS + (j)]
#define gO(i,j) gO[(i) * 1 * NH * HS + (j)]
#define gL(i) gL[(i) * NH]
#define gD(i) gD[(i) * NH]
#define sQ(i,j) sQ[(i) + (j) * TILE_SIZE]
#define sK_T(i,j) sK[(i) * TILE_SIZE + (j)]
#define sV(i,j) sV[(i) * HEAD_SIZE + (j)]
#define FLOAT4(value) reinterpret_cast<float4*>(&(value))[0]

__global__ __launch_bounds__(256)
void flash_attention_forward_kernel1(float* out, float* inp, float* l,
                                int B, int T, int NH, int HS) {

    // inp (B, T, 3, NH, HS)
    // out (B, T, NH, HS)
    // l (B, T, NH)

    // blockDim.x = NH
    // blockDim.y = T
    // blockDim.z = B

    // we use 256 threads = 8 warps in each threadblock
    // we use 64KB of shared memory for K, V so each uses 32KB of shared memory
    // Q is stored in registers
    // 32KB of shared memory can store 32 * 1024 / 4 = 8192 floats = 128 * 64 floats
    // so each threadblock computes a 128 * 64 tile of O, and each warp does a 16 * 64 tile of O
    // following flash attention 2, we only store (m + log l) instead of (m, l) for the backward pass

    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    int q_global_offset = blockIdx.z * T * 3 * NH * HS + blockIdx.y * TILE_SIZE * 3 * NH * HS + 0 * NH * HS + blockIdx.x * HS;
    int k_global_offset = blockIdx.z * T * 3 * NH * HS +                      0 * 3 * NH * HS + 1 * NH * HS + blockIdx.x * HS;
    int v_global_offset = blockIdx.z * T * 3 * NH * HS +                      0 * 3 * NH * HS + 2 * NH * HS + blockIdx.x * HS;
    int o_global_offset = blockIdx.z * T * 1 * NH * HS + blockIdx.y * TILE_SIZE * 1 * NH * HS + 0 * NH * HS + blockIdx.x * HS;
    int l_global_offset = blockIdx.z * T * NH + blockIdx.y * TILE_SIZE * NH + blockIdx.x;

    float* gQ = &inp[q_global_offset];
    float* gK = &inp[k_global_offset];
    float* gV = &inp[v_global_offset];
    float* gO = &out[o_global_offset];
    float* gL = &l[l_global_offset];

    extern __shared__ float sharedMemory[];

    float* sK = &sharedMemory[0 * TILE_SIZE * 64];
    float* sV = &sharedMemory[1 * TILE_SIZE * 64];

    int tile_increment = TILE_SIZE * 3 * NH * HS;

    // addresses for loading data from global to shared
    // as well as for register tiling

    int thread_row = warp_id * 16 + (lane_id / 16) * 4;
    int thread_col = (lane_id % 16) * 4;


    // main loop
    float tQ[8][4];
    float tK[8][4];
    float rQ[8] = {0.0f};
    float rK[8] = {0.0f};
    float rV[4] = {0.0f};
    float tS[8][8] = {0.0f};
    float (&tP)[8][8] = tS;
    //float tP[4][4] = {0.0f};
    float rP[8] = {0.0f};
    float rO[8][4] = {0.0f};
    float rM_old[8] = {-FLT_MAX, -FLT_MAX, -FLT_MAX, -FLT_MAX, -FLT_MAX, -FLT_MAX, -FLT_MAX, -FLT_MAX};
    float rM[8] = {0.0f};
    float rL_old[8] = {0.0f};
    float rL[8] = {0.0f};
    // this stores sum(rP) across the half-warps
    // in order to compute rL = exp(rM_old - rM) * rL_old + sum(rP)
    float rD[8] = {0.0f};
    unsigned mask = (lane_id < 16) ? 0xFFFF : 0xFFFF0000; // Mask for the two halves
    int lane_id_to_read_from = (lane_id < 16) ? 0 : 16; // Lane to read from

    // load gQ to sQ

    for (int i = 0; i < 4; i++) {
        FLOAT4(tQ[i][0]) = FLOAT4(gQ(thread_row + i, thread_col));
        FLOAT4(tQ[i+4][0]) = FLOAT4(gQ(thread_row + i + 8, thread_col));
    }


    // For auto regressive mask, need to check when kv_tile = blockIdx.y
    for (int tile = 0; tile <= blockIdx.y; tile++) {


        for (int i = 0; i < 4; i++) {
            FLOAT4(tK[i][0]) = FLOAT4(gK(thread_row + i, thread_col));
            FLOAT4(tK[i+4][0]) = FLOAT4(gK(thread_row + i + 8, thread_col));
        }

        for (int i = 0; i < 4; i++) {
            for (int j=0; j < 4; j++) {
                sK_T(thread_col + j, thread_row + i) = tK[i][j];
                sK_T(thread_col + j, thread_row + i + 8) = tK[i+4][j];
            }
        }



        // load gV to sV
        for (int i = 0; i < 4; i++) {
            FLOAT4(sV(thread_row + i, thread_col)) = FLOAT4(gV(thread_row + i, thread_col));
            FLOAT4(sV(thread_row + i + 8, thread_col)) = FLOAT4(gV(thread_row + i + 8, thread_col));
        }

        __syncthreads();



        //
        // compute rS
        //

        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j++) {
                tS[i][j] = 0;
            }
        }

        // We use warp shuffling to directly load data to each fragment from tQ to compute the outer product tS.
        // To do this, there is some array indexing involving the modulo operator for tQ to compute the lane id that we want to load data from.
        // For some reason, in this case the compiler will put tQ into local memory, causing register spillage,
        // even though the array indexing can be computed at compile time.
        // To resolve this, we use nested for loops to remove the use of modulo operator.
        for (int k_fragment_outer = 0; k_fragment_outer < 16; k_fragment_outer++) {
            for (int k_fragment_inner = 0; k_fragment_inner < 4; k_fragment_inner++) {
                // position is k_fragment_outer * 4 + k_fragment_inner
                int k_fragment = k_fragment_outer * 4 + k_fragment_inner;
                FLOAT4(rK[0]) = FLOAT4(sK_T(k_fragment, thread_col));
                FLOAT4(rK[4]) = FLOAT4(sK_T(k_fragment, thread_col + 64));
                for (int i = 0; i < 4; i++) {
                    //rdO[i] = sdO(thread_row_64_x_128 + i, k_fragment);
                    rQ[i] = __shfl_sync(mask, tQ[i][k_fragment_inner], (lane_id / 16) * 16  + k_fragment_outer);
                    rQ[i+4] = __shfl_sync(mask, tQ[i+4][k_fragment_inner], (lane_id / 16) * 16  + k_fragment_outer);
                }

                for (int i = 0; i < 4; i++) {
                    for (int j = 0; j < 4; j++) {
                        if (tile == blockIdx.y  && thread_row + i < thread_col + j) {
                            tS[i][j] = -FLT_MAX;
                        } else {
                            tS[i][j] += rQ[i] * rK[j];
                        }

                        if (tile == blockIdx.y  && thread_row + i + 8 < thread_col + j) {
                            tS[i + 4][j] = -FLT_MAX;
                        } else {
                            tS[i + 4][j] += rQ[i + 4] * rK[j];
                        }

                        if (tile == blockIdx.y  && thread_row + i < thread_col + j + 64) {
                            tS[i][j+4] = -FLT_MAX;
                        } else {
                            tS[i][j+4] += rQ[i] * rK[j+4];
                        }

                        if (tile == blockIdx.y  && thread_row + i + 8 < thread_col + j + 64) {
                            tS[i+4][j+4] = -FLT_MAX;
                        } else {
                            tS[i+4][j+4] += rQ[i+4] * rK[j+4];
                        }
                    }
                }
            }
        }


        // rescale preatt by 1/sqrt(HS)
        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j++) {
                if (tS[i][j] != -FLT_MAX) {
                    tS[i][j] *= 1.0f / sqrtf(HS);
                }
            }
        }

        //
        // compute m
        //


        // inter-thread reduction
        for (int i = 0; i < 8; i++) {
            rM[i] = rM_old[i];
            for (int j = 0; j < 8;j++) {
                rM[i] = fmaxf(rM[i], tS[i][j]);
            }
        }

        // inter-warp reduction
        for (int i=0; i < 8; i++) {
            for (int offset = 8; offset > 0; offset /= 2) {
               rM[i] = fmaxf(rM[i], __shfl_down_sync(mask, rM[i], offset));
            }
        }

        // now threads 0, 16 have the correct m[i],
        // so we broadcast m back to the other lanes in the half warp
        for (int i=0; i<8; i++) {
            rM[i] = __shfl_sync(mask, rM[i], lane_id_to_read_from);
        }

        //
        // compute P
        //
        for (int i=0;i<8;i++) {
            for (int j=0;j<8;j++){
                tP[i][j] = expf(tS[i][j] - rM[i]);
            }
        }

        //store to sP


        //
        // compute l
        //

        // rescale l and also reset rD to 0
        for (int i = 0; i < 8; i++) {
            rL[i] = expf(rM_old[i] - rM[i]) * rL_old[i];
            rD[i] = 0;
        }

        // inter-thread reduction
        for (int i = 0; i < 8; i++) {
            for (int j=0;j<8;j++){
                rD[i] += tP[i][j];
            }
        }

        // inter-warp reduction
        for (int i=0; i < 8; i++) {
            for (int offset = 8; offset > 0; offset /= 2) {
               rD[i] += __shfl_down_sync(mask, rD[i], offset);
            }
        }

        // now threads 0, 16 have the correct rD[i],
        // so we compute rL[i] and broadcast it back to the warp
        for (int i=0; i<8; i++) {
            rL[i] += rD[i];
            rL[i] = __shfl_sync(mask, rL[i], lane_id_to_read_from);
        }



        //
        // compute O
        //

        // first rescale O by exp(m_old - m)
        for (int i=0; i<8; i++) {
            for (int j=0;j<4;j++) {
                rO[i][j] = expf(rM_old[i] - rM[i]) * rO[i][j];
            }
        }

        // add PV to rO
        for (int step = 0; step < 2; step++) {
            for (int k_fragment_outer = 0; k_fragment_outer < 16; k_fragment_outer++) {
                for (int k_fragment_inner = 0; k_fragment_inner < 4; k_fragment_inner++) {
                    // position is h * 64 + l * 4 + k
                    int k_fragment = k_fragment_outer * 4 + k_fragment_inner;
                    FLOAT4(rV[0]) = FLOAT4(sV(step * 64 + k_fragment, thread_col));
                    for (int i=0;i<4;i++) {
                        rP[i] = __shfl_sync(mask, tP[i][k_fragment_inner + step * 4], (lane_id /16) * 16  + k_fragment_outer );
                        rP[i + 4] = __shfl_sync(mask, tP[i + 4][k_fragment_inner + step * 4], (lane_id /16) * 16  + k_fragment_outer);
                        //rV[i] = sV(step * 64 + k_fragment, thread_col + i);
                    }

                    for (int i = 0; i < 8; i++) {
                        for (int j = 0; j < 4; j++) {
                            rO[i][j] += rP[i] * rV[j];
                        }
                    }
                }
            }
        }


        // update m and l
        for (int i = 0; i < 8; i++) {
            rM_old[i] = rM[i];
            rL_old[i] = rL[i];
        }

        gK += tile_increment;
        gV += tile_increment;
        __syncthreads();
    }


    //rescale rO
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 4; j++) {
            rO[i][j] /= rL[i];
        }
    }


    // store l back to gL
    if (lane_id == 0 || lane_id == 16) {
        for (int i = 0; i < 4; i++) {
            gL(thread_row + i) = rM[i] + logf(rL[i]);
            gL(thread_row + 8 + i) = rM[i + 4] + logf(rL[i + 4]);
        }
    }

    // store rO to gO
    for (int i=0; i < 4; i++) {
        FLOAT4(gO(thread_row + i, thread_col)) = FLOAT4(rO[i][0]);
        FLOAT4(gO(thread_row + 8 + i, thread_col)) = FLOAT4(rO[i+4][0]);
    }

}

#undef sK_T

#define Q_TILE_SIZE 64
#define KV_TILE_SIZE 128
#define gdQ(i,j) gdQ[(i) * 3 * NH * HS + (j)]
#define gdK(i,j) gdK[(i) * 3 * NH * HS + (j)]
#define gdV(i,j) gdV[(i) * 3 * NH * HS + (j)]
#define gdO(i,j) gdO[(i) * 1 * NH * HS + (j)]
#define sK(i,j) sK[(i) * HEAD_SIZE + (j)]
#define sK_T(i,j) sK[(i) + (j) * HEAD_SIZE]
#define sQ_row(i,j) sQ[(i) * HEAD_SIZE + (j)]
#define sQ_col(i,j) sQ[(i) + (j) * Q_TILE_SIZE]
#define sdO_row(i,j) sdO[(i) * HEAD_SIZE + (j)]
#define sdO_col(i,j) sdO[(i) + (j) * Q_TILE_SIZE]
#define sdS(i,j) sdS[(i) + (j) * Q_TILE_SIZE]
#define sdQ(i,j) sdQ[(i) * HEAD_SIZE + (j)]

// preprocessing D = rowsum(dO * O)
__global__ void flash_attention_backward_preprocessing_kernel1(float* d, float* dout, float* out,
                                int B, int T, int NH, int HS) {
    // dout (B, T, NH, HS)
    // out (B, T, NH, HS)
    // d (B, T, NH)

    // blockDim.x = NH
    // blockDim.y = T / 256
    // blockDim.z = B

    // Each half-warps compute 4 rows,
    // so each warp computes 8 rows
    // We use 1024 threads = 32 warps per block, so each block computes 256 rows
    // so we have B * T / 256 * NH blocks

    int o_global_offset = blockIdx.z * T * NH * HS + blockIdx.y * 256 * NH * HS + blockIdx.x * HS;
    int d_global_offset = blockIdx.z * T * NH + blockIdx.y * 256 * NH + blockIdx.x;

    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    unsigned mask = (lane_id < 16) ? 0xFFFF : 0xFFFF0000; // Mask for the two halves

    float* gO = &out[o_global_offset];
    float* gdO = &dout[o_global_offset];
    float* gD = &d[d_global_offset];

    int thread_row = warp_id * 8 + (lane_id / 16) * 4;
    int thread_col = (lane_id % 16) * 4;

    float tO[4][4];
    float tdO[4][4];
    float sum[4] = {0.0f};

    for (int i=0;i<4;i++){
        FLOAT4(tO[i][0]) = FLOAT4(gO(thread_row + i, thread_col));
        FLOAT4(tdO[i][0]) = FLOAT4(gdO(thread_row + i, thread_col));
    }

    // inter-thread reduction
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4;j++) {
            sum[i] += tO[i][j] * tdO[i][j];
        }
    }

    // inter-warp reduction
    for (int i=0; i < 4; i++) {
        for (int offset = 8; offset > 0; offset /= 2) {
           sum[i] += __shfl_down_sync(mask, sum[i], offset);
        }
    }

    if (lane_id == 0 || lane_id == 16) {
        for (int i=0; i<4; i++) {
            gD(thread_row + i) = sum[i];
        }
    }
}


__global__ __launch_bounds__(256)
void flash_attention_backward_kernel1(float* dinp, float* inp, float* dout, float* out, float* l, float* d,
                                int B, int T, int NH, int HS) {
    // dinp (B, T, 3, NH, HS)
    // inp  (B, T, 3, NH, HS)
    // out  (B, T, NH, HS)
    // dout (B, T, NH, HS)
    // l    (B, T, NH)
    // d    (B, T, NH)

    // blockDim.x = NH
    // blockDim.y = T
    // blockDim.z = B

    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    // offset for the q,k,v of the corresponding head
    int q_global_offset = blockIdx.z * T * 3 * NH * HS + blockIdx.y * 2 * Q_TILE_SIZE * 3 * NH * HS + 0 * NH * HS + blockIdx.x * HS;
    int k_global_offset = blockIdx.z * T * 3 * NH * HS + blockIdx.y * KV_TILE_SIZE * 3 * NH * HS + 1 * NH * HS + blockIdx.x * HS;
    int v_global_offset = blockIdx.z * T * 3 * NH * HS + blockIdx.y * KV_TILE_SIZE * 3 * NH * HS + 2 * NH * HS + blockIdx.x * HS;
    int o_global_offset = blockIdx.z * T * 1 * NH * HS + blockIdx.y * 2 * Q_TILE_SIZE * 1 * NH * HS + 0 * NH * HS + blockIdx.x * HS;

    int ld_global_offset = blockIdx.z * T * NH + blockIdx.y * 2 * Q_TILE_SIZE * NH + blockIdx.x;

    int q_increment = Q_TILE_SIZE * 3 * NH * HS;
    int o_increment = Q_TILE_SIZE * NH * HS;
    int ld_increment = Q_TILE_SIZE * NH;

    float* gQ = &inp[q_global_offset];
    float* gK = &inp[k_global_offset];
    float* gV = &inp[v_global_offset];
    float* gdO = &dout[o_global_offset];
    float* gL = &l[ld_global_offset];
    float* gD = &d[ld_global_offset];

    // output
    float* gdQ = &dinp[q_global_offset];
    float* gdK = &dinp[k_global_offset];
    float* gdV = &dinp[v_global_offset];

    extern __shared__ float sharedMemory[];

    float* sQ = &sharedMemory[0];
    float* sdO = sQ + Q_TILE_SIZE * Q_TILE_SIZE;
    float* sK = sdO + Q_TILE_SIZE * Q_TILE_SIZE;
    float* sdS = sQ;
    float* sdQ = sQ;

    // offset for register tiling for dK and dV
    int thread_row_128_x_64 = warp_id * 16 + (lane_id / 16) * 4;
    int thread_col_128_x_64 = (lane_id % 16) * 4;

    // offset for register tiling for S and dP
    int thread_row_64_x_128 = thread_col_128_x_64;
    int thread_col_64_x_128 = thread_row_128_x_64;

    // offset for register tiling for dQ
    int thread_row_64_x_64 = warp_id * 8 + (lane_id / 16) * 4;
    int thread_col_64_x_64 = (lane_id % 16) * 4;


    // offset for atomic add for dQ
    int thread_row_atomic_add = warp_id * 8;
    int thread_col_atomic_add = lane_id;

    unsigned mask = (lane_id < 16) ? 0xFFFF : 0xFFFF0000; // Mask for the two halves

    float rL[4];
    float rD[4];
    float rQ[4];
    float rK[8];
    float rV[8];
    float rdO[4];
    float rP[8];
    float rdS[8];
    float tV[8][4];
    float tQ[4][4];
    float tdO[4][4];
    float tdQ[4][4] = {0.0f};
    float tdK[8][4] = {0.0f};
    float tdV[8][4] = {0.0f};
    float tS[4][8] = {0.0f};
    float (&tP)[4][8] = tS;
    float tdP[4][8] = {0.0f};
    float (&tdS)[4][8] = tdP;

    for (int i=0; i < 4;i ++){
        FLOAT4(sK(thread_row_128_x_64 + i, thread_col_128_x_64)) = FLOAT4(gK(thread_row_128_x_64 + i, thread_col_128_x_64));
        FLOAT4(sK(thread_row_128_x_64 + 8 + i,  thread_col_128_x_64)) = FLOAT4(gK(thread_row_128_x_64 + 8 + i, thread_col_128_x_64));
        FLOAT4(tV[i][0]) = FLOAT4(gV(thread_row_128_x_64 + i, thread_col_128_x_64));
        FLOAT4(tV[i+4][0]) = FLOAT4(gV(thread_row_128_x_64 + 8 + i, thread_col_128_x_64));
    }

    for (int q_tile = 2 * blockIdx.y; q_tile < T / Q_TILE_SIZE; q_tile++) {

        // load Q, dO into shared memory
        for (int i=0; i < 4;i ++){
            FLOAT4(tQ[i][0]) = FLOAT4(gQ(thread_row_64_x_64 + i, thread_col_64_x_64));
            FLOAT4(tdO[i][0]) = FLOAT4(gdO(thread_row_64_x_64 + i, thread_col_64_x_64));
        }

        for (int i=0;i<4;i++) {
            for (int j=0;j<4;j++) {
                sQ_col(thread_row_64_x_64 + i, thread_col_64_x_64+j) = tQ[i][j];
                sdO_col(thread_row_64_x_64 + i, thread_col_64_x_64+j) = tdO[i][j];
            }
        }


        // load l, d into registers
        for (int i=0; i< 4;i ++){
            rL[i] = gL(thread_row_64_x_128 + i);
            rD[i] = gD(thread_row_64_x_128 + i);
        }

        __syncthreads();


        //
        // compute S and P
        //

        // reset tS back to zero
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 8; j++) {
                tS[i][j] = 0;
            }
        }

        // compute S = Q * K^T
        for (int k_fragment = 0; k_fragment < HEAD_SIZE; k_fragment++) {
            FLOAT4(rQ[0]) = FLOAT4(sQ_col(thread_row_64_x_128, k_fragment));
            for (int i = 0; i < 4; i++) {
                rK[i] = sK_T(k_fragment, thread_col_64_x_128 + i);
                rK[i+4] = sK_T(k_fragment, thread_col_64_x_128 + 8 + i);
            }

            for (int i = 0; i < 4; i++) {
                for (int j = 0; j < 4; j++) {
                    if (q_tile * Q_TILE_SIZE + thread_row_64_x_128 + i < blockIdx.y * KV_TILE_SIZE + thread_col_64_x_128 + j) {
                        tS[i][j] = -FLT_MAX;
                    } else {
                        tS[i][j] += rQ[i] * rK[j];
                    }

                    if (q_tile * Q_TILE_SIZE + thread_row_64_x_128 + i < blockIdx.y * KV_TILE_SIZE + thread_col_64_x_128 + 8 + j) {
                        tS[i][j + 4] = -FLT_MAX;
                    } else {
                        tS[i][j + 4] += rQ[i] * rK[j + 4];
                    }
                }
            }
        }


        // rescale S by 1/sqrt(HS)
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 8; j++) {
                if (tS[i][j] != -FLT_MAX) {
                    tS[i][j] *= 1.0f / sqrtf(HS);
                }
            }
        }

        // compute P = exp(Q * K^T - l)
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 8; j++) {
               tP[i][j] = expf(tS[i][j] - rL[i]);
            }
        }

        //
        // compute dP and dS
        //

        // reset tdP back to zero
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 8; j++) {
                tdP[i][j] = 0;
            }
        }

        // compute dP = dO * V^T
        for (int k_fragment_outer = 0; k_fragment_outer < 16; k_fragment_outer++) {
            for (int k_fragment_inner = 0; k_fragment_inner < 4; k_fragment_inner++) {
                // position is k_fragment_outer * 4 + k_fragment_inner
                int k_fragment = k_fragment_outer * 4 + k_fragment_inner;
                FLOAT4(rdO[0]) = FLOAT4(sdO_col(thread_row_64_x_128, k_fragment));
                for (int i = 0; i < 4; i++) {
                    rV[i] = __shfl_sync(mask, tV[i][k_fragment_inner], (lane_id / 16) * 16  + k_fragment_outer);
                    rV[i+4] = __shfl_sync(mask, tV[i+4][k_fragment_inner], (lane_id / 16) * 16  + k_fragment_outer);
                }

                for (int i = 0; i < 4; i++) {
                    for (int j = 0; j < 4; j++) {
                        if (q_tile * Q_TILE_SIZE + thread_row_64_x_128 + i < blockIdx.y * KV_TILE_SIZE + thread_col_64_x_128 + j) {
                            tdP[i][j] = 0;
                        } else {
                            tdP[i][j] += rdO[i] * rV[j];
                        }
                    }
                    for (int j = 0; j < 4; j++) {
                        if (q_tile * Q_TILE_SIZE + thread_row_64_x_128 + i < blockIdx.y * KV_TILE_SIZE + thread_col_64_x_128 + 8 + j) {
                            tdP[i][j+4] = 0;
                        } else {
                            tdP[i][j+4] += rdO[i] * rV[j+4];
                        }
                    }

                }
            }
        }


        // compute dS = P \circ (dP - D)
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 8; j++) {
                tdS[i][j] = tP[i][j] * (tdP[i][j] - rD[i]);
            }
        }

        //
        // retile Q and dO to minimize bank conflicts
        //

        for (int i=0;i<4;i++) {
            for (int j=0;j<4;j++) {
                tQ[i][j] = sQ_col(thread_row_64_x_64 + i, thread_col_64_x_64+j);
                tdO[i][j] = sdO_col(thread_row_64_x_64 + i, thread_col_64_x_64+j);
            }
        }
        __syncthreads();

        for (int i=0;i<4;i++) {
            FLOAT4(sQ_row(thread_row_64_x_64 + i, thread_col_64_x_64)) = FLOAT4(tQ[i][0]);
            FLOAT4(sdO_row(thread_row_64_x_64 + i, thread_col_64_x_64)) = FLOAT4(tdO[i][0]);
        }
        __syncthreads();

        //
        //  compute dV
        //

        // compute dV = P^T * dO
        for (int k_fragment_outer = 0; k_fragment_outer < 16; k_fragment_outer++) {
            for (int k_fragment_inner = 0; k_fragment_inner < 4; k_fragment_inner++) {
                // position is k_fragment_outer * 4 + k_fragment_inner
                int k_fragment = k_fragment_outer * 4 + k_fragment_inner;
                for (int i = 0; i < 4; i++) {
                    rP[i] = __shfl_sync(mask, tP[k_fragment_inner][i], (lane_id / 16) * 16  + k_fragment_outer);
                    rP[i+4] = __shfl_sync(mask, tP[k_fragment_inner][i + 4], (lane_id / 16) * 16  + k_fragment_outer);
                }
                FLOAT4(rdO[0]) = FLOAT4(sdO_row(k_fragment, thread_col_128_x_64 ));
                for (int i = 0; i < 8; i++) {
                    for (int j = 0; j < 4; j++) {
                        tdV[i][j] += rP[i] * rdO[j];
                    }
                }
            }
        }

        //
        // dK
        //

        // compute dK = dS^T * Q
        for (int k_fragment_outer = 0; k_fragment_outer < 16; k_fragment_outer++) {
            for (int k_fragment_inner = 0; k_fragment_inner < 4; k_fragment_inner++) {
                // position is k_fragment_outer * 4 + k_fragment_inner
                int k_fragment = k_fragment_outer * 4 + k_fragment_inner;
                for (int i = 0; i < 4; i++) {
                    rdS[i] = __shfl_sync(mask, tdS[k_fragment_inner][i], (lane_id / 16) * 16  + k_fragment_outer);
                    rdS[i+4] = __shfl_sync(mask, tdS[k_fragment_inner][i + 4], (lane_id / 16) * 16  + k_fragment_outer);
                }
                FLOAT4(rQ[0]) = FLOAT4(sQ_row(k_fragment, thread_col_128_x_64 ));

                for (int i = 0; i < 8; i++) {
                    for (int j = 0; j < 4; j++) {
                        tdK[i][j] += rdS[i] * rQ[j];
                    }
                }
            }
        }


        __syncthreads();

        //
        // compute dQ
        //

        // reset tdQ back to zero
        for (int i=0;i<4;i++) {
            for (int j=0; j<4; j++) {
                tdQ[i][j] = 0;
            }
        }

        // store dS to shared memory
        for (int i = 0; i< 4; i++) {
            for (int j=0; j < 4;j++) {
                sdS(thread_row_64_x_128 + i, thread_col_64_x_128 + j) = tdS[i][j];
                sdS(thread_row_64_x_128 + i, thread_col_64_x_128 + j + 8) = tdS[i][j + 4];
            }
        }

        __syncthreads();


        //compute dQ = dS * K
        for (int k_fragment = 0; k_fragment < KV_TILE_SIZE; k_fragment++) {

            for (int i=0;i<4;i++) {
                //rdS[i] = sdS(thread_row_64_x_64 + i, k_fragment);
                //rK[i] = sK(k_fragment, thread_col_64_x_64 + i);
                FLOAT4(rdS[0]) = FLOAT4(sdS(thread_row_64_x_64, k_fragment));
                FLOAT4(rK[0]) = FLOAT4(sK(k_fragment, thread_col_64_x_64));
            }

            for (int i=0;i<4;i++) {
                for (int j=0; j<4; j++) {
                    tdQ[i][j] += rdS[i] * rK[j];
                }
            }
        }

        for (int i=0;i<4;i++) {
            for (int j=0; j<4; j++) {
                tdQ[i][j] *= 1.0f / sqrtf(HS);
            }
        }

        __syncthreads();

        // store dQ
        for (int i=0;i<4;i++) {
            for (int j=0; j<4; j++) {
                sdQ(thread_row_64_x_64 + i, thread_col_64_x_64 + j) = tdQ[i][j];
            }
        }
        __syncthreads();

        for (int i=0;i<8;i++) {
            atomicAdd(&gdQ(thread_row_atomic_add + i, thread_col_atomic_add ), sdQ(thread_row_atomic_add + i, thread_col_atomic_add));
            atomicAdd(&gdQ(thread_row_atomic_add + i, thread_col_atomic_add + 32), sdQ(thread_row_atomic_add + i, thread_col_atomic_add + 32));
        }

        gQ += q_increment;
        gdQ += q_increment;
        gdO += o_increment;
        gL += ld_increment;
        gD += ld_increment;
        __syncthreads();
    }

    // rescale dK
    for (int i=0;i<8;i++) {
        for (int j=0; j<4; j++) {
            tdK[i][j] *= 1.0f / sqrtf(HS);
        }
    }

    // store dK to global memory

    for (int i=0;i<4;i++) {
        FLOAT4(gdK(thread_row_128_x_64 + i ,thread_col_128_x_64)) = FLOAT4(tdK[i][0]);
        FLOAT4(gdK(thread_row_128_x_64 + 8 + i ,thread_col_128_x_64)) = FLOAT4(tdK[i+4][0]);
    }


    // store dV to global memory
    for (int i=0;i<4;i++) {
        FLOAT4(gdV(thread_row_128_x_64 + i ,thread_col_128_x_64)) = FLOAT4(tdV[i][0]);
        FLOAT4(gdV(thread_row_128_x_64 + 8 + i ,thread_col_128_x_64)) = FLOAT4(tdV[i+4][0]);
    }
}



// ----------------------------------------------------------------------------
// kernel launchers


// use att to store log l + m
void flash_attention_forward(float* out, float* inp, float* l,
                                int B, int T, int C, int NH) {
    // head size
    int HS = C / NH;

    // inp (B, T, 3, NH, HS)
    // out (B, T, NH, HS)
    // l (B, T, NH)

    dim3 dimGrid(NH, T / 128, B);
    dim3 dimBlock(256);
    int maxbytes = 65536;
    hipFuncSetAttribute(reinterpret_cast<const void*>(flash_attention_forward_kernel1), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    flash_attention_forward_kernel1<<<dimGrid, dimBlock, maxbytes>>>(out, inp, l, B, T, NH, HS);

    cudaCheck(hipGetLastError());
}


void flash_attention_backward(float *dinp, float* inp, float* dout, float* out, float* l, float* d,
                                int B, int T, int C, int NH) {

    // head size
    int HS = C / NH;

    // dinp (B, T, 3, NH, HS)
    // inp (B, T, 3, NH, HS)
    // dout (B, T, NH, HS)
    // out (B, T, NH, HS)
    // l (B, T, NH)
    // d (B, T, NH)

    // preprocess d = rowsum(dout * out)
    dim3 dimGrid_preprocessing(NH, T / 256, B);
    dim3 dimBlock_preprocessing(1024);
    flash_attention_backward_preprocessing_kernel1<<<dimGrid_preprocessing, dimBlock_preprocessing>>>(d, dout, out, B, T, NH, HS);

    dim3 dimGrid(NH, T / 128, B);
    dim3 dimBlock(256);
    int maxbytes = 65536;
    hipFuncSetAttribute(reinterpret_cast<const void*>(flash_attention_backward_kernel1), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    flash_attention_backward_kernel1<<<dimGrid, dimBlock, maxbytes>>>(dinp, inp, dout, out, l, d, B, T, NH, HS);

    cudaCheck(hipGetLastError());
}

void encoder_forward(float* out,
                     const int* inp, const float* wte, const float* wpe,
                     int B, int T, int C) {
    assert(C % 4 == 0);
    const int block_size = 512;
    const int N = B * T * C;
    const int grid_size = CEIL_DIV(N / 4, block_size);
    encoder_forward_kernel3<<<grid_size, block_size>>>((float4*) out, inp, (float4*) wte, (float4*) wpe, B, T, C);
    cudaCheck(hipGetLastError());
}

void encoder_backward(float* dwte, float* dwpe,
                    const float* dout, const int* inp,
                    int B, int T, int C) {
    const int N = B * T * C;
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    encoder_backward_kernel<<<grid_size, block_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward(float* out, float* mean, float* rstd,
                       float* inp, float* weight, float* bias,
                       int B, int T, int C) {
    const int block_size = 512;
    const int N = B * T;
    const int grid_size = CEIL_DIV(N * 32, block_size);
    layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

// kernel 1 is the most naive matmul kernel
void matmul_forward(float* out,
                    const float* inp, const float* weight, const float* bias,
                    int B, int T, int C, int OC) {
    // out is (B,T,OC). OC is short for "output channels", e.g. OC = 4 * C
    // inp is (B,T,C), weight is (OC, C), bias is (OC)
    int sqrt_block_size = 16;

    dim3 gridDim(CEIL_DIV(B * T, 8*sqrt_block_size), CEIL_DIV(OC, 8*sqrt_block_size));
    dim3 blockDim(sqrt_block_size, sqrt_block_size);
    matmul_forward_kernel4<<<gridDim, blockDim>>>(out, inp, weight, bias, C, OC);
    cudaCheck(hipGetLastError());
}

void attention_forward(float* out, float* qkvr, float* att,
                       float* inp,
                       int B, int T, int C, int NH) {
    // Note: `inp` is not needed for backward pass, so we re-use it as a scratch buffer.
    // Its contents will be overwritten by this function.
    const int block_size = 256;
    const int softmax_block_size = 256;

    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);
    cudaCheck(hipGetLastError());

    // batched matrix multiply with cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    float* preatt = inp;
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, &alpha, k, HS, T * HS, q, HS, T * HS, &beta, preatt, T, T * T, B * NH));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    int grid_size = CEIL_DIV(B * NH * T * 32, softmax_block_size);
    softmax_forward_kernel5<<<grid_size, softmax_block_size>>>(att, scale, preatt, B * NH, T);
    cudaCheck(hipGetLastError());

    // new approach: first cuBLAS another batched matmul
    float* vaccum = inp;
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, &alpha, v, HS, T * HS, att, T, T * T, &beta, vaccum, HS, T * HS, B * NH));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

void residual_forward(float* out, float* inp1, float* inp2, int N) {
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    residual_forward_kernel<<<grid_size, block_size>>>(out, inp1, inp2, N);
    cudaCheck(hipGetLastError());
}

void gelu_forward(float* out, const float* inp, int N) {
    const int block_size = 128;
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_forward_kernel<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void gelu_backward(float* dinp, const float* inp, const float* dout, const int N) {
    const int block_size = 128;
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_backward_kernel<<<grid_size, block_size>>>(dinp, inp, dout, N);
    cudaCheck(hipGetLastError());
}

void matmul_backward(float* dinp, float* dweight, float* dbias,
                     float* dout, float* inp, float* weight,
                     int B, int T, int C, int OC) {
    float one = 1.0f;
    float zero = 0.0f;
    // backward to input, uses = in the backward pass (set the gradient)
    cublasCheck(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C, B*T, OC, &one, weight, C, dout, OC, &zero, dinp, C));
    // backward to weight, uses += in the backward pass (accumulate the gradient)
    cublasCheck(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, C, OC, B*T, &one, inp, C, dout, OC, &one, dweight, C));
    // backward to bias, if given, does a +=
    if (dbias != NULL) {
        const int block_size = 1024;
        const int grid_size = OC / 32; // for now, OC must be divisible by 32 for this kernel to work
        matmul_backward_bias_kernel4<<<grid_size, block_size, block_size * sizeof(float)>>>(dbias, dout, B, T, OC);
        cudaCheck(hipGetLastError());
    }
}

void layernorm_backward(float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const  float* weight, const float* mean, const float* rstd,
                        int B, int T, int C) {
    const int block_size = 512;
    const int N = B * T;
    const int grid_size = CEIL_DIV(32*N, block_size);
    size_t shared_mem_size = 2 * C * sizeof(float);
    layernorm_backward_kernel2<<<grid_size, block_size, shared_mem_size>>>(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C);
    cudaCheck(hipGetLastError());
}

// the sequence of transformations in this compound op is:
// inp (B,T,3C) -> qkvr (B,T,3C) -> preatt (B,NH,T,T) -> att (B,NH,T,T) -> vaccum (B,T,C) -> out (B,T,C)
void attention_backward(float* dinp, float* dqkvr, float* dpreatt, float* datt, float* scratch,
                        const float* dout,
                        const float* qkvr, const float* att,
                        int B, int T, int C, int NH) {
    const int block_size = 256;
    int HS = C / NH; // head size
    const float one = 1.0f;
    const float zero = 0.0f; // note beta = 1.0f so that we accumulate gradients (+=)
    // unpack convenience pointers into q, k, v
    const float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    float *dq, *dk, *dv;
    dq = dqkvr + 0 * B * T * C;
    dk = dqkvr + 1 * B * T * C;
    dv = dqkvr + 2 * B * T * C;
    // backward through the unpermute operation
    int num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel_backward<<<num_blocks, block_size>>>(scratch, dout, B, T, NH, HS);
    cudaCheck(hipGetLastError());
    // backward into datt
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, &one, v, HS, T * HS, scratch, HS, T * HS, &zero, datt, T, T * T, B * NH));
    // backward into dv
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &one, scratch, HS, T * HS, att, T, T * T, &zero, dv, HS, T * HS, B * NH));
    // backward into preatt
    int hs = C / NH; // head size
    float scale = 1.0f / sqrtf(hs);
    softmax_autoregressive_backward_kernel<<<dim3(T / 4, B * NH), 256>>>(dpreatt, datt, att, B, T, C, scale);
    cudaCheck(hipGetLastError());
    // backward into q
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, &one, k, HS, T * HS, dpreatt, T, T * T, &zero, dq, HS, T * HS, B * NH));
    // backward into k
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &one, q, HS, T * HS, dpreatt, T, T * T, &zero, dk, HS, T * HS, B * NH));
    // backward into inp
    num_blocks = CEIL_DIV(B * NH * T * HS, block_size);
    permute_kernel_backward<<<num_blocks, block_size>>>(dinp, dq, dk, dv, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

// replaces logits with logit gradients
void fused_classifier3(float* logits, float* losses,
                      const float* dlosses, const int* targets,
                      int B, int T, int V, int P) {
    const int block_size = 1024;
    const int N = B * T;
    const int grid_size = N;
    fused_classifier_kernel3<<<grid_size, block_size>>>(logits, losses, NULL, dlosses, targets, B, T, V, P);
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------
// GPT-2 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int padded_vocab_size; // padded to e.g. %128==0, 50304
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

// the parameters of the model
#define NUM_PARAMETER_TENSORS 16
typedef struct {
    float* wte; // (V, C)
    float* wpe; // (maxT, C)
    float* ln1w; // (L, C)
    float* ln1b; // (L, C)
    float* qkvw; // (L, 3*C, C)
    float* qkvb; // (L, 3*C)
    float* attprojw; // (L, C, C)
    float* attprojb; // (L, C)
    float* ln2w; // (L, C)
    float* ln2b; // (L, C)
    float* fcw; // (L, 4*C, C)
    float* fcb; // (L, 4*C)
    float* fcprojw; // (L, C, 4*C)
    float* fcprojb; // (L, C)
    float* lnfw; // (C)
    float* lnfb; // (C)
} ParameterTensors;

void fill_in_parameter_sizes(size_t* param_sizes, GPT2Config config) {
    int Vp = config.padded_vocab_size;
    int C = config.channels;
    int maxT = config.max_seq_len;
    int L = config.num_layers;
    param_sizes[0] = Vp * C; // wte
    param_sizes[1] = maxT * C; // wpe
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b
    param_sizes[4] = L * (3 * C) * C; // qkvw
    param_sizes[5] = L * (3 * C); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * (4 * C) * C; // fcw
    param_sizes[11] = L * (4 * C); // fcb
    param_sizes[12] = L * C * (4 * C); // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb
}

// allocate memory for the parameters and point the individual tensors to the right places
float* malloc_and_point_parameters(ParameterTensors* params, size_t* param_sizes, int on_device) {
    // on_device: 0 = CPU, 1 = GPU
    // calculate the number of parameters
    size_t num_parameters = 0;
    for (size_t i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += param_sizes[i];
    }
    // malloc all parameters all at once on the device
    float* params_memory;
    if (on_device) {
        cudaCheck(hipMalloc((void**)&params_memory, num_parameters * sizeof(float)));
    } else {
        params_memory = (float*)mallocCheck(num_parameters * sizeof(float));
    }
    // assign all the tensors their place in the array
    float** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    float* params_memory_iterator = params_memory;
    for (size_t i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = params_memory_iterator;
        params_memory_iterator += param_sizes[i];
    }
    return params_memory;
}

#define NUM_ACTIVATION_TENSORS 21
typedef struct {
    float* encoded; // (B, T, C)
    float* ln1; // (L, B, T, C)
    float* ln1_mean; // (L, B, T)
    float* ln1_rstd; // (L, B, T)
    float* atty; // (L, B, T, C)
    float* att; // (L, B, NH, T, T)
    float* attproj; // (L, B, T, C)
    float* residual2; // (L, B, T, C)
    float* ln2; // (L, B, T, C)
    float* ln2_mean; // (L, B, T)
    float* ln2_rstd; // (L, B, T)
    float* fch; // (L, B, T, 4*C)
    float* fch_gelu; // (L, B, T, 4*C)
    float* fcproj; // (L, B, T, C)
    float* residual3; // (L, B, T, C)
    float* lnf; // (B, T, C)
    float* lnf_mean; // (B, T)
    float* lnf_rstd; // (B, T)

    float* losses; // (B, T)
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    float* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    float* output;
} ActivationTensors;

void fill_in_activation_sizes(size_t* act_sizes, int B, int T, GPT2Config config) {
    size_t Vp = config.padded_vocab_size;
    size_t L = config.num_layers;
    size_t NH = config.num_heads;
    size_t C = config.channels;
    act_sizes[0] = B * T * C; // encoded
    act_sizes[1] = L * B * T * C; // ln1
    act_sizes[2] = L * B * T; // ln1_mean
    act_sizes[3] = L * B * T; // ln1_rstd
    act_sizes[4] = L * B * T * C; // atty
    act_sizes[5] = L * B * NH * T * T; // att
    act_sizes[6] = L * B * T * C; // attproj
    act_sizes[7] = L * B * T * C; // residual2
    act_sizes[8] = L * B * T * C; // ln2
    act_sizes[9] = L * B * T; // ln2_mean
    act_sizes[10] = L * B * T; // ln2_rstd
    act_sizes[11] = L * B * T * 4*C; // fch
    act_sizes[12] = L * B * T * 4*C; // fch_gelu
    act_sizes[13] = L * B * T * C; // fcproj
    act_sizes[14] = L * B * T * C; // residual3
    act_sizes[15] = B * T * C; // lnf
    act_sizes[16] = B * T; // lnf_mean
    act_sizes[17] = B * T; // lnf_rstd
    act_sizes[18] = B * T; // losses
    act_sizes[19] = L * B * T * 3*C; // qkvr
    act_sizes[20] = B * T * max(3*C, max(NH*T, Vp)); // output / scratch
}

// Backward pass is conceptually quite different from forward, because we can discard
// the activations of a layer as soon as we're done with it. This lets us aggressively
// reuse memory, so that we need far fewer tensors for backward state.
#define NUM_BACKWARD_TENSORS 3
typedef struct {
    float* bt4c; // (B, T, 4*C)
    float* preatt; // (B, NH, T, T)
    float* residual3; // (B, T, C)
} GradActTensors;


void fill_in_grad_act_sizes(size_t* act_sizes, int B, int T, GPT2Config config) {
    size_t NH = config.num_heads;
    size_t C = config.channels;
    act_sizes[0] = B * T * 4 * C; // bt4c
    act_sizes[1] = B * NH * T * T; // preatt
    act_sizes[2] = B * T * C; // residual3
}


float* malloc_and_point(float** targets[], const size_t* act_sizes, int n) {
    size_t num_activations = 0;
    for (size_t i = 0; i < n; i++) {
        num_activations += act_sizes[i];
    }
    float* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, num_activations * sizeof(float)));
    float* acts_memory_iterator = acts_memory;
    for (size_t i = 0; i < n; i++) {
        *(targets[i]) = acts_memory_iterator;
        acts_memory_iterator += act_sizes[i];
    }
    return acts_memory;
}

float* malloc_and_point_activations(ActivationTensors* acts, const size_t* act_sizes) {
    float** ptrs[] = {
        &acts->encoded, &acts->ln1, &acts->ln1_mean, &acts->ln1_rstd, &acts->atty,
        &acts->att, &acts->attproj, &acts->residual2, &acts->ln2, &acts->ln2_mean,
        &acts->ln2_rstd, &acts->fch, &acts->fch_gelu, &acts->fcproj, &acts->residual3, &acts->lnf,
        &acts->lnf_mean, &acts->lnf_rstd, &acts->losses, &acts->qkvr, &acts->output
    };
    return malloc_and_point(ptrs, act_sizes, NUM_ACTIVATION_TENSORS);
}

float* malloc_and_point_backward(GradActTensors* acts, const size_t* act_sizes) {
    float** ptrs[] = {
        &acts->bt4c, &acts->preatt, &acts->residual3
    };
    return malloc_and_point(ptrs, act_sizes, NUM_BACKWARD_TENSORS);
}

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_sizes[NUM_PARAMETER_TENSORS];
    float* params_memory;
    size_t num_parameters;
    // gradients of the weights
    ParameterTensors grads;
    float* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    // the activations of the model, and their sizes
    ActivationTensors acts;
    size_t act_sizes[NUM_ACTIVATION_TENSORS];
    float* acts_memory;
    size_t num_activations;
    // gradients of the activations
    GradActTensors grads_acts;
    size_t num_grad_acts;
    float* grads_acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after a forward pass with targets, will be populated with the mean loss
    float* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
} GPT2;

void gpt2_build_from_checkpoint(GPT2 *model, const char* checkpoint_path) {

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { fprintf(stderr, "Bad magic model file\n"); exit(EXIT_FAILURE); }
    if (model_header[1] != 3) {
        // was bumped from 1 -> 3 to incorporate the padded vocab size
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];
    model->config.padded_vocab_size = model_header[7];

    // allocate space for all the parameters and read them in
    fill_in_parameter_sizes(model->param_sizes, model->config);

    // count the number of parameters
    size_t num_parameters = 0;
    for (size_t i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += model->param_sizes[i];
    }
    model->num_parameters = num_parameters;

    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_sizes, 1);

    // read in all the parameters from file and copy them to device
    float* params_memory_cpu = (float*)mallocCheck(num_parameters * sizeof(float));
    freadCheck(params_memory_cpu, sizeof(float), num_parameters, model_file);
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, num_parameters * sizeof(float), hipMemcpyHostToDevice));
    free(params_memory_cpu);
    fcloseCheck(model_file);

    // other inits
    model->acts_memory = NULL;
    model->grads_memory = NULL;
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->grads_acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->cpu_losses = NULL;
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f will designate no loss
}

void gpt2_forward(GPT2 *model, int* inputs, int* targets, int B, int T) {
    // targets are optional and could be NULL

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    int V = model->config.vocab_size;
    int Vp = model->config.padded_vocab_size;
    int L = model->config.num_layers;
    int NH = model->config.num_heads;
    int C = model->config.channels;

    // validate inputs, all indices must be in the range [0, V)
    for(int i = 0; i < B * T; i++) {
        assert(0 <= inputs[i] && inputs[i] < V);
        if (targets != NULL) {
            assert(0 <= targets[i] && targets[i] < V);
        }
    }

    // allocate space for all the activations if needed (done here, lazily)
    if(model->acts_memory == NULL) {
        // record the current B,T as well
        model->batch_size = B;
        model->seq_len = T;
        // and now allocate the space
        fill_in_activation_sizes(model->act_sizes, B, T, model->config);
        size_t num_activations = 0;
        for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
            num_activations += model->act_sizes[i];
        }
        model->num_activations = num_activations;
        model->acts_memory = malloc_and_point_activations(&model->acts, model->act_sizes);
        printf("allocated %zu MiB for activations\n", (num_activations * sizeof(float)) >> 20); // >> 20 is /(1024*1024)
        // also create memory for caching inputs and targets
        cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
        cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(float)));
    } else {
        // validate B,T is consistent with how we've allocated the memory before
        // in principle we could get more clever here in the future, for now this is safest
        if (B != model->batch_size || T != model->seq_len) {
            printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, B, T);
            exit(EXIT_FAILURE);
        }
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    if (targets != NULL) {
        cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    }

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    float* residual;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C); // encoding goes into residual[0]

    for (int l = 0; l < L; l++) {

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        float* l_ln1w = params.ln1w + l * C;
        float* l_ln1b = params.ln1b + l * C;
        float* l_qkvw = params.qkvw + l * 3*C * C;
        float* l_qkvb = params.qkvb + l * 3*C;
        float* l_attprojw = params.attprojw + l * C * C;
        float* l_attprojb = params.attprojb + l * C;
        float* l_ln2w = params.ln2w + l * C;
        float* l_ln2b = params.ln2b + l * C;
        float* l_fcw = params.fcw + l * 4*C * C;
        float* l_fcb = params.fcb + l * 4*C;
        float* l_fcprojw = params.fcprojw + l * C * 4*C;
        float* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        float* l_ln1 = acts.ln1 + l * B * T * C;
        float* l_ln1_mean = acts.ln1_mean + l * B * T;
        float* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        float* l_qkvr = acts.qkvr + l * B * T * 3*C;
        float* l_atty = acts.atty + l * B * T * C;
        float* l_att = acts.att + l * B * NH * T * T;
        float* l_attproj = acts.attproj + l * B * T * C;
        float* l_residual2 = acts.residual2 + l * B * T * C;
        float* l_ln2 = acts.ln2 + l * B * T * C;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        float* l_fch = acts.fch + l * B * T * 4*C;
        float* l_fch_gelu = acts.fch_gelu + l * B * T * 4*C;
        float* l_fcproj = acts.fcproj + l * B * T * C;
        float* l_residual3 = acts.residual3 + l * B * T * C;
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        float* scratch = acts.output;

        // now do the forward pass
        layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C);
//         matmul_forward(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
//         attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH);
        matmul_forward(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
        flash_attention_forward(l_atty, l_qkvr, l_att, B, T, C, NH);
        matmul_forward(l_attproj, l_atty, l_attprojw, l_attprojb, B, T, C, C);
        residual_forward(l_residual2, residual, l_attproj, B*T*C);
        layernorm_forward(l_ln2, l_ln2_mean, l_ln2_rstd, l_residual2, l_ln2w, l_ln2b, B, T, C);
        matmul_forward(l_fch, l_ln2, l_fcw, l_fcb, B, T, C, 4*C);
        gelu_forward(l_fch_gelu, l_fch, B*T*4*C);
        matmul_forward(l_fcproj, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C);
        residual_forward(l_residual3, l_residual2, l_fcproj, B*T*C);
    }

    residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    layernorm_forward(acts.lnf, acts.lnf_mean, acts.lnf_rstd, residual, params.lnfw, params.lnfb, B, T, C);
    matmul_forward(acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp);

    // also forward the cross-entropy loss function if we have the targets
    if (targets != NULL) {
        // fused classifier: does the forward pass and first part of the backward pass
        // we're passing dlosses = NULL, which will default them to 1.0f/(B*T), i.e. uniform loss
        fused_classifier3(acts.output, acts.losses, NULL, model->targets, B, T, V, Vp);
        // for convenience also evaluate the mean loss (TODO re-think this compute+sync point)
        // move the (B,T) losses to CPU
        cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(float), hipMemcpyDeviceToHost));
        float mean_loss = 0.0f;
        for (int i=0; i<B*T; i++) { mean_loss += model->cpu_losses[i]; }
        mean_loss /= B*T;
        model->mean_loss = mean_loss;

    } else {
        // if we don't have targets, we don't have loss
        model->mean_loss = -1.0f;
    }
}

void gpt2_zero_grad(GPT2 *model) {
    if (model->grads_acts_memory != NULL) { cudaCheck(hipMemset(model->grads_acts_memory, 0, model->num_grad_acts * sizeof(float))); }
    if (model->grads_memory != NULL) { cudaCheck(hipMemset(model->grads_memory, 0, model->num_parameters * sizeof(float))); }
}

void gpt2_backward(GPT2 *model) {

    // double check we forwarded previously, with targets
    if (model->mean_loss == -1.0f) {
        printf("Error: must forward with targets before backward\n");
        exit(EXIT_FAILURE);
    }

    // lazily allocate the memory for gradients of the weights and activations, if needed
    if (model->grads_memory == NULL) {
        // allocate buffers for weight gradients
        model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_sizes, 1);
        printf("allocated %zu MiB for parameter gradients\n", (model->num_parameters * sizeof(float)) >> 20);
        // we're going to be clever for the activations backward pass. we don't need to exactly
        // mirror the forward pass acrtivations and we will save memory.
        size_t bw_act_sizes[NUM_ACTIVATION_TENSORS];
        GPT2Config cfg = model->config;
        cfg.num_layers = 1; // copy the configuration but override number of layers to 1
        fill_in_grad_act_sizes(bw_act_sizes, model->batch_size, model->seq_len, cfg);
        // count up and allocate the space
        model->grads_acts_memory = malloc_and_point_backward(&model->grads_acts, bw_act_sizes);
        model->num_grad_acts = 0;
        for (int i = 0; i < NUM_BACKWARD_TENSORS; i++) {
            model->num_grad_acts += bw_act_sizes[i];
        }
        printf("allocated %zu MiB for activation gradients\n", (model->num_grad_acts * sizeof(float)) >> 20);
        // init gradients of parameters and activations to zero
        gpt2_zero_grad(model);
    }

    // convenience shortcuts
    int B = model->batch_size;
    int T = model->seq_len;
    int Vp = model->config.padded_vocab_size;
    int L = model->config.num_layers;
    int NH = model->config.num_heads;
    int C = model->config.channels;

    // backward pass: go in the reverse order of the forward pass, and call backward() functions
    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;
    GradActTensors grads_acts = model->grads_acts;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(grads_acts.bt4c, grads.wte, NULL, acts.output, acts.lnf, params.wte, B, T, C, Vp);
    // backward the final layernorm
    float* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    float* dresidual = grads_acts.residual3; // the main buffer holding the gradient in the backward pass
    layernorm_backward(dresidual, grads.lnfw, grads.lnfb, grads_acts.bt4c, residual, params.lnfw, acts.lnf_mean, acts.lnf_rstd, B, T, C);

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        float* l_ln1w = params.ln1w + l * C;
        float* l_qkvw = params.qkvw + l * 3*C * C;
        float* l_attprojw = params.attprojw + l * C * C;
        float* l_ln2w = params.ln2w + l * C;
        float* l_fcw = params.fcw + l * 4*C * C;
        float* l_fcprojw = params.fcprojw + l * C * 4*C;
        // get the pointers of the gradients of the weights for this layer
        float* dl_ln1w = grads.ln1w + l * C;
        float* dl_ln1b = grads.ln1b + l * C;
        float* dl_qkvw = grads.qkvw + l * 3*C * C;
        float* dl_qkvb = grads.qkvb + l * 3*C;
        float* dl_attprojw = grads.attprojw + l * C * C;
        float* dl_attprojb = grads.attprojb + l * C;
        float* dl_ln2w = grads.ln2w + l * C;
        float* dl_ln2b = grads.ln2b + l * C;
        float* dl_fcw = grads.fcw + l * 4*C * C;
        float* dl_fcb = grads.fcb + l * 4*C;
        float* dl_fcprojw = grads.fcprojw + l * C * 4*C;
        float* dl_fcprojb = grads.fcprojb + l * C;
        // get the pointers of the activations for this layer
        float* l_ln1 = acts.ln1 + l * B * T * C;
        float* l_ln1_mean = acts.ln1_mean + l * B * T;
        float* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        float* l_qkvr = acts.qkvr + l * B * T * 3*C;
        float* l_atty = acts.atty + l * B * T * C;
        float* l_att = acts.att + l * B * NH * T * T;
        float* l_residual2 = acts.residual2 + l * B * T * C;
        float* l_ln2 = acts.ln2 + l * B * T * C;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        float* l_fch = acts.fch + l * B * T * 4*C;
        float* l_fch_gelu = acts.fch_gelu + l * B * T * 4*C;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        // we need a B x T x C buffer; thankfully, the forward activation for lnf isn't needed anymore,
        // so we can co-opt it here.
        float* dl_btc = acts.lnf;
        float* dl_bt4c = grads_acts.bt4c;
        float* dl_preatt = grads_acts.preatt;

        // re-use scratch buffer of the forward pass
        float* scratch = acts.output;

        // backprop this layer
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_gelu, l_fcprojw, B, T, 4*C, C);
        gelu_backward(dl_bt4c, l_fch, dl_bt4c, B*T*4*C);
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c, l_ln2, l_fcw, B, T, C, 4 * C);
        // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, dl_ln2w, dl_ln2b, dl_btc, l_residual2, l_ln2w, l_ln2_mean, l_ln2_rstd, B, T, C);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, B, T, C, C);
        // we more B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        float* buffer_a = l_atty;
        float* buffer_b = l_fch;        // this is B x T x 4C, so even larger than what we need

//         attention_backward(dl_bt4c, buffer_b, dl_preatt, scratch, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH);
        flash_attention_backward(dl_bt4c, l_qkvr, dl_btc, l_atty, l_att, l_fch, B, T, C, NH);
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c, l_ln1, l_qkvw, B, T, C, 3 * C);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, dl_ln1w, dl_ln1b, dl_btc, residual, l_ln1w, l_ln1_mean, l_ln1_rstd, B, T, C);
    }
    encoder_backward(grads.wte, grads.wpe, dresidual, model->inputs, B, T, C);
}

void gpt2_update(GPT2 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, int t) {
    // reference: https://pytorch.org/docs/stable/generated/torch.optim.AdamW.html

    // lazily allocate the memory for m_memory and v_memory
    if (model->m_memory == NULL) {
        cudaCheck(hipMalloc((void**)&model->m_memory, model->num_parameters * sizeof(float)));
        cudaCheck(hipMalloc((void**)&model->v_memory, model->num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->m_memory, 0, model->num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->v_memory, 0, model->num_parameters * sizeof(float)));
        printf("allocated %zu MiB for AdamW optimizer state m\n", (model->num_parameters * sizeof(float)) >> 20);
        printf("allocated %zu MiB for AdamW optimizer state v\n", (model->num_parameters * sizeof(float)) >> 20);
    }

    int block_size = 512;
    int num_blocks = CEIL_DIV(model->num_parameters, block_size);
    float beta1_correction = 1.0f - powf(beta1, t);
    float beta2_correction = 1.0f - powf(beta2, t);
    adamw_kernel2<<<num_blocks, block_size>>>(model->params_memory, model->grads_memory, model->m_memory, model->v_memory,
                                              model->num_parameters,
                                              learning_rate, beta1, beta2, beta1_correction, beta2_correction, eps, weight_decay);
    cudaCheck(hipGetLastError());
}

void gpt2_free(GPT2 *model) {
    cudaCheck(hipFree(model->params_memory));
    cudaCheck(hipFree(model->grads_memory));
    cudaCheck(hipFree(model->m_memory));
    cudaCheck(hipFree(model->v_memory));
    cudaCheck(hipFree(model->acts_memory));
    cudaCheck(hipFree(model->grads_acts_memory));
    cudaCheck(hipFree(model->inputs));
    cudaCheck(hipFree(model->targets));
    hipHostFree(model->cpu_losses);
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip the int main below
// ----------------------------------------------------------------------------
// sampler: takes probabilities and samples integers from them

#define GPT2_EOT 50256

unsigned int random_u32(unsigned long long *state) {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    *state ^= *state >> 12;
    *state ^= *state << 25;
    *state ^= *state >> 27;
    return (*state * 0x2545F4914F6CDD1Dull) >> 32;
}
float random_f32(unsigned long long *state) { // random float32 in [0,1)
    return (random_u32(state) >> 8) / 16777216.0f;
}

int sample_softmax(const float* logits, int n, float coin) {
    // sample index from logits (converted to probabilities using softmax)
    // coin is a random number in [0, 1), usually from random_f32()
    double norm = 0;
    for (int i = 0; i < n; i++) {
        norm += expf(logits[i]);
    }
    // instead of dividing all exp(logits), we can just multiply coin.
    coin *= norm;
    float cdf = 0.0f;
    for (int i = 0; i < n; i++) {
        cdf += expf(logits[i]);
        if (coin < cdf) {
            return i;
        }
    }
    return n - 1; // in case of rounding errors
}

// ----------------------------------------------------------------------------
// Logger lite, will probably grow/change some over time

typedef struct {
    FILE *logfile;
    int flush_every; // every how many steps to flush the log
} Logger;

void logger_init(Logger *logger, const char *filename) {
    logger->flush_every = 20;
    logger->logfile = NULL;
    if (filename != NULL) { logger->logfile = fopenCheck(filename, "w"); }
}

void logger_log_val(Logger *logger, int step, float val_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d tel:%.4f\n", step, val_loss);
    }
}

void logger_log_train(Logger *logger, int step, float train_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d trl:%.4f\n", step, train_loss);
        if (step % 10 == 0) { fflush(logger->logfile); }
    }
}

void logger_free(Logger *logger) {
    if (logger->logfile != NULL) { fclose(logger->logfile); }
}

// ----------------------------------------------------------------------------
// CLI, poor man's argparse

void error_usage() {
    fprintf(stderr, "Usage:   ./train_gpt2fp32cu [options]\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -i <string> train data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_train.bin)\n");
    fprintf(stderr, "  -j <string> val data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_val.bin)\n");
    fprintf(stderr, "  -o <string> output log file (default = NULL)\n");
    fprintf(stderr, "  -b <int>    batch size B (default = 4)\n");
    fprintf(stderr, "  -t <int>    sequence length T (default = 1024)\n");
    fprintf(stderr, "  -l <float>  learning rate (default = 3e-4f)\n");
    fprintf(stderr, "  -v <int>    val_loss_every, how often we evaluate val loss (default = 20)\n");
    fprintf(stderr, "  -m <int>    val_max_steps, up to how many val batches to estimate val loss? (default = 20)\n");
    fprintf(stderr, "  -s <int>    sample_every, how often we inference the model (default = 20)\n");
    fprintf(stderr, "  -g <int>    genT, how many steps of inference we do (default = 64)\n");
    exit(EXIT_FAILURE);
}

// ----------------------------------------------------------------------------
// main training loop
int main(int argc, char *argv[]) {

    // read in the (optional) command line arguments
    const char* train_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
    const char* val_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
    const char* output_log_file = NULL;
    int B = 4; // batch size
    int T = 1024; // sequence length max
    float learning_rate = 3e-4f;
    int val_loss_every = 20; // every how many steps do we eval validation loss?
    int val_max_steps = 20; // how many batches max do we eval for validation loss?
    int sample_every = 20; // every how many steps to do inference?
    int genT = 64; // number of steps of inference we will do
    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (strlen(argv[i]) != 2) { error_usage(); } // must be -x (one dash, one letter)
        // read in the args
        if (argv[i][1] == 'i') { train_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'j') { val_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'o') { output_log_file = argv[i+1]; }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); }
        else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'l') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 's') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else { error_usage(); }
    }
    printf("+-----------------------+----------------------------------------------------+\n");
    printf("| Parameter             | Value                                              |\n");
    printf("+-----------------------+----------------------------------------------------+\n");
    printf("| train data pattern    | %-50s |\n", train_data_pattern);
    printf("| val data pattern      | %-50s |\n", val_data_pattern);
    printf("| output log file       | %-50s |\n", output_log_file == NULL ? "NULL" : output_log_file);
    printf("| batch size B          | %-50d |\n", B);
    printf("| sequence length T     | %-50d |\n", T);
    printf("| learning rate         | %-50f |\n", learning_rate);
    printf("| val_loss_every        | %-50d |\n", val_loss_every);
    printf("| val_max_steps         | %-50d |\n", val_max_steps);
    printf("| sample_every          | %-50d |\n", sample_every);
    printf("| genT                  | %-50d |\n", genT);
    printf("+-----------------------+----------------------------------------------------+\n");

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    printf("| device                | %-50s |\n", deviceProp.name);
    printf("| TF32                  | %-50s |\n", enable_tf32 ? "enabled" : "disabled");
    printf("+-----------------------+----------------------------------------------------+\n");

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, "gpt2_124M.bin");
    printf("| max_sequence_length T | %-50d |\n", model.config.max_seq_len);
    printf("| vocab_size V          | %-50d |\n", model.config.vocab_size);
    printf("| padded_vocab_size Vp  | %-50d |\n", model.config.padded_vocab_size);
    printf("| num_layers L          | %-50d |\n", model.config.num_layers);
    printf("| num_heads NH          | %-50d |\n", model.config.num_heads);
    printf("| channels C            | %-50d |\n", model.config.channels);
    printf("| num_parameters        | %-50zu |\n", model.num_parameters);
    printf("+-----------------------+----------------------------------------------------+\n");

    // build DataLoaders for both train and val
    DataLoader train_loader, val_loader;
    dataloader_init(&train_loader, train_data_pattern, B, T, 0, 1, 1);
    dataloader_init(&val_loader, val_data_pattern, B, T, 0, 1, 0);
    int train_num_batches = train_loader.num_tokens / (B*T); // let's do 1 epoch by default for now
    int val_num_batches = val_loader.num_tokens / (B*T);
    if (val_num_batches > val_max_steps) { val_num_batches = val_max_steps; }
    printf("| train_num_batches     | %-50d |\n", train_num_batches);
    printf("| val_num_batches       | %-50d |\n", val_num_batches);
    printf("+-----------------------+----------------------------------------------------+\n");

    // print model parameter allocations from gpt2_build_from_checkpoint down here to not mess up our table above
    printf("allocated %d MiB for model parameters\n", (int)round(model.num_parameters * sizeof(float) / (1024 * 1024)));

    // set up the Logger
    Logger logger;
    logger_init(&logger, output_log_file);

    // build the Tokenizer
    Tokenizer tokenizer;
    tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

    // some memory for generating samples from the model
    unsigned long long rng_state = 1337;
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    float* cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    // train
    struct timespec start, end;
    double total_sum_iteration_time_s = 0.0;
    for (int step = 0; step <= train_num_batches; step++) {
        int last_step = step == train_num_batches;

        // once in a while estimate the validation loss
        if (step % val_loss_every == 0 || last_step) {
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                gpt2_forward(&model, val_loader.inputs, val_loader.targets, B, T);
                val_loss += model.mean_loss;
            }
            val_loss /= val_num_batches;
            printf("val loss %f\n", val_loss);
            logger_log_val(&logger, step, val_loss);
        }

        // once in a while do model inference to print generated text
        if (step > 0 && step % sample_every == 0 || last_step) {
            // fill up gen_tokens with the GPT2_EOT, which kicks off the generation
            for(int i = 0; i < B * T; ++i) {
                gen_tokens[i] = GPT2_EOT;
            }
            // now sample from the model autoregressively
            printf("generating:\n---\n");
            for (int t = 1; t < genT; t++) {
                // note that inference is very wasteful here because for each token
                // we re-calculate the forward pass for all of (B,T) positions from scratch
                // but the inference here is just for sanity checking anyway
                // and we can maybe optimize a bit more later, with careful tests
                gpt2_forward(&model, gen_tokens, NULL, B, T);
                // furthermore, below we're only using b=0 (i.e. the first row) of all B rows
                // we're in principle running B "inference streams" in parallel here
                // only using position 0 because it's a bit faster (copy less probs from GPU -> CPU)
                // get the V-dimensional vector probs[0, t-1, :]
                float* logits = model.acts.output + (t - 1) * model.config.padded_vocab_size;
                // move probs back to CPU and sample (note we only move the first vocab_size logits, ignoring the padding)
                cudaCheck(hipMemcpy(cpu_logits, logits, model.config.vocab_size * sizeof(float), hipMemcpyDeviceToHost));
                float coin = random_f32(&rng_state);
                int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
                // print the generated token, either using the Tokenizer or a fallback
                if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                } else {
                    // fall back to printing the token id
                    printf("%d ", next_token);
                }
                fflush(stdout);
            }
            printf("\n---\n");
        }

        // bit confusing: we want to make sure to eval and sample on 0th iteration
        // but also after the very last iteration. so we loop for step <= train_num_batches
        // instead of just < train_num_batches (one extra due to <=), only to do
        // the validation/sampling one last time, and then we break right here as we're done.
        if (last_step) { break; }

        // do a training step
        clock_gettime(CLOCK_MONOTONIC, &start);
        dataloader_next_batch(&train_loader);
        gpt2_forward(&model, train_loader.inputs, train_loader.targets, B, T);
        gpt2_zero_grad(&model);
        gpt2_backward(&model);
        gpt2_update(&model, learning_rate, 0.9f, 0.999f, 1e-8f, 0.0f, step+1);
        cudaCheck(hipDeviceSynchronize()); // finish all CUDA work to get correct precise timings
        clock_gettime(CLOCK_MONOTONIC, &end);
        double time_elapsed_s = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
        total_sum_iteration_time_s += time_elapsed_s;
        int tokens_per_second = (B * T) / time_elapsed_s;
        printf("step %4d/%d: train loss %f (%f ms, %d tok/s)\n", step + 1, train_num_batches, model.mean_loss, time_elapsed_s * 1000, tokens_per_second);
        logger_log_train(&logger, step, model.mean_loss);
    }
    // add a total average, for optimizations that are only mild improvements
    printf("total average iteration time: %f ms\n", total_sum_iteration_time_s / train_num_batches * 1000);

    // free
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    gpt2_free(&model);
    free(cpu_logits);
    free(gen_tokens);
    cublasCheck(hipblasDestroy(cublas_handle));
    logger_free(&logger);

    return 0;
}
#endif