#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net training loop. See README.md for usage.
*/
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string>
#include <string_view>
#include <sys/stat.h>
#include <sys/types.h>
// ----------- CPU utilities -----------
// defines: fopenCheck, freadCheck, fcloseCheck, fseekCheck, mallocCheck
// defines: create_dir_if_not_exists, find_max_step, ends_with_bin
#include "llmc/utils.h"
// defines: tokenizer_init, tokenizer_decode, tokenizer_free
#include "llmc/tokenizer.h"
// defines: dataloader_init, dataloader_reset, dataloader_next_batch, dataloader_free
// defines: evalloader_init, evalloader_reset, evalloader_next_batch, evalloader_free
#include "llmc/dataloader.h"
// defines: manual_seed, normal_ (same as torch.manual_seed and torch.normal)
#include "llmc/rand.h"
// defines: lr_scheduler_init, get_learning_rate
#include "llmc/schedulers.h"
// defines: sample_softmax, random_f32
#include "llmc/sampler.h"
// defines: logger_init, logger_log_eval, logger_log_val, logger_log_train
#include "llmc/logger.h"
// defines: get_flops_promised
#include "llmc/mfu.h"
// defines: OutlierDetector, init_detector, update_detector
#include "llmc/outlier_detector.h"
// ----------- GPU utilities -----------
// defines:
// WARP_SIZE, MAX_1024_THREADS_BLOCKS, CEIL_DIV, cudaCheck, PRECISION_MODE
// NVTX_RANGE_FN
#include "llmc/cuda_common.h"
// defines:
// Packed128, f128, x128
// warpReduceSum, warpReduceMax, blockReduce, copy_and_cast_kernel, cudaMallocConditionallyManaged
#include "llmc/cuda_utils.cuh"
// defines: CUBLAS_LOWP, cublasCheck, cublaslt_workspace_size, cublaslt_workspace
// defines: cublas_compute, cublaslt_handle, cublas_handle
#include "llmc/cublas_common.h"
// ----------- Layer implementations in CUDA -----------
// defines: encoder_forward, encoder_backward
#include "llmc/encoder.cuh"
// defines: layernorm_forward, residual_forward, fused_residual_forward5, layernorm_backward
#include "llmc/layernorm.cuh"
// defines: matmul_cublaslt, matmul_forward, matmul_backward, gelu_forward, gelu_backward_inplace
#include "llmc/matmul.cuh"
#ifdef ENABLE_CUDNN
// defines: create_cudnn, destroy_cudnn, attention_forward_cudnn, attention_backward_cudnn
#include "llmc/cudnn_att.h"
#else
// defines: attention_forward, attention_backward
#include "llmc/attention.cuh"
#endif
// defines: fused_classifier
#include "llmc/fused_classifier.cuh"
// defines: adamw_kernel3
#include "llmc/adamw.cuh"
// defines: global_norm_squared
#include "llmc/global_norm.cuh"
// ----------- Multi-GPU support -----------
// defines: ncclFloatX, ncclCheck, MultiGpuConfig, ShardInfo
// defines: printf0, multi_gpu_config
// defines: multi_gpu_config_init, multi_gpu_config_free
// defines: set_zero_configs, multi_gpu_cpu_float_sum, multi_gpu_barrier
// defines: multi_gpu_get_shard_offset, multi_gpu_async_reduce_gradient
#include "llmc/zero.cuh"

// ----------------------------------------------------------------------------
// global vars for I/O
char filename_buffer[512];

// ----------------------------------------------------------------------------
// global vars containing information about the GPU this process is running on
hipDeviceProp_t deviceProp; // fills in common_start()
hipStream_t main_stream;
// buffer size to use for device <-> disk io
constexpr const size_t IO_BUF_SIZE = 32 * 1024 * 1024;

// ----------------------------------------------------------------------------
// GPT-2 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int padded_vocab_size; // padded to e.g. %128==0, 50304
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

// the parameters of the model
constexpr const int NUM_PARAMETER_TENSORS = 16;
typedef struct {
    floatX* wte; // (V, C)
    floatX* wpe; // (maxT, C)
    floatX* ln1w; // (L, C)
    floatX* ln1b; // (L, C)
    floatX* qkvw; // (L, 3*C, C)
    floatX* qkvb; // (L, 3*C)
    floatX* attprojw; // (L, C, C)
    floatX* attprojb; // (L, C)
    floatX* ln2w; // (L, C)
    floatX* ln2b; // (L, C)
    floatX* fcw; // (L, 4*C, C)
    floatX* fcb; // (L, 4*C)
    floatX* fcprojw; // (L, C, 4*C)
    floatX* fcprojb; // (L, C)
    floatX* lnfw; // (C)
    floatX* lnfb; // (C)
} ParameterTensors;
static_assert(sizeof(ParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

void fill_in_parameter_sizes(size_t* param_sizes, size_t* param_sizeof, GPT2Config config) {
    size_t Vp = config.padded_vocab_size;
    size_t C = config.channels;
    size_t maxT = config.max_seq_len;
    size_t L = config.num_layers;
    param_sizes[0] = Vp * C; // wte
    param_sizes[1] = maxT * C; // wpe
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b
    param_sizes[4] = L * (3 * C) * C; // qkvw
    param_sizes[5] = L * (3 * C); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * (4 * C) * C; // fcw
    param_sizes[11] = L * (4 * C); // fcb
    param_sizes[12] = L * C * (4 * C); // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb

    // populate the parameter sizes in bytes (all the same for now, keeping for future use)
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        param_sizeof[i] = sizeof(floatX);
    }
}

// allocate memory for the parameters and point the individual tensors to the right places
void* malloc_and_point_parameters(ParameterTensors* params, size_t* param_elements, size_t *param_sizeof) {
    // calculate the total number of parameters and bytes across all tensors
    size_t num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters_bytes += param_elements[i] * param_sizeof[i];
    }
    // malloc all parameters all at once on the device
    void* params_memory;
    cudaCheck(hipMalloc((void**)&params_memory, num_parameters_bytes));
    // assign all the tensors their place in the array
    floatX** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    char* params_memory_iterator = (char*)params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = (floatX*)params_memory_iterator;
        params_memory_iterator += param_elements[i] * param_sizeof[i];
    }
    return params_memory;
}

constexpr int NUM_ACTIVATION_TENSORS = 21;
typedef struct {
    floatX* encoded; // (B, T, C)
    floatX* ln1; // (L, B, T, C)
    float* ln1_mean; // (L, B, T)
    float* ln1_rstd; // (L, B, T)
    floatX* atty; // (L, B, T, C)
    // cuDNN saves only some statistics information
#if ENABLE_CUDNN
    float* att;  // (L, B, NH, T)
#else
    floatX* att; // (L, B, NH, T, T)
#endif

    floatX* residual2; // (L, B, T, C)
    floatX* ln2; // (L, B, T, C)
    float* ln2_mean; // (L, B, T)
    float* ln2_rstd; // (L, B, T)
    floatX* fch; // (L, B, T, 4*C)
    floatX* fch_gelu; // (L, B, T, 4*C)
    floatX* residual3; // (L, B, T, C)
    floatX* lnf; // (B, T, C);   if LN recomputation is enabled (-r 2 and above), will be used for _all_ layernorms
    float* lnf_mean; // (B, T)
    float* lnf_rstd; // (B, T)
    float* losses; // (B, T), will be accumulated in micro-steps
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    floatX* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    floatX* output;

    // some additional scratch buffers
    floatX* scratch_bt4c;   // (B, T, 4*C)
    floatX* scratch_btc;    // (B, T, C)
} ActivationTensors;


struct TensorSpec {
    void** ptr;
    size_t size;
    DType type;
};


#define TENSOR_SPEC(pointer, size) TensorSpec{(void**)(&pointer), (size), dtype_of(pointer)};

void fill_in_activation_sizes(const ActivationTensors* data, TensorSpec (&tensors)[NUM_ACTIVATION_TENSORS], size_t B, size_t T, GPT2Config config, int recompute) {
    size_t Vp = config.padded_vocab_size;
    size_t L = config.num_layers;
    size_t NH = config.num_heads;
    size_t C = config.channels;
    tensors[0] = TENSOR_SPEC(data->encoded, B * T * C);
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    tensors[1] = TENSOR_SPEC(data->ln1,  (recompute < 2) ? L * B * T * C : 0);
    tensors[2] = TENSOR_SPEC(data->ln1_mean, L * B * T);
    tensors[3] = TENSOR_SPEC(data->ln1_rstd, L * B * T);
    tensors[4] = TENSOR_SPEC(data->atty, L * B * T * C);
    #ifdef ENABLE_CUDNN
    // FP32 stats tensor for cuDNN to be passed to backward pass
    tensors[5] = TENSOR_SPEC(data->att, L * B * NH * T);
    #else
    tensors[5] = TENSOR_SPEC(data->att, L * B * NH * T * T);
    #endif
    tensors[6] = TENSOR_SPEC(data->residual2, L * B * T * C);
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    tensors[7] = TENSOR_SPEC(data->ln2, (recompute < 2) ? L * B * T * C : 0);
    tensors[8] = TENSOR_SPEC(data->ln2_mean, L * B * T);
    tensors[9] = TENSOR_SPEC(data->ln2_rstd, L * B * T);
    tensors[10] = TENSOR_SPEC(data->fch, L * B * T * 4*C);
    // if recompute >= 1 then we will recompute gelu_forward during backward and use this as scratch buffer
    tensors[11] = TENSOR_SPEC(data->fch_gelu, (recompute < 1) ? L * B * T * 4*C : B * T * 4*C);
    tensors[12] = TENSOR_SPEC(data->residual3, L * B * T * C);
    tensors[13] = TENSOR_SPEC(data->lnf, B * T * C);
    tensors[14] = TENSOR_SPEC(data->lnf_mean, B * T);
    tensors[15] = TENSOR_SPEC(data->lnf_rstd, B * T);
    tensors[16] = TENSOR_SPEC(data->losses, B * T);
    tensors[17] = TENSOR_SPEC(data->qkvr, L * B * T * 3*C);
    tensors[18] = TENSOR_SPEC(data->output, B * T * max(3*C, max(NH*T, Vp)));

    tensors[19] = TENSOR_SPEC(data->scratch_bt4c, B * T * 4 * C);
    tensors[20] = TENSOR_SPEC(data->scratch_btc, B * T * C);
}

void* malloc_and_point_activations(TensorSpec (&tensors)[NUM_ACTIVATION_TENSORS]) {
    size_t bytes = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        bytes += tensors[i].size * sizeof_dtype(tensors[i].type);
    }

    printf0("allocating %d MiB for activations\n", (int)round(bytes / (1024 * 1024)));

    void* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, bytes));

    // hipMalloc does not guarantee initial memory values so we memset the allocation here
    // this matters because e.g. non-cuDNN attention assumes the attention buffer is zeroed
    // todo - up to ~100ms on slow GPUs, could theoretically be more selective, but this is safer
    cudaCheck(hipMemset(acts_memory, 0, bytes));

    char* acts_memory_iterator = (char*)acts_memory;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        // extra protection so we don't accidentally use an empty buffer
        if(tensors[i].size == 0) {
            *(tensors[i].ptr) = NULL;
        }else {
            *(tensors[i].ptr) = acts_memory_iterator;
            acts_memory_iterator += tensors[i].size * sizeof_dtype(tensors[i].type);
        }
    }
    return acts_memory;
}

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_elements[NUM_PARAMETER_TENSORS];
    size_t param_sizeof[NUM_PARAMETER_TENSORS];
    void* params_memory;
    size_t num_parameters;
    size_t num_parameters_bytes;
    // gradients of the weights
    ParameterTensors grads;
    void* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    float* master_weights;     // is NULL unless fp32 weights is enabled.
    // the activations of the model, and their sizes
    ActivationTensors acts;
    TensorSpec acts_specs[NUM_ACTIVATION_TENSORS];
    void* acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after the last backward micro-batch, will be populated with mean loss across all GPUs and micro-steps
    float* accumulated_mean_loss; // GPU buffer used to accumulate loss across micro-steps
    float* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
    unsigned long long rng_state; // the RNG state for seeding stochastic rounding etc.
    unsigned long long rng_state_last_update; // RNG before last gpt2_update() to re-round identically from master weights
    int use_master_weights; // keep master weights copy in float for optim update? 0|1
    bool init_state;   // set to true if master weights need to be initialized
    int gelu_fusion; // fuse gelu via cuBLASLt (0=none, 1=forward, 2=forward+backward)
    int recompute; // recompute gelu | layernorm forward during model backward? 0|1|2
    // todo - if other functions need cpu scratch buffers in the future, reuse as generic scratch?
    int* workload_indices; // encoder_backward, B*T*num_c_groups (int)
    int4* bucket_info;     // encoder_backward, B*T*num_c_groups (int4) - size for worst case
} GPT2;

void gpt2_init_common(GPT2 *model) {
    // common inits outside of the model weights
    // memory lazily initialized in forward()
    model->acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->accumulated_mean_loss = NULL;
    model->cpu_losses = NULL;
    // the B,T params are determined and set, fixed on first batch in forward()
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f designates no loss, set at end of forward()
    model->params_memory = NULL;
    // memory lazily initialized in backward()
    model->grads_memory = NULL;
    model->workload_indices = NULL; // on cpu, for encoder_backward
    model->bucket_info = NULL; // on cpu, for encoder_backward
    // memory lazily initialized in update()
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    // other default settings
    model->rng_state = 13371337 + multi_gpu_config.process_rank; // used in stochastic rounding
    model->use_master_weights = 1; // safe default: do keep master weights in fp32
    model->init_state = true;
    model->recompute = 1; // good default: recompute gelu but not layernorm
    model->gelu_fusion = 0; //deviceProp.major >= 9 ? 2 : 0; // default: off for now (default must match main())
}

void gpt2_allocate_weights(GPT2 *model) {
    // fill in all the parameter tensor dimensions and types
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);
    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }
    // create memory for model parameters on the device
    assert(model->params_memory == nullptr);
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);
}

void gpt2_allocate_state(GPT2 *model, int B, int T) {
    printf0("allocating %d MiB for parameter gradients\n", (int)round(model->num_parameters * sizeof(floatX) / (1024 * 1024)));
    assert(model->grads_memory == nullptr);
    model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_elements, model->param_sizeof);

    // record the current B,T as well
    model->batch_size = B;
    model->seq_len = T;

    // allocate the space
    fill_in_activation_sizes(&model->acts, model->acts_specs, B, T, model->config, model->recompute);
    model->acts_memory = malloc_and_point_activations(model->acts_specs);
    // also create memory for caching inputs and targets
    cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
    cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
    cudaCheck(hipMalloc(((void**)&model->accumulated_mean_loss), sizeof(float)));
    cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(float)));

    // initialise cpu scratch buffers for encoder backward
    size_t num_c_groups = CEIL_DIV(model->config.channels, (WARP_SIZE * x128::size));
    assert((size_t)(model->batch_size * model->seq_len) * num_c_groups < (1ULL<<31ULL)); // todo - maybe an issue for llama3-400B(?)
    model->workload_indices = (int*)mallocCheck(sizeof(int) * model->batch_size * model->seq_len * num_c_groups);
    model->bucket_info = (int4*)mallocCheck(sizeof(int4) * model->batch_size * model->seq_len * num_c_groups);

    // cudaMallocConditionallyManaged can fall back to hipMallocManaged if not enough memory on device
    // and returns a status code of 1 if it had to fall back, in that case we want to print warning.
    int memory_status = 0;

    // we will now init the optimizer states and master weights
    // this is usually a substantial amount of memory allocation right here.
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters; // num parameters we are responsible for
    printf0("allocating %zu MiB for AdamW optimizer state m\n", (shard_num_parameters * sizeof(float)) >> 20);
    printf0("allocating %zu MiB for AdamW optimizer state v\n", (shard_num_parameters * sizeof(float)) >> 20);
    assert(model->m_memory == nullptr);
    assert(model->v_memory == nullptr);
    memory_status |= cudaMallocConditionallyManaged((void**)&model->m_memory, shard_num_parameters * sizeof(float));
    memory_status |= cudaMallocConditionallyManaged((void**)&model->v_memory, shard_num_parameters * sizeof(float));

    if (model->use_master_weights == 1) {
        assert(model->master_weights == nullptr);
        printf0("allocating %zu MiB for master copy of params\n", (shard_num_parameters * sizeof(float)) >> 20);
        memory_status |= cudaMallocConditionallyManaged((void**) &model->master_weights, shard_num_parameters * sizeof(float));
    }

    // report on mixed memory allocation status (re-using our float reduce function, bit awk ok)
    int reduced_memory_status = (int) multi_gpu_cpu_float_sum((float)memory_status, &multi_gpu_config);
    if (reduced_memory_status >= 1) {
        printf0("WARNING: Fell back to hipMallocManaged when initializing m,v,master_weights on %d GPUs\n", reduced_memory_status);
        printf0("         Prevents an OOM, but code may run much slower due to device <-> host memory movement\n");
    }
    // report on device memory usage
    size_t free, total;
    cudaCheck(hipMemGetInfo(&free, &total));
    printf0("device memory usage: %zd MiB / %zd MiB\n", (total-free) / 1024 / 1024, total / 1024 / 1024);
    // give an estimate of the maximum batch size
    size_t bytes_per_sequence = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        bytes_per_sequence += model->acts_specs[i].size * sizeof_dtype(model->acts_specs[i].type) / B;
    }
    printf0("memory per sequence: %zu MiB\n", bytes_per_sequence / 1024 / 1024);
    printf0(" -> estimated maximum batch size: %zu\n", B + free / bytes_per_sequence);
}

void gpt2_write_to_checkpoint(GPT2 *model, const char* checkpoint_path) {
    // write the model to a checkpoint file
    printf0("Writing model to %s\n", checkpoint_path);
    FILE *model_file = fopenCheck(checkpoint_path, "wb");
    // write the header first
    int model_header[256];
    memset(model_header, 0, sizeof(model_header));
    model_header[0] = 20240326; // magic number
    assert(PRECISION_MODE == PRECISION_FP32 || PRECISION_MODE == PRECISION_BF16);
    model_header[1] = PRECISION_MODE == PRECISION_FP32 ? 3 : 5; // version
    model_header[2] = model->config.max_seq_len;
    model_header[3] = model->config.vocab_size;
    model_header[4] = model->config.num_layers;
    model_header[5] = model->config.num_heads;
    model_header[6] = model->config.channels;
    model_header[7] = model->config.padded_vocab_size;
    fwriteCheck(model_header, sizeof(int), 256, model_file);
    // write the parameters
    device_to_file(model_file, model->params_memory, model->num_parameters_bytes,
                   IO_BUF_SIZE, main_stream);
    // close file, we're done
    fcloseCheck(model_file);
}

void gpt2_build_from_checkpoint(GPT2 *model, const char* checkpoint_path, bool weight_init=true) {
    // If weight_init is true, we will load the weights from this checkpoint .bin file
    // We sometimes want this to be false, if we are going to initialize these weights from
    // the master weights that are instead stored in the state .bin file.
    // In that case, this function mostly loads the model hyperparameters from the header.

    if (PRECISION_MODE == PRECISION_FP16) {
        // TODO for later perhaps, would require us dynamically converting the
        // model weights from fp32 to fp16 online, here in this function, or writing
        // the fp16 weights directly from Python, which we only do for fp32/bf16 atm.
        fprintf(stderr, "build_from_checkpoint() does not support fp16 right now.\n");
        exit(EXIT_FAILURE);
    }

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file\n"); exit(EXIT_FAILURE); }
    int version = model_header[1];
    if (!(version == 3 || version == 5)) {
        // 3 = fp32, padded vocab
        // 5 = bf16, padded vocab, layernorms also in bf16
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }

    // check if the precision mode of the checkpoing matches the model precision
    if (weight_init) {
        if (PRECISION_MODE == PRECISION_BF16 && version != 5) {
            fprintf(stderr, "Precision is configured as BF16 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: are you sure you're loading a _bf16.bin file?\n");
            exit(EXIT_FAILURE);
        }
        if (PRECISION_MODE == PRECISION_FP32 && version != 3) {
            fprintf(stderr, "Precision is configured as FP32 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: to turn on FP32 you have to compile like: `make train_gpt2cu PRECISION=FP32`\n");
            fprintf(stderr, "---> HINT: are you sure you're loading a .bin file without any _bf16 in the name?\n");
            exit(EXIT_FAILURE);
        }
    }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];
    model->config.padded_vocab_size = model_header[7];

    // allocate memory for the model parameters
    gpt2_allocate_weights(model);

    // read in the parameters if weight_init is true
    if (weight_init) {
        assert(model->params_memory != NULL);
        file_to_device(model->params_memory, model_file, model->num_parameters_bytes, IO_BUF_SIZE, main_stream);
    }
    fcloseCheck(model_file);

    // only return from this function once we are certain the params are ready on the GPU
    cudaCheck(hipDeviceSynchronize());
}

void gpt2_set_hyperparameters(GPT2Config* config, const char* depth_str) {
    int depth = atoi(depth_str);
    assert(depth > 0); // atoi returns 0 if not a number
    int channels, num_heads;
    if      (depth == 6)  { channels = 384; num_heads = 6; }   // (unofficial) gpt2-tiny (30M)
    else if (depth == 12) { channels = 768; num_heads = 12; }  // gpt2 (124M)
    else if (depth == 24) { channels = 1024; num_heads = 16; } // gpt2-medium (350M)
    else if (depth == 36) { channels = 1280; num_heads = 20; } // gpt2-large (774M)
    else if (depth == 48) { channels = 1600; num_heads = 25; } // gpt2-xl (1558M)
    else if (depth == 60) { channels = 1920; num_heads = 30; } // (unofficial) 2.7B
    else if (depth == 72) { channels = 2880; num_heads = 30; } // (unofficial) 7.3B
    else if (depth == 84) { channels = 3456; num_heads = 36; } // (unofficial) 12.2B
    else { fprintf(stderr, "Unsupported GPT-2 depth: %d\n", depth); exit(EXIT_FAILURE); }
    config->num_layers = depth;
    config->channels = channels;
    config->num_heads = num_heads;
    config->max_seq_len = 1024;
}

void gpt3_set_hyperparameters(GPT2Config* config, const char* channels_str) {
    // we use channels instead of depth for GPT-3 because GPT-3 model depths are not one-to-one
    // note that our models are not necessarily identical to GPT-3 because
    // we use dense attention, not the alternating dense/banded attention of GPT-3
    int channels = atoi(channels_str);
    assert(channels > 0); // atoi returns 0 if not a number
    int depth, head_size;
    if      (channels == 384)   { depth = 6;  head_size = 64; }  // (unofficial) gpt3-tiny (31M)
    else if (channels == 768)   { depth = 12; head_size = 64; }  // gpt3-small (125M)
    else if (channels == 1024)  { depth = 24; head_size = 64; }  // gpt3-medium (350M)
    else if (channels == 1536)  { depth = 24; head_size = 96; }  // gpt3-large (760M)
    else if (channels == 2048)  { depth = 24; head_size = 128; } // gpt3-xl (1.3B) [heads fixed]
    else if (channels == 2560)  { depth = 32; head_size = 80; }  // gpt3-2.7B
    else if (channels == 4096)  { depth = 32; head_size = 128; } // gpt3-6.7B
    else if (channels == 5140)  { depth = 40; head_size = 128; } // gpt3-13B
    else if (channels == 12288) { depth = 96; head_size = 128; } // gpt3 (175B)
    else { fprintf(stderr, "Unsupported GPT-3 channels: %d\n", channels); exit(EXIT_FAILURE); }
    assert(channels % head_size == 0);
    config->num_layers = depth;
    config->channels = channels;
    config->num_heads = channels / head_size;
    config->max_seq_len = 2048; // NOTE: GPT-3 uses context length of 2048 tokens, up from 1024 in GPT-2
}

void gpt_build_from_descriptor(GPT2 *model, const char* descriptor) {
    // The model descriptor can be:
    // - legacy format "dX", where X is number, e.g. "d12". This creates GPT-2 model with 12 layers.
    // - new explicit format "gpt2:dX", same as above, e.g. "gpt2:d48" for GPT-2 with 48 layers.
    // - "gpt3:cX", where X is now the channel count, e.g. "gpt3:c768" is the smallest GPT-3 model.

    // check the valid prexies and dispatch to the right setup function
    assert(descriptor != NULL);
    size_t len = strlen(descriptor);
    if (len > 1 && descriptor[0] == 'd') {
        gpt2_set_hyperparameters(&model->config, descriptor + 1); // pass along the depth str without the 'd'
    } else if (len > 6 && strncmp(descriptor, "gpt2:d", 6) == 0) {
        gpt2_set_hyperparameters(&model->config, descriptor + 6); // pass along the depth str without the 'gpt2:d'
    } else if (len > 6 && strncmp(descriptor, "gpt3:c", 6) == 0) {
        gpt3_set_hyperparameters(&model->config, descriptor + 6); // pass along the channels str without the 'gpt3:c'
    } else {
        fprintf(stderr, "Unsupported model descriptor: %s\n", descriptor); exit(EXIT_FAILURE);
    }

    // both GPT-2 and GPT-3 use the same tokenizer with 50257 tokens
    model->config.vocab_size = 50257;
    model->config.padded_vocab_size = 50304; // padded to 128 for CUDA kernel efficiency

    gpt2_allocate_weights(model);

    // allocate and random init the memory for all the parameters with GPT-2 schema
    // weights ~N(0, 0.02), biases 0, c_proj weights ~N(0, 0.02/(2*L)**0.5)
    // NOTE: assuming all parameters are of the type floatX, could be relaxed later
    mt19937_state init_rng;
    manual_seed(&init_rng, 42);
    floatX* params_memory_cpu = (floatX*)mallocCheck(model->num_parameters_bytes);
    memset(params_memory_cpu, 0, model->num_parameters_bytes);
    // fill in all the weights with random values
    float residual_scale = 1.0f / sqrtf(2.0f * model->config.num_layers);
    // we have to init all these tensors exactly in the order that PyTorch initializes them
    // so that we can match them up and get correctness and exactly the same initial conditions
    size_t L = model->config.num_layers;
    size_t offset = 0;
    for (int l = 0; l < L; l++) {
        offset = 0;
        for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
            // the layernorm parameters are all initialized to 1
            if (l == 0 && (i == 2 || i == 8 || i == 14)) { // only at l = 0 to init these just once
                for (size_t j = 0; j < model->param_elements[i]; j++) {
                    params_memory_cpu[offset + j] = 1.0f;
                }
            }
            // weights tensors are handled here
            if ((l == 0 && (i == 0 || i == 1)) // only at l = 0, init the wte and wpe tensors
              || i == 4 || i == 6 || i == 10 || i == 12) {
                size_t n = model->param_elements[i];
                size_t layer_offset = 0;
                if (i == 0) {
                    // for wte tensor (padded vocab) override to init V instead of Vp rows
                    n = model->config.vocab_size * model->config.channels;
                }
                if (i == 4 || i == 6 || i == 10 || i == 12) {
                    // weight tensors, we are only initializing layer l
                    assert(n % L == 0);
                    n = n / L;
                    layer_offset = l * n;
                }
                // in GPT-2, the projections back into the residual stream are additionally
                // scaled by 1/sqrt(2*L) for training stability
                float scale = (i == 6 || i == 12) ? 0.02f * residual_scale : 0.02f;
                // okay let's draw the random numbers and write them
                float *fp32_buffer = (float*)mallocCheck(n * sizeof(float));
                normal_(fp32_buffer, n, 0.0f, scale, &init_rng);
                for (size_t j = 0; j < n; j++) {
                    params_memory_cpu[offset + layer_offset + j] = (floatX)fp32_buffer[j];
                }
                free(fp32_buffer);
            }
            offset += model->param_elements[i];
        }
    }

    // copy them to GPU
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, model->num_parameters_bytes, hipMemcpyHostToDevice));
    free(params_memory_cpu);
}

// propagate inputs through the network to produce logits.
// right now, this function is fully synchronous with the host
void gpt2_forward(GPT2 *model, const int* inputs, size_t B, size_t T) {
    NVTX_RANGE_FN();
    // we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;

    // validate B,T are not larger than the values used at initialisation
    // (smaller B,T are okay for inference only)
    if (B > model->batch_size || T > model->seq_len) {
        printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
        exit(EXIT_FAILURE);
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    // validate inputs, all indices must be in the range [0, V)
    // we can do this while the copies are already underway
    tokenCheck(inputs, B*T, V);

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C, main_stream); // encoding goes into residual[0]

    // first layernorm isn't fused
    layernorm_forward((model->recompute < 2) ? acts.ln1 : acts.lnf, acts.ln1_mean, acts.ln1_rstd, acts.encoded, params.ln1w, params.ln1b, B, T, C, main_stream);

    for (int l = 0; l < L; l++) {
        NvtxRange layer_range("Layer", l);

        floatX* residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_qkvb = params.qkvb + l * 3*C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = params.attprojb + l * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcb = params.fcb + l * 4*C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        floatX* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        // reuse the same activation buffer at each layer, as we'll re-compute the gelu during backward
        // very useful because we dramatically reduce VRAM usage, and may be able to fit larger batch size
        floatX* l_fch_gelu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;
        floatX* scratch = (floatX*)acts.output; // used for non-cudnn attention, fcproj, attproj, etc.

        // now do the forward pass
        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        matmul_forward_cublaslt(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward_cudnn(l_atty, (float*)l_att, l_qkvr, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        if (T != model->seq_len) { // unused parts of attention buffer must be zeroed (T-dependent)
            cudaCheck(hipMemset(l_att, 0, B * NH * T * T * sizeof(floatX)));
        }
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH, main_stream);
        #endif

        matmul_forward_cublaslt(scratch, l_atty, l_attprojw, l_attprojb, B, T, C, C, main_stream);
        fused_residual_forward5(l_residual2, l_ln2, l_ln2_mean, l_ln2_rstd, residual, scratch, l_ln2w, l_ln2b, B*T, C, main_stream);
        matmul_forward_cublaslt(l_fch_gelu, l_ln2, l_fcw, l_fcb, B, T, C, 4*C, main_stream, l_fch, model->gelu_fusion);
        matmul_forward_cublaslt(scratch, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C, main_stream);
        // OK, fusion across blocks.
        if(l+1 != L) {
            floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + (l + 1) * B * T * C : acts.lnf;
            float* l_ln1_mean = acts.ln1_mean + (l + 1) * B * T;
            float* l_ln1_rstd = acts.ln1_rstd + (l + 1) * B * T;
            const floatX* l_ln1w = params.ln1w + (l + 1) * C;
            const floatX* l_ln1b = params.ln1b + (l + 1) * C;
            fused_residual_forward5(l_residual3, l_ln1, l_ln1_mean, l_ln1_rstd, l_residual2, scratch, l_ln1w, l_ln1b,
                                    B * T, C, main_stream);
        } else {
            fused_residual_forward5(l_residual3, acts.lnf, acts.lnf_mean, acts.lnf_rstd, l_residual2, scratch,
                                    params.lnfw, params.lnfb,
                                    B * T, C, main_stream);
        }
    }

    matmul_forward_cublaslt(acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp, main_stream);
    cudaCheck(hipDeviceSynchronize());
}


// Forwards both the model and the loss and is used for validation splits and evals.
// In particular it populates cpu_losses with loss at each token.
// Some of the evals (e.g. HellaSwag) require the per-token losses, which are produced here.
float gpt2_validate(GPT2 *model, const int* inputs, const int* targets, size_t B, size_t T) {
    assert(targets != NULL);
    // forward the model itself
    gpt2_forward(model, inputs, B, T);
    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;

    NvtxRange classifier_and_loss_range("classifier_and_loss");
    ActivationTensors acts = model->acts;
    float mean_loss = 0.0f;
    // fused classifier: does the forward pass and first part of the backward pass
    const float dloss = 1.0f / (B * T); // results in the uniform average loss over all elements
    // note: we don't need to generate dlogits here
    cudaCheck(hipMemset(acts.losses, 0, B*T*sizeof(float)));
    cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    tokenCheck(targets, B*T, V); // while the memcpy is underway, validate the targets
    fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp, False, main_stream);
    cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B*T; i++) {
        mean_loss += model->cpu_losses[i];
    }
    mean_loss /= B*T;
    cudaCheck(hipDeviceSynchronize());
    return mean_loss;
}

void gpt2_backward_and_reduce(GPT2 *model, int* inputs, const int* targets, int grad_accum_steps, int micro_step) {
    if(model->grads_memory == nullptr) {
        fprintf(stderr, "Need to allocate gradients before backward");
        exit(EXIT_FAILURE);
    }
    NVTX_RANGE_FN();
    bool last_step = micro_step == grad_accum_steps - 1;
    // on the first micro-step zero the gradients, as we're about to += accumulate into them
    if (micro_step == 0) {
        // there are currently two state vars during the gradient accumulation inner loop:
        // 1) the losses accumulate += into acts.losses, reset here
        // 2) the gradients accumulate += into grads_memory, reset here
        cudaCheck(hipMemsetAsync(model->acts.losses, 0, model->batch_size * model->seq_len * sizeof(float), main_stream));
        cudaCheck(hipMemsetAsync(model->grads_memory, 0, model->num_parameters * sizeof(floatX), main_stream));
    }

    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    const size_t B = model->batch_size;
    const size_t T = model->seq_len;
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;

    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;

    // accumulate the losses inside acts.losses, and kick off the backward pass inside the fused classifier
    NvtxRange classifier_and_loss_range("classifier_and_loss");
    const float dloss = 1.0f / (float)(B * T * grad_accum_steps); // results in the uniform average loss over all elements
    cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    tokenCheck(targets, B*T, V);
    fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp, True, main_stream);

    // backward pass: go in the reverse order of the forward pass, and call backward() functions

    // reset residual stream gradients (put here to work with gradient accumulation)
    floatX* dresidual = (floatX*)model->acts.scratch_btc; // the main buffer holding the gradient in the backward pass
    cudaCheck(hipMemset(dresidual, 0, B * T * C * sizeof(floatX)));

    // re-use the output buffer of the forward pass as a scratchpad during backward pass
    float*  scratchF = (float*)acts.output;
    floatX* scratchX = (floatX*)acts.output;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(model->acts.scratch_bt4c, grads.wte, NULL, acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp, main_stream);
    // backward the final layernorm
    floatX* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    layernorm_backward(dresidual, grads.lnfw, grads.lnfb, scratchF, model->acts.scratch_bt4c, residual, params.lnfw, acts.lnf_mean, acts.lnf_rstd, B, T, C, main_stream);

    // from this point on, we no longer need the values stored in the last residual, so we can reuse that memory as generic
    // scratch for backward computations
    floatX* dl_btc = residual;

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        NvtxRange layer_range("Layer", l);

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_ln1w = params.ln1w + l * C;
        floatX* l_ln1b = params.ln1b + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        // get the pointers of the gradients of the weights for this layer
        floatX* dl_ln1w = grads.ln1w + l * C;
        floatX* dl_ln1b = grads.ln1b + l * C;
        floatX* dl_qkvw = grads.qkvw + l * 3*C * C;
        floatX* dl_qkvb = grads.qkvb + l * 3*C;
        floatX* dl_attprojw = grads.attprojw + l * C * C;
        floatX* dl_attprojb = grads.attprojb + l * C;
        floatX* dl_ln2w = grads.ln2w + l * C;
        floatX* dl_ln2b = grads.ln2b + l * C;
        floatX* dl_fcw = grads.fcw + l * 4*C * C;
        floatX* dl_fcb = grads.fcb + l * 4*C;
        floatX* dl_fcprojw = grads.fcprojw + l * C * 4*C;
        floatX* dl_fcprojb = grads.fcprojb + l * C;
        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        float* l_ln1_mean = acts.ln1_mean + l * B * T;
        float* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch_pre_gelu = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        floatX* dl_bt4c = (floatX*)model->acts.scratch_bt4c;

        // start the backward pass for this layer
        if(model->recompute >= 1) {
            // recompute >= 1 means we recompute gelu. in this case,
            // l_fch_gelu is just a buffer, so re-compute the gelu from l_fch here
            gelu_forward(l_fch_gelu, l_fch_pre_gelu, B*T*4*C, main_stream);
        }
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_gelu, l_fcprojw, scratchF, B, T, 4*C, C, main_stream, l_fch_pre_gelu, model->gelu_fusion);
        if(model->recompute >= 2) {
            // same as gelu above, l_ln1 and l_ln2 are just buffers if recompute >= 2, recompute them here on demand
            layernorm_forward(l_ln2, l_ln2_mean, l_ln2_rstd, l_residual2, l_ln2w, l_ln2b, B, T, C, main_stream);
        }
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c, l_ln2, l_fcw, scratchF, B, T, C, 4 * C, main_stream);
        // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, dl_ln2w, dl_ln2b, scratchF, dl_btc, l_residual2, l_ln2w, l_ln2_mean, l_ln2_rstd, B, T, C, main_stream);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, scratchF, B, T, C, C, main_stream);

        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        attention_backward_cudnn(dl_bt4c, dl_btc, l_qkvr, l_atty, (float*)l_att, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // we need B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        floatX* buffer_a = l_atty;
        floatX* buffer_b = l_fch_pre_gelu;        // this is B x T x 4C, so even larger than what we need
        attention_backward(dl_bt4c, buffer_b, scratchX, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH, main_stream);
        #endif
        if(model->recompute >= 2) {
            layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C, main_stream);
        }
        // QKV parameter gradients
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c, l_ln1, l_qkvw, scratchF, B, T, C, 3 * C, main_stream);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, dl_ln1w, dl_ln1b, scratchF, dl_btc, residual, l_ln1w, l_ln1_mean, l_ln1_rstd, B, T, C, main_stream);

        // Accumulate gradients from this layer in a background stream.
        if(last_step) {
            floatX* const pointers[] = {
                dl_ln1w, dl_ln1b,
                dl_qkvw, dl_qkvb,
                dl_attprojw, dl_attprojb,
                dl_ln2w, dl_ln2b,
                dl_fcw, dl_fcb,
                dl_fcprojw, dl_fcprojb
            };
            const size_t nelem[] = {
                C, C,
                3 * C * C, 3 * C,
                C * C, C,
                C, C,
                4 * C * C, 4 * C,
                C * 4 * C, C
            };
            multi_gpu_async_reduce_gradient(pointers, nelem, &multi_gpu_config, main_stream);
        }
    }
    encoder_backward(grads.wte, grads.wpe, scratchX, model->workload_indices, model->bucket_info,
                     dresidual, model->inputs, inputs, B, T, C, random_u32(&model->rng_state), main_stream);

    // Aggregate all gradients that are not part of the transformer blocks
    if(last_step) {
        // reduce all the losses within the current GPU (across all microsteps)
        global_sum_deterministic(model->accumulated_mean_loss, acts.losses, B*T, main_stream);
        // reduce loss across GPUs to a single, final float across all microsteps and GPUs
        #if MULTI_GPU
        ncclCheck(ncclAllReduce(model->accumulated_mean_loss, model->accumulated_mean_loss, sizeof(float), ncclFloat, ncclAvg, multi_gpu_config.nccl_comm, main_stream));
        #endif
        cudaCheck(hipMemcpyAsync(&model->mean_loss, model->accumulated_mean_loss, sizeof(float), hipMemcpyDeviceToHost, main_stream));
        // reduce the gradients for non-transformer block parameters
        floatX* const pointers[] = {grads.wte, grads.wpe, grads.lnfw, grads.lnfb};
        const size_t nelem[] = {Vp * C, T * C, C, C};
        multi_gpu_async_reduce_gradient(pointers, nelem, &multi_gpu_config, main_stream);
    }

    cudaCheck(hipDeviceSynchronize());
    if(last_step) {
        model->mean_loss /= B*T*grad_accum_steps;
    } else {
        model->mean_loss = -1.f; // no loss available yet
    }
}

// Gets the offset of a specific tensor for a specific layer in the GPT2 model
// layer_id is ignored for weights that are not part of a transformer block
ShardInfo gpt2_get_tensor_at_layer(const GPT2 *model, int layer_id, int param_tensor_id) {
    // first offset our way to the parameter tensor start
    ptrdiff_t offset = 0;
    for (int i = 0; i < param_tensor_id; i++) {
        offset += (ptrdiff_t)model->param_elements[i];
    }
    size_t size = model->param_elements[param_tensor_id] ;
    // if we are in the transformer block, we need to additionally offset by the layer id
    if(2 <= param_tensor_id && param_tensor_id <= 13) {
        size /= model->config.num_layers;
        offset += (ptrdiff_t)(layer_id * size);
    }
    return {offset, size};
}

float gpt2_calculate_grad_norm(GPT2 *model, MultiGpuConfig* multi_gpu_config) {
    NVTX_RANGE_FN();
    floatX* grads_memory = (floatX*)model->grads_memory;

    // repurposing this buffer (which isn't needed now) to write grad norm into it
    float* grad_norm_squared = (float*)model->acts.output;
    float grad_norm_squared_cpu = 0.0f;

    int num_slices[2] = {1, model->config.num_layers};
    int max_num_block_sums = get_max_num_block_sums(num_slices, 2);
    if (multi_gpu_config->zero_stage == 1) {
        // because of the ncclReduceScatter() in backward,
        // grads_memory only contains the averaged gradients at the local shards,
        // so we only calculate the grad norm at the grads_memory belonging to the local shards
        for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
            ShardInfo tensor = gpt2_get_tensor_at_layer(model, 0, i);
            ShardInfo shard = multi_gpu_get_shard_offset(tensor.size, multi_gpu_config, 1);
            ptrdiff_t offset = tensor.offset + shard.offset;
            bool is_first_pass = (i == 0);
            if((i < 2 || i > 13)) {
                global_norm_squared(grad_norm_squared, grads_memory + offset, shard.size, 0, 1,
                                    max_num_block_sums, is_first_pass, main_stream);
            } else {
                global_norm_squared(grad_norm_squared, grads_memory + offset, shard.size, tensor.size, model->config.num_layers,
                                    max_num_block_sums, is_first_pass, main_stream);
            }
        }
        global_sum_deterministic(grad_norm_squared, grad_norm_squared, max_num_block_sums, main_stream);
#if MULTI_GPU
        // further sum the (partial) squared norm across all GPUs
        ncclCheck(ncclAllReduce(grad_norm_squared, grad_norm_squared, sizeof(float), ncclFloat, ncclSum, multi_gpu_config->nccl_comm, main_stream));
#endif
    } else {
        // in regular DDP, backward has averaged the gradients across all GPUs
        // so each GPU can compute the squared norm over the whole grad vector, with no added comms needed
        global_norm_squared(grad_norm_squared, grads_memory, model->num_parameters, 0, 1, max_num_block_sums, true, main_stream);
        global_sum_deterministic(grad_norm_squared, grad_norm_squared, max_num_block_sums, main_stream);
    }
    cudaCheck(hipMemcpy(&grad_norm_squared_cpu, grad_norm_squared, sizeof(float), hipMemcpyDeviceToHost));
    float grad_norm_cpu = sqrtf(grad_norm_squared_cpu);
    return grad_norm_cpu;
}

void gpt2_update(GPT2 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, float grad_scale, int t,
                 MultiGpuConfig* multi_gpu_config, bool init_from_master_only=false) {
    // update the model parameters using the AdamW optimizer
    // keep in mind that optimizer sharding (ZeRO-1) assigns different parameters to different GPUs
    // so we may not be responsible for the entire parameter tensor
    // also, this function was very simple a while back but become very complex, only because we want to
    // selectively weight decay some, but not all tensors :(
    // TODO: revisit and probably refactor this entire function
    NVTX_RANGE_FN();
    if(model->grads_memory == nullptr || model->m_memory == nullptr || model->v_memory == nullptr) {
        fprintf(stderr, "Need to allocate optimizer state before update");
        exit(EXIT_FAILURE);
    }

    bool init_state = model->init_state;
    if(init_state) {
        model->init_state = false;
        NvtxRange rng("InitOpt");
        cudaCheck(hipMemset(model->m_memory, 0, multi_gpu_config->shard_num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->v_memory, 0, multi_gpu_config->shard_num_parameters * sizeof(float)));
    }

    // save RNG state at this point so we can round from master weights identically when restoring from a checkpoint
    model->rng_state_last_update = model->rng_state;

    // AdamW update
    // handle adamw for all the transformer blocks
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        // generate a unique seed for each tensor
        unsigned int seed = random_u32(&model->rng_state);

        int num_layers = model->config.num_layers;
        if((i < 2 || i > 13)) {
            num_layers = 1;
        }

        ShardInfo tensor = gpt2_get_tensor_at_layer(model, 0, i);
        ShardInfo shard = multi_gpu_get_shard_offset(tensor.size, multi_gpu_config, 1);
        ptrdiff_t local_offset_full = tensor.offset + shard.offset;
        ptrdiff_t local_offset_partial = tensor.offset / multi_gpu_config->num_processes;

        // we only want to weight decay the 2D tensors and leave all 1D tensors alone
        // in particular this also decays the embedding weights, but this is ok:
        // - the token embeddings are weight shared and participate in the final projection to logits
        // - the position embeddings actively participate at every forward/backward pass
        float wd = (i == 0 || i == 1 || i == 4 || i == 6 || i == 10 || i == 12) ? weight_decay : 0.0f;
        floatX* param_ptr = (floatX*)model->params_memory + local_offset_full;
        floatX* grad_ptr = (floatX*)model->grads_memory + local_offset_full;

        ptrdiff_t opt_state_offset = multi_gpu_config->zero_stage < 1 ?  local_offset_full : local_offset_partial;
        float* m_ptr = model->m_memory + opt_state_offset;
        float* v_ptr = model->v_memory + opt_state_offset;
        float* master_ptr = nullptr;
        if (model->master_weights != nullptr) { master_ptr = model->master_weights + opt_state_offset; }
        if(init_state && model->master_weights != nullptr ) {
            size_t grid_size = CEIL_DIV(shard.size, 512);
            copy_and_cast_kernel<<<dim3(grid_size, num_layers), 512, 0, main_stream>>>(master_ptr, param_ptr, shard.size,
                                                                     shard.size, tensor.size);
            cudaCheck(hipGetLastError());
        }

        if (init_from_master_only) {
            // when resuming training from a checkpoint with master weights (allows changing precision)
            init_from_master(param_ptr, master_ptr, shard.size, tensor.size, shard.size, num_layers, seed, main_stream);
        } else {
            // ok finally call the kernel to update the weights with AdamW
            adamw_update(param_ptr, master_ptr, grad_ptr,
                        m_ptr, v_ptr,
                        shard.size, tensor.size, tensor.size, shard.size, num_layers,
                        learning_rate,
                        beta1, beta2, t, eps, wd, grad_scale, seed, main_stream);
        }

        if (multi_gpu_config->zero_stage == 1) {
#if MULTI_GPU
            ncclCheck(ncclGroupStart());
            for(int l = 0; l < num_layers; ++l) {
                // gather updated shards of model->params_memory from each process
                ncclCheck(ncclAllGather(param_ptr + l * tensor.size,
                                        (floatX*) model->params_memory + tensor.offset + l * tensor.size,
                                        shard.size, ncclFloatX,
                                        multi_gpu_config->nccl_comm, multi_gpu_config->nccl_stream));
            }
            ncclCheck(ncclGroupEnd());
#endif
        }
    }

    cudaCheck(hipDeviceSynchronize());
}

float gpt2_estimate_mfu(GPT2 *model, int num_tokens, float dt) {
    /*
    Estimate model flops utilization (MFU)
    ref: Section 2.1 of https://arxiv.org/pdf/2001.08361
    Note: Ideally, the N here would be only the parameters that actually
    participate in matrix multiplications. In this N, we are over-estimating by
    including LayerNorm params, biases, and the position embedding weights,
    but these are very small terms. Also keep in mind that we would want to exclude
    the token embedding weights, but in GPT-2 these are weight shared, so they
    participate in the classifier matmul, so they are correct to be included in N.
    Note 2: The first term (6 * N) in flops_per_token is all weight matmuls, the
    second is the attention matmul, which is also usually a small contribution.
    */
    size_t N = model->num_parameters;
    int L = model->config.num_layers;
    int C = model->config.channels;
    int T = model->seq_len;
    size_t flops_per_token = 6 * N + (size_t)6 * L * C * T;
    size_t flops_per_step = flops_per_token * num_tokens;
    // express our flops throughput as ratio of A100 bfloat16 peak flops
    float flops_achieved = (float)flops_per_step * (1.0f / dt); // per second
    float flops_promised = get_flops_promised(deviceProp.name, PRECISION_MODE) * 1e12f;
    if(flops_promised < 0) {
        return -1.f;   // don't know
    }
    float mfu = flops_achieved / flops_promised;
    return mfu;
}

void gpt2_free(GPT2 *model) {
    cudaFreeCheck(&model->params_memory);
    cudaFreeCheck(&model->grads_memory);
    cudaFreeCheck(&model->m_memory);
    cudaFreeCheck(&model->v_memory);
    cudaFreeCheck(&model->master_weights);
    cudaFreeCheck(&model->acts_memory);
    cudaFreeCheck(&model->inputs);
    cudaFreeCheck(&model->targets);
    cudaFreeCheck(&model->accumulated_mean_loss);
    cudaCheck(hipHostFree(model->cpu_losses));
    free(model->workload_indices);
    free(model->bucket_info);
}

// ----------------------------------------------------------------------------
// common init & free code for all of train/test/profile

void common_start(bool override_enable_tf32 = true, bool print_device_info = true) {

    // get CUDA device infos
    cudaCheck(hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx));
    if (print_device_info) {
        printf("[System]\n");
        printf("Device %d: %s\n", multi_gpu_config.local_device_idx, deviceProp.name);
    }

    // set up the cuda streams. atm everything is on the single main stream
    cudaCheck(hipStreamCreate(&main_stream));
    nvtxNameCudaStreamA(main_stream, "main stream");

    // set up cuBLAS and cuBLASLt
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    bool enable_tf32 = PRECISION_MODE == PRECISION_FP32 && deviceProp.major >= 8 && override_enable_tf32;
    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;

    #ifdef ENABLE_CUDNN
    create_cudnn();
    #endif
}

void common_free(GPT2 &model) {
    cudaCheck(hipStreamDestroy(main_stream));
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    #ifdef ENABLE_CUDNN
    destroy_cudnn();
    #endif
}


void save_state(const char* filename, int step, GPT2* model, DataLoader* loader) {
    printf("Writing state to %s\n", filename);
    FILE *state_file = fopenCheck(filename, "wb");
    int state_header[256];
    memset(state_header, 0, sizeof(state_header));
    // basic identifying information
    state_header[0] = 20240527; // magic number
    state_header[1] = 1; // version number
    state_header[2] = multi_gpu_config.num_processes; // number of processes
    state_header[3] = multi_gpu_config.process_rank; // rank of this process
    state_header[4] = model->use_master_weights;  // whether we're using fp32 master weights
    state_header[5] = loader->should_shuffle; // shuffle state of the dataloader
    // int main state, start at 10 to leave some padding
    state_header[10] = step; // step of the optimization
    // model rng state, start at 20 to leave some padding
    *((unsigned long long*)&state_header[20]) = model->rng_state; // random number generator state
    *((unsigned long long*)&state_header[22]) = model->rng_state_last_update; // last gpt2_update
    // dataloader state, start at 30 to leave some padding
    *((size_t*)&state_header[30]) = loader->current_shard_idx; // shard of the dataset
    *((size_t*)&state_header[32]) = loader->current_sample_idx; // position in shard
    fwriteCheck(state_header, sizeof(int), 256, state_file);

    // write AdamW m, v, and master_weights here (they are all float)
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    device_to_file(state_file, model->m_memory, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    device_to_file(state_file, model->v_memory, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    if(model->use_master_weights) {
        device_to_file(state_file, model->master_weights, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    }

    // write dataloader state if we are using the Permuted version of it
    if (loader->should_shuffle) {
        fwriteCheck(&loader->glob_result.gl_pathc, sizeof(size_t), 1, state_file);  // number of shards
        fwriteCheck(loader->shard_indices, sizeof(int), loader->glob_result.gl_pathc, state_file);
        fwriteCheck(&loader->shard_num_samples, sizeof(size_t), 1, state_file);
        fwriteCheck(loader->intra_shard_indices, sizeof(int), loader->shard_num_samples, state_file);
        fwriteCheck(&loader->shuffle_rng, sizeof(mt19937_state), 1, state_file);
    }
    fcloseCheck(state_file);
}

void load_state(int* step, GPT2* model, DataLoader* loader, const char* filename) {
    FILE *state_file = fopenCheck(filename, "rb");
    int state_header[256];
    freadCheck(state_header, sizeof(int), 256, state_file);
    assert(state_header[0] == 20240527); // magic number
    assert(state_header[1] == 1); // version number
    assert(state_header[2] == multi_gpu_config.num_processes); // number of processes
    assert(state_header[3] == multi_gpu_config.process_rank); // rank of this process
    int use_master_weights = state_header[4];  // whether we're using fp32 master weights
    int should_shuffle = state_header[5]; // shuffle state of the dataloader
    *step = state_header[10]; // step of the optimization
    model->rng_state = *((unsigned long long*)&state_header[20]); // random number generator state
    model->rng_state_last_update = *((unsigned long long*)&state_header[22]); // last gpt2_update
    size_t current_shard_idx = *((size_t*)&state_header[30]); // shard index
    size_t current_sample_idx = *((size_t*)&state_header[32]); // position in shard

    // read AdamW m, v, master_weights (they are all float)
    // allocate all the needed memory as necessary
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    if(use_master_weights == 1 && !model->use_master_weights) {
        printf0("Warning: Master weights are present in state, but not enabled for current run.");
    } else if (use_master_weights == 0 && model->use_master_weights) {
        printf0("Error: Master weights requested, but not present in state file.");
        exit(EXIT_FAILURE);
    }

    model->init_state = false;      // we just got the state from file, no need to do first-touch init
    assert(model->m_memory != nullptr);
    assert(model->v_memory != nullptr);
    file_to_device(model->m_memory, state_file, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    file_to_device(model->v_memory, state_file, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    if(model->use_master_weights) {
        assert(model->master_weights != nullptr);
        file_to_device(model->master_weights, state_file, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
        // restore weights from the master weights using the RNG state before last weight update
        model->rng_state = model->rng_state_last_update;
        gpt2_update(model, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0, &multi_gpu_config, /* init_from_master_only*/ true);
        model->rng_state = *((unsigned long long*)&state_header[20]); // use final RNG state from checkpoint after this
    }

    // revive the DataLoader object and its state
    loader->should_shuffle = should_shuffle;
    if (should_shuffle == 1) {
        // ensure the number of shards matches
        size_t glob_result_gl_pathc;
        freadCheck(&glob_result_gl_pathc, sizeof(size_t), 1, state_file);
        assert(glob_result_gl_pathc == loader->glob_result.gl_pathc);
        // read the shard indices
        loader->shard_indices = (int*)mallocCheck(loader->glob_result.gl_pathc * sizeof(int));
        freadCheck(loader->shard_indices, sizeof(int), loader->glob_result.gl_pathc, state_file);
        // ensure the number of samples matches
        size_t shard_num_samples;
        freadCheck(&shard_num_samples, sizeof(size_t), 1, state_file);
        assert(shard_num_samples == loader->shard_num_samples);
        // read the intra-shard indices
        loader->intra_shard_indices = (int*)mallocCheck(loader->shard_num_samples * sizeof(int));
        freadCheck(loader->intra_shard_indices, sizeof(int), loader->shard_num_samples, state_file);
        // read the shuffle rng state
        freadCheck(&loader->shuffle_rng, sizeof(mt19937_state), 1, state_file);
    }
    dataloader_resume(loader, current_shard_idx, current_sample_idx);

    // all done, close state file
    fcloseCheck(state_file);
}

void write_checkpoint(const char* output_log_dir, int step, GPT2* model, DataLoader* train_loader, MultiGpuConfig* multi_gpu_config) {
    // a checkpoint contains: model weights, optimizer/dataloader state, and a DONE file
    printf0("Writing checkpoint at step %d\n", step);
    int rank = multi_gpu_config->process_rank;
    // only rank 0 writes the model file because it is the same across all ranks
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, step);
        gpt2_write_to_checkpoint(model, filename_buffer);
    }
    // all ranks write their state file
    snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, step, rank);
    save_state(filename_buffer, step, model, train_loader);
    // DONE file is a signal that this checkpoint as a whole is complete
    multi_gpu_barrier(multi_gpu_config);
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/DONE_%08d", output_log_dir, step);
        FILE* done_file = fopenCheck(filename_buffer, "w");
        fcloseCheck(done_file);
    }
}

void delete_checkpoint(const char* output_log_dir, int step, MultiGpuConfig* multi_gpu_config) {
    // mirrors write_checkpoint function, cleans up checkpoint from disk
    printf0("Deleting checkpoint at step %d\n", step);
    int rank = multi_gpu_config->process_rank;
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, step);
        remove(filename_buffer);
    }
    snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, step, rank);
    remove(filename_buffer);
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/DONE_%08d", output_log_dir, step);
        remove(filename_buffer);
    }
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip everything below this point

// ----------------------------------------------------------------------------
// training resumption logic, very useful when jobs crash once in a while
// the goal is that we can resume optimization from any checkpoint, bit-perfect
// note that "state" refers to things not already saved in the model checkpoint file

// ----------------------------------------------------------------------------
// CLI, poor man's argparse
// (all single letters have been claimed now)

void error_usage() {
    fprintf(stderr, "Usage:   ./train_gpt2cu [options]\n");
    fprintf(stderr, "Options:\n");
    // file system input / output
    fprintf(stderr, "  -i <string> train data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_train.bin)\n");
    fprintf(stderr, "  -j <string> val data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_val.bin)\n");
    fprintf(stderr, "  -e <string> input .bin filename or descriptor, see code comments as docs. (default = gpt2_124M_bf16.bin)\n");
    fprintf(stderr, "  -o <string> output log dir (default = NULL, no logging)\n");
    fprintf(stderr, "  -lg <int>   log gpu info every x steps (default = -1; disabled)\n");
    fprintf(stderr, "  -n <int>    write optimization checkpoints every how many steps? (default 0, don't)\n");
    fprintf(stderr, "  -nk <int>   max number of checkpoints to keep in the directory, removing old ones (0 = disable, default)\n");
    fprintf(stderr, "  -nm <int>   every how many step checkpoints are considered major? major checkpoints never get deleted.\n");
    fprintf(stderr, "  -y <int>    resume optimization found inside output log dir? (0=restart/overwrite, 1=resume/append)\n");
    // token layout for each step of the optimization
    fprintf(stderr, "  -b <int>    (per-GPU, micro) batch size B (default = 4)\n");
    fprintf(stderr, "  -t <int>    sequence length T (default = 1024)\n");
    fprintf(stderr, "  -d <int>    total desired batch size (default = B * T * num_processes, i.e. no grad accumulation\n");
    // workload (number of steps)
    fprintf(stderr, "  -x <int>    max_steps of optimization to run (-1 (default) = disable, run 1 epoch)\n");
    // optimization
    fprintf(stderr, "  -k <string> learning rate scheduler (default = cosine)\n");
    fprintf(stderr, "  -l <float>  learning rate (default = 3e-4f)\n");
    fprintf(stderr, "  -u <int>    learning rate warmup iterations (default = 0, no warmup)\n");
    fprintf(stderr, "  -q <float>  learning rate decay: final fraction, at end of training (default = 1.0 (no decay))\n");
    fprintf(stderr, "  -c <float>  weight decay (default = 0.0f)\n");
    fprintf(stderr, "  -sl <float> outlier stability: skip update if loss goes above this in zscore (0.0f=off)\n");
    fprintf(stderr, "  -sg <float> outlier stability: skip update if grad_norm goes above this in zscore (0.0f=off)\n");
    // evaluation
    fprintf(stderr, "  -v <int>    val_loss_every, how often we evaluate val loss (default = 20)\n");
    fprintf(stderr, "  -m <int>    val_max_steps, up to how many val batches to estimate val loss? (default = 20)\n");
    fprintf(stderr, "  -s <int>    sample_every, how often we inference the model (default = 20)\n");
    fprintf(stderr, "  -g <int>    genT, how many steps of inference we do (default = 64)\n");
    fprintf(stderr, "  -h <int>    hellaswag eval run? (default = 0)\n");
    // debugging
    fprintf(stderr, "  -a <int>    overfit a single batch? 0/1. useful for debugging\n");
    // numerics
    fprintf(stderr, "  -f <int>    enable_tf32 override (default: 1, set to 0 to disable tf32)\n");
    fprintf(stderr, "  -w <int>    keep f32 copy of weights for the optimizer? (default: 1)\n");
    fprintf(stderr, "  -ge <int>   gelu fusion: 0=none, 1=forward, 2=forward+backward (default: 2 for >=SM90, 0 for older GPUs)\n");
    // memory management
    fprintf(stderr, "  -z <int>    zero_stage, Zero Optimization Stage, 0,1,2,3 (default = 0)\n");
    fprintf(stderr, "  -r <int>    recompute: less memory but less speed. (default = 1), 0|1|2 = none,gelu,gelu+ln\n");
    // multi-node settings
    fprintf(stderr, "  -pn <int>    num_processes (default = 1)\n");
    fprintf(stderr, "  -pr <int>    process_rank (default = 0)\n");
    fprintf(stderr, "  -pg <int>    gpus_per_node (default = 8)\n");
    fprintf(stderr, "  -pm <string> nccl_init_method: tcp,fs,mpi (default = mpi)\n");
    fprintf(stderr, "  -ps <string> server_ip - used only when nccl_init_method is tcp (default = -1)\n");
    fprintf(stderr, "  -pp <string> fs_path - used only when nccl_init_method is fs (default = /tmp)\n");
    exit(EXIT_FAILURE);
}

// ----------------------------------------------------------------------------
// main training loop
int main(int argc, char *argv[]) {
    // read in the (optional) command line arguments
    const char* train_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
    const char* val_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
    const char* load_filename = "gpt2_124M_bf16.bin"; // bf16 weights of the model
    const char* lr_scheduler_type = "cosine";
    const char* output_log_dir = NULL;
    int checkpoint_every = 0; // write checkpoints every how many steps?
    int checkpoints_keep = 0; // how long checkpoint history do we keep? (in units of checkpoints)
    int major_checkpoint_every = 0; // major checkpoints never get deleted when maintaining history
    int resume = 0; // resume the optimization, if one is found inside output_log_dir?
    int B = 4; // batch size
    int T = 1024; // sequence length max
    int total_batch_size = -1; // will be calculated down below later, if not provided
    float learning_rate = 3e-4f;
    int log_gpu_every = -1;
    int warmup_iterations = 0;
    float final_learning_rate_frac = 1.0f; // final fraction of learning rate, at end of training
    float weight_decay = 0.0f;
    float skip_update_lossz = 0.0f; // skip update if loss goes above this in zscore
    float skip_update_gradz = 0.0f; // skip update if grad_norm goes above this in zscore
    int val_loss_every = 20; // every how many steps do we eval validation loss?
    int val_max_steps = 20; // how many batches max do we eval for validation loss?
    int sample_every = 20; // every how many steps to do inference?
    int genT = 64; // number of steps of inference we will do
    int overfit_single_batch = 0; // useful for debugging, 1 = only load a single data batch once
    int max_steps = -1;
    int override_enable_tf32 = 1;
    int use_master_weights = 1;
    int gelu_fusion = -1; // 0 = none, 1 = forward, 2 = forward+backward (-1 => per-GPU default)
    int recompute = 1; // recompute during backward setting, 0 = none, 1 = recompute gelu
    int zero_stage = 0; // Zero Optimization Stage for Multi-GPU training
    int hellaswag_eval = 0;
    // multi-node settings
    int num_processes = 1;  // this should be set by the slurm environment
    int process_rank = 0;  // this should be set by the slurm environment
    int gpus_per_node = 8;  // this should be set by the slurm environment
    char nccl_init_method[256] = "mpi";  // "tcp" or "fs" or "mpi"
    char server_ip[256] = "";  // used if init_method set to "tcp" -> set to your server ip address
    char fs_path[256] = "";  // used if init_method set to "fs" -> set to a shared filesystem path
    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (!(strlen(argv[i]) == 2 || strlen(argv[i]) == 3)) { error_usage(); } // must be -x[y] (one dash, one or two letters)
        // read in the args
        if (argv[i][1] == 'i') { train_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'j') { val_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'e') { load_filename = argv[i+1]; }
        else if (argv[i][1] == 'o') { output_log_dir = argv[i+1]; }
        else if (argv[i][1] == 'n' && argv[i][2] == '\0') { checkpoint_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'y') { resume = atoi(argv[i+1]); }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); } // Per-GPU (micro) batch size
        else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'd') { total_batch_size = atoi(argv[i+1]); }
        else if (argv[i][1] == 'l' && argv[i][2] == '\0') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'l' && argv[i][2] == 'g') { log_gpu_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'u') { warmup_iterations = atoi(argv[i+1]); }
        else if (argv[i][1] == 'q') { final_learning_rate_frac = atof(argv[i+1]); }
        else if (argv[i][1] == 'c') { weight_decay = atof(argv[i+1]); }
        else if (argv[i][1] == 'x') { max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 's' && argv[i][2] == '\0') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g' && argv[i][2] == 'e') { gelu_fusion = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else if (argv[i][1] == 'a') { overfit_single_batch = atoi(argv[i+1]); }
        else if (argv[i][1] == 'f') { override_enable_tf32 = atoi(argv[i+1]); }
        else if (argv[i][1] == 'w') { use_master_weights = atoi(argv[i+1]); }
        else if (argv[i][1] == 'z') { zero_stage = atoi(argv[i+1]); }
        else if (argv[i][1] == 'r') { recompute = atoi(argv[i+1]); }
        else if (argv[i][1] == 'h') { hellaswag_eval = atoi(argv[i+1]); }
        else if (argv[i][1] == 'k') { lr_scheduler_type = argv[i+1]; }
        else if (argv[i][1] == 'p' && argv[i][2] == 'i') { strcpy(nccl_init_method, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'f') { strcpy(fs_path, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 's') { strcpy(server_ip, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'n') { num_processes = atoi(argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'r') { process_rank = atoi(argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'g') { gpus_per_node = atoi(argv[i+1]); }
        else if (argv[i][1] == 's' && argv[i][2] == 'l') { skip_update_lossz = atof(argv[i+1]); }
        else if (argv[i][1] == 's' && argv[i][2] == 'g') { skip_update_gradz = atof(argv[i+1]); }
        else if (argv[i][1] == 'n' && argv[i][2] == 'k') { checkpoints_keep = atoi(argv[i+1]); }
        else if (argv[i][1] == 'n' && argv[i][2] == 'm') { major_checkpoint_every = atoi(argv[i+1]); }
        else { error_usage(); }
    }

    multi_gpu_config = multi_gpu_config_init(num_processes, process_rank, gpus_per_node, server_ip, fs_path, nccl_init_method);
    common_start(override_enable_tf32, false); // common init code for train/test/profile

    // should do a bit more error checking here
    assert(warmup_iterations >= 0);
    if (output_log_dir != NULL) {
        assert(strlen(output_log_dir) < 400); // careful bunch of hardcoded snprintf around this
    }
    int tokens_per_fwdbwd = B * T * multi_gpu_config.num_processes; // one micro-batch processes this many tokens
    // calculate sensible default for total batch size as assuming no gradient accumulation
    if (total_batch_size == -1) { total_batch_size = tokens_per_fwdbwd; }
    // in the future, we might want to set gelu fusion to 2 for SM90+ and 0 for other GPUs
    if (gelu_fusion == -1) { gelu_fusion = 0; } // (deviceProp.major >= 9) ? 2 : 0; } // in gpt2_init_common for test_gpt2cu...
    // calculate the number of gradient accumulation steps from the desired total batch size
    assert(total_batch_size % tokens_per_fwdbwd == 0);
    int grad_accum_steps = total_batch_size / tokens_per_fwdbwd;
    // if we're only overfitting a single batch for debugging, let's overfit the first batch
    // from val instead of train split, because val is smaller and faster. (train_gpt2.py does the same)
    if (overfit_single_batch == 1) { train_data_pattern = val_data_pattern; }
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| Parameter             | Value                                              |\n");
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| train data pattern    | %-50s |\n", train_data_pattern);
    printf0("| val data pattern      | %-50s |\n", val_data_pattern);
    printf0("| output log dir        | %-50s |\n", output_log_dir == NULL ? "NULL" : output_log_dir);
    printf0("| checkpoint_every      | %-50d |\n", checkpoint_every);
    printf0("| resume                | %-50d |\n", resume);
    printf0("| micro batch size B    | %-50d |\n", B);
    printf0("| sequence length T     | %-50d |\n", T);
    printf0("| total batch size      | %-50d |\n", total_batch_size);
    printf0("| LR scheduler          | %-50s |\n", lr_scheduler_type);
    printf0("| learning rate (LR)    | %-50e |\n", learning_rate);
    printf0("| warmup iterations     | %-50d |\n", warmup_iterations);
    printf0("| final LR fraction     | %-50e |\n", final_learning_rate_frac);
    printf0("| weight decay          | %-50e |\n", weight_decay);
    printf0("| skip update lossz     | %-50f |\n", skip_update_lossz);
    printf0("| skip update gradz     | %-50f |\n", skip_update_gradz);
    printf0("| max_steps             | %-50d |\n", max_steps);
    printf0("| val_loss_every        | %-50d |\n", val_loss_every);
    printf0("| val_max_steps         | %-50d |\n", val_max_steps);
    printf0("| sample_every          | %-50d |\n", sample_every);
    printf0("| genT                  | %-50d |\n", genT);
    printf0("| overfit_single_batch  | %-50d |\n", overfit_single_batch);
    printf0("| use_master_weights    | %-50s |\n", use_master_weights ? "enabled" : "disabled");
    printf0("| gelu_fusion           | %-50d |\n", gelu_fusion);
    printf0("| recompute             | %-50d |\n", recompute);
    printf0("+-----------------------+----------------------------------------------------+\n");
    const char* precision_str = (PRECISION_MODE == PRECISION_FP32)
                              ? (cublas_compute == HIPBLAS_COMPUTE_32F_FAST_TF32 ? "TF32" : "FP32")
                              : (PRECISION_MODE == PRECISION_FP16 ? "FP16" : "BF16");
    printf0("| device                | %-50s |\n", deviceProp.name);
    printf0("| peak TFlops           | %-50.1f |\n", get_flops_promised(deviceProp.name, PRECISION_MODE));
    printf0("| precision             | %-50s |\n", precision_str);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // figure out if we are going to be resuming the optimization
    int resuming = 0;
    // find the DONE file with the highest step count
    int resume_max_step = find_max_step(output_log_dir);
    if (resume == 1) { // is -y 1 resume flag set?
        assert(output_log_dir != NULL);
        if (resume_max_step != -1) {
            resuming = 1; // -y 1 is set, and we found a checkpoint we can resume from
            snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, resume_max_step);
        }
    }

    // build the GPT-2 model
    GPT2 model;
    gpt2_init_common(&model);
    if (resuming == 1) {
        // if `-y 1` was set, then we are resuming from the latest checkpoint
        // if we are using master weights, we'll init them later inside load_state()
        bool weight_init = !use_master_weights;
        gpt2_build_from_checkpoint(&model, filename_buffer, weight_init);
    } else if (ends_with_bin(load_filename)) {
        // otherwise, if this is a .bin file, we assume it's a model, let's init from it
        gpt2_build_from_checkpoint(&model, load_filename);
    } else {
        // if it's not .bin, it could be a "special descriptor". This descriptor is used to
        // construct GPT-2 / GPT-3 models in a convenient format. See the function for docs.
        gpt_build_from_descriptor(&model, load_filename);
    }

    model.use_master_weights = use_master_weights;
    model.gelu_fusion = gelu_fusion;
    model.recompute = recompute;
    printf0("| weight init method    | %-50s |\n", resuming == 1 ? "intermediate checkpoint" : load_filename);
    printf0("| max_sequence_length T | %-50d |\n", model.config.max_seq_len);
    printf0("| vocab_size V          | %-50d |\n", model.config.vocab_size);
    printf0("| padded_vocab_size Vp  | %-50d |\n", model.config.padded_vocab_size);
    printf0("| num_layers L          | %-50d |\n", model.config.num_layers);
    printf0("| num_heads NH          | %-50d |\n", model.config.num_heads);
    printf0("| channels C            | %-50d |\n", model.config.channels);
    printf0("| num_parameters        | %-50zu |\n", model.num_parameters);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build DataLoaders for both train and val
    int permute_train_loader = (overfit_single_batch == 1) ? 0 : 1;
    DataLoader train_loader, val_loader;
    dataloader_init(&train_loader, train_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes, permute_train_loader);
    dataloader_init(&val_loader, val_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes, 0);
    // figure out the number of training steps we will run for
    int train_num_batches = max_steps; // passed in from command line
    if (train_num_batches == -1) {
        // sensible default is to train for exactly one epoch
        size_t ntok = train_loader.num_tokens;
        // the number of (outer loop) steps each process should take for us to reach one epoch
        train_num_batches = ntok / total_batch_size;
    }
    // figure out the number of validation steps to run for
    int val_num_batches = val_max_steps; // passed in from command line
    if (val_num_batches == -1) {
        // sensible default is to evaluate the full validation split
        size_t ntok = val_loader.num_tokens;
        // note that unlike the training loop, there is no gradient accumulation inner loop here
        val_num_batches = ntok / tokens_per_fwdbwd;
    }
    printf0("| train_num_batches     | %-50d |\n", train_num_batches);
    printf0("| val_num_batches       | %-50d |\n", val_num_batches);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build an EvalLoader for HellaSwag
    EvalLoader eval_loader;
    const char* hellaswag_path = "dev/data/hellaswag/hellaswag_val.bin";
    const bool hellaswag_available = access(hellaswag_path, F_OK) == 0;
    const bool run_hellaswag = hellaswag_eval && hellaswag_available;
    if (run_hellaswag) {
        evalloader_init(&eval_loader, hellaswag_path, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    }
    printf0("| run hellaswag         | %-50s |\n", run_hellaswag ? "yes" : "no");
    printf0("+-----------------------+----------------------------------------------------+\n");

    // pretty print in a table the multi-gpu configuration as well
    set_zero_configs(&multi_gpu_config, zero_stage, model.num_parameters);
    printf0("| num_processes         | %-50d |\n", multi_gpu_config.num_processes);
    printf0("| zero_stage            | %-50d |\n", multi_gpu_config.zero_stage);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // prints outside of pretty table to here and below
    if (!hellaswag_available) {
        printf0("HellaSwag eval not found at %s, skipping its evaluation\n", hellaswag_path);
        printf0("You can run `python dev/data/hellaswag.py` to export and use it with `-h 1`.\n");
    }
    // more prints related to allocations from gpt2_build_from_checkpoint down here to not mess up our table above
    printf0("num_parameters: %zu => bytes: %zu\n", model.num_parameters, model.num_parameters_bytes);
    printf0("allocated %d MiB for model parameters\n", (int)round(model.num_parameters_bytes / (1024 * 1024)));
    // few more prints for gradient accumulation math up above
    printf0("batch_size B=%d * seq_len T=%d * num_processes=%d and total_batch_size=%d\n",
            B, T, multi_gpu_config.num_processes, total_batch_size);
    printf0("=> setting grad_accum_steps=%d\n", grad_accum_steps);

    // set up logging
    if (multi_gpu_config.process_rank == 0) { create_dir_if_not_exists(output_log_dir); }
    Logger logger;
    logger_init(&logger, output_log_dir, multi_gpu_config.process_rank, resume);

    // set up the Tokenizer
    Tokenizer tokenizer;
    tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

    // set up learning rate scheduler
    LearningRateScheduler lr_scheduler;
    lr_scheduler_init(&lr_scheduler, lr_scheduler_type, learning_rate,
                      warmup_iterations, train_num_batches, final_learning_rate_frac);

    // some memory for generating samples from the model
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float*  cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    // if we found a checkpoint to resume from, load the optimization state
    int step = 0;
    gpt2_allocate_state(&model, B, T);
    if (resuming == 1) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, resume_max_step, multi_gpu_config.process_rank);
        load_state(&step, &model, &train_loader, filename_buffer);
    }

    // init an OutlierDetector the training loss
    OutlierDetector loss_outlier_detector, grad_norm_outlier_detector;
    init_detector(&loss_outlier_detector);
    init_detector(&grad_norm_outlier_detector);

    // do some checks here before we kick off training
    // cross-check the desired sequence length T with the model's max sequence length
    if (T < model.config.max_seq_len) {
        printf0("!!!!!!!!\n");
        printf0("WARNING:\n");
        printf0("- The training sequence length is: T=%d (set with -t)\n", T);
        printf0("- The model's max sequence length is: max_seq_len=%d\n", model.config.max_seq_len);
        printf0("You are attempting to train with a sequence length shorter than the model's max.\n");
        printf0("This will lead to unused parameters in the wpe position embedding weights.\n");
        printf0("If you know what you're doing you can ignore this warning.\n");
        printf0("If you're like ???, you are most likely misconfiguring your training run.\n");
        printf0("---> HINT: If you're training GPT-2 use -t 1024. If GPT-3, use -t 2048.\n");
        printf0("!!!!!!!!\n");
    }
    // in any case, this must be true or we'd index beyond the model's wpe (position embedding table)
    assert(T <= model.config.max_seq_len);

    // train
    hipEvent_t start, end;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&end));
    cudaCheck(hipProfilerStart());
    double total_sum_iteration_time_s = 0.0;
    float ema_tokens_per_second = 0.0f;
    for (; step <= train_num_batches; step++) {
        NvtxRange step_range("Train step", step);

        int last_step = step == train_num_batches;

        // once in a while estimate the validation loss (all processes collaborate)
        if (step % val_loss_every == 0 || last_step) {
            NvtxRange validation_range("validation");
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                val_loss += gpt2_validate(&model, val_loader.inputs, val_loader.targets, B, T);
            }
            val_loss /= val_num_batches;
            val_loss = multi_gpu_cpu_float_sum(val_loss, &multi_gpu_config) / multi_gpu_config.num_processes;
            printf0("val loss %f\n", val_loss);
            logger_log_val(&logger, step, val_loss);
        }

        // once in a while estimate HellaSwag accuracy (all processes collaborate)
        if (run_hellaswag &&
           ((step > 0 && step % val_loss_every == 0) || last_step)) {
            NvtxRange evaluation_range("evaluation");
            float eval_acc_norm = 0.0f;
            evalloader_reset(&eval_loader);
            for (int i = 0; i < eval_loader.num_batches; i++) {
                if (i % 10 == 0) { printf("evaluating HellaSwag: %d/%d\r", i, eval_loader.num_batches); }
                evalloader_next_batch(&eval_loader);
                gpt2_validate(&model, eval_loader.inputs, eval_loader.targets, B, T);
                int correct = evalloader_stat_losses(&eval_loader, model.cpu_losses);
                eval_acc_norm += (float)correct;
            }
            // careful because not all ranks may have the exact same allocation of number of examples
            eval_acc_norm = multi_gpu_cpu_float_sum(eval_acc_norm, &multi_gpu_config);
            printf0("HellaSwag: %d/%d = %f\n", (int)eval_acc_norm, eval_loader.num_examples, eval_acc_norm / eval_loader.num_examples);
            logger_log_eval(&logger, step, eval_acc_norm / eval_loader.num_examples);
        }

        // once in a while do model inference to print generated text (only rank 0)
        if (multi_gpu_config.process_rank == 0 && sample_every > 0 &&
           (step > 0 && (step % sample_every) == 0 || last_step)) {
            NvtxRange generation_range("generation");
            unsigned long long sample_rng_state = 1337;
            // fill up gen_tokens with the <|endoftext|> token, which kicks off the generation
            int eot_token = tokenizer.eot_token;
            for(int i = 0; i < B * T; ++i) {
                gen_tokens[i] = eot_token;
            }
            // now sample from the model autoregressively
            printf("generating:\n---\n");
            for (int t = 1; t < genT; t++) {
                NvtxRange generation_range("Generation step", t);
                // we try not to be too wasteful for inference by not calculating all of B,T
                // Using a smaller B is always bit-for-bit identical, but T is more tricky
                // for non-CUDNN, we need to make sure the attention buffer is memset to 0
                // for cuDNN, it might suddenly decide to use a slightly different algorithm...
                // on cuDNN 9.2.1 with cuDNN FrontEnd 1.5.2, T >= 256 seems bit-for-bit identical
                // (but even if it wasn't fully identical that's probably not the end of the world)
                // note this is still somewhat wasteful because we don't have a KV cache!
                gpt2_forward(&model, gen_tokens, 1, CEIL_DIV(t, min(T,256)) * min(T,256));
                // get the V-dimensional vector probs[0, t-1, :]
                floatX* logits = model.acts.output + (t - 1) * model.config.padded_vocab_size;
                // move probs back to CPU and sample (note we only move the first vocab_size logits, ignoring the padding)
                cudaCheck(hipMemcpy(cpu_logits_raw, logits, model.config.vocab_size * sizeof(floatX), hipMemcpyDeviceToHost));
                // convert to FP32 into cpu_logits (this does nothing useful if floatX == float)
                for (int i = 0; i < model.config.vocab_size; i++) {
                    cpu_logits[i] = (float)cpu_logits_raw[i];
                }
                // sample the next token
                float coin = random_f32(&sample_rng_state);
                int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
                // print the generated token, either using the Tokenizer or a fallback
                if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                } else {
                    // fall back to printing the token id
                    printf("%d ", next_token);
                }
                fflush(stdout);
            }
            printf("\n---\n");
        }

        // once in a while checkpoint the optimization state (all ranks)
        if ((checkpoint_every > 0 && output_log_dir != NULL && resuming == 0) &&
            ((step > 0 && step % checkpoint_every == 0) || last_step)) {
            // writes model .bin file, state .bin files, and DONE file for step
            write_checkpoint(output_log_dir, step, &model, &train_loader, &multi_gpu_config);
            // we only keep checkpoints_keep checkpoints on disk to save space
            // so now that we wrote a new checkpoint, delete one old one (unless it is a "major" checkpoint)
            // we only do this is checkpoint keeping is turned on (checkpoints_keep > 0)
            int step_delete = step - checkpoints_keep * checkpoint_every;
            if (checkpoints_keep > 0 && step_delete > 0 &&
               (major_checkpoint_every == 0 || step_delete % major_checkpoint_every != 0)
                ) {
                delete_checkpoint(output_log_dir, step_delete, &multi_gpu_config);
            }
        }
        resuming = 0;

        // bit confusing: we want to make sure to eval and sample on 0th iteration
        // but also after the very last iteration. so we loop for step <= train_num_batches
        // instead of just < train_num_batches (one extra due to <=), only to do
        // the validation/sampling one last time, and then we break right here as we're done.
        if (last_step) { break; }

        // --------------- TRAINING SECTION BEGIN -----------------
        if (overfit_single_batch == 1) {
            // if we are trying to overfit a single batch, we reset the loader here
            dataloader_reset(&train_loader);
        }
        // do one training step, doing forward/backward/update on total_batch_size tokens
        cudaCheck(hipEventRecord(start));
        // gradient and loss accumulation loop over micro-batches
        for (int micro_step = 0; micro_step < grad_accum_steps; micro_step++) {
            // fetch the next data batch
            dataloader_next_batch(&train_loader);
            // forward pass. note that we pass in grad_accum_steps, which scales down the loss
            gpt2_forward(&model, train_loader.inputs, B, T);
            // backward pass. all model params accumulate gradients with += inside this inner loop
            gpt2_backward_and_reduce(&model, train_loader.inputs, train_loader.targets, grad_accum_steps, micro_step);
        }
        float zloss = (float)(update_detector(&loss_outlier_detector, (double)model.mean_loss)); // loss z-score
        // fetch the next learning rate
        float step_learning_rate = get_learning_rate(&lr_scheduler, step);
        // calculate the gradient norm and how much we wish to scale the gradient
        float grad_norm = gpt2_calculate_grad_norm(&model, &multi_gpu_config);
        float zgrad = (float)(update_detector(&grad_norm_outlier_detector, (double)grad_norm)); // grad z-score
        // update the model parameters
        if (isfinite(zloss) && skip_update_lossz != 0.0f && zloss > skip_update_lossz) {
            printf0("skipping update due to loss z-score of %f\n", zloss);
        } else if (isfinite(zgrad) && skip_update_gradz != 0.0f && zgrad > skip_update_gradz) {
            printf0("skipping update due to grad z-score of %f\n", zgrad);
        } else {
            // clip the gradient norm to a maximum value
            float grad_clip = 1.0f;
            float grad_scale = (grad_norm > grad_clip) ? grad_clip / grad_norm : 1.0f;
            gpt2_update(&model, step_learning_rate, 0.9f, 0.95f, 1e-8f, weight_decay, grad_scale, step+1, &multi_gpu_config);
        }
        cudaCheck(hipEventRecord(end));
        cudaCheck(hipEventSynchronize(end)); // wait for the end event to finish to get correct timings
        // --------------- TRAINING SECTION END -------------------
        // everything that follows now is just diagnostics, prints, logging, etc.

        // todo - move or double-buffer all of this timing logic to avoid idling the GPU at this point!
        float time_elapsed_ms;
        cudaCheck(hipEventElapsedTime(&time_elapsed_ms, start, end));
        size_t tokens_processed = (size_t)multi_gpu_config.num_processes * B * T * grad_accum_steps;
        float tokens_per_second = tokens_processed / time_elapsed_ms * 1000.0f;
        float bias_corrected_ema_tokens_per_second = tokens_per_second; // by default set to non-ema version
        if (step > 0) { // consider the first batch to be a warmup (e.g. cuBLAS/cuDNN initialisation)
            total_sum_iteration_time_s += time_elapsed_ms / 1000.0f;
            // smooth out the tok/s with an exponential moving average, and bias correct just like in AdamW
            ema_tokens_per_second = 0.95f * ema_tokens_per_second + 0.05f * tokens_per_second;
            bias_corrected_ema_tokens_per_second = ema_tokens_per_second / (1.0f - powf(0.95f, step));
        }
        float mfu = gpt2_estimate_mfu(&model, B * T * grad_accum_steps, time_elapsed_ms / 1000.0f);
        printf0("step %4d/%d | loss %7.6f (%+.2fz)| norm %6.4f (%+.2fz)| lr %.2e | %.2f ms | %.1f%% bf16 MFU | %.0f tok/s\n",
                step + 1, train_num_batches, model.mean_loss, zloss, grad_norm, zgrad, step_learning_rate,
                time_elapsed_ms, 100*mfu, bias_corrected_ema_tokens_per_second);
        if(log_gpu_every > 0 && (step + 1) % log_gpu_every == 0) {
            GPUUtilInfo gpu_info = get_gpu_utilization_info();
            printf0("                  compute %2.1f%% | memory: %2.1f%% | fan: %2d%% | %4d MHz / %4d MHz | %3d W / %3d W | %d°C / %d°C | %s\n",
                    gpu_info.gpu_utilization, gpu_info.mem_utilization, gpu_info.fan, gpu_info.clock, gpu_info.max_clock, gpu_info.power / 1000, gpu_info.power_limit / 1000,
                    gpu_info.temperature, gpu_info.temp_slowdown, gpu_info.throttle_reason);
        }
        logger_log_train(&logger, step, model.mean_loss, step_learning_rate, grad_norm);

        // disable the profiler after 3 steps of optimization
        if (step == 3) { hipProfilerStop(); }
    }
    // add a total average, for optimizations that are only mild improvements (excluding 1st batch as warmup)
    printf0("total average iteration time: %f ms\n", total_sum_iteration_time_s / (train_num_batches-1) * 1000);

    // free and destroy everything
    cudaCheck(hipEventDestroy(end));
    cudaCheck(hipEventDestroy(start));
    if (run_hellaswag) { evalloader_free(&eval_loader); }
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    multi_gpu_config_free(&multi_gpu_config);
    gpt2_free(&model);
    common_free(model);
    return 0;
}
#endif
