#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net training loop. See README.md for usage.
*/
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string>
#include <string_view>
#include <sys/stat.h>
#include <sys/types.h>
// ----------- CPU utilities -----------
// defines: fopenCheck, freadCheck, fcloseCheck, fseekCheck, mallocCheck
// defines: create_dir_if_not_exists, find_max_step, ends_with_bin
#include "llmc/utils.h"
// defines: tokenizer_init, tokenizer_decode, tokenizer_free
#include "llmc/tokenizer.h"
// defines: dataloader_init, dataloader_reset, dataloader_next_batch, dataloader_free
// defines: evalloader_init, evalloader_reset, evalloader_next_batch, evalloader_free
#include "llmc/dataloader.h"
// defines: manual_seed, normal_ (same as torch.manual_seed and torch.normal)
#include "llmc/rand.h"
// defines: lr_scheduler_init, get_learning_rate
#include "llmc/schedulers.h"
// defines: sample_softmax, random_f32
#include "llmc/sampler.h"
// defines: logger_init, logger_log_eval, logger_log_val, logger_log_train
#include "llmc/logger.h"
// defines: get_flops_promised
#include "llmc/mfu.h"
// defines: OutlierDetector, init_detector, update_detector
#include "llmc/outlier_detector.h"
// ----------- GPU utilities -----------
// defines:
// WARP_SIZE, MAX_1024_THREADS_BLOCKS, CEIL_DIV, cudaCheck, PRECISION_MODE
// NVTX_RANGE_FN
#include "llmc/cuda_common.h"
// defines:
// Packed128, f128, x128
// warpReduceSum, warpReduceMax, blockReduce, copy_and_cast_kernel, cudaMallocConditionallyManaged
#include "llmc/cuda_utils.cuh"
// defines: CUBLAS_LOWP, cublasCheck, cublaslt_workspace_size, cublaslt_workspace
// defines: cublas_compute, cublaslt_handle, cublas_handle
#include "llmc/cublas_common.h"
// ----------- Layer implementations in CUDA -----------
// defines: encoder_forward, encoder_backward
#include "llmc/encoder.cuh"
// defines: layernorm_forward, residual_forward, fused_residual_forward5, layernorm_backward
#include "llmc/layernorm.cuh"
// defines: rmsnorm_forward, fused_residual_rmsnorm_forward5, rmsnorm_backward
#include "llmc/rmsnorm.cuh"
// defines: matmul_cublaslt, matmul_forward, matmul_backward, gelu_forward, gelu_backward_inplace
#include "llmc/matmul.cuh"
#ifdef ENABLE_CUDNN
// defines: create_cudnn, destroy_cudnn, attention_forward_cudnn, attention_backward_cudnn
#include "llmc/cudnn_att.h"
#else
// defines: attention_forward, attention_backward
#include "llmc/attention.cuh"
#endif
// defines: fused_classifier
#include "llmc/fused_classifier.cuh"
// defines: adamw_kernel3
#include "llmc/adamw.cuh"
// defines: global_norm_squared
#include "llmc/global_norm.cuh"
// defines: repkv_forward, repkv_backward
#include "llmc/repkv.cuh"
// defines: precompute_freqs_cis, rope_forward, rope_backward_inplace
#include "llmc/rope.cuh"
// defines: swiglu_forward, swiglu_backward
#include "llmc/swiglu.cuh"
// ----------- Multi-GPU support -----------
// defines: ncclFloatX, ncclCheck, MultiGpuConfig, ShardInfo
// defines: printf0, multi_gpu_config
// defines: multi_gpu_config_init, multi_gpu_config_free
// defines: set_zero_configs, multi_gpu_cpu_float_sum, multi_gpu_barrier
// defines: multi_gpu_get_shard_offset, multi_gpu_async_reduce_gradient
#include "llmc/zero.cuh"

// ----------------------------------------------------------------------------
// global vars for I/O
char filename_buffer[512];

// ----------------------------------------------------------------------------
// global vars containing information about the GPU this process is running on
hipDeviceProp_t deviceProp; // fills in common_start()
hipStream_t main_stream;
// buffer size to use for device <-> disk io
constexpr const size_t IO_BUF_SIZE = 32 * 1024 * 1024;

// ----------------------------------------------------------------------------
// LLama-3 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 128256
    int padded_vocab_size; // padded to e.g. %128==0, 128256
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of query heads in attention, e.g. 32
    int num_kv_heads; // number of key and value heads in attention, e.g. 8 (<-- new in Llama 3)
    int channels; // number of channels, e.g. 2048
    int multiple_of; // used in feedforward layer sizing, e.g. 1024 (<-- new in Llama 3)
    int use_scaled_rope; // whether to use scaled rope
    float ffn_dim_multiplier; // multiplier used in feedforward layer, e.g. 1.3 (<-- new in Llama 3)
    float norm_eps; // epsilon used in layernorm, e.g. 1e-5
    float rope_theta; // theta used in ROPE attention, e.g. 500000.0 (<-- new in Llama 3)
    bool use_biases;  // we always allocate memory for biases; to match llama3 they are not used
} LLama3Config;

// the parameters of the model
constexpr const int NUM_PARAMETER_TENSORS = 16;
typedef struct {
    floatX* wte; // (V, C)
    floatX* wlmhead; // (V, C)
    floatX* ln1w; // (L, C)
    floatX* ln1b; // (L, C)
    floatX* qkvw; // (L, 3*C, C)
    floatX* qkvb; // (L, 3*C)
    floatX* attprojw; // (L, C, C)
    floatX* attprojb; // (L, C)
    floatX* ln2w; // (L, C)
    floatX* ln2b; // (L, C)
    floatX* fcw; // (L, 4*C, C)
    floatX* fcb; // (L, 4*C)
    floatX* fcprojw; // (L, C, 4*C)
    floatX* fcprojb; // (L, C)
    floatX* lnfw; // (C)
    floatX* lnfb; // (C)
} ParameterTensors;
static_assert(sizeof(ParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

void fill_in_parameter_sizes(size_t* param_sizes, size_t* param_sizeof, LLama3Config config) {
    // see train_llama3.py write_tensors() function for detailed docs of some of the trickery here
    // trick 1: all biases are still present but set to zero
    // trick 2: the SwiGLU weights are "packed" into one, concatenated
    // trick 3: the positional embedding is replaced with the final classifier layer weights
    size_t Vp = config.padded_vocab_size;
    size_t C = config.channels;
    size_t L = config.num_layers;
    // calculation following the .py code inside CausalSelfAttention
    // we have to calculate the number of channels in the QKV projection
    size_t n_head = config.num_heads;
    size_t n_kv_head = config.num_kv_heads;
    size_t hd = C / n_head; // head dimension
    size_t qkv_channels = (n_head + 2*n_kv_head) * hd; // Q, K, V channels
    // calculation following the .py code inside MLP
    // we have to calculate the number of channels in the SwiGLU projections c_fc + c_fc2
    size_t hidden_dim = 4 * C;
    hidden_dim = (2 * hidden_dim) / 3;
    hidden_dim = config.ffn_dim_multiplier * hidden_dim;
    hidden_dim = config.multiple_of * ((hidden_dim + config.multiple_of - 1) / config.multiple_of);
    size_t ffn_channels = hidden_dim * 2; // c_fc + c_fc2 concatenated
    // now populate the parameter sizes
    param_sizes[0] = Vp * C; // wte
    param_sizes[1] = Vp * C; // (3) lm_head (final classifier layer weights)
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b; (1) all biases are zero it's ok
    param_sizes[4] = L * (qkv_channels) * C; // qkvw
    param_sizes[5] = L * (qkv_channels); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * ffn_channels * C; // fcw; (2) this is twice the size
    param_sizes[11] = L * ffn_channels; // fcb
    param_sizes[12] = L * C * hidden_dim; // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb
    // populate the parameter sizes in bytes (all the same for now, keeping for future use)
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        param_sizeof[i] = sizeof(floatX);
    }
}

// allocate memory for the parameters and point the individual tensors to the right places
void* malloc_and_point_parameters(ParameterTensors* params, size_t* param_elements, size_t *param_sizeof) {
    // calculate the total number of parameters and bytes across all tensors
    size_t num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters_bytes += param_elements[i] * param_sizeof[i];
    }
    // malloc all parameters all at once on the device
    void* params_memory;
    cudaCheck(hipMalloc((void**)&params_memory, num_parameters_bytes));
    // assign all the tensors their place in the array
    floatX** ptrs[] = {
        &params->wte, &params->wlmhead, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    char* params_memory_iterator = (char*)params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = (floatX*)params_memory_iterator;
        params_memory_iterator += param_elements[i] * param_sizeof[i];
    }
    return params_memory;
}

constexpr int NUM_ACTIVATION_TENSORS = 22;
typedef struct {
    floatX* encoded; // (B, T, C)
    floatX* ln1; // (L, B, T, C)
    float* ln1_mean; // (L, B, T)
    float* ln1_rstd; // (L, B, T)
    floatX* atty; // (L, B, T, C)
    // cuDNN saves only some statistics information
#if ENABLE_CUDNN
    float* att;  // (L, B, NH, T)
#else
    floatX* att; // (L, B, NH, T, T)
#endif

    floatX* residual2; // (L, B, T, C)
    floatX* ln2; // (L, B, T, C)
    float* ln2_mean; // (L, B, T)
    float* ln2_rstd; // (L, B, T)
    floatX* fch; // (L, B, T, 4*C)
    floatX* fch_gelu; // (L, B, T, 4*C)
    floatX* residual3; // (L, B, T, C)
    floatX* lnf; // (B, T, C);   if LN recomputation is enabled (-r 2 and above), will be used for _all_ layernorms
    float* lnf_mean; // (B, T)
    float* lnf_rstd; // (B, T)
    float* losses; // (B, T), will be accumulated in micro-steps
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    floatX* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    floatX* output;

    // some additional scratch buffers
    floatX* scratch_bt4c;   // (B, T, 4*C)
    floatX* scratch_btc;    // (B, T, C)
    floatX* scratch_bt4c2;  // (B, T, 4*C), for simplicify use this one for backward pass too, probably not needed
} ActivationTensors;


struct TensorSpec {
    void** ptr;
    size_t size;
    DType type;
};


#define TENSOR_SPEC(pointer, size) TensorSpec{(void**)(&pointer), (size), dtype_of(pointer)};

void fill_in_activation_sizes(const ActivationTensors* data, TensorSpec (&tensors)[NUM_ACTIVATION_TENSORS], size_t B, size_t T, LLama3Config config, int recompute) {
    const size_t Vp = config.padded_vocab_size;
    const size_t L = config.num_layers;
    const size_t NH = config.num_heads;
    const size_t C = config.channels;
    const size_t n_head = config.num_heads; // num query heads
    const size_t n_kv_head = config.num_kv_heads; // num key and value heads
    const size_t hd = C / n_head; // the size of each head
    const size_t qkv_channels = (n_head + 2*n_kv_head) * hd; // Q, K, V channels
    // SwiGLU-related calculation to determine the number of channels here
    size_t hidden_dim = 4 * C;
    hidden_dim = (2 * hidden_dim) / 3;
    hidden_dim = config.ffn_dim_multiplier * hidden_dim;
    hidden_dim = config.multiple_of * ((hidden_dim + config.multiple_of - 1) / config.multiple_of);
    size_t ffn_channels = hidden_dim * 2; // c_fc + c_fc2 concatenated
    size_t ffn_channels_post_gelu = hidden_dim; // swiglu will halve the channels

    tensors[0] = TENSOR_SPEC(data->encoded, B * T * C);
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    tensors[1] = TENSOR_SPEC(data->ln1,  (recompute < 2) ? L * B * T * C : 0);
    tensors[2] = TENSOR_SPEC(data->ln1_mean, 0); // Llama 3 does not use this activation
    tensors[3] = TENSOR_SPEC(data->ln1_rstd, L * B * T);
    tensors[4] = TENSOR_SPEC(data->atty, L * B * T * C);
    #ifdef ENABLE_CUDNN
    // FP32 stats tensor for cuDNN to be passed to backward pass
    tensors[5] = TENSOR_SPEC(data->att, L * B * NH * T);
    #else
    tensors[5] = TENSOR_SPEC(data->att, L * B * NH * T * T);
    #endif
    tensors[6] = TENSOR_SPEC(data->residual2, L * B * T * C);
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    tensors[7] = TENSOR_SPEC(data->ln2, (recompute < 2) ? L * B * T * C : 0);
    tensors[8] = TENSOR_SPEC(data->ln2_mean, L * B * T);
    tensors[9] = TENSOR_SPEC(data->ln2_rstd, L * B * T);
    tensors[10] = TENSOR_SPEC(data->fch, L * B * T * ffn_channels);
    // if recompute >= 1 then we will recompute gelu_forward during backward and use this as scratch buffer
    tensors[11] = TENSOR_SPEC(data->fch_gelu, (recompute < 1) ? L * B * T * ffn_channels_post_gelu : B * T * ffn_channels_post_gelu);
    tensors[12] = TENSOR_SPEC(data->residual3, L * B * T * C);
    tensors[13] = TENSOR_SPEC(data->lnf, B * T * C);
    tensors[14] = TENSOR_SPEC(data->lnf_mean, B * T);
    tensors[15] = TENSOR_SPEC(data->lnf_rstd, B * T);
    tensors[16] = TENSOR_SPEC(data->losses, B * T);
    tensors[17] = TENSOR_SPEC(data->qkvr, L * B * T * 3*C); // 3*C is correct - this is QKV after replication of KV
    tensors[18] = TENSOR_SPEC(data->output, B * T * max(qkv_channels, max(ffn_channels, max(NH*T, Vp))));
    tensors[19] = TENSOR_SPEC(data->scratch_bt4c, B * T * ffn_channels);
    tensors[20] = TENSOR_SPEC(data->scratch_btc, B * T * C);
    tensors[21] = TENSOR_SPEC(data->scratch_bt4c2, B * T * ffn_channels);
}

void* malloc_and_point_activations(TensorSpec (&tensors)[NUM_ACTIVATION_TENSORS]) {
    size_t bytes = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        bytes += tensors[i].size * sizeof_dtype(tensors[i].type);
    }
    printf0("allocating %d MiB for activations\n", (int)round(bytes / (1024 * 1024)));
    void* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, bytes));
    // hipMalloc does not guarantee initial memory values so we memset the allocation here
    // this matters because e.g. non-cuDNN attention assumes the attention buffer is zeroed
    // todo - up to ~100ms on slow GPUs, could theoretically be more selective, but this is safer
    cudaCheck(hipMemset(acts_memory, 0, bytes));
    char* acts_memory_iterator = (char*)acts_memory;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        // extra protection so we don't accidentally use an empty buffer
        if(tensors[i].size == 0) {
            *(tensors[i].ptr) = NULL;
        }else {
            *(tensors[i].ptr) = acts_memory_iterator;
            acts_memory_iterator += tensors[i].size * sizeof_dtype(tensors[i].type);
        }
    }
    return acts_memory;
}

typedef struct {
    LLama3Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_elements[NUM_PARAMETER_TENSORS];
    size_t param_sizeof[NUM_PARAMETER_TENSORS];
    void* params_memory;
    size_t num_parameters;
    size_t num_parameters_bytes;
    // gradients of the weights
    ParameterTensors grads;
    void* grads_memory;
    // buffers for the AdamW optimizer
    floatOpt* m_memory;
    floatOpt* v_memory;
    float* master_weights;     // is NULL unless fp32 weights is enabled.
    // the activations of the model, and their sizes
    ActivationTensors acts;
    TensorSpec acts_specs[NUM_ACTIVATION_TENSORS];
    void* acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after the last backward micro-batch, will be populated with mean loss across all GPUs and micro-steps
    float* accumulated_mean_loss; // GPU buffer used to accumulate loss across micro-steps
    float* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
    unsigned long long rng_state; // the RNG state for seeding stochastic rounding etc.
    unsigned long long rng_state_last_update; // RNG before last llama3_update() to re-round identically from master weights
    int use_master_weights; // keep master weights copy in float for optim update? 0|1
    bool init_state;   // set to true if master weights need to be initialized
    int gelu_fusion; // fuse gelu via cuBLASLt (0=none, 1=forward, 2=forward+backward)
    int recompute; // recompute gelu | layernorm forward during model backward? 0|1|2
    // todo - if other functions need cpu scratch buffers in the future, reuse as generic scratch?
    int* workload_indices; // encoder_backward, B*T*num_c_groups (int)
    int4* bucket_info;     // encoder_backward, B*T*num_c_groups (int4) - size for worst case
    floatX* freqs_cis; // (T, hd) for RoPE
} LLama3;

void llama3_init_common(LLama3 *model) {
    // common inits outside of the model weights
    // memory lazily initialized in forward()
    model->acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->accumulated_mean_loss = NULL;
    model->cpu_losses = NULL;
    // the B,T params are determined and set, fixed on first batch in forward()
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f designates no loss, set at end of forward()
    model->params_memory = NULL;
    // memory lazily initialized in backward()
    model->grads_memory = NULL;
    model->workload_indices = NULL; // on cpu, for encoder_backward
    model->bucket_info = NULL; // on cpu, for encoder_backward
    // memory lazily initialized in update()
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    // other default settings
    model->rng_state = 13371337 + multi_gpu_config.process_rank; // used in stochastic rounding
    model->use_master_weights = 1; // safe default: do keep master weights in fp32
    model->init_state = true;
    model->recompute = 1; // good default: recompute gelu but not layernorm
    model->gelu_fusion = 0; //deviceProp.major >= 9 ? 2 : 0; // default: off for now (default must match main())
    model->freqs_cis = NULL;
}

void llama3_allocate_weights(LLama3 *model) {
    // fill in all the parameter tensor dimensions and types
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);
    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }

    // TODO TAKE OUT ----------------------------------------------------------
    // DEBUGGING: print out the sizes of the parameters
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        printf("param_elements[%d] = %zu\n", i, model->param_elements[i]);
    }
    printf("num_parameters = %zu\n", model->num_parameters);
    printf("num_parameters_bytes = %zu\n", model->num_parameters_bytes);
    // ------------------------------------------------------------------------

    // create memory for model parameters on the device
    assert(model->params_memory == nullptr);
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);
}

void llama3_allocate_state(LLama3 *model, int B, int T) {
    printf0("allocating %d MiB for parameter gradients\n", (int)round(model->num_parameters * sizeof(floatX) / (1024 * 1024)));
    assert(model->grads_memory == nullptr);
    model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_elements, model->param_sizeof);

    // record the current B,T as well
    model->batch_size = B;
    model->seq_len = T;

    // allocate the space
    fill_in_activation_sizes(&model->acts, model->acts_specs, B, T, model->config, model->recompute);
    model->acts_memory = malloc_and_point_activations(model->acts_specs);
    // also create memory for caching inputs and targets
    cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
    cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
    cudaCheck(hipMalloc(((void**)&model->accumulated_mean_loss), sizeof(float)));
    cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(float)));

    // initialise cpu scratch buffers for encoder backward
    size_t num_c_groups = CEIL_DIV(model->config.channels, (WARP_SIZE * x128::size));
    assert((size_t)(model->batch_size * model->seq_len) * num_c_groups < (1ULL<<31ULL)); // todo - maybe an issue for llama3-400B(?)
    model->workload_indices = (int*)mallocCheck(sizeof(int) * model->batch_size * model->seq_len * num_c_groups);
    model->bucket_info = (int4*)mallocCheck(sizeof(int4) * model->batch_size * model->seq_len * num_c_groups);

    // precompute freqs_cis for RoPE
    int hd = model->config.channels / model->config.num_heads;
    printf("calculating and allocating %zu KiB for freqs_cis\n", (T * hd * sizeof(floatX)) >> 10);
    floatX* freqs_cis_cpu = (floatX*)mallocCheck(T * hd * sizeof(floatX));
    precompute_freqs_cis(freqs_cis_cpu, hd, T, model->config.rope_theta, model->config.use_scaled_rope);
    cudaCheck(hipMalloc((void**)&model->freqs_cis, T * hd * sizeof(floatX)));
    cudaCheck(hipMemcpy(model->freqs_cis, freqs_cis_cpu, T * hd * sizeof(floatX), hipMemcpyHostToDevice));
    free(freqs_cis_cpu);

    // cudaMallocConditionallyManaged can fall back to hipMallocManaged if not enough memory on device
    // and returns a status code of 1 if it had to fall back, in that case we want to print warning.
    int memory_status = 0;

    // we will now init the optimizer states and master weights
    // this is usually a substantial amount of memory allocation right here.
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters; // num parameters we are responsible for
    printf0("allocating %zu MiB for AdamW optimizer state m\n", (shard_num_parameters * sizeof(floatOpt)) >> 20);
    printf0("allocating %zu MiB for AdamW optimizer state v\n", (shard_num_parameters * sizeof(floatOpt)) >> 20);
    assert(model->m_memory == nullptr);
    assert(model->v_memory == nullptr);
    memory_status |= cudaMallocConditionallyManaged((void**)&model->m_memory, shard_num_parameters * sizeof(floatOpt));
    memory_status |= cudaMallocConditionallyManaged((void**)&model->v_memory, shard_num_parameters * sizeof(floatOpt));

    if (model->use_master_weights == 1) {
        assert(model->master_weights == nullptr);
        printf0("allocating %zu MiB for master copy of params\n", (shard_num_parameters * sizeof(float)) >> 20);
        memory_status |= cudaMallocConditionallyManaged((void**) &model->master_weights, shard_num_parameters * sizeof(float));
    }

    // report on mixed memory allocation status (re-using our float reduce function, bit awk ok)
    int reduced_memory_status = (int) multi_gpu_cpu_float_sum((float)memory_status, &multi_gpu_config);
    if (reduced_memory_status >= 1) {
        printf0("WARNING: Fell back to hipMallocManaged when initializing m,v,master_weights on %d GPUs\n", reduced_memory_status);
        printf0("         Prevents an OOM, but code may run much slower due to device <-> host memory movement\n");
    }
    // report on device memory usage
    size_t free, total;
    cudaCheck(hipMemGetInfo(&free, &total));
    printf0("device memory usage: %zd MiB / %zd MiB\n", (total-free) / 1024 / 1024, total / 1024 / 1024);
    // give an estimate of the maximum batch size
    size_t bytes_per_sequence = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        bytes_per_sequence += model->acts_specs[i].size * sizeof_dtype(model->acts_specs[i].type) / B;
    }
    printf0("memory per sequence: %zu MiB\n", bytes_per_sequence / 1024 / 1024);
    printf0(" -> estimated maximum batch size: %zu\n", B + free / bytes_per_sequence);
}

void llama3_write_to_checkpoint(LLama3 *model, const char* checkpoint_path) {
    // write the model to a checkpoint file
    printf0("Writing model to %s\n", checkpoint_path);
    FILE *model_file = fopenCheck(checkpoint_path, "wb");
    // write the header first
    int model_header[256];
    memset(model_header, 0, sizeof(model_header));
    model_header[0] = 20240803; // magic number
    assert(PRECISION_MODE == PRECISION_FP32 || PRECISION_MODE == PRECISION_BF16);
    model_header[1] = PRECISION_MODE == PRECISION_FP32 ? 3 : 5; // version
    model_header[2] = model->config.max_seq_len;
    model_header[3] = model->config.vocab_size;
    model_header[4] = model->config.num_layers;
    model_header[5] = model->config.num_heads;
    model_header[6] = model->config.num_kv_heads;
    model_header[7] = model->config.channels;
    model_header[8] = model->config.multiple_of;
    model_header[9] = model->config.use_scaled_rope;
    model_header[10] = 3;
    model_header[11] = 1;
    fwriteCheck(model_header, sizeof(int), 256, model_file);
    float float_header[256];
    float_header[0] = model->config.ffn_dim_multiplier;
    float_header[1] = model->config.norm_eps;
    float_header[2] = model->config.rope_theta;
    fwriteCheck(float_header, sizeof(float), 256, model_file);
    // write the parameters
    device_to_file(model_file, model->params_memory, model->num_parameters_bytes,
                   IO_BUF_SIZE, main_stream);
    // close file, we're done
    fcloseCheck(model_file);
}

void llama3_build_from_checkpoint(LLama3 *model, const char* checkpoint_path, bool weight_init=true) {
    // If weight_init is true, we will load the weights from this checkpoint .bin file
    // We sometimes want this to be false, if we are going to initialize these weights from
    // the master weights that are instead stored in the state .bin file.
    // In that case, this function mostly loads the model hyperparameters from the header.

    if (PRECISION_MODE == PRECISION_FP16) {
        // TODO for later perhaps, would require us dynamically converting the
        // model weights from fp32 to fp16 online, here in this function, or writing
        // the fp16 weights directly from Python, which we only do for fp32/bf16 atm.
        fprintf(stderr, "build_from_checkpoint() does not support fp16 right now.\n");
        exit(EXIT_FAILURE);
    }

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int header_int[256]; // int section of the header
    freadCheck(header_int, sizeof(int), 256, model_file);
    assert(sizeof(int) == 4); // i think the python export code currently assumes this is int32
    float header_float[256]; // float section of the header
    freadCheck(header_float, sizeof(float), 256, model_file);
    assert(sizeof(float) == 4); // i think the python export code currently assumes this is float32
    if (header_int[0] != 20240803) { printf("Bad magic model file\n"); exit(EXIT_FAILURE); }
    int version = header_int[1];
    if (!(version == 3 || version == 5)) {
        // 3 = fp32, padded vocab
        // 5 = bf16, padded vocab, layernorms also in bf16
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_llama3.py`\n");
        exit(EXIT_FAILURE);
    }

    // check if the precision mode of the checkpoing matches the model precision
    if (weight_init) {
        if (PRECISION_MODE == PRECISION_BF16 && version != 5) {
            fprintf(stderr, "Precision is configured as BF16 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: are you sure you're loading a _bf16.bin file?\n");
            exit(EXIT_FAILURE);
        }
        if (PRECISION_MODE == PRECISION_FP32 && version != 3) {
            fprintf(stderr, "Precision is configured as FP32 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: to turn on FP32 you have to compile like: `make train_llama3cu PRECISION=FP32`\n");
            fprintf(stderr, "---> HINT: are you sure you're loading a .bin file without any _bf16 in the name?\n");
            exit(EXIT_FAILURE);
        }
    }

    // read in hyperparameters from the header
    // first the integer section
    model->config.max_seq_len = header_int[2];
    model->config.vocab_size = header_int[3];
    model->config.padded_vocab_size = model->config.vocab_size; // in Llama 3 there is no need for padding
    model->config.num_layers = header_int[4];
    model->config.num_heads = header_int[5];
    model->config.num_kv_heads = header_int[6];
    model->config.channels = header_int[7];
    model->config.multiple_of = header_int[8];
    model->config.use_scaled_rope = header_int[9];
    model->config.use_biases = false;
    int major_version = header_int[10]; // currently unused, e.g. 3
    int minor_version = header_int[11]; // currently unused, e.g. 1 (so Llama 3.1)
    // now the float section
    model->config.ffn_dim_multiplier = header_float[0];
    model->config.norm_eps = header_float[1];
    model->config.rope_theta = header_float[2];

    // ------------------------------------------------------------------------
    // TODO TAKE OUT ----------------------------------------------------------
    // Debugging: print all of the values above to check visually and EXIT
    printf("CHECK:\n");
    printf("max_seq_len: %d\n", model->config.max_seq_len);
    printf("vocab_size: %d\n", model->config.vocab_size);
    printf("padded_vocab_size: %d\n", model->config.padded_vocab_size);
    printf("num_layers: %d\n", model->config.num_layers);
    printf("num_heads: %d\n", model->config.num_heads);
    printf("num_kv_heads: %d\n", model->config.num_kv_heads);
    printf("channels: %d\n", model->config.channels);
    printf("multiple_of: %d\n", model->config.multiple_of);
    printf("use_scaled_rope: %d\n", model->config.use_scaled_rope);
    printf("major version: %d\n", major_version);
    printf("minor version: %d\n", minor_version);
    printf("ffn_dim_multiplier: %f\n", model->config.ffn_dim_multiplier);
    printf("norm_eps: %f\n", model->config.norm_eps);
    printf("rope_theta: %f\n", model->config.rope_theta);
    // ------------------------------------------------------------------------

    // allocate memory for the model parameters
    llama3_allocate_weights(model);

    // read in the parameters if weight_init is true
    if (weight_init) {
        assert(model->params_memory != NULL);
        file_to_device(model->params_memory, model_file, model->num_parameters_bytes, IO_BUF_SIZE, main_stream);
    }
    fcloseCheck(model_file);

    // only return from this function once we are certain the params are ready on the GPU
    cudaCheck(hipDeviceSynchronize());
}

// propagate inputs through the network to produce logits.
// right now, this function is fully synchronous with the host
void llama3_forward(LLama3 *model, const int* inputs, size_t B, size_t T) {
    NVTX_RANGE_FN();
    // we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;
    const size_t n_head = model->config.num_heads;
    const size_t n_kv_head = model->config.num_kv_heads;
    const size_t hd = C / n_head; // head dimension
    const size_t qkv_channels = (n_head + 2*n_kv_head) * hd; // Q, K, V channels
    size_t hidden_dim = 4 * C;
    hidden_dim = (2 * hidden_dim) / 3;
    hidden_dim = model->config.ffn_dim_multiplier * hidden_dim;
    hidden_dim = model->config.multiple_of * ((hidden_dim + model->config.multiple_of - 1) / model->config.multiple_of);
    size_t ffn_channels = hidden_dim * 2; // c_fc + c_fc2 concatenated
    size_t ffn_channels_post_gelu = hidden_dim; // swiglu halves the channels

    // validate B,T are not larger than the values used at initialisation
    // (smaller B,T are okay for inference only)
    if (B > model->batch_size || T > model->seq_len) {
        printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
        exit(EXIT_FAILURE);
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    // validate inputs, all indices must be in the range [0, V)
    // we can do this while the copies are already underway
    tokenCheck(inputs, B*T, V);

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    encoder_forward(acts.encoded, model->inputs, params.wte, NULL, B, T, C, main_stream); // encoding goes into residual[0]
    // first rmsnorm isn't fused
    rmsnorm_forward((model->recompute < 2) ? acts.ln1 : acts.lnf, acts.ln1_rstd, acts.encoded, params.ln1w, B, T, C, main_stream);

    for (int l = 0; l < L; l++) {
        NvtxRange layer_range("Layer", l);

        floatX* residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_qkvw = params.qkvw + l * qkv_channels * C;
        floatX* l_qkvb = model->config.use_biases ? params.qkvb + l * qkv_channels: nullptr;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = model->config.use_biases ? params.attprojb + l * C : nullptr;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_fcw = params.fcw + l * ffn_channels * C;
        floatX* l_fcb = model->config.use_biases ? params.fcb + l * ffn_channels : nullptr;
        floatX* l_fcprojw = params.fcprojw + l * C * ffn_channels_post_gelu;
        floatX* l_fcprojb = model->config.use_biases ? params.fcprojb + l * C : nullptr;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * ffn_channels;
        // reuse the same activation buffer at each layer, as we'll re-compute the gelu during backward
        // very useful because we dramatically reduce VRAM usage, and may be able to fit larger batch size
        floatX* l_fch_swiglu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * ffn_channels_post_gelu : acts.fch_gelu;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;
        floatX* scratch = (floatX*)acts.output; // used for non-cudnn attention, fcproj, attproj, etc.
        floatX* qkv_rep_scratch = (floatX*)acts.scratch_bt4c; // we can use the BT4C scratch for qkv replication

        // Attention block
        // The input l_ln1 now holds the (already layernormed) input
        #ifdef ENABLE_CUDNN
            printf("cuDNN path TODO\n"); exit(0);
            matmul_forward_cublaslt(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, qkv_channels, main_stream);
            float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
            attention_forward_cudnn(l_atty, (float*)l_att, l_qkvr, B, T, NH, C, main_stream);
        #else
            // unused parts of attention buffer must be zeroed (T-dependent)
            floatX* l_att = acts.att + l * B * NH * T * T;
            if (T != model->seq_len) { cudaCheck(hipMemset(l_att, 0, B * NH * T * T * sizeof(floatX))); }
            // 1) projection to QKV vectors (note k,v may be fewer heads than q)
            matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, qkv_channels, main_stream);
            // 2) replicate k,v so that all of q,k,v have the same number of heads. done for simplicity, for now
            repkv_forward(qkv_rep_scratch, scratch, B, T, n_head, n_kv_head, hd, main_stream);
            // 3) apply RoPE to q,k in place
            rope_forward(qkv_rep_scratch, qkv_rep_scratch, model->freqs_cis, B, T, n_head, hd, main_stream);
            // 4) attention: att <- softmax(qk^T)v
            attention_forward(l_atty, l_qkvr, l_att, qkv_rep_scratch, B, T, C, NH, main_stream);
        #endif

        matmul_forward_cublaslt(scratch, l_atty, l_attprojw, l_attprojb, B, T, C, C, main_stream);
        fused_residual_rmsnorm_forward5(l_residual2, l_ln2, l_ln2_rstd, residual, scratch, l_ln2w, B*T, C, main_stream);
        matmul_forward_cublaslt(l_fch, l_ln2, l_fcw, l_fcb, B, T, C, ffn_channels, main_stream);
        swiglu_forward(l_fch_swiglu, l_fch, B, T, ffn_channels_post_gelu, main_stream);
        matmul_forward_cublaslt(scratch, l_fch_swiglu, l_fcprojw, l_fcprojb, B, T, ffn_channels_post_gelu, C, main_stream);

        // OK, fusion across blocks.
        if(l+1 != L) {
            floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + (l + 1) * B * T * C : acts.lnf;
            float* l_ln1_rstd = acts.ln1_rstd + (l + 1) * B * T;
            const floatX* l_ln1w = params.ln1w + (l + 1) * C;
            fused_residual_rmsnorm_forward5(l_residual3, l_ln1, l_ln1_rstd, l_residual2, scratch, l_ln1w, B * T, C, main_stream);
        } else {
            fused_residual_rmsnorm_forward5(l_residual3, acts.lnf, acts.lnf_rstd, l_residual2, scratch, params.lnfw, B * T, C, main_stream);
        }
    }

    matmul_forward_cublaslt(acts.output, acts.lnf, params.wlmhead, NULL, B, T, C, Vp, main_stream);
    cudaCheck(hipDeviceSynchronize());
}


// Forwards both the model and the loss and is used for validation splits and evals.
// In particular it populates cpu_losses with loss at each token.
// Some of the evals (e.g. HellaSwag) require the per-token losses, which are produced here.
float llama3_validate(LLama3 *model, const int* inputs, const int* targets, size_t B, size_t T) {
    assert(targets != NULL);
    // forward the model itself
    llama3_forward(model, inputs, B, T);
    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;

    NvtxRange classifier_and_loss_range("classifier_and_loss");
    ActivationTensors acts = model->acts;
    float mean_loss = 0.0f;
    // fused classifier: does the forward pass and first part of the backward pass
    const float dloss = 1.0f / (B * T); // results in the uniform average loss over all elements
    // note: we don't need to generate dlogits here
    cudaCheck(hipMemset(acts.losses, 0, B*T*sizeof(float)));
    cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    tokenCheck(targets, B*T, V); // while the memcpy is underway, validate the targets
    fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp, False, main_stream);
    cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B*T; i++) {
        mean_loss += model->cpu_losses[i];
    }
    mean_loss /= B*T;
    cudaCheck(hipDeviceSynchronize());
    return mean_loss;
}

void llama3_backward_and_reduce(LLama3 *model, int* inputs, const int* targets, int grad_accum_steps, int micro_step) {
    if(model->grads_memory == nullptr) {
        fprintf(stderr, "Need to allocate gradients before backward");
        exit(EXIT_FAILURE);
    }
    NVTX_RANGE_FN();
    bool last_step = micro_step == grad_accum_steps - 1;
    // on the first micro-step zero the gradients, as we're about to += accumulate into them
    if (micro_step == 0) {
        // there are currently two state vars during the gradient accumulation inner loop:
        // 1) the losses accumulate += into acts.losses, reset here
        // 2) the gradients accumulate += into grads_memory, reset here
        cudaCheck(hipMemsetAsync(model->acts.losses, 0, model->batch_size * model->seq_len * sizeof(float), main_stream));
        cudaCheck(hipMemsetAsync(model->grads_memory, 0, model->num_parameters * sizeof(floatX), main_stream));
    }

    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    const size_t B = model->batch_size;
    const size_t T = model->seq_len;
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;
    const size_t n_head = model->config.num_heads;
    const size_t n_kv_head = model->config.num_kv_heads;
    const size_t hd = C / n_head; // head dimension
    const size_t qkv_channels = (n_head + 2*n_kv_head) * hd; // Q, K, V channels
    size_t hidden_dim = 4 * C;
    hidden_dim = (2 * hidden_dim) / 3;
    hidden_dim = model->config.ffn_dim_multiplier * hidden_dim;
    hidden_dim = model->config.multiple_of * ((hidden_dim + model->config.multiple_of - 1) / model->config.multiple_of);
    size_t ffn_channels = hidden_dim * 2; // c_fc + c_fc2 concatenated
    size_t ffn_channels_post_gelu = hidden_dim; // swiglu halves the channels

    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;

    // accumulate the losses inside acts.losses, and kick off the backward pass inside the fused classifier
    NvtxRange classifier_and_loss_range("classifier_and_loss");
    const float dloss = 1.0f / (float)(B * T * grad_accum_steps); // results in the uniform average loss over all elements
    cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    tokenCheck(targets, B*T, V);
    fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp, True, main_stream);
    // ------------------------------------------------------------------------
    // backward pass: go in the reverse order of the forward pass, and call backward() functions

    // reset residual stream gradients (put here to work with gradient accumulation)
    floatX* dresidual = (floatX*)model->acts.scratch_btc; // the main buffer holding the gradient in the backward pass
    cudaCheck(hipMemset(dresidual, 0, B * T * C * sizeof(floatX)));

    // re-use the output buffer of the forward pass as a scratchpad during backward pass
    float*  scratchF = (float*)acts.output;
    floatX* scratchX = (floatX*)acts.output;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(model->acts.scratch_bt4c, grads.wlmhead, NULL, acts.output, acts.lnf, params.wlmhead, NULL, B, T, C, Vp, main_stream);
    // backward the final layernorm
    floatX* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    rmsnorm_backward(dresidual, grads.lnfw, scratchF, model->acts.scratch_bt4c, residual, params.lnfw, acts.lnf_rstd, B, T, C, main_stream);
    // from this point on, we no longer need the values stored in the last residual, so we can reuse that memory as generic
    // scratch for backward computations
    floatX* dl_btc = residual;

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        NvtxRange layer_range("Layer", l);

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_ln1w = params.ln1w + l * C;
        floatX* l_qkvw = params.qkvw + l * qkv_channels * C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_fcw = params.fcw + l * ffn_channels * C;
        floatX* l_fcprojw = params.fcprojw + l * C * ffn_channels_post_gelu;
        // get the pointers of the gradients of the weights for this layer
        floatX* dl_ln1w = grads.ln1w + l * C;
        floatX* dl_ln1b = grads.ln1b + l * C;
        floatX* dl_qkvw = grads.qkvw + l * qkv_channels * C;
        floatX* dl_qkvb = model->config.use_biases ? grads.qkvb + l * qkv_channels : nullptr;
        floatX* dl_attprojw = grads.attprojw + l * C * C;
        floatX* dl_attprojb = model->config.use_biases ? grads.attprojb + l * C : nullptr;
        floatX* dl_ln2w = grads.ln2w + l * C;
        floatX* dl_ln2b = model->config.use_biases ? grads.ln2b + l * C : nullptr;
        floatX* dl_fcw = grads.fcw + l * ffn_channels * C;
        floatX* dl_fcb = model->config.use_biases ? grads.fcb + l * ffn_channels : nullptr;
        floatX* dl_fcprojw = grads.fcprojw + l * C * ffn_channels_post_gelu;
        floatX* dl_fcprojb = model->config.use_biases ? grads.fcprojb + l * C : nullptr;
        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        float* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch_pre_gelu = acts.fch + l * B * T * ffn_channels;
        floatX* l_fch_swiglu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * ffn_channels_post_gelu : acts.fch_gelu;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        floatX* dl_bt4c = (floatX*)model->acts.scratch_bt4c;
        floatX* dl_bt4c2 = (floatX*)model->acts.scratch_bt4c2; // same size as dl_bt4c, just a second buffer

        // start the backward pass for this layer
        if(model->recompute >= 1) {
            // recompute >= 1 means we recompute gelu. in this case,
            // l_fch_swiglu is just a buffer, so re-compute the gelu from l_fch here
            swiglu_forward(l_fch_swiglu, l_fch_pre_gelu, B, T, ffn_channels_post_gelu, main_stream);
        }
        // backward the 2nd matmul of MLP
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_swiglu, l_fcprojw, scratchF, B, T, ffn_channels_post_gelu, C, main_stream);
        // backward the swiglu here, use scratchX to hold the grad because SwiGLU can't be inplace
        swiglu_backward(dl_bt4c2, dl_bt4c, l_fch_pre_gelu, B, T, ffn_channels_post_gelu, main_stream);
        // backward the 1st matmul of MLP
        if(model->recompute >= 2) {
            // same as gelu above, l_ln1 and l_ln2 are just buffers if recompute >= 2, recompute them here on demand
            rmsnorm_forward(l_ln2, l_ln2_rstd, l_residual2, l_ln2w, B, T, C, main_stream);
        }
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c2, l_ln2, l_fcw, scratchF, B, T, C, ffn_channels, main_stream);
        // rmsnorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        rmsnorm_backward(dresidual, dl_ln2w, scratchF, dl_btc, l_residual2, l_ln2w, l_ln2_rstd, B, T, C, main_stream);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, scratchF, B, T, C, C, main_stream);

        // <--- gradient here matches OK

        #ifdef ENABLE_CUDNN
        printf("cuDNN path TODO\n"); exit(0);
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        attention_backward_cudnn(dl_bt4c, dl_btc, l_qkvr, l_atty, (float*)l_att, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // we need B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        floatX* buffer_a = l_atty;
        floatX* buffer_b = l_fch_pre_gelu;        // this is B x T x 4C, so even larger than what we need
        attention_backward(dl_bt4c, buffer_b, scratchX, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH, main_stream);
        #endif
        // backward rope (this can be done in-place)
        rope_backward_inplace(dl_bt4c, dl_bt4c, model->freqs_cis, B, T, NH, hd, main_stream);
        // backward repkv (use scratchX as gradient buffer here)
        repkv_backward(dl_bt4c2, dl_bt4c, B, T, NH, n_kv_head, hd, main_stream);
        // backward QKV projection
        if(model->recompute >= 2) {
            rmsnorm_forward(l_ln1, l_ln1_rstd, residual, l_ln1w, B, T, C, main_stream);
        }
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c2, l_ln1, l_qkvw, scratchF, B, T, C, qkv_channels, main_stream);
        rmsnorm_backward(dresidual, dl_ln1w, scratchF, dl_btc, residual, l_ln1w, l_ln1_rstd, B, T, C, main_stream);

        // Accumulate gradients from this layer in a background stream.
        if(last_step) {
            floatX* const pointers[] = {
                dl_ln1w, dl_ln1b,
                dl_qkvw, dl_qkvb,
                dl_attprojw, dl_attprojb,
                dl_ln2w, dl_ln2b,
                dl_fcw, dl_fcb,
                dl_fcprojw, dl_fcprojb
            };
            const size_t nelem[] = {
                C, C,
                qkv_channels * C, qkv_channels,
                C * C, C,
                C, C,
                ffn_channels * C, ffn_channels,
                C * ffn_channels_post_gelu, C
            };
            multi_gpu_async_reduce_gradient(pointers, nelem, &multi_gpu_config, main_stream);
        }
    }

    encoder_backward(grads.wte, NULL, scratchX, model->workload_indices, model->bucket_info,
                     dresidual, model->inputs, inputs, B, T, C, random_u32(&model->rng_state), main_stream);

    // Aggregate all gradients that are not part of the transformer blocks
    if(last_step) {
        // reduce all the losses within the current GPU (across all microsteps)
        global_sum_deterministic(model->accumulated_mean_loss, acts.losses, B*T, main_stream);
        // reduce loss across GPUs to a single, final float across all microsteps and GPUs
        #if MULTI_GPU
        ncclCheck(ncclAllReduce(model->accumulated_mean_loss, model->accumulated_mean_loss, sizeof(float), ncclFloat, ncclAvg, multi_gpu_config.nccl_comm, main_stream));
        #endif
        cudaCheck(hipMemcpyAsync(&model->mean_loss, model->accumulated_mean_loss, sizeof(float), hipMemcpyDeviceToHost, main_stream));
        // reduce the gradients for non-transformer block parameters
        floatX* const pointers[] = {grads.wte, grads.wlmhead, grads.lnfw, grads.lnfb};
        const size_t nelem[] = {Vp * C, Vp * C, C, C};
        multi_gpu_async_reduce_gradient(pointers, nelem, &multi_gpu_config, main_stream);
    }

    cudaCheck(hipDeviceSynchronize());
    if(last_step) {
        model->mean_loss /= B*T*grad_accum_steps;
    } else {
        model->mean_loss = -1.f; // no loss available yet
    }
}

// Gets the offset of a specific tensor for a specific layer in the LLama3 model
// layer_id is ignored for weights that are not part of a transformer block
ShardInfo llama3_get_tensor_at_layer(const LLama3 *model, int layer_id, int param_tensor_id) {
    // first offset our way to the parameter tensor start
    ptrdiff_t offset = 0;
    for (int i = 0; i < param_tensor_id; i++) {
        offset += (ptrdiff_t)model->param_elements[i];
    }
    size_t size = model->param_elements[param_tensor_id] ;
    // if we are in the transformer block, we need to additionally offset by the layer id
    if(2 <= param_tensor_id && param_tensor_id <= 13) {
        size /= model->config.num_layers;
        offset += (ptrdiff_t)(layer_id * size);
    }
    return {offset, size};
}

float llama3_calculate_grad_norm(LLama3 *model, MultiGpuConfig* multi_gpu_config) {
    NVTX_RANGE_FN();
    floatX* grads_memory = (floatX*)model->grads_memory;

    // repurposing this buffer (which isn't needed now) to write grad norm into it
    float* grad_norm_squared = (float*)model->acts.output;
    float grad_norm_squared_cpu = 0.0f;

    int num_slices[2] = {1, model->config.num_layers};
    int max_num_block_sums = get_max_num_block_sums(num_slices, 2);
    if (multi_gpu_config->zero_stage == 1) {
        // because of the ncclReduceScatter() in backward,
        // grads_memory only contains the averaged gradients at the local shards,
        // so we only calculate the grad norm at the grads_memory belonging to the local shards
        for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
            ShardInfo tensor = llama3_get_tensor_at_layer(model, 0, i);
            ShardInfo shard = multi_gpu_get_shard_offset(tensor.size, multi_gpu_config, 1);
            ptrdiff_t offset = tensor.offset + shard.offset;
            bool is_first_pass = (i == 0);
            if((i < 2 || i > 13)) {
                global_norm_squared(grad_norm_squared, grads_memory + offset, shard.size, 0, 1,
                                    max_num_block_sums, is_first_pass, main_stream);
            } else {
                global_norm_squared(grad_norm_squared, grads_memory + offset, shard.size, tensor.size, model->config.num_layers,
                                    max_num_block_sums, is_first_pass, main_stream);
            }
        }
        global_sum_deterministic(grad_norm_squared, grad_norm_squared, max_num_block_sums, main_stream);
#if MULTI_GPU
        // further sum the (partial) squared norm across all GPUs
        ncclCheck(ncclAllReduce(grad_norm_squared, grad_norm_squared, sizeof(float), ncclFloat, ncclSum, multi_gpu_config->nccl_comm, main_stream));
#endif
    } else {
        // in regular DDP, backward has averaged the gradients across all GPUs
        // so each GPU can compute the squared norm over the whole grad vector, with no added comms needed
        global_norm_squared(grad_norm_squared, grads_memory, model->num_parameters, 0, 1, max_num_block_sums, true, main_stream);
        global_sum_deterministic(grad_norm_squared, grad_norm_squared, max_num_block_sums, main_stream);
    }
    cudaCheck(hipMemcpy(&grad_norm_squared_cpu, grad_norm_squared, sizeof(float), hipMemcpyDeviceToHost));
    float grad_norm_cpu = sqrtf(grad_norm_squared_cpu);
    return grad_norm_cpu;
}

void llama3_update(LLama3 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, float grad_scale, int t,
                   MultiGpuConfig* multi_gpu_config, bool init_from_master_only=false) {
    // update the model parameters using the AdamW optimizer
    // keep in mind that optimizer sharding (ZeRO-1) assigns different parameters to different GPUs
    // so we may not be responsible for the entire parameter tensor
    // also, this function was very simple a while back but become very complex, only because we want to
    // selectively weight decay some, but not all tensors :(
    // TODO: revisit and probably refactor this entire function
    NVTX_RANGE_FN();
    if(model->grads_memory == nullptr || model->m_memory == nullptr || model->v_memory == nullptr) {
        fprintf(stderr, "Need to allocate optimizer state before update");
        exit(EXIT_FAILURE);
    }

    bool init_state = model->init_state;
    if(init_state) {
        model->init_state = false;
        NvtxRange rng("InitOpt");
        cudaCheck(hipMemset(model->m_memory, 0, multi_gpu_config->shard_num_parameters * sizeof(floatOpt)));
        cudaCheck(hipMemset(model->v_memory, 0, multi_gpu_config->shard_num_parameters * sizeof(floatOpt)));
    }

    // save RNG state at this point so we can round from master weights identically when restoring from a checkpoint
    model->rng_state_last_update = model->rng_state;

    // AdamW update
    // handle adamw for all the transformer blocks
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        // generate a unique seed for each tensor
        unsigned int seed = random_u32(&model->rng_state);

        int num_layers = model->config.num_layers;
        if((i < 2 || i > 13)) {
            num_layers = 1;
        }

        ShardInfo tensor = llama3_get_tensor_at_layer(model, 0, i);
        ShardInfo shard = multi_gpu_get_shard_offset(tensor.size, multi_gpu_config, 1);
        ptrdiff_t local_offset_full = tensor.offset + shard.offset;
        ptrdiff_t local_offset_partial = tensor.offset / multi_gpu_config->num_processes;

        // we only want to weight decay the 2D tensors and leave all 1D tensors alone
        // in particular this also decays the embedding weights, but this is ok:
        // - the token embeddings are weight shared and participate in the final projection to logits
        // - the position embeddings actively participate at every forward/backward pass
        float wd = (i == 0 || i == 1 || i == 4 || i == 6 || i == 10 || i == 12) ? weight_decay : 0.0f;
        floatX* param_ptr = (floatX*)model->params_memory + local_offset_full;
        floatX* grad_ptr = (floatX*)model->grads_memory + local_offset_full;

        ptrdiff_t opt_state_offset = multi_gpu_config->zero_stage < 1 ?  local_offset_full : local_offset_partial;
        floatOpt* m_ptr = model->m_memory + opt_state_offset;
        floatOpt* v_ptr = model->v_memory + opt_state_offset;
        float* master_ptr = nullptr;
        if (model->master_weights != nullptr) { master_ptr = model->master_weights + opt_state_offset; }
        if(init_state && model->master_weights != nullptr ) {
            size_t grid_size = CEIL_DIV(shard.size, 512);
            copy_and_cast_kernel<<<dim3(grid_size, num_layers), 512, 0, main_stream>>>(master_ptr, param_ptr, shard.size,
                                                                     shard.size, tensor.size);
            cudaCheck(hipGetLastError());
        }

        if (init_from_master_only) {
            // when resuming training from a checkpoint with master weights (allows changing precision)
            init_from_master(param_ptr, master_ptr, shard.size, tensor.size, shard.size, num_layers, seed, main_stream);
        } else {
            // ok finally call the kernel to update the weights with AdamW
            adamw_update(param_ptr, master_ptr, grad_ptr,
                        m_ptr, v_ptr,
                        shard.size, tensor.size, tensor.size, shard.size, num_layers,
                        learning_rate,
                        beta1, beta2, t, eps, wd, grad_scale, seed, main_stream);
        }

        if (multi_gpu_config->zero_stage == 1) {
#if MULTI_GPU
            ncclCheck(ncclGroupStart());
            for(int l = 0; l < num_layers; ++l) {
                // gather updated shards of model->params_memory from each process
                ncclCheck(ncclAllGather(param_ptr + l * tensor.size,
                                        (floatX*) model->params_memory + tensor.offset + l * tensor.size,
                                        shard.size, ncclFloatX,
                                        multi_gpu_config->nccl_comm, multi_gpu_config->nccl_stream));
            }
            ncclCheck(ncclGroupEnd());
#endif
        }
    }

    cudaCheck(hipDeviceSynchronize());
}

float llama3_estimate_mfu(LLama3 *model, int num_tokens, float dt) {
    /*
    Estimate model flops utilization (MFU)
    ref: Section 2.1 of https://arxiv.org/pdf/2001.08361
    Note: Ideally, the N here would be only the parameters that actually
    participate in matrix multiplications. In this N, we are over-estimating by
    including LayerNorm params, biases, and the position embedding weights,
    but these are very small terms. Also keep in mind that we would want to exclude
    the token embedding weights, but in GPT-2 these are weight shared, so they
    participate in the classifier matmul, so they are correct to be included in N.
    Note 2: The first term (6 * N) in flops_per_token is all weight matmuls, the
    second is the attention matmul, which is also usually a small contribution.
    */
    size_t N = model->num_parameters;
    int L = model->config.num_layers;
    int C = model->config.channels;
    int T = model->seq_len;
    size_t flops_per_token = 6 * N + (size_t)6 * L * C * T;
    size_t flops_per_step = flops_per_token * num_tokens;
    // express our flops throughput as ratio of A100 bfloat16 peak flops
    float flops_achieved = (float)flops_per_step * (1.0f / dt); // per second
    float flops_promised = get_flops_promised(deviceProp.name, PRECISION_MODE) * 1e12f;
    if(flops_promised < 0) {
        return -1.f;   // don't know
    }
    float mfu = flops_achieved / flops_promised;
    return mfu;
}

void llama3_free(LLama3 *model) {
    cudaFreeCheck(&model->params_memory);
    cudaFreeCheck(&model->grads_memory);
    cudaFreeCheck(&model->m_memory);
    cudaFreeCheck(&model->v_memory);
    cudaFreeCheck(&model->master_weights);
    cudaFreeCheck(&model->acts_memory);
    cudaFreeCheck(&model->inputs);
    cudaFreeCheck(&model->targets);
    cudaFreeCheck(&model->accumulated_mean_loss);
    cudaCheck(hipHostFree(model->cpu_losses));
    free(model->workload_indices);
    free(model->bucket_info);
}

// ----------------------------------------------------------------------------
// common init & free code for all of train/test/profile

void common_start(bool override_enable_tf32 = true, bool print_device_info = true) {

    // get CUDA device infos
    cudaCheck(hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx));
    if (print_device_info) {
        printf("[System]\n");
        printf("Device %d: %s\n", multi_gpu_config.local_device_idx, deviceProp.name);
    }

    // set up the cuda streams. atm everything is on the single main stream
    cudaCheck(hipStreamCreate(&main_stream));
    nvtxNameCudaStreamA(main_stream, "main stream");

    // set up cuBLAS and cuBLASLt
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    bool enable_tf32 = PRECISION_MODE == PRECISION_FP32 && deviceProp.major >= 8 && override_enable_tf32;
    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;

    #ifdef ENABLE_CUDNN
    create_cudnn();
    #endif
}

void common_free(LLama3 &model) {
    cudaCheck(hipStreamDestroy(main_stream));
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    #ifdef ENABLE_CUDNN
    destroy_cudnn();
    #endif
}


void save_state(const char* filename, int step, LLama3* model, DataLoader* loader) {
    printf("Writing state to %s\n", filename);
    FILE *state_file = fopenCheck(filename, "wb");
    int state_header[256];
    memset(state_header, 0, sizeof(state_header));
    // basic identifying information
    state_header[0] = 20240527; // magic number
    state_header[1] = 1; // version number
    state_header[2] = multi_gpu_config.num_processes; // number of processes
    state_header[3] = multi_gpu_config.process_rank; // rank of this process
    state_header[4] = model->use_master_weights;  // whether we're using fp32 master weights
    state_header[5] = loader->should_shuffle; // shuffle state of the dataloader
    // int main state, start at 10 to leave some padding
    state_header[10] = step; // step of the optimization
    // model rng state, start at 20 to leave some padding
    *((unsigned long long*)&state_header[20]) = model->rng_state; // random number generator state
    *((unsigned long long*)&state_header[22]) = model->rng_state_last_update; // last llama3_update
    // dataloader state, start at 30 to leave some padding
    *((size_t*)&state_header[30]) = loader->current_shard_idx; // shard of the dataset
    *((size_t*)&state_header[32]) = loader->current_sample_idx; // position in shard
    fwriteCheck(state_header, sizeof(int), 256, state_file);

    // write AdamW m, v, and master_weights here (they are all float, unless OPTIMIZER_LOW_PRECISION is defined)
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    device_to_file(state_file, model->m_memory, shard_num_parameters * sizeof(floatOpt), IO_BUF_SIZE, main_stream);
    device_to_file(state_file, model->v_memory, shard_num_parameters * sizeof(floatOpt), IO_BUF_SIZE, main_stream);
    if(model->use_master_weights) {
        device_to_file(state_file, model->master_weights, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    }

    // write dataloader state if we are using the Permuted version of it
    if (loader->should_shuffle) {
        fwriteCheck(&loader->glob_result.gl_pathc, sizeof(size_t), 1, state_file);  // number of shards
        fwriteCheck(loader->shard_indices, sizeof(int), loader->glob_result.gl_pathc, state_file);
        fwriteCheck(&loader->shard_num_samples, sizeof(size_t), 1, state_file);
        fwriteCheck(loader->intra_shard_indices, sizeof(int), loader->shard_num_samples, state_file);
        fwriteCheck(&loader->shuffle_rng, sizeof(mt19937_state), 1, state_file);
    }
    fcloseCheck(state_file);
}

void load_state(int* step, LLama3* model, DataLoader* loader, const char* filename) {
    FILE *state_file = fopenCheck(filename, "rb");
    int state_header[256];
    freadCheck(state_header, sizeof(int), 256, state_file);
    assert(state_header[0] == 20240527); // magic number
    assert(state_header[1] == 1); // version number
    assert(state_header[2] == multi_gpu_config.num_processes); // number of processes
    assert(state_header[3] == multi_gpu_config.process_rank); // rank of this process
    int use_master_weights = state_header[4];  // whether we're using fp32 master weights
    int should_shuffle = state_header[5]; // shuffle state of the dataloader
    *step = state_header[10]; // step of the optimization
    model->rng_state = *((unsigned long long*)&state_header[20]); // random number generator state
    model->rng_state_last_update = *((unsigned long long*)&state_header[22]); // last llama3_update
    size_t current_shard_idx = *((size_t*)&state_header[30]); // shard index
    size_t current_sample_idx = *((size_t*)&state_header[32]); // position in shard

    // read AdamW m, v, master_weights (they are all float)
    // allocate all the needed memory as necessary
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    if(use_master_weights == 1 && !model->use_master_weights) {
        printf0("Warning: Master weights are present in state, but not enabled for current run.");
    } else if (use_master_weights == 0 && model->use_master_weights) {
        printf0("Error: Master weights requested, but not present in state file.");
        exit(EXIT_FAILURE);
    }

    model->init_state = false;      // we just got the state from file, no need to do first-touch init
    assert(model->m_memory != nullptr);
    assert(model->v_memory != nullptr);
    file_to_device(model->m_memory, state_file, shard_num_parameters * sizeof(floatOpt), IO_BUF_SIZE, main_stream);
    file_to_device(model->v_memory, state_file, shard_num_parameters * sizeof(floatOpt), IO_BUF_SIZE, main_stream);
    if(model->use_master_weights) {
        assert(model->master_weights != nullptr);
        file_to_device(model->master_weights, state_file, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
        // restore weights from the master weights using the RNG state before last weight update
        model->rng_state = model->rng_state_last_update;
        llama3_update(model, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0, &multi_gpu_config, /* init_from_master_only*/ true);
        model->rng_state = *((unsigned long long*)&state_header[20]); // use final RNG state from checkpoint after this
    }

    // revive the DataLoader object and its state
    loader->should_shuffle = should_shuffle;
    if (should_shuffle == 1) {
        // ensure the number of shards matches
        size_t glob_result_gl_pathc;
        freadCheck(&glob_result_gl_pathc, sizeof(size_t), 1, state_file);
        assert(glob_result_gl_pathc == loader->glob_result.gl_pathc);
        // read the shard indices
        loader->shard_indices = (int*)mallocCheck(loader->glob_result.gl_pathc * sizeof(int));
        freadCheck(loader->shard_indices, sizeof(int), loader->glob_result.gl_pathc, state_file);
        // ensure the number of samples matches
        size_t shard_num_samples;
        freadCheck(&shard_num_samples, sizeof(size_t), 1, state_file);
        assert(shard_num_samples == loader->shard_num_samples);
        // read the intra-shard indices
        loader->intra_shard_indices = (int*)mallocCheck(loader->shard_num_samples * sizeof(int));
        freadCheck(loader->intra_shard_indices, sizeof(int), loader->shard_num_samples, state_file);
        // read the shuffle rng state
        freadCheck(&loader->shuffle_rng, sizeof(mt19937_state), 1, state_file);
    }
    dataloader_resume(loader, current_shard_idx, current_sample_idx);

    // all done, close state file
    fcloseCheck(state_file);
}

void write_checkpoint(const char* output_log_dir, int step, LLama3* model, DataLoader* train_loader, MultiGpuConfig* multi_gpu_config) {
    // a checkpoint contains: model weights, optimizer/dataloader state, and a DONE file
    printf0("Writing checkpoint at step %d\n", step);
    int rank = multi_gpu_config->process_rank;
    // only rank 0 writes the model file because it is the same across all ranks
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, step);
        llama3_write_to_checkpoint(model, filename_buffer);
    }
    // all ranks write their state file
    snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, step, rank);
    save_state(filename_buffer, step, model, train_loader);
    // DONE file is a signal that this checkpoint as a whole is complete
    multi_gpu_barrier(multi_gpu_config);
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/DONE_%08d", output_log_dir, step);
        FILE* done_file = fopenCheck(filename_buffer, "w");
        fcloseCheck(done_file);
    }
}

void delete_checkpoint(const char* output_log_dir, int step, MultiGpuConfig* multi_gpu_config) {
    // mirrors write_checkpoint function, cleans up checkpoint from disk
    printf0("Deleting checkpoint at step %d\n", step);
    int rank = multi_gpu_config->process_rank;
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, step);
        remove(filename_buffer);
    }
    snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, step, rank);
    remove(filename_buffer);
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/DONE_%08d", output_log_dir, step);
        remove(filename_buffer);
    }
}

#ifndef TESTING
// if we are TESTING (see test_llama3.cu), we'll skip everything below this point

// ----------------------------------------------------------------------------
// training resumption logic, very useful when jobs crash once in a while
// the goal is that we can resume optimization from any checkpoint, bit-perfect
// note that "state" refers to things not already saved in the model checkpoint file

// ----------------------------------------------------------------------------
// CLI, poor man's argparse
// (all single letters have been claimed now)

void error_usage() {
    fprintf(stderr, "Usage:   ./train_llama3cu [options]\n");
    fprintf(stderr, "Options:\n");
    // file system input / output
    fprintf(stderr, "  -i <string> train data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_train.bin)\n");
    fprintf(stderr, "  -j <string> val data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_val.bin)\n");
    fprintf(stderr, "  -e <string> input .bin filename or descriptor, see code comments as docs. (default = llama3.2_1B_bf16.bin)\n");
    fprintf(stderr, "  -o <string> output log dir (default = NULL, no logging)\n");
    fprintf(stderr, "  -lg <int>   log gpu info every x steps (default = -1; disabled)\n");
    fprintf(stderr, "  -n <int>    write optimization checkpoints every how many steps? (default 0, don't)\n");
    fprintf(stderr, "  -nk <int>   max number of checkpoints to keep in the directory, removing old ones (0 = disable, default)\n");
    fprintf(stderr, "  -nm <int>   every how many step checkpoints are considered major? major checkpoints never get deleted.\n");
    fprintf(stderr, "  -y <int>    resume optimization found inside output log dir? (0=restart/overwrite, 1=resume/append)\n");
    // token layout for each step of the optimization
    fprintf(stderr, "  -b <int>    (per-GPU, micro) batch size B (default = 4)\n");
    fprintf(stderr, "  -t <int>    sequence length T (default = 1024)\n");
    fprintf(stderr, "  -d <int>    total desired batch size (default = B * T * num_processes, i.e. no grad accumulation\n");
    // workload (number of steps)
    fprintf(stderr, "  -x <int>    max_steps of optimization to run (-1 (default) = disable, run 1 epoch)\n");
    // optimization
    fprintf(stderr, "  -k <string> learning rate scheduler (default = cosine)\n");
    fprintf(stderr, "  -l <float>  learning rate (default = 3e-4f)\n");
    fprintf(stderr, "  -u <int>    learning rate warmup iterations (default = 0, no warmup)\n");
    fprintf(stderr, "  -q <float>  learning rate decay: final fraction, at end of training (default = 1.0 (no decay))\n");
    fprintf(stderr, "  -c <float>  weight decay (default = 0.0f)\n");
    fprintf(stderr, "  -sl <float> outlier stability: skip update if loss goes above this in zscore (0.0f=off)\n");
    fprintf(stderr, "  -sg <float> outlier stability: skip update if grad_norm goes above this in zscore (0.0f=off)\n");
    // evaluation
    fprintf(stderr, "  -v <int>    val_loss_every, how often we evaluate val loss (default = 20)\n");
    fprintf(stderr, "  -m <int>    val_max_steps, up to how many val batches to estimate val loss? (default = 20)\n");
    fprintf(stderr, "  -s <int>    sample_every, how often we inference the model (default = 20)\n");
    fprintf(stderr, "  -g <int>    genT, how many steps of inference we do (default = 64)\n");
    fprintf(stderr, "  -h <int>    hellaswag eval run? (default = 0)\n");
    // debugging
    fprintf(stderr, "  -a <int>    overfit a single batch? 0/1. useful for debugging\n");
    // numerics
    fprintf(stderr, "  -f <int>    enable_tf32 override (default: 1, set to 0 to disable tf32)\n");
    fprintf(stderr, "  -w <int>    keep f32 copy of weights for the optimizer? (default: 1)\n");
    fprintf(stderr, "  -ge <int>   gelu fusion: 0=none, 1=forward, 2=forward+backward (default: 2 for >=SM90, 0 for older GPUs)\n");
    // memory management
    fprintf(stderr, "  -z <int>    zero_stage, Zero Optimization Stage, 0,1,2,3 (default = 0)\n");
    fprintf(stderr, "  -r <int>    recompute: less memory but less speed. (default = 1), 0|1|2 = none,gelu,gelu+ln\n");
    // multi-node settings
    fprintf(stderr, "  -pn <int>    num_processes (default = 1)\n");
    fprintf(stderr, "  -pr <int>    process_rank (default = 0)\n");
    fprintf(stderr, "  -pg <int>    gpus_per_node (default = 8)\n");
    fprintf(stderr, "  -pm <string> nccl_init_method: tcp,fs,mpi (default = mpi)\n");
    fprintf(stderr, "  -ps <string> server_ip - used only when nccl_init_method is tcp (default = -1)\n");
    fprintf(stderr, "  -pp <string> fs_path - used only when nccl_init_method is fs (default = /tmp)\n");
    exit(EXIT_FAILURE);
}

// ----------------------------------------------------------------------------
// main training loop
int main(int argc, char *argv[]) {
    // read in the (optional) command line arguments
    const char* train_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
    const char* val_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
    const char* load_filename = "llama3.2_1B_bf16.bin"; // bf16 weights of the Llama 3.2 1B model
    const char* lr_scheduler_type = "cosine";
    const char* output_log_dir = NULL;
    int checkpoint_every = 0; // write checkpoints every how many steps?
    int checkpoints_keep = 0; // how long checkpoint history do we keep? (in units of checkpoints)
    int major_checkpoint_every = 0; // major checkpoints never get deleted when maintaining history
    int resume = 0; // resume the optimization, if one is found inside output_log_dir?
    int B = 4; // batch size
    int T = 64; // sequence length max
    int total_batch_size = -1; // will be calculated down below later, if not provided
    float learning_rate = 1e-5f;
    int log_gpu_every = -1;
    int warmup_iterations = 0;
    float final_learning_rate_frac = 1.0f; // final fraction of learning rate, at end of training
    float weight_decay = 0.0f;
    float skip_update_lossz = 0.0f; // skip update if loss goes above this in zscore
    float skip_update_gradz = 0.0f; // skip update if grad_norm goes above this in zscore
    int val_loss_every = 20; // every how many steps do we eval validation loss?
    int val_max_steps = 20; // how many batches max do we eval for validation loss?
    int sample_every = 20; // every how many steps to do inference?
    int genT = 64; // number of steps of inference we will do
    int overfit_single_batch = 1; // useful for debugging, 1 = only load a single data batch once
    int max_steps = 10;
    int override_enable_tf32 = 1;
    int use_master_weights = 1;
    int gelu_fusion = -1; // 0 = none, 1 = forward, 2 = forward+backward (-1 => per-GPU default)
    int recompute = 1; // recompute during backward setting, 0 = none, 1 = recompute gelu
    int zero_stage = 0; // Zero Optimization Stage for Multi-GPU training
    int hellaswag_eval = 0;
    // multi-node settings
    int num_processes = 1;  // this should be set by the slurm environment
    int process_rank = 0;  // this should be set by the slurm environment
    int gpus_per_node = 8;  // this should be set by the slurm environment
    char nccl_init_method[256] = "mpi";  // "tcp" or "fs" or "mpi"
    char server_ip[256] = "";  // used if init_method set to "tcp" -> set to your server ip address
    char fs_path[256] = "";  // used if init_method set to "fs" -> set to a shared filesystem path
    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (!(strlen(argv[i]) == 2 || strlen(argv[i]) == 3)) { error_usage(); } // must be -x[y] (one dash, one or two letters)
        // read in the args
        if (argv[i][1] == 'i') { train_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'j') { val_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'e') { load_filename = argv[i+1]; }
        else if (argv[i][1] == 'o') { output_log_dir = argv[i+1]; }
        else if (argv[i][1] == 'n' && argv[i][2] == '\0') { checkpoint_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'y') { resume = atoi(argv[i+1]); }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); } // Per-GPU (micro) batch size
        else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'd') { total_batch_size = atoi(argv[i+1]); }
        else if (argv[i][1] == 'l' && argv[i][2] == '\0') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'l' && argv[i][2] == 'g') { log_gpu_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'u') { warmup_iterations = atoi(argv[i+1]); }
        else if (argv[i][1] == 'q') { final_learning_rate_frac = atof(argv[i+1]); }
        else if (argv[i][1] == 'c') { weight_decay = atof(argv[i+1]); }
        else if (argv[i][1] == 'x') { max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 's' && argv[i][2] == '\0') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g' && argv[i][2] == 'e') { gelu_fusion = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else if (argv[i][1] == 'a') { overfit_single_batch = atoi(argv[i+1]); }
        else if (argv[i][1] == 'f') { override_enable_tf32 = atoi(argv[i+1]); }
        else if (argv[i][1] == 'w') { use_master_weights = atoi(argv[i+1]); }
        else if (argv[i][1] == 'z') { zero_stage = atoi(argv[i+1]); }
        else if (argv[i][1] == 'r') { recompute = atoi(argv[i+1]); }
        else if (argv[i][1] == 'h') { hellaswag_eval = atoi(argv[i+1]); }
        else if (argv[i][1] == 'k') { lr_scheduler_type = argv[i+1]; }
        else if (argv[i][1] == 'p' && argv[i][2] == 'i') { strcpy(nccl_init_method, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'f') { strcpy(fs_path, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 's') { strcpy(server_ip, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'n') { num_processes = atoi(argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'r') { process_rank = atoi(argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'g') { gpus_per_node = atoi(argv[i+1]); }
        else if (argv[i][1] == 's' && argv[i][2] == 'l') { skip_update_lossz = atof(argv[i+1]); }
        else if (argv[i][1] == 's' && argv[i][2] == 'g') { skip_update_gradz = atof(argv[i+1]); }
        else if (argv[i][1] == 'n' && argv[i][2] == 'k') { checkpoints_keep = atoi(argv[i+1]); }
        else if (argv[i][1] == 'n' && argv[i][2] == 'm') { major_checkpoint_every = atoi(argv[i+1]); }
        else { error_usage(); }
    }

    multi_gpu_config = multi_gpu_config_init(num_processes, process_rank, gpus_per_node, server_ip, fs_path, nccl_init_method);
    common_start(override_enable_tf32, false); // common init code for train/test/profile

    // should do a bit more error checking here
    assert(warmup_iterations >= 0);
    if (output_log_dir != NULL) {
        assert(strlen(output_log_dir) < 400); // careful bunch of hardcoded snprintf around this
    }
    int tokens_per_fwdbwd = B * T * multi_gpu_config.num_processes; // one micro-batch processes this many tokens
    // calculate sensible default for total batch size as assuming no gradient accumulation
    if (total_batch_size == -1) { total_batch_size = tokens_per_fwdbwd; }
    // in the future, we might want to set gelu fusion to 2 for SM90+ and 0 for other GPUs
    if (gelu_fusion == -1) { gelu_fusion = 0; } // (deviceProp.major >= 9) ? 2 : 0; } // in llama3_init_common for test_llama3cu...
    // calculate the number of gradient accumulation steps from the desired total batch size
    assert(total_batch_size % tokens_per_fwdbwd == 0);
    int grad_accum_steps = total_batch_size / tokens_per_fwdbwd;
    // if we're only overfitting a single batch for debugging, let's overfit the first batch
    // from val instead of train split, because val is smaller and faster. (train_llama3.py does the same)
    if (overfit_single_batch == 1) { train_data_pattern = val_data_pattern; }
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| Parameter             | Value                                              |\n");
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| train data pattern    | %-50s |\n", train_data_pattern);
    printf0("| val data pattern      | %-50s |\n", val_data_pattern);
    printf0("| output log dir        | %-50s |\n", output_log_dir == NULL ? "NULL" : output_log_dir);
    printf0("| checkpoint_every      | %-50d |\n", checkpoint_every);
    printf0("| resume                | %-50d |\n", resume);
    printf0("| micro batch size B    | %-50d |\n", B);
    printf0("| sequence length T     | %-50d |\n", T);
    printf0("| total batch size      | %-50d |\n", total_batch_size);
    printf0("| LR scheduler          | %-50s |\n", lr_scheduler_type);
    printf0("| learning rate (LR)    | %-50e |\n", learning_rate);
    printf0("| warmup iterations     | %-50d |\n", warmup_iterations);
    printf0("| final LR fraction     | %-50e |\n", final_learning_rate_frac);
    printf0("| weight decay          | %-50e |\n", weight_decay);
    printf0("| skip update lossz     | %-50f |\n", skip_update_lossz);
    printf0("| skip update gradz     | %-50f |\n", skip_update_gradz);
    printf0("| max_steps             | %-50d |\n", max_steps);
    printf0("| val_loss_every        | %-50d |\n", val_loss_every);
    printf0("| val_max_steps         | %-50d |\n", val_max_steps);
    printf0("| sample_every          | %-50d |\n", sample_every);
    printf0("| genT                  | %-50d |\n", genT);
    printf0("| overfit_single_batch  | %-50d |\n", overfit_single_batch);
    printf0("| use_master_weights    | %-50s |\n", use_master_weights ? "enabled" : "disabled");
    printf0("| gelu_fusion           | %-50d |\n", gelu_fusion);
    printf0("| recompute             | %-50d |\n", recompute);
    printf0("+-----------------------+----------------------------------------------------+\n");
    const char* precision_str = (PRECISION_MODE == PRECISION_FP32)
                              ? (cublas_compute == HIPBLAS_COMPUTE_32F_FAST_TF32 ? "TF32" : "FP32")
                              : (PRECISION_MODE == PRECISION_FP16 ? "FP16" : "BF16");
    printf0("| device                | %-50s |\n", deviceProp.name);
    printf0("| peak TFlops           | %-50.1f |\n", get_flops_promised(deviceProp.name, PRECISION_MODE));
    printf0("| precision             | %-50s |\n", precision_str);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // figure out if we are going to be resuming the optimization
    int resuming = 0;
    // find the DONE file with the highest step count
    int resume_max_step = find_max_step(output_log_dir);
    if (resume == 1) { // is -y 1 resume flag set?
        assert(output_log_dir != NULL);
        if (resume_max_step != -1) {
            resuming = 1; // -y 1 is set, and we found a checkpoint we can resume from
            snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, resume_max_step);
        }
    }

    // build the GPT-2 model
    LLama3 model;
    llama3_init_common(&model);
    if (resuming == 1) {
        // if `-y 1` was set, then we are resuming from the latest checkpoint
        // if we are using master weights, we'll init them later inside load_state()
        bool weight_init = !use_master_weights;
        llama3_build_from_checkpoint(&model, filename_buffer, weight_init);
    } else if (ends_with_bin(load_filename)) {
        // otherwise, if this is a .bin file, we assume it's a model, let's init from it
        llama3_build_from_checkpoint(&model, load_filename);
    } else {
        // For Llama 3.1 we currently demand a .bin file to load the model from, and
        // initializing from scratch is currently not supported (but can be added later)
        printf0("Error: Llama 3 cannot be initialized from scratch right now\n");
        exit(EXIT_FAILURE);
    }

    model.use_master_weights = use_master_weights;
    model.gelu_fusion = gelu_fusion;
    model.recompute = recompute;
    printf0("| weight init method    | %-50s |\n", resuming == 1 ? "intermediate checkpoint" : load_filename);
    printf0("| max_sequence_length T | %-50d |\n", model.config.max_seq_len);
    printf0("| vocab_size V          | %-50d |\n", model.config.vocab_size);
    printf0("| padded_vocab_size Vp  | %-50d |\n", model.config.padded_vocab_size);
    printf0("| num_layers L          | %-50d |\n", model.config.num_layers);
    printf0("| num_heads NH          | %-50d |\n", model.config.num_heads);
    printf0("| channels C            | %-50d |\n", model.config.channels);
    printf0("| num_parameters        | %-50zu |\n", model.num_parameters);
    printf0("+-----------------------+----------------------------------------------------+\n");
    assert(T <= model.config.max_seq_len);

    // build DataLoaders for both train and val
    int permute_train_loader = (overfit_single_batch == 1) ? 0 : 1;
    DataLoader train_loader, val_loader;
    dataloader_init(&train_loader, train_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes, permute_train_loader);
    dataloader_init(&val_loader, val_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes, 0);
    // figure out the number of training steps we will run for
    int train_num_batches = max_steps; // passed in from command line
    if (train_num_batches == -1) {
        // sensible default is to train for exactly one epoch
        size_t ntok = train_loader.num_tokens;
        // the number of (outer loop) steps each process should take for us to reach one epoch
        train_num_batches = ntok / total_batch_size;
    }
    // figure out the number of validation steps to run for
    int val_num_batches = val_max_steps; // passed in from command line
    if (val_num_batches == -1) {
        // sensible default is to evaluate the full validation split
        size_t ntok = val_loader.num_tokens;
        // note that unlike the training loop, there is no gradient accumulation inner loop here
        val_num_batches = ntok / tokens_per_fwdbwd;
    }
    printf0("| train_num_batches     | %-50d |\n", train_num_batches);
    printf0("| val_num_batches       | %-50d |\n", val_num_batches);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build an EvalLoader for HellaSwag
    EvalLoader eval_loader;
    const char* hellaswag_path = "dev/data/hellaswag/hellaswag_val.bin";
    const bool hellaswag_available = access(hellaswag_path, F_OK) == 0;
    const bool run_hellaswag = hellaswag_eval && hellaswag_available;
    if (run_hellaswag) {
        evalloader_init(&eval_loader, hellaswag_path, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    }
    printf0("| run hellaswag         | %-50s |\n", run_hellaswag ? "yes" : "no");
    printf0("+-----------------------+----------------------------------------------------+\n");

    // pretty print in a table the multi-gpu configuration as well
    set_zero_configs(&multi_gpu_config, zero_stage, model.num_parameters);
    printf0("| num_processes         | %-50d |\n", multi_gpu_config.num_processes);
    printf0("| zero_stage            | %-50d |\n", multi_gpu_config.zero_stage);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // prints outside of pretty table to here and below
    if (!hellaswag_available) {
        printf0("HellaSwag eval not found at %s, skipping its evaluation\n", hellaswag_path);
        printf0("You can run `python dev/data/hellaswag.py` to export and use it with `-h 1`.\n");
    }
    // more prints related to allocations from llama3_build_from_checkpoint down here to not mess up our table above
    printf0("num_parameters: %zu => bytes: %zu\n", model.num_parameters, model.num_parameters_bytes);
    printf0("allocated %d MiB for model parameters\n", (int)round(model.num_parameters_bytes / (1024 * 1024)));
    // few more prints for gradient accumulation math up above
    printf0("batch_size B=%d * seq_len T=%d * num_processes=%d and total_batch_size=%d\n",
            B, T, multi_gpu_config.num_processes, total_batch_size);
    printf0("=> setting grad_accum_steps=%d\n", grad_accum_steps);

    // set up logging
    if (multi_gpu_config.process_rank == 0) { create_dir_if_not_exists(output_log_dir); }
    Logger logger;
    logger_init(&logger, output_log_dir, multi_gpu_config.process_rank, resume);

    // set up the Tokenizer
    Tokenizer tokenizer;
    // tokenizer_init(&tokenizer, "llama3_tokenizer.bin"); // TODO: port tokenizer later from GPT2 -> Llama 3

    // set up learning rate scheduler
    LearningRateScheduler lr_scheduler;
    lr_scheduler_init(&lr_scheduler, lr_scheduler_type, learning_rate,
                      warmup_iterations, train_num_batches, final_learning_rate_frac);

    // some memory for generating samples from the model
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float*  cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    // if we found a checkpoint to resume from, load the optimization state
    int step = 0;
    llama3_allocate_state(&model, B, T);
    if (resuming == 1) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, resume_max_step, multi_gpu_config.process_rank);
        load_state(&step, &model, &train_loader, filename_buffer);
    }

    // init an OutlierDetector the training loss
    OutlierDetector loss_outlier_detector, grad_norm_outlier_detector;
    init_detector(&loss_outlier_detector);
    init_detector(&grad_norm_outlier_detector);

    // train
    hipEvent_t start, end;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&end));
    cudaCheck(hipProfilerStart());
    double total_sum_iteration_time_s = 0.0;
    float ema_tokens_per_second = 0.0f;
    for (; step <= train_num_batches; step++) {
        NvtxRange step_range("Train step", step);

        int last_step = step == train_num_batches;

        if(0) { // TODO DELETE; START: IGNORE ALL THIS BLOCK WHILE GETTING STUFF TO WORK

        // once in a while estimate the validation loss (all processes collaborate)
        if (step % val_loss_every == 0 || last_step) {
            NvtxRange validation_range("validation");
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                val_loss += llama3_validate(&model, val_loader.inputs, val_loader.targets, B, T);
            }
            val_loss /= val_num_batches;
            val_loss = multi_gpu_cpu_float_sum(val_loss, &multi_gpu_config) / multi_gpu_config.num_processes;
            printf0("val loss %f\n", val_loss);
            logger_log_val(&logger, step, val_loss);
        }

        // once in a while estimate HellaSwag accuracy (all processes collaborate)
        if (run_hellaswag &&
           ((step > 0 && step % val_loss_every == 0) || last_step)) {
            NvtxRange evaluation_range("evaluation");
            float eval_acc_norm = 0.0f;
            evalloader_reset(&eval_loader);
            for (int i = 0; i < eval_loader.num_batches; i++) {
                if (i % 10 == 0) { printf("evaluating HellaSwag: %d/%d\r", i, eval_loader.num_batches); }
                evalloader_next_batch(&eval_loader);
                llama3_validate(&model, eval_loader.inputs, eval_loader.targets, B, T);
                int correct = evalloader_stat_losses(&eval_loader, model.cpu_losses);
                eval_acc_norm += (float)correct;
            }
            // careful because not all ranks may have the exact same allocation of number of examples
            eval_acc_norm = multi_gpu_cpu_float_sum(eval_acc_norm, &multi_gpu_config);
            printf0("HellaSwag: %d/%d = %f\n", (int)eval_acc_norm, eval_loader.num_examples, eval_acc_norm / eval_loader.num_examples);
            logger_log_eval(&logger, step, eval_acc_norm / eval_loader.num_examples);
        }

        // once in a while do model inference to print generated text (only rank 0)
        if (multi_gpu_config.process_rank == 0 && sample_every > 0 &&
           (step > 0 && (step % sample_every) == 0 || last_step)) {
            NvtxRange generation_range("generation");
            unsigned long long sample_rng_state = 1337;
            // fill up gen_tokens with the <|endoftext|> token, which kicks off the generation
            int eot_token = tokenizer.eot_token;
            for(int i = 0; i < B * T; ++i) {
                gen_tokens[i] = eot_token;
            }
            // now sample from the model autoregressively
            printf("generating:\n---\n");
            for (int t = 1; t < genT; t++) {
                NvtxRange generation_range("Generation step", t);
                // we try not to be too wasteful for inference by not calculating all of B,T
                // Using a smaller B is always bit-for-bit identical, but T is more tricky
                // for non-CUDNN, we need to make sure the attention buffer is memset to 0
                // for cuDNN, it might suddenly decide to use a slightly different algorithm...
                // on cuDNN 9.2.1 with cuDNN FrontEnd 1.5.2, T >= 256 seems bit-for-bit identical
                // (but even if it wasn't fully identical that's probably not the end of the world)
                // note this is still somewhat wasteful because we don't have a KV cache!
                llama3_forward(&model, gen_tokens, 1, CEIL_DIV(t, min(T,256)) * min(T,256));
                // get the V-dimensional vector probs[0, t-1, :]
                floatX* logits = model.acts.output + (t - 1) * model.config.padded_vocab_size;
                // move probs back to CPU and sample (note we only move the first vocab_size logits, ignoring the padding)
                cudaCheck(hipMemcpy(cpu_logits_raw, logits, model.config.vocab_size * sizeof(floatX), hipMemcpyDeviceToHost));
                // convert to FP32 into cpu_logits (this does nothing useful if floatX == float)
                for (int i = 0; i < model.config.vocab_size; i++) {
                    cpu_logits[i] = (float)cpu_logits_raw[i];
                }
                // sample the next token
                float coin = random_f32(&sample_rng_state);
                int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
                // print the generated token, either using the Tokenizer or a fallback
                if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                } else {
                    // fall back to printing the token id
                    printf("%d ", next_token);
                }
                fflush(stdout);
            }
            printf("\n---\n");
        }

        // once in a while checkpoint the optimization state (all ranks)
        if ((checkpoint_every > 0 && output_log_dir != NULL && resuming == 0) &&
            ((step > 0 && step % checkpoint_every == 0) || last_step)) {
            // writes model .bin file, state .bin files, and DONE file for step
            write_checkpoint(output_log_dir, step, &model, &train_loader, &multi_gpu_config);
            // we only keep checkpoints_keep checkpoints on disk to save space
            // so now that we wrote a new checkpoint, delete one old one (unless it is a "major" checkpoint)
            // we only do this is checkpoint keeping is turned on (checkpoints_keep > 0)
            int step_delete = step - checkpoints_keep * checkpoint_every;
            if (checkpoints_keep > 0 && step_delete > 0 &&
               (major_checkpoint_every == 0 || step_delete % major_checkpoint_every != 0)
                ) {
                delete_checkpoint(output_log_dir, step_delete, &multi_gpu_config);
            }
        }
        resuming = 0;
        } // TODO DELETE; END: IGNORE ALL THIS BLOCK WHILE GETTING STUFF TO WORK

        // bit confusing: we want to make sure to eval and sample on 0th iteration
        // but also after the very last iteration. so we loop for step <= train_num_batches
        // instead of just < train_num_batches (one extra due to <=), only to do
        // the validation/sampling one last time, and then we break right here as we're done.
        if (last_step) { break; }

        // --------------- TRAINING SECTION BEGIN -----------------
        if (overfit_single_batch == 1) {
            // if we are trying to overfit a single batch, we reset the loader here
            dataloader_reset(&train_loader);
        }
        // do one training step, doing forward/backward/update on total_batch_size tokens
        cudaCheck(hipEventRecord(start));
        // gradient and loss accumulation loop over micro-batches
        for (int micro_step = 0; micro_step < grad_accum_steps; micro_step++) {
            // fetch the next data batch
            dataloader_next_batch(&train_loader);
            // forward pass. note that we pass in grad_accum_steps, which scales down the loss
            llama3_forward(&model, train_loader.inputs, B, T);
            // backward pass. all model params accumulate gradients with += inside this inner loop
            llama3_backward_and_reduce(&model, train_loader.inputs, train_loader.targets, grad_accum_steps, micro_step);
        }
        float zloss = (float)(update_detector(&loss_outlier_detector, (double)model.mean_loss)); // loss z-score
        // fetch the next learning rate
        float step_learning_rate = get_learning_rate(&lr_scheduler, step);
        // calculate the gradient norm and how much we wish to scale the gradient
        float grad_norm = llama3_calculate_grad_norm(&model, &multi_gpu_config);
        float zgrad = (float)(update_detector(&grad_norm_outlier_detector, (double)grad_norm)); // grad z-score
        // update the model parameters
        if (isfinite(zloss) && skip_update_lossz != 0.0f && zloss > skip_update_lossz) {
            printf0("skipping update due to loss z-score of %f\n", zloss);
        } else if (isfinite(zgrad) && skip_update_gradz != 0.0f && zgrad > skip_update_gradz) {
            printf0("skipping update due to grad z-score of %f\n", zgrad);
        } else {
            // clip the gradient norm to a maximum value
            float grad_clip = 1.0f;
            float grad_scale = (grad_norm > grad_clip) ? grad_clip / grad_norm : 1.0f;
            llama3_update(&model, step_learning_rate, 0.9f, 0.95f, 1e-8f, weight_decay, grad_scale, step+1, &multi_gpu_config);
        }
        cudaCheck(hipEventRecord(end));
        cudaCheck(hipEventSynchronize(end)); // wait for the end event to finish to get correct timings
        // --------------- TRAINING SECTION END -------------------
        // everything that follows now is just diagnostics, prints, logging, etc.

        // todo - move or double-buffer all of this timing logic to avoid idling the GPU at this point!
        float time_elapsed_ms;
        cudaCheck(hipEventElapsedTime(&time_elapsed_ms, start, end));
        size_t tokens_processed = (size_t)multi_gpu_config.num_processes * B * T * grad_accum_steps;
        float tokens_per_second = tokens_processed / time_elapsed_ms * 1000.0f;
        float bias_corrected_ema_tokens_per_second = tokens_per_second; // by default set to non-ema version
        if (step > 0) { // consider the first batch to be a warmup (e.g. cuBLAS/cuDNN initialisation)
            total_sum_iteration_time_s += time_elapsed_ms / 1000.0f;
            // smooth out the tok/s with an exponential moving average, and bias correct just like in AdamW
            ema_tokens_per_second = 0.95f * ema_tokens_per_second + 0.05f * tokens_per_second;
            bias_corrected_ema_tokens_per_second = ema_tokens_per_second / (1.0f - powf(0.95f, step));
        }
        float mfu = llama3_estimate_mfu(&model, B * T * grad_accum_steps, time_elapsed_ms / 1000.0f);
        printf0("step %4d/%d | loss %7.6f (%+.2fz)| norm %6.4f (%+.2fz)| lr %.2e | %.2f ms | %.1f%% bf16 MFU | %.0f tok/s\n",
                step + 1, train_num_batches, model.mean_loss, zloss, grad_norm, zgrad, step_learning_rate,
                time_elapsed_ms, 100*mfu, bias_corrected_ema_tokens_per_second);
        if(log_gpu_every > 0 && (step + 1) % log_gpu_every == 0) {
            GPUUtilInfo gpu_info = get_gpu_utilization_info();
            printf0("                  compute %2.1f%% | memory: %2.1f%% | fan: %2d%% | %4d MHz / %4d MHz | %3d W / %3d W | %d°C / %d°C | %s\n",
                    gpu_info.gpu_utilization, gpu_info.mem_utilization, gpu_info.fan, gpu_info.clock, gpu_info.max_clock, gpu_info.power / 1000, gpu_info.power_limit / 1000,
                    gpu_info.temperature, gpu_info.temp_slowdown, gpu_info.throttle_reason);
        }
        logger_log_train(&logger, step, model.mean_loss, step_learning_rate, grad_norm);

        // disable the profiler after 3 steps of optimization
        if (step == 3) { hipProfilerStop(); }
    }
    // add a total average, for optimizations that are only mild improvements (excluding 1st batch as warmup)
    printf0("total average iteration time: %f ms\n", total_sum_iteration_time_s / (train_num_batches-1) * 1000);

    // free and destroy everything
    cudaCheck(hipEventDestroy(end));
    cudaCheck(hipEventDestroy(start));
    if (run_hellaswag) { evalloader_free(&eval_loader); }
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    multi_gpu_config_free(&multi_gpu_config);
    llama3_free(&model);
    common_free(model);
    return 0;
}
#endif
