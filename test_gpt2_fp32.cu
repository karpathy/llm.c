#include "hip/hip_runtime.h"
#define TESTING
#include "train_gpt2_fp32.cu"

// poor man's tensor checker
int check_tensor(float *a, float *b, int n, const char* label) {
    int print_upto = 5;
    int ok = 1;
    printf("%s\n", label);
    for (int i = 0; i < n; i++) {
        if (fabsf(a[i] - b[i]) <= 1e-2) {
            if (i < print_upto) { printf("OK "); }
        } else {
            if (i < print_upto) { printf("NOT OK "); }
            ok = 0;
        }
        if (i < print_upto) { printf("%f %f\n", a[i], b[i]); }
    }
    // print the final result
    if (ok) {
        printf("TENSOR OK\n");
    } else {
        printf("TENSOR NOT OK\n");
    }
    return ok;
}

int main(int argc, char *argv[]) {

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    printf("[System]\n");
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = 0; // NOTE: disable TF32 for testing!!!
    custom_matmul_kernel = 2; // force forward matmul to use FP32 cuBLAS for testing
    printf("enable_tf32: %d\n", enable_tf32);
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, "gpt2_124M.bin");

    // int C = model.config.channels;
    int V = model.config.vocab_size;
    int Vp = model.config.padded_vocab_size;
    int maxT = model.config.max_seq_len;
    // int L = model.config.num_layers;

    // load additional information that we will use for debugging and error checking
    FILE *state_file = fopenCheck("gpt2_124M_debug_state.bin", "rb");
    int state_header[256];
    freadCheck(state_header, sizeof(int), 256, state_file);
    if (state_header[0] != 20240327) { printf("Bad magic state file\n"); exit(EXIT_FAILURE); }
    if (state_header[1] != 2) {
        fprintf(stderr, "Bad version in state file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }
    int B = state_header[2]; // batch size, e.g. 4
    int T = state_header[3]; // time / sequence length (e.g. 64, up to maxT)
    assert(0 <= T && T <= maxT);
    printf("[State]\n");
    printf("batch_size: %d\n", B);
    printf("seq_len: %d\n", T);

    ParameterTensors expected_grads; // will be read from file (from PyTorch)
    ParameterTensors calculated_grads; // will be calculated by us
    float* expected_grads_memory = malloc_and_point_parameters(&expected_grads, model.param_sizes, 0);
    float* calculated_grads_memory = malloc_and_point_parameters(&calculated_grads, model.param_sizes, 0);

    // inputs and expected outputs, only used for error checking
    int* x = (int*)mallocCheck(B * T * sizeof(int));
    int* y = (int*)mallocCheck(B * T * sizeof(int));
    float* expected_logits = (float*) mallocCheck(B * T * V * sizeof(float));
    float* expected_loss = (float*) mallocCheck(1 * sizeof(float));

    // read reference information from Python
    freadCheck(x, sizeof(int), B*T, state_file);
    freadCheck(y, sizeof(int), B*T, state_file);
    freadCheck(expected_logits, sizeof(float), B*T*V, state_file);
    freadCheck(expected_loss, sizeof(float), 1, state_file);
    freadCheck(expected_grads_memory, sizeof(float), model.num_parameters, state_file);
    fcloseCheck(state_file);

    // overall OK signal for the test
    int allok = 1;

    // First, do target-free forward pass to validate logits
    gpt2_forward(&model, x, NULL, B, T);
    // at this point, target should be equal to expected_logits, let's compare
    // copy logits to CPU so we can compare them
    float* logits_cpu = (float*)mallocCheck(B * T * Vp * sizeof(float));
    cudaCheck(hipMemcpy(logits_cpu, model.acts.output, B * T * Vp * sizeof(float), hipMemcpyDeviceToHost));

    // compare the output logits from the forward pass
    // also careful that we don't access and compare the padded columns of logits
    int logits_ok = 1;
    float max_diff = 0.0f;
    for (int bt = 0; bt < B*T; bt++) {
        for (int v = 0; v < V; v++) {
            int i = bt * Vp + v; // linearized index
            if (i < 10) {
                printf("%f, %f\n", expected_logits[i], logits_cpu[i]);
            }
            float diff = fabsf(expected_logits[bt*V + v] - logits_cpu[i]);
            max_diff = fmaxf(max_diff, diff);
            if (diff >= 1e-2f) {
                printf("MISMATCH AT INDEX %d,%d: ", bt, v);
                printf("%f %f\n", expected_logits[bt*V + v], logits_cpu[i]);
                logits_ok = 0;
                bt = B*T; // to break out of both loops
                break;
            }
        }
    }
    allok = allok && logits_ok;
    if(!logits_ok) { printf("NOT "); }
    printf("OK (LOGITS)\n");

    // let's do 10 training iterations, following the pytorch code
    float losses[10];
    for (int step = 0; step < 10; step++) {
        struct timespec start, end;
        clock_gettime(CLOCK_MONOTONIC, &start);
        gpt2_forward(&model, x, y, B, T);
        gpt2_zero_grad(&model);
        gpt2_backward(&model);
        clock_gettime(CLOCK_MONOTONIC, &end);
        double time_elapsed_s = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

        if (step == 0) {
            // error checking at step 0 for reference activations
            free(logits_cpu);

            // compare the achieved loss
            if (fabsf(model.mean_loss - *expected_loss) >= 1e-2) {
                printf("LOSS MISMATCH: %f %f\n", model.mean_loss, *expected_loss);
                allok = 0;
            } else {
                printf("LOSS OK: %f %f\n", model.mean_loss, *expected_loss);
            }

            // and now compare the gradients on the parameters
            // hipMemcpy(calculated_grads.lnfw, model.grads.lnfw, C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.lnfb, model.grads.lnfb, C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.fcprojw, model.grads.fcprojw, L * C * 4*C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.fcprojb, model.grads.fcprojb, L * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.fcw, model.grads.fcw, L * 4*C * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.fcb, model.grads.fcb, L * 4*C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.ln2w, model.grads.ln2w, L * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.ln2b, model.grads.ln2b, L * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.attprojw, model.grads.attprojw, L * C * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.attprojb, model.grads.attprojb, L * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.qkvw, model.grads.qkvw, L * 3*C * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.qkvb, model.grads.qkvb, L * 3*C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.ln1w, model.grads.ln1w, L * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.ln1b, model.grads.ln1b, L * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.wte, model.grads.wte, V * C * sizeof(float), hipMemcpyDeviceToHost);
            // hipMemcpy(calculated_grads.wpe, model.grads.wpe, maxT * C * sizeof(float), hipMemcpyDeviceToHost);
            // check_tensor(calculated_grads.lnfb, expected_grads.lnfb, C, "lnfb");
            // check_tensor(calculated_grads.lnfw, expected_grads.lnfw, C, "lnfw");
            // check_tensor(calculated_grads.fcprojw, expected_grads.fcprojw, L * C * 4*C, "fcprojw");
            // check_tensor(calculated_grads.fcprojb, expected_grads.fcprojb, L * C, "fcprojb");
            // check_tensor(calculated_grads.fcw, expected_grads.fcw, L * 4*C * C, "fcw");
            // check_tensor(calculated_grads.fcb, expected_grads.fcb, L * 4*C, "fcb");
            // check_tensor(calculated_grads.ln2w, expected_grads.ln2w, L * C, "ln2w");
            // check_tensor(calculated_grads.ln2b, expected_grads.ln2b, L * C, "ln2b");
            // check_tensor(calculated_grads.attprojw, expected_grads.attprojw, L * C * C, "attprojw");
            // check_tensor(calculated_grads.attprojb, expected_grads.attprojb, L * C, "attprojb");
            // check_tensor(calculated_grads.qkvw, expected_grads.qkvw, L * 3*C * C, "qkvw");
            // check_tensor(calculated_grads.qkvb, expected_grads.qkvb, L * 3*C, "qkvb");
            // check_tensor(calculated_grads.ln1w, expected_grads.ln1w, L * C, "ln1w");
            // check_tensor(calculated_grads.ln1b, expected_grads.ln1b, L * C, "ln1b");
            // check_tensor(calculated_grads.wte, expected_grads.wte, V * C, "wte");
            // check_tensor(calculated_grads.wpe, expected_grads.wpe, maxT * C, "wpe");

            // compare the gradients ona the parameters all at once
            hipMemcpy(calculated_grads_memory, model.grads_memory, model.num_parameters * sizeof(float), hipMemcpyDeviceToHost);
            check_tensor(calculated_grads_memory, expected_grads_memory, model.num_parameters, "grads");
        }

        gpt2_update(&model, 1e-4f, 0.9f, 0.999f, 1e-8f, 0.01f, step+1);

        // print the timing information at the end
        printf("step %d: loss %f (took %f ms)\n", step, model.mean_loss, time_elapsed_s * 1000);
        losses[step] = model.mean_loss;
    }

    // expected losses are as follows, from Python
    float expected_losses[10] = {
        5.270007133483887f,
        4.059706687927246f,
        3.3751230239868164f,
        2.8007826805114746f,
        2.315382242202759f,
        1.8490285873413086f,
        1.3946564197540283f,
        0.9991465210914612f,
        0.6240804195404053f,
        0.37651097774505615f
    };

    // compare
    for (int i = 0; i < 10; i++) {
        if (fabsf(losses[i] - expected_losses[i]) >= 1e-2) {
            printf("LOSS MISMATCH AT STEP %d: %f %f\n", i, losses[i], expected_losses[i]);
            allok = 0;
        } else {
            printf("loss ok at step %d: %f %f\n", i, losses[i], expected_losses[i]);
        }
    }

    // final approval
    printf("overall okay: %d\n", allok);

    // free everything
    free(x);
    free(y);
    free(expected_logits);
    free(expected_loss);
    free(expected_grads_memory);
    free(calculated_grads_memory);
    gpt2_free(&model);
    cublasCheck(hipblasDestroy(cublas_handle));

    return 0;
}